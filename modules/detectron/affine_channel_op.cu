#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2016-present, Facebook, Inc.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "affine_channel_op.h"
#include "caffe2/core/context_gpu.h"

namespace caffe2 {

namespace {
template <typename T>
__global__ void ScaleBiasForward(
    const int n,
    const T* in,
    const T* scale,
    const T* bias,
    const int scale_dim,
    const int hxw_dim,
    T* out) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    const int scale_index = (index / hxw_dim) % scale_dim;
    out[index] = in[index] * scale[scale_index] + bias[scale_index];
  }
}

template <typename T>
__global__ void ScaleForward(
    const int n,
    const T* in,
    const T* scale,
    const int scale_dim,
    const int hxw_dim,
    T* out) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    const int scale_index = (index / hxw_dim) % scale_dim;
    out[index] = in[index] * scale[scale_index];
  }
}
} // namespace

template <>
bool AffineChannelOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);
  auto& scale = Input(1);
  auto& bias = Input(2);
  auto* Y = Output(0);

  Y->ResizeLike(X);
  const int output_size = Y->size();
  ScaleBiasForward<float>
      <<<CAFFE_GET_BLOCKS(output_size),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context_.hip_stream()>>>(
          output_size,
          X.data<float>(),
          scale.data<float>(),
          bias.data<float>(),
          X.dim32(1),
          X.dim32(2) * X.dim32(3),
          Y->mutable_data<float>());
  return true;
}

template <>
bool AffineChannelGradientOp<float, HIPContext>::RunOnDevice() {
  auto& scale = Input(0);
  auto& dY = Input(1);
  auto* dX = Output(0);

  dX->ResizeLike(dY);
  ScaleForward<float>
      <<<CAFFE_GET_BLOCKS(dY.size()),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context_.hip_stream()>>>(
          dY.size(),
          dY.data<float>(),
          scale.data<float>(),
          dY.dim32(1),
          dY.dim32(2) * dY.dim32(3),
          dX->mutable_data<float>());
  return true;
}

REGISTER_HIP_OPERATOR(AffineChannel, AffineChannelOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(
    AffineChannelGradient,
    AffineChannelGradientOp<float, HIPContext>);
} // namespace caffe2
