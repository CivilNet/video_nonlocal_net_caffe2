#include "hip/hip_runtime.h"
#include <algorithm>
#include <atomic>
#include <cstdlib>
#include <string>
#include <unordered_map>

#include "cub/util_allocator.cuh"

#include "caffe2/core/asan.h"
#include "caffe2/core/hip/common_miopen.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/core/init.h"
#include "caffe2/core/logging.h"
#include "caffe2/core/tensor.h"
#include "caffe2/utils/string_utils.h"

CAFFE2_DEFINE_string(caffe2_cuda_memory_pool, "",
              "Sets the memory pool used by caffe2. Possible values are "
              "none, cnmen and cub.");

// For description of CUB caching allocator configuration, see
// https://nvlabs.github.io/cub/structcub_1_1_caching_device_allocator.html
CAFFE2_DEFINE_int(caffe2_cub_bin_growth, 8,
             "If using cub as the memory allocator, sets the growth of bins "
             "used by the cub pool.");
CAFFE2_DEFINE_int(caffe2_cub_min_bin, 3,
             "If using cub as the memory allocator, sets the min number of "
             "bins.");
CAFFE2_DEFINE_int(caffe2_cub_max_bin, 10,
             "If using cub as the memory allocator, sets the max number of "
             "bins.");
CAFFE2_DEFINE_int(caffe2_cub_max_managed_mb, 10 * 1024,
             "If using cub as the memory allocators, sets the maximum amount "
             "of memory managed in gigabytes");
CAFFE2_DEFINE_bool(
    caffe2_cub_print_allocation_events,
    false,
    "If true CachingDeviceAllocator will print allocation and deallocation "
    "events to stdout.");

CAFFE2_DEFINE_bool(
    caffe2_gpu_memory_tracking,
    false,
    "If set, logs changes in GPU memory allocations");
CAFFE2_DEFINE_int(
    caffe2_gpu_memory_report_interval_mb,
    128,
    "The threshold in MB on how frequently to report memory changes");

namespace caffe2 {

CAFFE_KNOWN_TYPE(Tensor<HIPContext>);

thread_local ThreadLocalCUDAObjects HIPContext::cuda_objects_;

// TODO(jiayq): these variables shouldn't be currently accessed during static
// initialization. We should consider moving them to a Mayer's singleton to
// be totally safe against SIOF.

// Static global variables for setting up the memory pool.
CudaMemoryPoolType g_cuda_memory_pool_type;

// For cub allocator
unique_ptr<hipcub::CachingDeviceAllocator> g_cub_allocator;
// an unordered map that holds the map from the cuda memory pointer to the
// device id that it is allocated from. This is used in the cuda memory pool
// cases, where we need the device id to carry out the deletion.
// Note(jiayq): an alternate approach is to use cudaGetPointerAttributes, but
// that is usually quite slow. We might want to benchmark the speed difference
// though.
// Note(jiayq): another alternate approach is to augment the Tensor class that
// would allow one to record the device id. However, this does not address any
// non-tensor allocation and deallocation.
// Ideally, a memory pool should already have the device id information, as
// long as we are using UVA (as of CUDA 5 and later) so the addresses are
// unique.
static std::unordered_map<void*, uint8_t> g_cuda_device_affiliation;

// Data structures for optional memory tracking. Access to these structures
// is garded by the HIPContext::mutex.
static std::unordered_map<void*, long> g_size_map;
static std::vector<long> g_total_by_gpu_map(CAFFE2_COMPILE_TIME_MAX_GPUS, 0);
static std::vector<long> g_max_by_gpu_map(CAFFE2_COMPILE_TIME_MAX_GPUS, 0);

static long g_total_mem = 0;
static long g_last_rep = 0;

CudaMemoryPoolType GetCudaMemoryPoolType() {
  return g_cuda_memory_pool_type;
}

vector<TIndex> GetCUDATensorInfo(
    const void* c,
    bool* shares_data,
    size_t* capacity,
    DeviceOption* device) {
  vector<TIndex> dims =
      GetTensorInfo<HIPContext>(c, shares_data, capacity, device);
  const Tensor<HIPContext>* tc = static_cast<const Tensor<HIPContext>*>(c);
  device->set_device_type(CUDA);
  device->set_cuda_gpu_id(GetGPUIDForPointer(tc->raw_data()));
  return dims;
}

///////////////////////////////////////////////////////////////////////////////
// A wrapper to allow us to lazily initialize all cuda environments that Caffe
// uses. This gets done the first time a caffe2::HIPContext::New() gets called
// which is probably the decisive indication that this caffe2 run is going to
// use GPUs. We avoid cuda initialization with core/init.h functionalities so
// that we have minimal resource impact in case we will need to run multiple
// caffe2 instances on a GPU machine.
///////////////////////////////////////////////////////////////////////////////

static void Caffe2InitializeCuda() {
  // If the current run does not have any cuda devices, do nothing.
  if (!HasCudaGPU()) {
    VLOG(1) << "No cuda gpu present. Skipping.";
    return;
  }
  // Check if the number of GPUs matches the expected compile-time max number
  // of GPUs.
  CAFFE_ENFORCE_LE(
      NumCudaDevices(),
      CAFFE2_COMPILE_TIME_MAX_GPUS,
      "Number of CUDA devices on the machine is larger than the compiled "
      "max number of gpus expected (",
      CAFFE2_COMPILE_TIME_MAX_GPUS,
      "). Increase that and recompile the caffe binary.");

  for (int i = 0; i < NumCudaDevices(); ++i) {
    DeviceGuard g(i);
    // Enable peer access.
    const int peer_group = i / CAFFE2_CUDA_MAX_PEER_SIZE;
    const int peer_start = peer_group * CAFFE2_CUDA_MAX_PEER_SIZE;
    const int peer_end = std::min(
        NumCudaDevices(), (peer_group + 1) * CAFFE2_CUDA_MAX_PEER_SIZE);
    VLOG(1) << "Enabling peer access within group #" << peer_group
            << ", from gpuid " << peer_start << " to " << peer_end - 1
            << ", for gpuid " << i << ".";

    for (int j = peer_start; j < peer_end; ++j) {
      if (i == j) continue;
      int can_access;
      CUDA_ENFORCE(hipDeviceCanAccessPeer(&can_access, i, j));
      if (can_access) {
        VLOG(1) << "Enabling peer access from " << i << " to " << j;
        // Note: just for future reference, the 0 here is not a gpu id, it is
        // a reserved flag for hipDeviceEnablePeerAccess that should always be
        // zero currently.
        CUDA_ENFORCE(hipDeviceEnablePeerAccess(j, 0));
      }
    }
  }

  RegisterTypeCallFunction(
    TypeMeta::Id<Tensor<HIPContext>>(),
    GetTensorType<HIPContext>
  );

  RegisterTensorInfoFunction(
      TypeMeta::Id<Tensor<HIPContext>>(), GetCUDATensorInfo);

  // Check the versions of cuDNN that were compiled and linked with are compatible
  CheckCuDNNVersions();
}

static void SetUpCub() {
  VLOG(1) << "Setting up cub memory pool.";
  // Sets up the cub memory pool
  try {
    g_cub_allocator.reset(new hipcub::CachingDeviceAllocator(
        FLAGS_caffe2_cub_bin_growth,
        FLAGS_caffe2_cub_min_bin,
        FLAGS_caffe2_cub_max_bin,
        size_t(FLAGS_caffe2_cub_max_managed_mb) * 1024L * 1024L,
        false,
        FLAGS_caffe2_cub_print_allocation_events));
  } catch (...) {
    CAFFE_THROW("Some error happened at cub initialization.");
  }
  VLOG(1) << "Done setting up cub memory pool.";
}

static void Caffe2SetCUDAMemoryPool() {
  if (FLAGS_caffe2_cuda_memory_pool == "" ||
      FLAGS_caffe2_cuda_memory_pool == "none") {
    g_cuda_memory_pool_type = CudaMemoryPoolType::NONE;
  } else if (FLAGS_caffe2_cuda_memory_pool == "cnmem") {
    CAFFE_THROW("CNMEM is no longer used by Caffe2. Use cub instead. "
                "This error message may go away in the future.");
  } else if (FLAGS_caffe2_cuda_memory_pool == "cub") {
    // Sets up cub.
    g_cuda_memory_pool_type = CudaMemoryPoolType::CUB;
    SetUpCub();
  } else {
    CAFFE_THROW("Unrecognized cuda memory pool type: ",
                FLAGS_caffe2_cuda_memory_pool);
  }
}

// An initialization function that sets the CPU side to use pinned cpu
// allocator.
void Caffe2UsePinnedCPUAllocator() {
#if CAFFE2_ASAN_ENABLED
  // Note(jiayq): for more details, see
  //     https://github.com/google/sanitizers/issues/629
  LOG(WARNING) << "There are known issues between address sanitizer and "
                  "hipHostMalloc. As a result, caffe2 will not enable pinned "
                  "memory allocation in asan mode. If you are expecting any "
                  "behavior that depends on asan, be advised that it is not "
                  "turned on.";
#else
  if (!HasCudaGPU()) {
    VLOG(1) << "No GPU present. I won't use pinned allocator then.";
    return;
  }
  VLOG(1) << "Caffe2 gpu: setting CPUAllocator to PinnedCPUAllocator.";
  SetCPUAllocator(new PinnedCPUAllocator());
#endif
}

// Caffe2CudaInitializerHelper is a minimal struct whose sole purpose is to
// detect the first hint that this Caffe2 run is going to use GPU: either
// HIPContext is initialized or HIPContext::New is called. It then runs
// all the related cuda initialization functions.
namespace {
struct Caffe2CudaInitializerHelper {
  Caffe2CudaInitializerHelper() {
    // We cannot use bool because nvcc changes bool to __nv_bool which does
    // not have a std::atomic instantiation.
    static std::atomic<char> first_call(1);
    if (first_call.fetch_and((char)0)) {
      Caffe2InitializeCuda();
      Caffe2SetCUDAMemoryPool();
      Caffe2UsePinnedCPUAllocator();
    }
  }
};
}  // namespace

/**
 * A utility function to rectify the gpu id. If the context specifies the
 * gpu id to be -1, it means that we will just use the current gpu id when
 * the function is being called.
 */
static inline int RectifyGPUID(const int gpu_id) {
  return gpu_id == -1 ? CaffeCudaGetDevice() : gpu_id;
}

HIPContext::HIPContext(const int gpu_id)
    : gpu_id_(RectifyGPUID(gpu_id)), random_seed_(RandomNumberSeed()) {
  static Caffe2CudaInitializerHelper g_cuda_initializer_;
}

HIPContext::HIPContext(const DeviceOption& option)
    : gpu_id_(
          option.has_cuda_gpu_id() ? RectifyGPUID(option.cuda_gpu_id())
                                   : CaffeCudaGetDevice()),
      random_seed_(
          option.has_random_seed() ? option.random_seed()
                                   : RandomNumberSeed()) {
  static Caffe2CudaInitializerHelper g_cuda_initializer_;
  DCHECK_EQ(option.device_type(), CUDA);
}

// shared mutex to lock out alloc / free during NCCL launches
std::mutex& HIPContext::mutex() {
  static std::mutex m;
  return m;
}

std::vector<long> HIPContext::TotalMemoryByGpu() {
  std::lock_guard<std::mutex> lock(HIPContext::mutex());
  CAFFE_ENFORCE(
      FLAGS_caffe2_gpu_memory_tracking,
      "Pass --caffe2_gpu_memory_tracking to enable memory stats");
  return g_total_by_gpu_map;
}

std::vector<long> HIPContext::MaxMemoryByGpu() {
  std::lock_guard<std::mutex> lock(HIPContext::mutex());
  CAFFE_ENFORCE(
      FLAGS_caffe2_gpu_memory_tracking,
      "Pass --caffe2_gpu_memory_tracking to enable memory stats");
  return g_max_by_gpu_map;
}

namespace {
void TrackMemoryAlloc(size_t nbytes) {
  int this_gpu = CaffeCudaGetDevice();
  g_total_by_gpu_map[this_gpu] += nbytes;
  g_max_by_gpu_map[this_gpu] =
      max(g_max_by_gpu_map[this_gpu], g_total_by_gpu_map[this_gpu]);
  g_total_mem += nbytes;
  if (g_total_mem - g_last_rep >
      FLAGS_caffe2_gpu_memory_report_interval_mb * 1024 * 1024) {
    for (int gpu = 0; gpu < g_total_by_gpu_map.size(); gpu++) {
      long t = g_total_by_gpu_map[gpu];
      long max_t = g_max_by_gpu_map[gpu];
      if (max_t > 0) {
        if (max_t != t) {
          LOG(INFO) << "GPU " << gpu << ": " << t / 1024 / 1024 << " MB"
                    << " (max: " << max_t / 1024 / 1024 << " MB)";
        } else {
          LOG(INFO) << "GPU " << gpu << ": " << t / 1024 / 1024 << " MB";
        }
      }
    }
    LOG(INFO) << "Total: " << g_total_mem / 1024 / 1024 << " MB";
    g_last_rep = g_total_mem;
  }
}
}

std::pair<void*, MemoryDeleter> HIPContext::New(size_t nbytes) {
  // Lock the mutex
  std::lock_guard<std::mutex> lock(HIPContext::mutex());
  // A one-time caffe2 cuda initializer.
  static Caffe2CudaInitializerHelper g_cuda_initializer_;
  void* ptr = nullptr;

  if (FLAGS_caffe2_gpu_memory_tracking) {
    TrackMemoryAlloc(nbytes);
  }
  switch (g_cuda_memory_pool_type) {
  case CudaMemoryPoolType::NONE:
    CUDA_ENFORCE(hipMalloc(&ptr, nbytes));
    if (FLAGS_caffe2_gpu_memory_tracking) {
      g_size_map[ptr] = nbytes;
      g_cuda_device_affiliation[ptr] = CaffeCudaGetDevice();
    }
    return {ptr, Delete};
  case CudaMemoryPoolType::CUB:
    CUDA_ENFORCE(g_cub_allocator->DeviceAllocate(&ptr, nbytes));
    g_cuda_device_affiliation[ptr] = CaffeCudaGetDevice();
    VLOG(2) << "CUB allocating pointer " << ptr << " on device "
            << CaffeCudaGetDevice();
    if (FLAGS_caffe2_gpu_memory_tracking) {
      g_size_map[ptr] = nbytes;
    }
    return {ptr, Delete};
  }
  return {nullptr, Delete};
}

void HIPContext::Delete(void* ptr) {
  // lock the mutex
  std::lock_guard<std::mutex> lock(HIPContext::mutex());

  if (FLAGS_caffe2_gpu_memory_tracking) {
    auto sz_it = g_size_map.find(ptr);
    DCHECK(sz_it != g_size_map.end());
    auto aff_it = g_cuda_device_affiliation.find(ptr);
    DCHECK(aff_it != g_cuda_device_affiliation.end());
    g_total_mem -= sz_it->second;
    g_total_by_gpu_map[aff_it->second] -= sz_it->second;
    g_size_map.erase(sz_it);
  }

  switch (g_cuda_memory_pool_type) {
  case CudaMemoryPoolType::NONE: {
    // If memory pool is not set up, use simple hipFree.
    hipError_t error = hipFree(ptr);
    // For some reason, in Python runtime we sometimes delete a data pointer
    // after the cuda runtime exits - this is odd but is probably caused by
    // a static workspace that pycaffe2 uses, and the destruction got
    // entangled in some race condition. Anyway, since cuda runtime is exiting
    // anyway, we will not need to worry about memory leak, so we basically
    // ignore it. This is definitely not ideal but works for now.
    if (error != hipSuccess && error != hipErrorDeinitialized) {
      LOG(FATAL) << "Error at: " << __FILE__ << ":" << __LINE__ << ": "
                 << hipGetErrorString(error);
    }

    if (FLAGS_caffe2_gpu_memory_tracking) {
      g_cuda_device_affiliation.erase(g_cuda_device_affiliation.find(ptr));
    }

    break; }
  case CudaMemoryPoolType::CUB: {
    auto it = g_cuda_device_affiliation.find(ptr);
    DCHECK(it != g_cuda_device_affiliation.end());
    VLOG(2) << "CUB freeing pointer " << ptr << " on device " << it->second;
    CUDA_ENFORCE(g_cub_allocator->DeviceFree(it->second, ptr));
    g_cuda_device_affiliation.erase(it);
    break;
  }
  }
}

}  // namespace caffe2
