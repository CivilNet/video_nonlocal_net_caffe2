#include "hip/hip_runtime.h"
/*
 * Copyright 2014 Google Inc. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *    http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <vector>

#include "../include/cudaconv2.cuh"

/*
 * Block size: 16x16.
 * blockIdx.x determines case in batches of 16*imgsPerThread.
 * blockIdx.y determines 4x4 image region in target image.
 *
 * threadIdx.x determines case.
 * threadIdx.y determines pixel.
 *
 * hidActs:     (numFilters, numModulesY, numModulesX, numImages)
 * filters:     (numColors, filterPixels, numFilters)                               if conv
 *              (numModulesY, numModulesX, numColors, filterPixels, numFilters)     otherwise
 * targets:     (numColors, imgSizeY, imgSizeX, numImages)
 *
 * Each block reconstructs one 4x4 pixels from 16*imgsPerThread cases.
 *
 * Number of filters must be divisible by 16.
 * Number of images must be divisible by 16*imgsPerThread  if checkCaseBounds is false.
 * 16 * imgsPerThread must be divisible by 32.
 *
 * This version loads 32 cases at a time, so it gets full coalescing on that load.
 * It only loads 16 weights at a time, so those aren't fully coalesced.
 * This version conserves shared memory by loading 16 filters at a time rather than 32.
 */
template <int imgsPerThread, int numColors, bool scale, bool checkCaseBounds, bool conv>
__global__ void img_acts_color(const float* hidActs, const float* filters, float* targets,
                                   const int numModulesY, const int numModulesX, const int numImages, const int numFilters,
                                   const int filterSize, const int imgSizeY, const int imgSizeX,
                                   const int paddingStart, const int moduleStride,
                                   const float scaleTargets, const float scaleOutputs) {
    __shared__ float shFilters[numColors*16][16 + 1];
    __shared__ float shHidActs[16][16*imgsPerThread];

    const int blockCaseIdx = blockIdx.x * 16*imgsPerThread;
    const int numRegionsX = DIVUP(imgSizeX, 4);
    const int blockRegionIdx = blockIdx.y;
    const int blockRegionIdxX = blockRegionIdx % numRegionsX;
    const int blockRegionIdxY = blockRegionIdx / numRegionsX;
    const int blockRegionLeft = blockRegionIdxX * 4;
    const int blockRegionTop = blockRegionIdxY * 4;
    const int pxYInRegion = threadIdx.y / 4, pxXInRegion = threadIdx.y % 4;
    const int pxY = blockRegionTop + pxYInRegion;
    const int pxX = blockRegionLeft + pxXInRegion;
    const int pxIdx = pxY * imgSizeX + pxX;
    const bool isPxInImg = pxY < imgSizeY && pxX < imgSizeX;
    const int numModules = numModulesY * numModulesX;
    const int filterPixels = filterSize * filterSize;
    const int imgPixels = imgSizeX * imgSizeY;
    const int tidx = threadIdx.y * 16 + threadIdx.x;
    const int loadY = tidx / 32, loadX = tidx % 32;

    hidActs += blockCaseIdx + loadY * numImages * numModules + loadX;
    filters += threadIdx.x;
    targets += pxIdx * numImages + blockCaseIdx + threadIdx.x;


    float prod[numColors][imgsPerThread];
    #pragma unroll
    for (int c = 0; c < numColors; c++) {
        #pragma unroll
        for (int i = 0; i < imgsPerThread; i++) {
            prod[c][i] = 0;
        }
    }
    const int startY = blockRegionTop - paddingStart < filterSize ? 0
                        : 1 + (blockRegionTop - paddingStart - filterSize) / moduleStride;
    const int endY = MIN(numModulesY, 1 + (blockRegionTop + 3 - paddingStart) / moduleStride);
    const int startX = blockRegionLeft - paddingStart < filterSize ? 0
                        : 1 + (blockRegionLeft - paddingStart - filterSize) / moduleStride;
    const int endX = MIN(numModulesX, 1 + (blockRegionLeft + 3 - paddingStart) / moduleStride);

    float* shilterLoad = &shFilters[threadIdx.y][threadIdx.x];
    float* shHidActLoad = &shHidActs[loadY][loadX];

    for (int my = startY; my < endY; my++) {
        const int moduleTop = paddingStart + my * moduleStride;
        const int pxInModuleY = pxY - moduleTop;

        for (int mx = startX; mx < endX; mx++) {
            const int moduleIdx = my * numModulesX + mx;
            const int moduleLeft = paddingStart + mx * moduleStride;
            const int pxInModuleX = pxX - moduleLeft;

            const bool isPxInModule = pxInModuleY >= 0 && pxInModuleY < filterSize && pxInModuleX >= 0 && pxInModuleX < filterSize;
            const int pxIdxInModule = pxInModuleY * filterSize + pxInModuleX;

            for (int f = 0; f < numFilters; f += 16) { // multiply with 16 filters at a time
                // Now the threads split up into half-warps, and each half-warp decides if it's interested.
                const float* hLoad = &hidActs[(moduleIdx + f * numModules) * numImages];
                #pragma unroll
                for (int i = 0; i < imgsPerThread * 16; i += 32) {
                    if (!checkCaseBounds || blockCaseIdx + i + loadX < numImages) {
                        #pragma unroll
                        for (int j = 0; j < 16; j += 8) { // load 16 rows of imgsPerThread*16 cols, 8 * 32 elements at a time.
                            shHidActLoad[j * 16 * imgsPerThread + i] = hLoad[j * numModules * numImages + i];
                        }
                    } else {
                        #pragma unroll
                        for (int j = 0; j < 16; j += 8) { // load 16 rows of imgsPerThread*16 cols, 8 * 32 elements at a time.
                            shHidActLoad[j * 16 * imgsPerThread + i] = 0;
                        }
                    }
                }

                if (isPxInImg && isPxInModule) {
                    // This half-warp is interested, so it's going to load the weights from this module to its pixel.
                    // Not fully coalesced read :(
                    // But taking out this read entirely only reduces the runtime by ~2.8%, so it isn't costing me much.
                    const float* fLoad = conv ? &filters[pxIdxInModule * numFilters + f]
                                              : &filters[(moduleIdx * numColors * filterPixels + pxIdxInModule) * numFilters + f];
                    #pragma unroll
                    for (int c = 0; c < numColors; c++) {
                        shilterLoad[c * 16 * (16 + 1)] = fLoad[c * filterPixels * numFilters];
                    }


                }

                __syncthreads();
                // Do some actual computation
                if (isPxInImg && isPxInModule) {
                    #pragma unroll
                    for (int c = 0; c < numColors; c++) {
                        #pragma unroll
                        for (int w = 0; w < 16; w++) {
                            #pragma unroll
                            for (int i = 0; i < imgsPerThread; i++) {
                                prod[c][i] += shFilters[threadIdx.y + c * 16][w] * shHidActs[w][threadIdx.x + i * 16];
                            }
                        }
                    }
                }
                __syncthreads();
            }
        }
    }
    // Not fully coalesced write :(... shmem (and fully coalesced) version is actually slightly slower, though
    if (isPxInImg) {
        if (scale) {
            #pragma unroll
            for (int i = 0; i < imgsPerThread; i++) {
                if (!checkCaseBounds || blockCaseIdx + threadIdx.x + i * 16 < numImages) {
                    #pragma unroll
                    for (int c = 0; c < numColors; c++) {
                        targets[c * imgPixels * numImages + i * 16] = scaleTargets * targets[c * imgPixels * numImages + i * 16] + scaleOutputs * prod[c][i];
                    }
                }
            }
        } else {
            #pragma unroll
            for (int i = 0; i < imgsPerThread; i++) {
                if (!checkCaseBounds || blockCaseIdx + threadIdx.x + i * 16 < numImages) {
                    #pragma unroll
                    for (int c = 0; c < numColors; c++) {
                        targets[c * imgPixels * numImages + i * 16] = scaleOutputs * prod[c][i];
                    }
                }
            }
        }
    }
}

/*
 * Block size: 16x16.
 * blockIdx.x determines case in batches of 16*imgsPerThread, also color in batches of colorsPerThread.
 *  In essence, blockIdx.x.x = 1..numImages/(16*imgsPerThread)
 *              blockIdx.x.y = 1..numImgColors/colorsPerThread
 * blockIdx.y determines 4x4 image region in target image.
 *
 * threadIdx.x determines case.
 * threadIdx.y determines pixel.
 *
 * hidActs:     (numFilters, numModulesY, numModulesX, numImages)
 * filters:     (numFilterColors, filterPixels, numFilters)                             if conv
 *              (numModulesY, numModulesX, numFilterColors, filterPixels, numFilters)   otherwise
 * targets:     (numImageColors, imgSizeY, imgSizeX, numImages)
 *
 * Each block reconstructs one 4x4 pixels from 16*imgsPerThread cases.
 *
 * numImages must be divisible by 16*imgsPerThread if checkCaseBounds is false.
 * 16 * imgsPerThread must be divisible by 32.
 * numImageColors/numGroups must be divisible by colorsPerThread.
 *
 * This version loads 32 cases at a time, so it gets full coalescing on that load.
 * It only loads 16 weights at a time, so those aren't fully coalesced.
 * This version conserves shared memory by loading 16 filters at a time rather than 32.
 *
 * To be used when there are 4-16 color channels.
 */
template <int imgsPerThread, int colorsPerThread,  bool scale, bool checkCaseBounds, bool conv>
__global__ void img_acts_mediumcolor(const float* hidActs, const float* filters, float* targets,
                                       const int numModulesY, const int numModulesX, const int numImages, const int numFilters,
                                       const int filterSize, const int imgSizeY, const int imgSizeX, const int paddingStart,
                                       const int moduleStride, const int numImgColors, const int numGroups,
                                       const float scaleTargets, const float scaleOutputs) {
    __shared__ float shFilters[colorsPerThread*16][16 + 1];
    __shared__ float shHidActs[16][16*imgsPerThread];

    const int numImgBlocks = DIVUP(numImages,16*imgsPerThread);
    const int blockCaseIdx = (blockIdx.x % numImgBlocks) * 16*imgsPerThread;

    const int imgColorIdx = (blockIdx.x / numImgBlocks) * colorsPerThread; // color idx globally
    const int numFilterColors = numImgColors / numGroups;
    const int blockGroupIdx = imgColorIdx / numFilterColors;
    const int filterColorIdx = imgColorIdx % numFilterColors; // color idx within group
    const int numFiltersPerGroup = numFilters / numGroups;
    const int blockFilterIdx = blockGroupIdx * numFiltersPerGroup;

    const int numRegionsX = DIVUP(imgSizeX, 4);
    const int blockRegionIdx = blockIdx.y;
    const int blockRegionIdxX = blockRegionIdx % numRegionsX;
    const int blockRegionIdxY = blockRegionIdx / numRegionsX;
    const int blockRegionLeft = blockRegionIdxX * 4;
    const int blockRegionTop = blockRegionIdxY * 4;
    const int pxYInRegion = threadIdx.y / 4, pxXInRegion = threadIdx.y % 4;
    const int pxY = blockRegionTop + pxYInRegion;
    const int pxX = blockRegionLeft + pxXInRegion;
    const int pxIdx = pxY * imgSizeX + pxX;
    const bool isPxInImg = pxY < imgSizeY && pxX < imgSizeX;
    const uint numModules = numModulesY * numModulesX;
    const int filterPixels = filterSize * filterSize;
    const int imgPixels = imgSizeY * imgSizeX;
    const int tidx = threadIdx.y * 16 + threadIdx.x;
    const int loadY = tidx / 32, loadX = tidx % 32;

    hidActs += blockCaseIdx + (blockFilterIdx + loadY) * numImages * numModules + loadX;
    filters += blockFilterIdx + filterColorIdx * filterPixels * numFilters + threadIdx.x;
    targets += imgColorIdx * imgPixels * numImages + pxIdx * numImages + blockCaseIdx + threadIdx.x;

    float prod[colorsPerThread][imgsPerThread];
    #pragma unroll
    for (int c = 0; c < colorsPerThread; c++) {
        #pragma unroll
        for (int i = 0; i < imgsPerThread; i++) {
            prod[c][i] = 0;
        }
    }
    const int startY = blockRegionTop - paddingStart < filterSize ? 0
                        : 1 + (blockRegionTop - paddingStart - filterSize) / moduleStride;
    const int endY = MIN(numModulesY, 1 + (blockRegionTop + 3 - paddingStart) / moduleStride);
    const int startX = blockRegionLeft - paddingStart < filterSize ? 0
                        : 1 + (blockRegionLeft - paddingStart - filterSize) / moduleStride;
    const int endX = MIN(numModulesX, 1 + (blockRegionLeft + 3 - paddingStart) / moduleStride);

    float* shFilterLoad = &shFilters[threadIdx.y][threadIdx.x];
    float* shHidActLoad = &shHidActs[loadY][loadX];

    for (int my = startY; my < endY; my++) {
        const int moduleTop = paddingStart + my * moduleStride;
        const int pxInModuleY = pxY - moduleTop;

        for (int mx = startX; mx < endX; mx++) {
            const int moduleIdx = my * numModulesX + mx;
            const int moduleLeft = paddingStart + mx * moduleStride;
            const int pxInModuleX = pxX - moduleLeft;

            const bool isPxInModule = pxInModuleY >= 0 && pxInModuleY < filterSize && pxInModuleX >= 0 && pxInModuleX < filterSize;
            const int pxIdxInModule = pxInModuleY * filterSize + pxInModuleX;

            for (int f = 0; f < numFiltersPerGroup; f += 16) { // multipply with 16 filters at a time
                // Now the threads split up into half-warps, and each half-warp decides if it's interested.
                const float* hLoad = &hidActs[(moduleIdx + f * numModules) * numImages];
                #pragma unroll
                for (int i = 0; i < imgsPerThread * 16; i += 32) {
                    if (!checkCaseBounds || blockCaseIdx + loadX + i < numImages) {
                        #pragma unroll
                        for (int j = 0; j < 16; j += 8) { // load 16 rows of imgsPerThread*16 cols, 8 * 32 elements at a time.
                            shHidActLoad[j * 16 * imgsPerThread + i] = hLoad[j * numModules * numImages + i];
                        }
                    } else {
                        #pragma unroll
                        for (int j = 0; j < 16; j += 8) { // load 16 rows of imgsPerThread*16 cols, 8 * 32 elements at a time.
                            shHidActLoad[j * 16 * imgsPerThread + i] = 0;
                        }
                    }
                }

                if (isPxInImg && isPxInModule) {
                    // This half-warp is interested, so it's going to load the weights from this module to its pixel.

                    // Not fully coalesced read :(
                    // But taking out this read entirely only reduces the runtime by ~2.8%, so it isn't costing me much.
                    const float* fLoad = conv ? &filters[pxIdxInModule * numFilters + f]
                                              : &filters[moduleIdx * numFilterColors * filterPixels * numFilters + pxIdxInModule * numFilters + f];
                    #pragma unroll
                    for (int c = 0; c < colorsPerThread; c++) {
                        shFilterLoad[c * 16 * (16 + 1)] = fLoad[c * filterPixels * numFilters];
                    }
                }

                __syncthreads();
                // Do some actual computation
                if (isPxInImg && isPxInModule) {
                    #pragma unroll
                    for (int c = 0; c < colorsPerThread; c++) {
                        #pragma unroll
                        for (int w = 0; w < 16; w++) {
                            #pragma unroll
                            for (int i = 0; i < imgsPerThread; i++) {
                                prod[c][i] += shFilters[threadIdx.y + c * 16][w] * shHidActs[w][threadIdx.x + i * 16];
                            }
                        }
                    }
                }
                __syncthreads();
            }
        }
    }
    // Not fully coalesced write :(... shmem (and fully coalesced) version is actually slightly slower, though
    if (isPxInImg) {
        if (scale) {
            #pragma unroll
            for (int i = 0; i < imgsPerThread; i++) {
                if (!checkCaseBounds || blockCaseIdx + threadIdx.x + i * 16 < numImages) {
                    #pragma unroll
                    for (int c = 0; c < colorsPerThread; c++) {
                        targets[c * imgPixels * numImages + i * 16] = scaleTargets * targets[c * imgPixels * numImages + i * 16] + scaleOutputs * prod[c][i];
                    }
                }
            }
        } else {
            #pragma unroll
            for (int i = 0; i < imgsPerThread; i++) {
                if (!checkCaseBounds || blockCaseIdx + threadIdx.x + i * 16 < numImages) {
                    #pragma unroll
                    for (int c = 0; c < colorsPerThread; c++) {
                        targets[c * imgPixels * numImages + i * 16] = scaleOutputs * prod[c][i];
                    }
                }
            }
        }
    }
}

/*
 * Block size: B_YxB_X.
 * blockIdx.x determines case in batches of B_X*imgsPerThread, also color in batches of B_Y*colorsPerThread.
 *  In essence, blockIdx.x.x = 1..numImages/(B_X*imgsPerThread)
 *              blockIdx.x.y = 1..numImgColors/(B_Y*colorsPerThread)
 * blockIdx.y determines image pixel in target image.
 *
 * threadIdx.x determines case.
 * threadIdx.y determines color.
 *
 * hidActs:     (numFilters, numModulesY, numModulesX, numImages)
 * filters:     (numFilterColors, filterPixels, numFilters)                             if conv
 *              (numModulesY, numModulesX, numFilterColors, filterPixels, numFilters)   otherwise
 * targets:     (numImageColors, imgSizeY, imgSizeX, numImages)
 *
 * Each block reconstructs one B_Y*colorsPerThread colors from 1 pixel from B_X*imgsPerThread cases.
 *
 * numImages must be divisible by B_X*imgsPerThread if checkCaseBounds is false.
 * numFiltersPerGroup must be divisible by filterCache.
 *
 * B_X * imgsPerThread must be divisible by 32.
 * numFilterColors must be divisible by B_Y*colorsPerThread.
 * B_X*B_Y must be divisible by 32.
 * filterCache must be divisible by B_X*B_Y/32
 * B_X*B_Y must be divisible by filterCache

 * This version loads 32 cases at a time, so it gets full coalescing on that load.
 * It only loads filterCache weights at a time, so those aren't fully coalesced (depending on size of filterCache).
 *
 * To be used when there are >= 16 color channels.
 */
template <int B_Y, int B_X, int imgsPerThread, int colorsPerThread, int filterCache, bool scale, bool checkCaseBounds, bool conv>
__global__ void conv_img_acts_manycolor(const float* hidActs, const float* filters, float* targets,
                                          const int numModulesY, const int numModulesX, const int numImages, const int numFilters,
                                          const int filterSize, const int imgSizeY, const int imgSizeX, const int paddingStart, const int moduleStride,
                                          const int numImgColors, const int numGroups,
                                          const float scaleTargets, const float scaleOutputs) {
    __shared__ float shFilters[colorsPerThread*B_Y][filterCache + 1];
    __shared__ float shHidActs[filterCache][B_X*imgsPerThread];

    const int numImgBlocks = DIVUP(numImages,B_X*imgsPerThread);
    const int blockCaseIdx = (blockIdx.x % numImgBlocks) * B_X*imgsPerThread;

    const int imgColorIdx = (blockIdx.x / numImgBlocks) * B_Y*colorsPerThread; // color idx globally
    const int numFilterColors = numImgColors / numGroups;
    const int blockGroupIdx = imgColorIdx / numFilterColors;
    const int filterColorIdx = imgColorIdx % numFilterColors; // color idx within group
    const int numFiltersPerGroup = numFilters / numGroups;
    const int blockFilterIdx = blockGroupIdx * numFiltersPerGroup;

    const int blockPixelIdx = blockIdx.y;
    const int blockPixelIdxX = blockPixelIdx % imgSizeX;
    const int blockPixelIdxY = blockPixelIdx / imgSizeX;

    const int filterPixels = filterSize * filterSize;
    const int imgPixels = imgSizeY * imgSizeX;
    const int tidx = threadIdx.y * B_X + threadIdx.x;
    const int hidActLoadY = tidx / 32, hidActLoadX = tidx % 32;
    const int filtersLoadY = tidx / filterCache, filtersLoadX = tidx % filterCache;
    const int numModules = numModulesY * numModulesX;

    hidActs += blockCaseIdx + (blockFilterIdx + hidActLoadY) * numImages * numModules + hidActLoadX;
    filters += blockFilterIdx + (filterColorIdx + filtersLoadY) * filterPixels * numFilters + filtersLoadX;
    targets += (imgColorIdx + threadIdx.y) * imgPixels * numImages + blockPixelIdx * numImages + blockCaseIdx + threadIdx.x;

    float prod[colorsPerThread][imgsPerThread];
    #pragma unroll
    for (int c = 0; c < colorsPerThread; c++) {
        #pragma unroll
        for (int i = 0; i < imgsPerThread; i++) {
            prod[c][i] = 0;
        }
    }

    const int startY = blockPixelIdxY - paddingStart < filterSize ? 0
                        : 1 + (blockPixelIdxY - paddingStart - filterSize) / moduleStride;
    const int endY = MIN(numModulesY, 1 + (blockPixelIdxY - paddingStart) / moduleStride);
    const int startX = blockPixelIdxX - paddingStart < filterSize ? 0
                        : 1 + (blockPixelIdxX - paddingStart - filterSize) / moduleStride;
    const int endX = MIN(numModulesX, 1 + (blockPixelIdxX - paddingStart) / moduleStride);

    float* shFilterLoad = &shFilters[filtersLoadY][filtersLoadX];
    float* shHidActLoad = &shHidActs[hidActLoadY][hidActLoadX];

    for (int my = startY; my < endY; my++) {
        const int moduleTop = paddingStart + my * moduleStride;
        const int pxInFilterY = blockPixelIdxY - moduleTop;

        for (int mx = startX; mx < endX; mx++) {
            const int moduleIdx = my * numModulesX + mx;
            const int moduleLeft = paddingStart + mx * moduleStride;
            const int pxInFilterX = blockPixelIdxX - moduleLeft;

            const int pxIdxInFilter = pxInFilterY * filterSize + pxInFilterX;

            for (int f = 0; f < numFiltersPerGroup; f += filterCache) { // multiply with filterCache filters at a time
                const float* hLoad = &hidActs[(moduleIdx + f * numModules) * numImages];
                #pragma unroll
                for (int i = 0; i < imgsPerThread * B_X; i += 32) {
                    if (!checkCaseBounds || blockCaseIdx + hidActLoadX + i < numImages) {
                        #pragma unroll
                        for (int j = 0; j < filterCache; j += B_X*B_Y/32) { // load filterCache rows of imgsPerThread*B_X cols, 8 * 32 elements at a time.
                            shHidActLoad[j * B_X * imgsPerThread + i] = hLoad[j * numModules * numImages + i];
                        }
                    } else {
                        #pragma unroll
                        for (int j = 0; j < filterCache; j += B_X*B_Y/32) { // load filterCache rows of imgsPerThread*B_X cols, 8 * 32 elements at a time.
                            shHidActLoad[j * B_X * imgsPerThread + i] = 0;
                        }
                    }
                }
                const float* fLoad = conv ? &filters[pxIdxInFilter * numFilters + f]
                                          : &filters[moduleIdx * numFilterColors * filterPixels * numFilters + pxIdxInFilter * numFilters + f];
                #pragma unroll
                for (int i = 0; i < colorsPerThread*B_Y; i+= B_X*B_Y/filterCache) {
                    if ((colorsPerThread*B_Y) % (B_X*B_Y/filterCache) == 0 || i + filtersLoadY < colorsPerThread*B_Y) {
                        shFilterLoad[i * (filterCache + 1)] = fLoad[i * filterPixels * numFilters];
                    }
                }

                __syncthreads();
                // Do some actual computation
                #pragma unroll
                for (int i = 0; i < imgsPerThread; i++) {
                    #pragma unroll
                    for (int w = 0; w < filterCache; w++) {
                        #pragma unroll
                        for (int c = 0; c < colorsPerThread; c++) {
                            prod[c][i] += shFilters[c * B_Y + threadIdx.y][w] * shHidActs[w][threadIdx.x + i * B_X];
                        }
                    }
                }
                __syncthreads();
            }
        }
    }
    if (scale) {
        #pragma unroll
        for (int i = 0; i < imgsPerThread; i++) {
            if (!checkCaseBounds || blockCaseIdx + threadIdx.x + i * B_X < numImages) {
                #pragma unroll
                for (int c = 0; c < colorsPerThread; c++) {
                    targets[c * B_Y * imgPixels * numImages + i * B_X] = scaleTargets * targets[c * B_Y * imgPixels * numImages + i * B_X] + scaleOutputs * prod[c][i];
                }
            }
        }
    } else {
        #pragma unroll
        for (int i = 0; i < imgsPerThread; i++) {
            if (!checkCaseBounds || blockCaseIdx + threadIdx.x + i * B_X < numImages) {
                #pragma unroll
                for (int c = 0; c < colorsPerThread; c++) {
                    targets[c * B_Y * imgPixels * numImages + i * B_X] = scaleOutputs * prod[c][i];
                }
            }
        }
    }
}


/*
 * Block size: B_YxB_X.
 * blockIdx.x determines case in batches of B_X*imgsPerThread, also color in batches of B_Y*colorsPerThread.
 *  In essence, blockIdx.x.x = 1..numImages/(B_X*imgsPerThread)
 *              blockIdx.x.y = 1..numImgColors/(B_Y*colorsPerThread)
 * blockIdx.y determines image pixel in target image.
 *
 * threadIdx.x determines case.
 * threadIdx.y determines color.
 *
 * hidActs:     (numFilters, numModulesY, numModulesX, numImages)
 * filters:     (numFilterColors, filterPixels, numFilters)                             if conv
 *              (numModulesY, numModulesX, numFilterColors, filterPixels, numFilters)   otherwise
 * targets:     (numImageColors, imgSizeY, imgSizeX, numImages)
 *
 * Each block reconstructs one B_Y*colorsPerThread colors from 1 pixel from B_X*imgsPerThread cases.
 *
 * numImages must be divisible by B_X*imgsPerThread if checkCaseBounds is false.
 * numFiltersPerGroup must be divisible by filterCacheF.
 *
 * numFilterColors must be divisible by B_Y*colorsPerThread.
 * B_X*B_Y must be divisible by filterCacheF
 * filterCacheF must be divisible by filterCacheH
 *
 * This version loads 32 cases at a time, so it gets full coalescing on that load.
 * It only loads filterCacheF weights at a time, so those aren't fully coalesced (depending on size of filterCacheF).
 *
 * To be used when there are >= 16 color channels.
 */
template <int B_Y, int B_X, int imgsPerThread, int colorsPerThread, int filterCacheF, int filterCacheH, bool scale, bool checkCaseBounds, bool conv>
__global__ void conv_img_acts_manycolor_kepler(const float* hidActs, const float* filters, float* targets,
                                          const int numModulesY, const int numModulesX, const int numImages, const int numFilters,
                                          const int filterSize, const int imgSizeY, const int imgSizeX, const int paddingStart, const int moduleStride,
                                          const int numImgColors, const int numGroups,
                                          const float scaleTargets, const float scaleOutputs) {
    __shared__ float shFilters[colorsPerThread*B_Y][filterCacheF];
    __shared__ float shHidActs[filterCacheH][B_X*imgsPerThread];

    const int numImgBlocks = DIVUP(numImages,B_X*imgsPerThread);
    const int blockCaseIdx = (blockIdx.x % numImgBlocks) * B_X*imgsPerThread;

    const int imgColorIdx = (blockIdx.x / numImgBlocks) * B_Y*colorsPerThread; // color idx globally
    const int numFilterColors = numImgColors / numGroups;
    const int blockGroupIdx = imgColorIdx / numFilterColors;
    const int filterColorIdx = imgColorIdx % numFilterColors; // color idx within group
    const int numFiltersPerGroup = numFilters / numGroups;
    const int blockFilterIdx = blockGroupIdx * numFiltersPerGroup;

    const int blockPixelIdx = blockIdx.y;
    const int blockPixelIdxX = blockPixelIdx % imgSizeX;
    const int blockPixelIdxY = blockPixelIdx / imgSizeX;

    const int filterPixels = filterSize * filterSize;
    const int imgPixels = imgSizeY * imgSizeX;
    const int tidx = threadIdx.y * B_X + threadIdx.x;
    const int hidActLoadY = threadIdx.y, hidActLoadX = threadIdx.x;
    //const int hidActLoadY = tidx / (B_X*imgsPerThread), hidActLoadX = tidx % (B_X*imgsPerThread);
    const int filtersLoadY = tidx / filterCacheF, filtersLoadX = tidx % filterCacheF;
    // nvcc is behaving idiotically again, these useless declarations save registers
    //const int outputY = threadIdx.y, outputX = threadIdx.x;
    //const int ty = threadIdx.y, tx = threadIdx.x;
    const int numModules = numModulesY * numModulesX;

    hidActs += blockCaseIdx + (blockFilterIdx + hidActLoadY) * numImages * numModules + hidActLoadX;
    filters += blockFilterIdx + (filterColorIdx + filtersLoadY) * filterPixels * numFilters + filtersLoadX;
    targets += (imgColorIdx + threadIdx.y) * imgPixels * numImages + blockPixelIdx * numImages + blockCaseIdx + threadIdx.x;

    float prod[colorsPerThread][imgsPerThread];
    #pragma unroll
    for (int c = 0; c < colorsPerThread; c++) {
        #pragma unroll
        for (int i = 0; i < imgsPerThread; i++) {
            prod[c][i] = 0;
        }
    }

    const int startY = blockPixelIdxY - paddingStart < filterSize ? 0
                        : 1 + (blockPixelIdxY - paddingStart - filterSize) / moduleStride;
    const int endY = min(numModulesY, 1 + (blockPixelIdxY - paddingStart) / moduleStride);
    const int startX = blockPixelIdxX - paddingStart < filterSize ? 0
                        : 1 + (blockPixelIdxX - paddingStart - filterSize) / moduleStride;
    const int endX = min(numModulesX, 1 + (blockPixelIdxX - paddingStart) / moduleStride);

    float* shFilterLoad = &shFilters[filtersLoadY][filtersLoadX];
    float* shHidActLoad = &shHidActs[hidActLoadY][hidActLoadX];
    //const bool noFLoop = filterCacheF == filterCacheH;
    for (int my = startY; my < endY; my++) {
        const int moduleTop = paddingStart + my * moduleStride;
        const int pxInFilterY = blockPixelIdxY - moduleTop;

        for (int mx = startX; mx < endX; mx++) {
            const int moduleIdx = my * numModulesX + mx;
            const int moduleLeft = paddingStart + mx * moduleStride;
            const int pxInFilterX = blockPixelIdxX - moduleLeft;

            const int pxIdxInFilter = pxInFilterY * filterSize + pxInFilterX;

            for (int f = 0; f < numFiltersPerGroup; f += filterCacheF) { // multiply with filterCacheF filters at a time
                const float* fLoad = conv ? &filters[pxIdxInFilter * numFilters + f]
                                          : &filters[moduleIdx * numFilterColors * filterPixels * numFilters + pxIdxInFilter * numFilters + f];
                #pragma unroll
                for (int i = 0; i < colorsPerThread*B_Y; i+= B_X*B_Y/filterCacheF) {
                    if ((colorsPerThread*B_Y) % (B_X*B_Y/filterCacheF) == 0 || i + filtersLoadY < colorsPerThread*B_Y) {
                        shFilterLoad[i * filterCacheF] = fLoad[i * filterPixels * numFilters];
                    }
                }
                //#pragma unroll

                for (int fh = f; fh < f + filterCacheF; fh += filterCacheH) {
                    //conv_img_acts_manycolor_dummy_fhLoop<B_Y, B_X, imgsPerThread, colorsPerThread, filterCacheF, filterCacheH, checkCaseBounds>(hidActs, shHidActLoad, shHidActs, shFilters, moduleIdx, numImages, hidActLoadY, hidActLoadX, blockCaseIdx, numModules, f, fh, prod);

                    const float* hLoad = &hidActs[(moduleIdx + fh * numModules) * numImages];

                    #pragma unroll
                    for (int j = 0; j < filterCacheH; j += B_Y) {
                        if (filterCacheH % B_Y == 0 || hidActLoadY + j < filterCacheH) {
                            #pragma unroll
                            for (int i = 0; i < imgsPerThread*B_X; i += B_X) {
                                if (!checkCaseBounds || blockCaseIdx + hidActLoadX + i < numImages) {
                                    shHidActLoad[j * B_X * imgsPerThread + i] = hLoad[j * numModules * numImages + i];
                                } else {
                                    shHidActLoad[j * B_X * imgsPerThread + i] = 0;
                                }
                            }
                        }
                    }

                    __syncthreads();

                    // Do some actual computation
                    // Using these variables causes register usage to go from 161 --> 123.
                    // But nonetheless, the high-register version is faster.
                    //const float* shF = &shFilters[threadIdx.y][fh-f];
                    //const float* const shF2 = &shFilters[threadIdx.y][fh];
                    //const float*  shH = &shHidActs[0][threadIdx.x];
                    #pragma unroll
                    for (int w = 0; w < filterCacheH; w++) {
                        #pragma unroll
                        for (int c = 0; c < colorsPerThread; c++) {
                            #pragma unroll
                            for (int i = 0; i < imgsPerThread; i++) {
                                prod[c][i] += shFilters[c * B_Y + threadIdx.y][fh-f + w] * shHidActs[w][threadIdx.x + i * B_X];

                            }
                        }
                    }
                    __syncthreads();

                }
            }
        }
    }
    if (scale) {
        #pragma unroll
        for (int i = 0; i < imgsPerThread; i++) {
            if (!checkCaseBounds || blockCaseIdx + threadIdx.x + i * B_X < numImages) {
                #pragma unroll
                for (int c = 0; c < colorsPerThread; c++) {
                    targets[c * B_Y * imgPixels * numImages + i * B_X] = scaleTargets * targets[c * B_Y * imgPixels * numImages + i * B_X] + scaleOutputs * prod[c][i];
                }
            }
        }
    } else {
        #pragma unroll
        for (int i = 0; i < imgsPerThread; i++) {
            if (!checkCaseBounds || blockCaseIdx + threadIdx.x + i * B_X < numImages) {
                #pragma unroll
                for (int c = 0; c < colorsPerThread; c++) {
                    targets[c * B_Y * imgPixels * numImages + i * B_X] = scaleOutputs * prod[c][i];
                }
            }
        }
    }
}

/*
 * New Titan-optimized stuff.
 */

__device__ __forceinline__ void conv_img_acts_manycolor_preload_ty_8_tx_32_c_8_ff_32_fh_16_setCoords(const int my, const int mx, const int numModulesX,
        const int paddingStart, const int moduleStride, const int blockPixelIdxY, const int blockPixelIdxX, const int filterSize, int &moduleIdx, int &pxIdxInFilter) {
    const int moduleTop = paddingStart + my * moduleStride;
    const int pxInFilterY = blockPixelIdxY - moduleTop;

    moduleIdx = my * numModulesX + mx; // out
    const int moduleLeft = paddingStart + mx * moduleStride;
    const int pxInFilterX = blockPixelIdxX - moduleLeft;

    pxIdxInFilter = pxInFilterY * filterSize + pxInFilterX; // out
}

#define IA_PRELOAD_LOOP(w,offset) _Pragma("unroll") \
for (int i = 0; i < imgsPerThread; i++) { \
    _Pragma("unroll") \
    for (int c = 0; c < colorsPerThread; c++) { \
        prod[c][i] += shFilters[c * B_Y + threadIdx.y][(w)+(offset)] * shHidActs[w][threadIdx.x * imgsPerThread + i]; \
    } \
} \

/*
 * Same loop as above but inverted.
 */
#define IA_PRELOAD_LOOP2(w,offset) _Pragma("unroll") \
for (int c = 0; c < colorsPerThread; c++) { \
    _Pragma("unroll") \
    for (int i = 0; i < imgsPerThread; i++) { \
        prod[c][i] += shFilters[c * B_Y + threadIdx.y][(w)+(offset)] * shHidActs[w][threadIdx.x * imgsPerThread + i]; \
    } \
} \

#define IA_PRELOAD_LOOP3(i,offset) _Pragma("unroll") \
for (int w = 0; w < filterCacheH; w++) { \
    _Pragma("unroll") \
    for (int c = 0; c < colorsPerThread; c++) { \
        prod[c][i] += shFilters[c * B_Y + threadIdx.y][(w)+(offset)] * shHidActs[w][threadIdx.x * imgsPerThread + i]; \
    } \
} \

#define IA_PRELOAD_W(z) wPreload[z] = fLoad[(z) * B_X*B_Y/filterCacheF * filterPixels * numFilters];
#define IA_PRELOAD_W_TX(z) wPreload[z] = tex1Dfetch<float>(filters, filtersLoadOffset + (z) * B_X*B_Y/filterCacheF * filterPixels * numFilters);
#define IA_PRELOAD_H(y,x) if (!checkCaseBounds || myCaseIdx + (x) * B_X < numImages) { \
    hPreload[y][x] =  hLoad[(y) * B_Y * numModules * numImages + (x) * B_X]; \
}
#define IA_PRELOAD_H_TX(y,x) if (!checkCaseBounds || myCaseIdx + (x) * B_X < numImages) { \
    hPreload[y][x] =  tex1Dfetch<float>(hidActs, hidActsLoadOffset + (y) * B_Y * numModules * numImages + (x) * B_X); \
}

template <int B_Y, int B_X, int imgsPerThread, int colorsPerThread, int filterCacheF, int filterCacheH, bool scale, bool checkCaseBounds, bool conv>
__global__ void
__launch_bounds__(256, 2)   // 256 threads per block, 2 blocks per multiprocessor
                            // These launch bounds ensure 25% occupancy (128 registers used)
                            // as oppposed to 13% (130 registers) achieved by defaults.
conv_img_acts_manycolor_preloadfh_ty_8_tx_32_c_8_ff_32_fh_16_tex(hipTextureObject_t hidActs, hipTextureObject_t filters, float* targets,
                                          const int numModulesY, const int numModulesX, const int numImages, const int numFilters,
                                          const int filterSize, const int imgSizeY, const int imgSizeX, const int paddingStart, const int moduleStride,
                                          const int numImgColors, const int numGroups,
                                          const float scaleTargets, const float scaleOutputs) {
    __shared__ float shFilters[colorsPerThread*B_Y][filterCacheF];
    __shared__ float shHidActs[filterCacheH][B_X*imgsPerThread];

    const int numImgBlocks = DIVUP(numImages,B_X*imgsPerThread);
    const int blockCaseIdx = (blockIdx.x % numImgBlocks) * B_X*imgsPerThread;
    const int myCaseIdx = blockCaseIdx + threadIdx.x;

    const int imgColorIdx = (blockIdx.x / numImgBlocks) * B_Y*colorsPerThread; // color idx globally
    const int numFilterColors = numImgColors / numGroups;
    const int blockGroupIdx = imgColorIdx / numFilterColors;
    const int filterColorIdx = imgColorIdx % numFilterColors; // color idx within group
    const int numFiltersPerGroup = numFilters / numGroups;
    const int blockFilterIdx = blockGroupIdx * numFiltersPerGroup;

    const int blockPixelIdx = blockIdx.y;
    const int blockPixelIdxX = blockPixelIdx % imgSizeX;
    const int blockPixelIdxY = blockPixelIdx / imgSizeX;

    const int filterPixels = filterSize * filterSize;
    const int imgPixels = imgSizeY * imgSizeX;
    const int tidx = threadIdx.y * B_X + threadIdx.x;
//    const int hidActLoadY = threadIdx.y % B_Y, hidActLoadX = threadIdx.x % B_X;
    //const int hidActLoadY = tidx / (B_X*imgsPerThread), hidActLoadX = tidx % (B_X*imgsPerThread);
    const int filtersLoadY = tidx / filterCacheF, filtersLoadX = tidx % filterCacheF;
    // nvcc is behaving idiotically again, these useless declarations save registers
    //const int outputY = threadIdx.y, outputX = threadIdx.x;
    //const int ty = threadIdx.y, tx = threadIdx.x;
    const int numModules = numModulesY * numModulesX;
    const int hidActsOffset = (blockFilterIdx + threadIdx.y) * numImages * numModules + myCaseIdx;
    const int filtersOffset = blockFilterIdx + (filterColorIdx + filtersLoadY) * filterPixels * numFilters + filtersLoadX;
//    hidActs += (blockFilterIdx + threadIdx.y) * numImages * numModules + myCaseIdx;
//    filters += blockFilterIdx + (filterColorIdx + filtersLoadY) * filterPixels * numFilters + filtersLoadX;
    targets += (imgColorIdx + threadIdx.y) * imgPixels * numImages + blockPixelIdx * numImages + myCaseIdx;

    float prod[colorsPerThread][imgsPerThread];
    #pragma unroll
    for (int i = 0; i < imgsPerThread; i++) {
        #pragma unroll
        for (int c = 0; c < colorsPerThread; c++) {
            prod[c][i] = 0;
        }
    }

    const int startY = blockPixelIdxY - paddingStart < filterSize ? 0
                        : 1 + (blockPixelIdxY - paddingStart - filterSize) / moduleStride;
    const int endY = min(numModulesY, 1 + (blockPixelIdxY - paddingStart) / moduleStride);
    const int startX = blockPixelIdxX - paddingStart < filterSize ? 0
                        : 1 + (blockPixelIdxX - paddingStart - filterSize) / moduleStride;
    const int endX = min(numModulesX, 1 + (blockPixelIdxX - paddingStart) / moduleStride);

    float* shFilterLoad = &shFilters[filtersLoadY][filtersLoadX];
    float* shHidActLoad = &shHidActs[threadIdx.y][threadIdx.x * imgsPerThread];
    //const bool noFLoop = filterCacheF == filterCacheH;

    /*
     * Initial preload
     */
    float hPreload[filterCacheH/B_Y][imgsPerThread]; // [2][4]
    float wPreload[filterCacheF*colorsPerThread/B_X]; // [8]

    int moduleIdx, pxIdxInFilter;
    conv_img_acts_manycolor_preload_ty_8_tx_32_c_8_ff_32_fh_16_setCoords(startY, startX, numModulesX, paddingStart, moduleStride, blockPixelIdxY,
                                                                         blockPixelIdxX, filterSize, moduleIdx, pxIdxInFilter);
//    const float* fLoad = conv ? &filters[pxIdxInFilter * numFilters + 0]
//                              : &filters[moduleIdx * numFilterColors * filterPixels * numFilters + pxIdxInFilter * numFilters + 0];
    int filtersLoadOffset = filtersOffset + (conv ? pxIdxInFilter * numFilters + 0
                                                  : moduleIdx * numFilterColors * filterPixels * numFilters + pxIdxInFilter * numFilters);
    #pragma unroll
    for (int i = 0; i < colorsPerThread*B_Y; i+= B_X*B_Y/filterCacheF) {
        if ((colorsPerThread*B_Y) % (B_X*B_Y/filterCacheF) == 0 || i + filtersLoadY < colorsPerThread*B_Y) {
            wPreload[i * filterCacheF/(B_X*B_Y)] = tex1Dfetch<float>(filters, filtersLoadOffset + i * filterPixels * numFilters);
        }
    }

//    const float* hLoad = &hidActs[(moduleIdx + 0 * numModules) * numImages];
    int hidActsLoadOffset = hidActsOffset + (moduleIdx + 0 * numModules) * numImages;
    #pragma unroll
    for (int j = 0; j < filterCacheH; j += B_Y) {
        if (filterCacheH % B_Y == 0 || threadIdx.y + j < filterCacheH) {
            #pragma unroll
            for (int i = 0; i < imgsPerThread; i++) {
                if (!checkCaseBounds || myCaseIdx + i * B_X < numImages) {
                    hPreload[j/B_Y][i] = tex1Dfetch<float>(hidActs, hidActsLoadOffset + j * numModules * numImages + i * B_X);
                }
            }
        }
    }

    for (int my = startY; my < endY; my++) {
        const int moduleTop = paddingStart + my * moduleStride;
        const int pxInFilterY = blockPixelIdxY - moduleTop;

        for (int mx = startX; mx < endX; mx++) {
            moduleIdx = my * numModulesX + mx;
            const int moduleLeft = paddingStart + mx * moduleStride;
            const int pxInFilterX = blockPixelIdxX - moduleLeft;

            pxIdxInFilter = pxInFilterY * filterSize + pxInFilterX;
            int myNext = my, mxNext = mx, moduleIdxNext, pxIdxInFilterNext;
            const bool lastModule = my == endY - 1 && mx == endX - 1;
            if (!lastModule) {
                mxNext = mx + 1 == endX ? startX : mx + 1;
                myNext = my + (mx + 1 == endX);
            }
            conv_img_acts_manycolor_preload_ty_8_tx_32_c_8_ff_32_fh_16_setCoords(myNext, mxNext, numModulesX, paddingStart, moduleStride, blockPixelIdxY,
                                                                                 blockPixelIdxX, filterSize, moduleIdxNext, pxIdxInFilterNext);
            for (int f = 0; f < numFiltersPerGroup; f += filterCacheF) { // multiply with filterCacheF filters at a time
                #pragma unroll
                for (int i = 0; i < colorsPerThread*B_Y; i+= B_X*B_Y/filterCacheF) {
                    if ((colorsPerThread*B_Y) % (B_X*B_Y/filterCacheF) == 0 || i + filtersLoadY < colorsPerThread*B_Y) {
                        shFilterLoad[i * filterCacheF] = wPreload[i * filterCacheF/(B_X*B_Y)];
                    }
                }

                filtersLoadOffset = filtersOffset + (conv ? pxIdxInFilter * numFilters + f + filterCacheF
                                                          : moduleIdx * numFilterColors * filterPixels * numFilters + pxIdxInFilter * numFilters + f + filterCacheF);
                if (f == numFiltersPerGroup - filterCacheF) {
                    filtersLoadOffset = filtersOffset + (conv ? pxIdxInFilterNext * numFilters
                                                              : moduleIdxNext * numFilterColors * filterPixels * numFilters + pxIdxInFilterNext * numFilters);
                }

                #pragma unroll
                for (int j = 0; j < filterCacheH; j += B_Y) {
                    if (filterCacheH % B_Y == 0 || threadIdx.y + j < filterCacheH) {
                        #pragma unroll
                        for (int i = 0; i < imgsPerThread; i++) {
                            // NOTE: bank conflicts here!
                            if (!checkCaseBounds || myCaseIdx + i * B_X < numImages) {
                                shHidActLoad[j * B_X * imgsPerThread + i] = hPreload[j/B_Y][i];
                            }
                        }
                    }
                }

                __syncthreads();

                hidActsLoadOffset = hidActsOffset + (moduleIdx + (f + filterCacheH) * numModules) * numImages;

                #pragma unroll
                for (int z = 0; z < 4; ++z) {
                    IA_PRELOAD_LOOP(z,0);
                    IA_PRELOAD_W_TX(z);
                }

                #pragma unroll
                for (int z = 4; z < 12; ++z) {
                    IA_PRELOAD_LOOP(z,0);
                    IA_PRELOAD_H_TX((z-4)/4,z%4);
                }

                #pragma unroll
                for (int z = 12; z < 16; ++z) {
                    IA_PRELOAD_LOOP(z,0);
                }

                __syncthreads();

                #pragma unroll
                for (int j = 0; j < filterCacheH; j += B_Y) {
                    if (filterCacheH % B_Y == 0 || threadIdx.y + j < filterCacheH) {
                        #pragma unroll
                        for (int i = 0; i < imgsPerThread; i++) {
                            if (!checkCaseBounds || myCaseIdx + i * B_X < numImages) {
                                shHidActLoad[j * B_X * imgsPerThread + i] = hPreload[j/B_Y][i];
                            }
                        }
                    }
                }

                __syncthreads();

                hidActsLoadOffset = hidActsOffset + (moduleIdx + (f + filterCacheF) * numModules) * numImages;
                if (f == numFiltersPerGroup - filterCacheF) {
                    hidActsLoadOffset = hidActsOffset + moduleIdxNext * numImages;
                }

                #pragma unroll
                for (int z = 0; z < 4; ++z) {
                    IA_PRELOAD_LOOP(z,filterCacheH);
                    IA_PRELOAD_W_TX(z+4);
                }

                #pragma unroll
                for (int z = 4; z < 12; ++z) {
                    IA_PRELOAD_LOOP(z,filterCacheH);
                    IA_PRELOAD_H_TX((z-4)/4, z%4);
                }

                #pragma unroll
                for (int z = 12; z < 16; ++z) {
                    IA_PRELOAD_LOOP(z,filterCacheH);
                }

                __syncthreads();
            }
        }
    }
    if (scale) {
        #pragma unroll
        for (int c = 0; c < colorsPerThread; c++) {
            #pragma unroll
            for (int i = 0; i < imgsPerThread; i++) {
                if (!checkCaseBounds || myCaseIdx + i * B_X < numImages) {
                    targets[c * B_Y * imgPixels * numImages + i * B_X] = scaleTargets * targets[c * B_Y * imgPixels * numImages + i * B_X] + scaleOutputs * prod[c][i];
                }
            }
        }
    } else {
        #pragma unroll
        for (int c = 0; c < colorsPerThread; c++) {
            #pragma unroll
            for (int i = 0; i < imgsPerThread; i++) {
                if (!checkCaseBounds || myCaseIdx + i * B_X < numImages) {
                    targets[c * B_Y * imgPixels * numImages + i * B_X] = scaleOutputs * prod[c][i];
                }
            }
        }
    }
}


template <int B_Y, int B_X, int imgsPerThread, int colorsPerThread, int filterCacheF, int filterCacheH, bool scale, bool checkCaseBounds, bool conv>
__global__ void
//__launch_bounds__(128, 3)   // 128 threads per block, 3 blocks per multiprocessor
conv_img_acts_manycolor_preloadfh_ty_4_tx_32_c_12_ff_16_fh_16(hipTextureObject_t hidActs, hipTextureObject_t filters, float* targets,
                                          const int numModulesY, const int numModulesX, const int numImages, const int numFilters,
                                          const int filterSize, const int imgSizeY, const int imgSizeX, const int paddingStart, const int moduleStride,
                                          const int numImgColors, const int numGroups,
                                          const float scaleTargets, const float scaleOutputs) {
    __shared__ float shFilters[colorsPerThread*B_Y][filterCacheF];
    __shared__ float shHidActs[filterCacheH][B_X*imgsPerThread];

    const int numImgBlocks = DIVUP(numImages,B_X*imgsPerThread);
    const int blockCaseIdx = (blockIdx.x % numImgBlocks) * B_X*imgsPerThread;
    const int myCaseIdx = blockCaseIdx + threadIdx.x;

    const int imgColorIdx = (blockIdx.x / numImgBlocks) * B_Y*colorsPerThread; // color idx globally
    const int numFilterColors = numImgColors / numGroups;
    const int blockGroupIdx = imgColorIdx / numFilterColors;
    const int filterColorIdx = imgColorIdx % numFilterColors; // color idx within group
    const int numFiltersPerGroup = numFilters / numGroups;
    const int blockFilterIdx = blockGroupIdx * numFiltersPerGroup;

    const int blockPixelIdx = blockIdx.y;
    const int blockPixelIdxX = blockPixelIdx % imgSizeX;
    const int blockPixelIdxY = blockPixelIdx / imgSizeX;

    const int filterPixels = filterSize * filterSize;
    const int imgPixels = imgSizeY * imgSizeX;
    const int tidx = threadIdx.y * B_X + threadIdx.x;
//    const int hidActLoadY = threadIdx.y % B_Y, hidActLoadX = threadIdx.x % B_X;
    //const int hidActLoadY = tidx / (B_X*imgsPerThread), hidActLoadX = tidx % (B_X*imgsPerThread);
    const int filtersLoadY = tidx / filterCacheF, filtersLoadX = tidx % filterCacheF;
    // nvcc is behaving idiotically again, these useless declarations save registers
    //const int outputY = threadIdx.y, outputX = threadIdx.x;
    //const int ty = threadIdx.y, tx = threadIdx.x;
    const int numModules = numModulesY * numModulesX;

    const int hidActsOffset = (blockFilterIdx + threadIdx.y) * numImages * numModules + myCaseIdx;
    const int filtersOffset = blockFilterIdx + (filterColorIdx + filtersLoadY) * filterPixels * numFilters + filtersLoadX;

//    hidActs += (blockFilterIdx + threadIdx.y) * numImages * numModules + myCaseIdx;
//    filters += blockFilterIdx + (filterColorIdx + filtersLoadY) * filterPixels * numFilters + filtersLoadX;
    targets += (imgColorIdx + threadIdx.y) * imgPixels * numImages + blockPixelIdx * numImages + myCaseIdx;

    float prod[colorsPerThread][imgsPerThread];
    #pragma unroll
    for (int i = 0; i < imgsPerThread; i++) {
        #pragma unroll
        for (int c = 0; c < colorsPerThread; c++) {
            prod[c][i] = 0;
        }
    }

    const int startY = blockPixelIdxY - paddingStart < filterSize ? 0
                        : 1 + (blockPixelIdxY - paddingStart - filterSize) / moduleStride;
    const int endY = min(numModulesY, 1 + (blockPixelIdxY - paddingStart) / moduleStride);
    const int startX = blockPixelIdxX - paddingStart < filterSize ? 0
                        : 1 + (blockPixelIdxX - paddingStart - filterSize) / moduleStride;
    const int endX = min(numModulesX, 1 + (blockPixelIdxX - paddingStart) / moduleStride);

    float* shFilterLoad = &shFilters[filtersLoadY][filtersLoadX];
    float* shHidActLoad = &shHidActs[threadIdx.y][threadIdx.x * imgsPerThread];
    //const bool noFLoop = filterCacheF == filterCacheH;

    /*
     * Initial preload
     */
    float hPreload[filterCacheH/B_Y][imgsPerThread]; // [4][4]
    float wPreload[filterCacheF*colorsPerThread/B_X]; // [6]

    int moduleIdx, pxIdxInFilter;
    conv_img_acts_manycolor_preload_ty_8_tx_32_c_8_ff_32_fh_16_setCoords(startY, startX, numModulesX, paddingStart, moduleStride, blockPixelIdxY,
                                                                         blockPixelIdxX, filterSize, moduleIdx, pxIdxInFilter);
//    const float* fLoad = conv ? &filters[pxIdxInFilter * numFilters + 0]
//                              : &filters[moduleIdx * numFilterColors * filterPixels * numFilters + pxIdxInFilter * numFilters + 0];
    int filtersLoadOffset = filtersOffset + (conv ? pxIdxInFilter * numFilters
                                                : moduleIdx * numFilterColors * filterPixels * numFilters + pxIdxInFilter * numFilters);
    #pragma unroll
    for (int i = 0; i < colorsPerThread*B_Y; i+= B_X*B_Y/filterCacheF) {
        if ((colorsPerThread*B_Y) % (B_X*B_Y/filterCacheF) == 0 || i + filtersLoadY < colorsPerThread*B_Y) {
            wPreload[i * filterCacheF/(B_X*B_Y)] = tex1Dfetch<float>(filters, filtersLoadOffset + i * filterPixels * numFilters);
        }
    }

//    const float* hLoad = &hidActs[moduleIdx * numImages];
    int hidActsLoadOffset = hidActsOffset + moduleIdx * numImages;
    #pragma unroll
    for (int j = 0; j < filterCacheH; j += B_Y) {
        if (filterCacheH % B_Y == 0 || threadIdx.y + j < filterCacheH) {
            #pragma unroll
            for (int i = 0; i < imgsPerThread; i++) {
                if (!checkCaseBounds || myCaseIdx + i * B_X < numImages) {
                    hPreload[j/B_Y][i] = tex1Dfetch<float>(hidActs, hidActsLoadOffset + j * numModules * numImages + i * B_X);
                }
            }
        }
    }

    for (int my = startY; my < endY; my++) {
        const int moduleTop = paddingStart + my * moduleStride;
        const int pxInFilterY = blockPixelIdxY - moduleTop;

        for (int mx = startX; mx < endX; mx++) {
            moduleIdx = my * numModulesX + mx;
            const int moduleLeft = paddingStart + mx * moduleStride;
            const int pxInFilterX = blockPixelIdxX - moduleLeft;

            pxIdxInFilter = pxInFilterY * filterSize + pxInFilterX;
            int myNext = my, mxNext = mx, moduleIdxNext, pxIdxInFilterNext;
            const bool lastModule = my == endY - 1 && mx == endX - 1;
            if (!lastModule) {
                mxNext = mx + 1 == endX ? startX : mx + 1;
                myNext = my + (mx + 1 == endX);
            }
            conv_img_acts_manycolor_preload_ty_8_tx_32_c_8_ff_32_fh_16_setCoords(myNext, mxNext, numModulesX, paddingStart, moduleStride, blockPixelIdxY,
                                                                                 blockPixelIdxX, filterSize, moduleIdxNext, pxIdxInFilterNext);
            for (int f = 0; f < numFiltersPerGroup; f += filterCacheF) { // multiply with filterCacheF filters at a time
                #pragma unroll
                for (int i = 0; i < colorsPerThread*B_Y; i+= B_X*B_Y/filterCacheF) {
                    if ((colorsPerThread*B_Y) % (B_X*B_Y/filterCacheF) == 0 || i + filtersLoadY < colorsPerThread*B_Y) {
                        shFilterLoad[i * filterCacheF] = wPreload[i * filterCacheF/(B_X*B_Y)];
                    }
                }

                filtersLoadOffset = filtersOffset + (conv ? pxIdxInFilter * numFilters + f + filterCacheF
                                                          : moduleIdx * numFilterColors * filterPixels * numFilters + pxIdxInFilter * numFilters + f + filterCacheF);
                if (f == numFiltersPerGroup - filterCacheF) {
                    filtersLoadOffset = filtersOffset + (conv ? pxIdxInFilterNext * numFilters
                                                              : moduleIdxNext * numFilterColors * filterPixels * numFilters + pxIdxInFilterNext * numFilters);
                }

                #pragma unroll
                for (int j = 0; j < filterCacheH; j += B_Y) {
                    if (filterCacheH % B_Y == 0 || threadIdx.y + j < filterCacheH) {
                        #pragma unroll
                        for (int i = 0; i < imgsPerThread; i++) {
                            // NOTE: bank conflicts here!
                            if (!checkCaseBounds || myCaseIdx + i * B_X < numImages) {
                                shHidActLoad[j * B_X * imgsPerThread + i] = hPreload[j/B_Y][i];
                            }
                        }
                    }
                }
                hidActsLoadOffset = hidActsOffset + (moduleIdx + (f + filterCacheF) * numModules) * numImages;
                if (f == numFiltersPerGroup - filterCacheF) {
                    hidActsLoadOffset = hidActsOffset + moduleIdxNext * numImages;
                }

                __syncthreads();

                // It seems that there is no point explicitly interleaving loads
                // and computations because the scheduler does that anyway.

                IA_PRELOAD_LOOP2(0,0);
                IA_PRELOAD_LOOP2(1,0);
                IA_PRELOAD_LOOP2(2,0);
                IA_PRELOAD_LOOP2(3,0);
                IA_PRELOAD_LOOP2(4,0);
                IA_PRELOAD_LOOP2(5,0);
                IA_PRELOAD_LOOP2(6,0);
                IA_PRELOAD_LOOP2(7,0);
                IA_PRELOAD_LOOP2(8,0);
                IA_PRELOAD_LOOP2(9,0);
                IA_PRELOAD_LOOP2(10,0);
                IA_PRELOAD_LOOP2(11,0);
                IA_PRELOAD_LOOP2(12,0);
                IA_PRELOAD_LOOP2(13,0);
                IA_PRELOAD_LOOP2(14,0);
                IA_PRELOAD_LOOP2(15,0);

                IA_PRELOAD_W_TX(0);
                IA_PRELOAD_W_TX(1);
                IA_PRELOAD_W_TX(2);
                IA_PRELOAD_W_TX(3);
                IA_PRELOAD_W_TX(4);
                IA_PRELOAD_W_TX(5);

                IA_PRELOAD_H_TX(0,0);
                IA_PRELOAD_H_TX(0,1);
                IA_PRELOAD_H_TX(0,2);
                IA_PRELOAD_H_TX(0,3);
                IA_PRELOAD_H_TX(1,0);
                IA_PRELOAD_H_TX(1,1);
                IA_PRELOAD_H_TX(1,2);
                IA_PRELOAD_H_TX(1,3);
                IA_PRELOAD_H_TX(2,0);
                IA_PRELOAD_H_TX(2,1);
                IA_PRELOAD_H_TX(2,2);
                IA_PRELOAD_H_TX(2,3);
                IA_PRELOAD_H_TX(3,0);
                IA_PRELOAD_H_TX(3,1);
                IA_PRELOAD_H_TX(3,2);
                IA_PRELOAD_H_TX(3,3);

                __syncthreads();
            }
        }
    }
    if (scale) {
        #pragma unroll
        for (int c = 0; c < colorsPerThread; c++) {
            #pragma unroll
            for (int i = 0; i < imgsPerThread; i++) {
                if (!checkCaseBounds || myCaseIdx + i * B_X < numImages) {
                    targets[c * B_Y * imgPixels * numImages + i * B_X] = scaleTargets * targets[c * B_Y * imgPixels * numImages + i * B_X] + scaleOutputs * prod[c][i];
                }
            }
        }
    } else {
        #pragma unroll
        for (int c = 0; c < colorsPerThread; c++) {
            #pragma unroll
            for (int i = 0; i < imgsPerThread; i++) {
                if (!checkCaseBounds || myCaseIdx + i * B_X < numImages) {
                    targets[c * B_Y * imgPixels * numImages + i * B_X] = scaleOutputs * prod[c][i];
                }
            }
        }
    }
}

/*
 * hidActs:         (numFilters, numModules, numImages)
 * filters:         (numFilterColors, filterPixels, numFilters)               if conv
 *                  (numModules, numFilterColors, filterPixels, numFilters)   otherwise
 * targets:         (overSample, numImgColors, imgPixels, numImages)
 *
 * Note: all of these convolution routines are optimized for the case when
 * the number of images (i.e. the minibatch size) is a multiple of 128.
 * Other batch sizes will work, but but I made no attempt whatsoever
 * to make them work fast.
 */
void _imgActs(caffe2::HIPContext* context, caffe2::TensorCUDA* hidActs, caffe2::TensorCUDA* filters, caffe2::TensorCUDA* targets,
              int imgSizeY, int imgSizeX, int numModulesY, int paddingStart, int moduleStride, int numImgColors, int numGroups,
              float scaleTargets, float scaleOutput, bool conv) {
    CAFFE_ENFORCE(hidActs->ndim() == 2);
    CAFFE_ENFORCE(filters->ndim() == 2);
    CAFFE_ENFORCE(targets->ndim() == 2);

    int numFilterColors = numImgColors / numGroups;
    int numImages = hidActs->dim32(1);
    int numFilters = filters->dim32(1);
    int numModules = hidActs->dim32(0) / numFilters;
    int filterModuleMult = conv ? 1 : numModules;
    int filterPixels = filters->dim32(0) / (filterModuleMult * numFilterColors);
    int filterSize = sqrt(filterPixels);
    int imgPixels = imgSizeY * imgSizeX;
    int numModulesX = numModules / numModulesY;

    CAFFE_ENFORCE(numImgColors % numGroups == 0);
    CAFFE_ENFORCE(numFilters % (16*numGroups) == 0); // TODO: insisting on 32 filters due to bug in calling code below. fix that.
    CAFFE_ENFORCE(numGroups > 1 || (numImgColors > 0 && (numImgColors <= 3 || numImgColors % 2 == 0)));
    CAFFE_ENFORCE(numGroups == 1 || numFilterColors % 4 == 0);

    CAFFE_ENFORCE(filterPixels == filterSize * filterSize);
    CAFFE_ENFORCE(hidActs->dim32(0) == numModules * numFilters);
    CAFFE_ENFORCE(filters->dim32(0) == filterModuleMult * numFilterColors * filterPixels);
    CAFFE_ENFORCE(numModules == numModulesY * numModulesX);

    // These routines don't handle the case when only part of the image is visited in the convolution
    CAFFE_ENFORCE(paddingStart <= 0);
    CAFFE_ENFORCE(paddingStart + (numModulesX-1)*moduleStride + filterSize >= imgSizeX);
    CAFFE_ENFORCE(paddingStart + (numModulesY-1)*moduleStride + filterSize >= imgSizeY);
    CAFFE_ENFORCE(moduleStride <= filterSize);

    dim3 blocks;
    dim3 threads;
    int colorsPerThread, imgsPerThread;
    if (numFilterColors % 8 == 0) {
        threads = dim3(32, numFilterColors % 64 == 0 ? 8 : 4);
        colorsPerThread = numFilterColors % 64 == 0 ? 8
                        : numFilterColors % 48 == 0 ? 12
                        : numFilterColors % 32 == 0 ? 8
                        : numFilterColors % 16 == 0 ? 4
                        : 2;
        imgsPerThread = numImages % 128 == 0 ? 4 : numImages % 64 == 0 ? 2 : 1;
        CAFFE_ENFORCE(numFilterColors % (threads.y * colorsPerThread) == 0);

        blocks = dim3(DIVUP(numImages, threads.x*imgsPerThread) * (numImgColors/(threads.y*colorsPerThread)), imgPixels);
        // NOTE: the case when channels % 32 == 0 but channels % 48 != 0 and channels % 64 != 0 has not been optimized!!
    } else if (numFilterColors > 3) {
        // NOTE: THIS CASE HAS NOT BEEN OPTIMIZED FOR KEPLER!!
        imgsPerThread = numImages % 128 == 0 ? 8 : numImages % 64 == 0 ? 4 : 2;
        threads = dim3(16, 16);
        colorsPerThread = numFilterColors % 4 == 0 ? 4 : 2;
        blocks = dim3(DIVUP(numImages,threads.x*imgsPerThread) * (numImgColors / colorsPerThread), DIVUP(imgSizeY,4) * DIVUP(imgSizeX,4));
    } else {
        // NOTE: THIS CASE HAS NOT BEEN OPTIMIZED FOR KEPLER!!
        imgsPerThread = numImages % 128 == 0 ? 8 : numImages % 64 == 0 ? 4 : 2;
        threads = dim3(16, 16);
        blocks = dim3(DIVUP(numImages,threads.x*imgsPerThread), DIVUP(imgSizeY,4) * DIVUP(imgSizeX,4));
    }
    bool checkCaseBounds = numImages % (threads.x * imgsPerThread) != 0;

    if (scaleTargets == 0) { // do not scale or use targets matrix
        targets->Resize(std::vector<int>{numImgColors*imgPixels, numImages});
    } else {
        CAFFE_ENFORCE(targets->dim32(0) == numImgColors * imgPixels);
        CAFFE_ENFORCE(targets->dim32(1) == numImages);
    }
    const bool scale = scaleTargets != 0;

    hipTextureObject_t tex_hidacts = GetTensorTextureObject(hidActs);
    hipTextureObject_t tex_filters = GetTensorTextureObject(filters);
    float* hidacts_data = hidActs->mutable_data<float>();
    float* filters_data = filters->mutable_data<float>();
    float* targets_data = targets->mutable_data<float>();

    hipStream_t stream = context->hip_stream();
//    hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_preloadfh_ty_4_tx_32_c_12_ff_16_fh_16< 4), 32, 4, 12, 16, 16, false, false, true >, hipFuncCachePreferShared);
//    conv_img_acts_manycolor_preloadfh_ty_4_tx_32_c_12_ff_16_fh_16< 4, 32, 4, 12, 16, 16, false, false, true ><<<blocks, threads, 0, stream>>>(
//            tex_hidacts, tex_filters, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize,
//            imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);

    //return;
//    printf("conv: %d\n", conv);
//    printf("scale: %d\n", scale);
//    printf("checkCaseBounds: %d\n", checkCaseBounds);
//    printf("numFilterColors: %d\n", numFilterColors);
//    printf("numImages: %d\n", numImages);
//    hipStream_t stream = NVMatrix::getDefaultStream();

    if (conv == true) {
        if (scale == false) {
            if (checkCaseBounds == false) {
                if (numFilterColors % 8 == 0) {
                    if (numFilterColors % 64 == 0) {
                        if (numFilters % 32 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_preloadfh_ty_8_tx_32_c_8_ff_32_fh_16_tex< 8), 32, 4, 8, 32, 16, false, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_preloadfh_ty_8_tx_32_c_8_ff_32_fh_16_tex< 8, 32, 4, 8, 32, 16, false, false, true ><<<blocks, threads, 0, stream>>>(tex_hidacts, tex_filters, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 2, 8, 32, 16, false, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 8, 32, 2, 8, 32, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 1, 8, 32, 16, false, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 8, 32, 1, 8, 32, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 1, 8, 32, 16, false, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 8, 32, 1, 8, 32, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                        else if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 4, 8, 16, 16, false, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 8, 32, 4, 8, 16, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 2, 8, 16, 16, false, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 8, 32, 2, 8, 16, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 1, 8, 16, 16, false, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 8, 32, 1, 8, 16, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 1, 8, 16, 16, false, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 8, 32, 1, 8, 16, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors % 48 == 0) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_preloadfh_ty_4_tx_32_c_12_ff_16_fh_16< 4), 32, 4, 12, 16, 16, false, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_preloadfh_ty_4_tx_32_c_12_ff_16_fh_16< 4, 32, 4, 12, 16, 16, false, false, true ><<<blocks, threads, 0, stream>>>(tex_hidacts, tex_filters, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 2, 12, 16, 16, false, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 2, 12, 16, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 12, 16, 16, false, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 12, 16, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 12, 16, 16, false, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 12, 16, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors % 32 == 0) {
                        if (numFilters % 32 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 4, 8, 32, 16, false, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 4, 8, 32, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 2, 8, 32, 16, false, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 2, 8, 32, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 8, 32, 16, false, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 8, 32, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 8, 32, 16, false, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 8, 32, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                        else if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 4, 8, 16, 16, false, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 4, 8, 16, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 2, 8, 16, 16, false, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 2, 8, 16, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 8, 16, 16, false, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 8, 16, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 8, 16, 16, false, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 8, 16, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors % 16 == 0) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 4, 4, 16, 16, false, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 4, 4, 16, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 2, 4, 16, 16, false, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 2, 4, 16, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 4, 16, 16, false, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 4, 16, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 4, 16, 16, false, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 4, 16, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors % 8 == 0) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 4, 2, 16, 16, false, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 4, 2, 16, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 2, 2, 16, 16, false, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 2, 2, 16, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 2, 16, 16, false, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 2, 16, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 2, 16, 16, false, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 2, 16, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                }
                else if (numFilterColors > 3) {
                    if (numFilterColors == 4) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor < 8), 4, false, false, true >, hipFuncCachePreferShared);
                                img_acts_mediumcolor < 8, 4, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor < 4), 4, false, false, true >, hipFuncCachePreferShared);
                                img_acts_mediumcolor < 4, 4, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor < 2), 4, false, false, true >, hipFuncCachePreferShared);
                                img_acts_mediumcolor < 2, 4, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor < 2), 4, false, false, true >, hipFuncCachePreferShared);
                                img_acts_mediumcolor < 2, 4, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors == 2) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 8), 2, false, false, true >, hipFuncCachePreferShared);
                                img_acts_color < 8, 2, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 4), 2, false, false, true >, hipFuncCachePreferShared);
                                img_acts_color < 4, 2, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 2, false, false, true >, hipFuncCachePreferShared);
                                img_acts_color < 2, 2, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 2, false, false, true >, hipFuncCachePreferShared);
                                img_acts_color < 2, 2, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                        }
                    }
                }
                else if (numFilterColors <= 3) {
                    if (numFilterColors == 3) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 8), 3, false, false, true >, hipFuncCachePreferShared);
                                img_acts_color < 8, 3, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 4), 3, false, false, true >, hipFuncCachePreferShared);
                                img_acts_color < 4, 3, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 3, false, false, true >, hipFuncCachePreferShared);
                                img_acts_color < 2, 3, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 3, false, false, true >, hipFuncCachePreferShared);
                                img_acts_color < 2, 3, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors == 2) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 8), 2, false, false, true >, hipFuncCachePreferShared);
                                img_acts_color < 8, 2, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 4), 2, false, false, true >, hipFuncCachePreferShared);
                                img_acts_color < 4, 2, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 2, false, false, true >, hipFuncCachePreferShared);
                                img_acts_color < 2, 2, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 2, false, false, true >, hipFuncCachePreferShared);
                                img_acts_color < 2, 2, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors == 1) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 8), 1, false, false, true >, hipFuncCachePreferShared);
                                img_acts_color < 8, 1, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 4), 1, false, false, true >, hipFuncCachePreferShared);
                                img_acts_color < 4, 1, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 1, false, false, true >, hipFuncCachePreferShared);
                                img_acts_color < 2, 1, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 1, false, false, true >, hipFuncCachePreferShared);
                                img_acts_color < 2, 1, false, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                        }
                    }
                }
            }
            else if (checkCaseBounds == true) {
                if (numFilterColors % 8 == 0) {
                    if (numFilterColors % 64 == 0) {
                        if (numFilters % 32 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 1, 8, 32, 16, false, true, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 8, 32, 1, 8, 32, 16, false, true, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                        else if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 1, 8, 16, 16, false, true, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 8, 32, 1, 8, 16, 16, false, true, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors % 48 == 0) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 12, 16, 16, false, true, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 12, 16, 16, false, true, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors % 32 == 0) {
                        if (numFilters % 32 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 8, 32, 16, false, true, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 8, 32, 16, false, true, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                        else if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 8, 16, 16, false, true, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 8, 16, 16, false, true, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors % 16 == 0) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 4, 16, 16, false, true, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 4, 16, 16, false, true, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors % 8 == 0) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 2, 16, 16, false, true, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 2, 16, 16, false, true, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                }
                else if (numFilterColors > 3) {
                    if (numFilterColors == 4) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor < 2), 4, false, true, true >, hipFuncCachePreferShared);
                                img_acts_mediumcolor < 2, 4, false, true, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors == 2) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 2, false, true, true >, hipFuncCachePreferShared);
                                img_acts_color < 2, 2, false, true, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                        }
                    }
                }
                else if (numFilterColors <= 3) {
                    if (numFilterColors == 3) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 3, false, true, true >, hipFuncCachePreferShared);
                                img_acts_color < 2, 3, false, true, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors == 2) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 2, false, true, true >, hipFuncCachePreferShared);
                                img_acts_color < 2, 2, false, true, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors == 1) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 1, false, true, true >, hipFuncCachePreferShared);
                                img_acts_color < 2, 1, false, true, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                        }
                    }
                }
            }
        }
        else if (scale == true) {
            if (checkCaseBounds == false) {
                if (numFilterColors % 8 == 0) {
                    if (numFilterColors % 64 == 0) {
                        if (numFilters % 32 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_preloadfh_ty_8_tx_32_c_8_ff_32_fh_16_tex< 8), 32, 4, 8, 32, 16, true, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_preloadfh_ty_8_tx_32_c_8_ff_32_fh_16_tex< 8, 32, 4, 8, 32, 16, true, false, true ><<<blocks, threads, 0, stream>>>(tex_hidacts, tex_filters, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 2, 8, 32, 16, true, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 8, 32, 2, 8, 32, 16, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 1, 8, 32, 16, true, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 8, 32, 1, 8, 32, 16, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 1, 8, 32, 16, true, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 8, 32, 1, 8, 32, 16, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                        else if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 4, 8, 16, 16, true, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 8, 32, 4, 8, 16, 16, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 2, 8, 16, 16, true, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 8, 32, 2, 8, 16, 16, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 1, 8, 16, 16, true, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 8, 32, 1, 8, 16, 16, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 1, 8, 16, 16, true, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 8, 32, 1, 8, 16, 16, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors % 48 == 0) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_preloadfh_ty_4_tx_32_c_12_ff_16_fh_16< 4), 32, 4, 12, 16, 16, true, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_preloadfh_ty_4_tx_32_c_12_ff_16_fh_16< 4, 32, 4, 12, 16, 16, true, false, true ><<<blocks, threads, 0, stream>>>(tex_hidacts, tex_filters, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 2, 12, 16, 16, true, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 2, 12, 16, 16, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 12, 16, 16, true, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 12, 16, 16, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 12, 16, 16, true, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 12, 16, 16, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors % 32 == 0) {
                        if (numFilters % 32 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 4, 8, 32, 16, true, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 4, 8, 32, 16, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 2, 8, 32, 16, true, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 2, 8, 32, 16, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 8, 32, 16, true, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 8, 32, 16, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 8, 32, 16, true, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 8, 32, 16, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                        else if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 4, 8, 16, 16, true, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 4, 8, 16, 16, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 2, 8, 16, 16, true, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 2, 8, 16, 16, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 8, 16, 16, true, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 8, 16, 16, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 8, 16, 16, true, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 8, 16, 16, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors % 16 == 0) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 4, 4, 16, 16, true, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 4, 4, 16, 16, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 2, 4, 16, 16, true, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 2, 4, 16, 16, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 4, 16, 16, true, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 4, 16, 16, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 4, 16, 16, true, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 4, 16, 16, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors % 8 == 0) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 4, 2, 16, 16, true, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 4, 2, 16, 16, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 2, 2, 16, 16, true, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 2, 2, 16, 16, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 2, 16, 16, true, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 2, 16, 16, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 2, 16, 16, true, false, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 2, 16, 16, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                }
                else if (numFilterColors > 3) {
                    if (numFilterColors == 4) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor < 8), 4, true, false, true >, hipFuncCachePreferShared);
                                img_acts_mediumcolor < 8, 4, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor < 4), 4, true, false, true >, hipFuncCachePreferShared);
                                img_acts_mediumcolor < 4, 4, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor < 2), 4, true, false, true >, hipFuncCachePreferShared);
                                img_acts_mediumcolor < 2, 4, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor < 2), 4, true, false, true >, hipFuncCachePreferShared);
                                img_acts_mediumcolor < 2, 4, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors == 2) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 8), 2, true, false, true >, hipFuncCachePreferShared);
                                img_acts_color < 8, 2, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 4), 2, true, false, true >, hipFuncCachePreferShared);
                                img_acts_color < 4, 2, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 2, true, false, true >, hipFuncCachePreferShared);
                                img_acts_color < 2, 2, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 2, true, false, true >, hipFuncCachePreferShared);
                                img_acts_color < 2, 2, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                        }
                    }
                }
                else if (numFilterColors <= 3) {
                    if (numFilterColors == 3) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 8), 3, true, false, true >, hipFuncCachePreferShared);
                                img_acts_color < 8, 3, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 4), 3, true, false, true >, hipFuncCachePreferShared);
                                img_acts_color < 4, 3, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 3, true, false, true >, hipFuncCachePreferShared);
                                img_acts_color < 2, 3, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 3, true, false, true >, hipFuncCachePreferShared);
                                img_acts_color < 2, 3, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors == 2) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 8), 2, true, false, true >, hipFuncCachePreferShared);
                                img_acts_color < 8, 2, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 4), 2, true, false, true >, hipFuncCachePreferShared);
                                img_acts_color < 4, 2, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 2, true, false, true >, hipFuncCachePreferShared);
                                img_acts_color < 2, 2, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 2, true, false, true >, hipFuncCachePreferShared);
                                img_acts_color < 2, 2, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors == 1) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 8), 1, true, false, true >, hipFuncCachePreferShared);
                                img_acts_color < 8, 1, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 4), 1, true, false, true >, hipFuncCachePreferShared);
                                img_acts_color < 4, 1, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 1, true, false, true >, hipFuncCachePreferShared);
                                img_acts_color < 2, 1, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 1, true, false, true >, hipFuncCachePreferShared);
                                img_acts_color < 2, 1, true, false, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                        }
                    }
                }
            }
            else if (checkCaseBounds == true) {
                if (numFilterColors % 8 == 0) {
                    if (numFilterColors % 64 == 0) {
                        if (numFilters % 32 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 1, 8, 32, 16, true, true, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 8, 32, 1, 8, 32, 16, true, true, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                        else if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 1, 8, 16, 16, true, true, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 8, 32, 1, 8, 16, 16, true, true, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors % 48 == 0) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 12, 16, 16, true, true, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 12, 16, 16, true, true, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors % 32 == 0) {
                        if (numFilters % 32 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 8, 32, 16, true, true, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 8, 32, 16, true, true, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                        else if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 8, 16, 16, true, true, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 8, 16, 16, true, true, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors % 16 == 0) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 4, 16, 16, true, true, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 4, 16, 16, true, true, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors % 8 == 0) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 2, 16, 16, true, true, true >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 2, 16, 16, true, true, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                }
                else if (numFilterColors > 3) {
                    if (numFilterColors == 4) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor < 2), 4, true, true, true >, hipFuncCachePreferShared);
                                img_acts_mediumcolor < 2, 4, true, true, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors == 2) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 2, true, true, true >, hipFuncCachePreferShared);
                                img_acts_color < 2, 2, true, true, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                        }
                    }
                }
                else if (numFilterColors <= 3) {
                    if (numFilterColors == 3) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 3, true, true, true >, hipFuncCachePreferShared);
                                img_acts_color < 2, 3, true, true, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors == 2) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 2, true, true, true >, hipFuncCachePreferShared);
                                img_acts_color < 2, 2, true, true, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors == 1) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 1, true, true, true >, hipFuncCachePreferShared);
                                img_acts_color < 2, 1, true, true, true ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                        }
                    }
                }
            }
        }
    }
    else if (conv == false) {
        if (scale == false) {
            if (checkCaseBounds == false) {
                if (numFilterColors % 8 == 0) {
                    if (numFilterColors % 64 == 0) {
                        if (numFilters % 32 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_preloadfh_ty_8_tx_32_c_8_ff_32_fh_16_tex< 8), 32, 4, 8, 32, 16, false, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_preloadfh_ty_8_tx_32_c_8_ff_32_fh_16_tex< 8, 32, 4, 8, 32, 16, false, false, false ><<<blocks, threads, 0, stream>>>(tex_hidacts, tex_filters, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 2, 8, 32, 16, false, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 8, 32, 2, 8, 32, 16, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 1, 8, 32, 16, false, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 8, 32, 1, 8, 32, 16, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 1, 8, 32, 16, false, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 8, 32, 1, 8, 32, 16, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                        else if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 4, 8, 16, 16, false, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 8, 32, 4, 8, 16, 16, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 2, 8, 16, 16, false, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 8, 32, 2, 8, 16, 16, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 1, 8, 16, 16, false, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 8, 32, 1, 8, 16, 16, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 1, 8, 16, 16, false, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 8, 32, 1, 8, 16, 16, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors % 48 == 0) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_preloadfh_ty_4_tx_32_c_12_ff_16_fh_16< 4), 32, 4, 12, 16, 16, false, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_preloadfh_ty_4_tx_32_c_12_ff_16_fh_16< 4, 32, 4, 12, 16, 16, false, false, false ><<<blocks, threads, 0, stream>>>(tex_hidacts, tex_filters, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 2, 12, 16, 16, false, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 2, 12, 16, 16, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 12, 16, 16, false, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 12, 16, 16, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 12, 16, 16, false, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 12, 16, 16, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors % 32 == 0) {
                        if (numFilters % 32 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 4, 8, 32, 16, false, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 4, 8, 32, 16, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 2, 8, 32, 16, false, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 2, 8, 32, 16, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 8, 32, 16, false, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 8, 32, 16, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 8, 32, 16, false, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 8, 32, 16, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                        else if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 4, 8, 16, 16, false, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 4, 8, 16, 16, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 2, 8, 16, 16, false, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 2, 8, 16, 16, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 8, 16, 16, false, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 8, 16, 16, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 8, 16, 16, false, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 8, 16, 16, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors % 16 == 0) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 4, 4, 16, 16, false, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 4, 4, 16, 16, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 2, 4, 16, 16, false, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 2, 4, 16, 16, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 4, 16, 16, false, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 4, 16, 16, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 4, 16, 16, false, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 4, 16, 16, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors % 8 == 0) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 4, 2, 16, 16, false, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 4, 2, 16, 16, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 2, 2, 16, 16, false, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 2, 2, 16, 16, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 2, 16, 16, false, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 2, 16, 16, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 2, 16, 16, false, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 2, 16, 16, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                }
                else if (numFilterColors > 3) {
                    if (numFilterColors == 4) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor < 8), 4, false, false, false >, hipFuncCachePreferShared);
                                img_acts_mediumcolor < 8, 4, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor < 4), 4, false, false, false >, hipFuncCachePreferShared);
                                img_acts_mediumcolor < 4, 4, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor < 2), 4, false, false, false >, hipFuncCachePreferShared);
                                img_acts_mediumcolor < 2, 4, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor < 2), 4, false, false, false >, hipFuncCachePreferShared);
                                img_acts_mediumcolor < 2, 4, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors == 2) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 8), 2, false, false, false >, hipFuncCachePreferShared);
                                img_acts_color < 8, 2, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 4), 2, false, false, false >, hipFuncCachePreferShared);
                                img_acts_color < 4, 2, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 2, false, false, false >, hipFuncCachePreferShared);
                                img_acts_color < 2, 2, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 2, false, false, false >, hipFuncCachePreferShared);
                                img_acts_color < 2, 2, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                        }
                    }
                }
                else if (numFilterColors <= 3) {
                    if (numFilterColors == 3) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 8), 3, false, false, false >, hipFuncCachePreferShared);
                                img_acts_color < 8, 3, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 4), 3, false, false, false >, hipFuncCachePreferShared);
                                img_acts_color < 4, 3, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 3, false, false, false >, hipFuncCachePreferShared);
                                img_acts_color < 2, 3, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 3, false, false, false >, hipFuncCachePreferShared);
                                img_acts_color < 2, 3, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors == 2) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 8), 2, false, false, false >, hipFuncCachePreferShared);
                                img_acts_color < 8, 2, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 4), 2, false, false, false >, hipFuncCachePreferShared);
                                img_acts_color < 4, 2, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 2, false, false, false >, hipFuncCachePreferShared);
                                img_acts_color < 2, 2, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 2, false, false, false >, hipFuncCachePreferShared);
                                img_acts_color < 2, 2, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors == 1) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 8), 1, false, false, false >, hipFuncCachePreferShared);
                                img_acts_color < 8, 1, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 4), 1, false, false, false >, hipFuncCachePreferShared);
                                img_acts_color < 4, 1, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 1, false, false, false >, hipFuncCachePreferShared);
                                img_acts_color < 2, 1, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 1, false, false, false >, hipFuncCachePreferShared);
                                img_acts_color < 2, 1, false, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                        }
                    }
                }
            }
            else if (checkCaseBounds == true) {
                if (numFilterColors % 8 == 0) {
                    if (numFilterColors % 64 == 0) {
                        if (numFilters % 32 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 1, 8, 32, 16, false, true, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 8, 32, 1, 8, 32, 16, false, true, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                        else if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 1, 8, 16, 16, false, true, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 8, 32, 1, 8, 16, 16, false, true, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors % 48 == 0) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 12, 16, 16, false, true, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 12, 16, 16, false, true, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors % 32 == 0) {
                        if (numFilters % 32 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 8, 32, 16, false, true, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 8, 32, 16, false, true, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                        else if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 8, 16, 16, false, true, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 8, 16, 16, false, true, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors % 16 == 0) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 4, 16, 16, false, true, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 4, 16, 16, false, true, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors % 8 == 0) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 2, 16, 16, false, true, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 2, 16, 16, false, true, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                }
                else if (numFilterColors > 3) {
                    if (numFilterColors == 4) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor < 2), 4, false, true, false >, hipFuncCachePreferShared);
                                img_acts_mediumcolor < 2, 4, false, true, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors == 2) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 2, false, true, false >, hipFuncCachePreferShared);
                                img_acts_color < 2, 2, false, true, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                        }
                    }
                }
                else if (numFilterColors <= 3) {
                    if (numFilterColors == 3) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 3, false, true, false >, hipFuncCachePreferShared);
                                img_acts_color < 2, 3, false, true, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors == 2) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 2, false, true, false >, hipFuncCachePreferShared);
                                img_acts_color < 2, 2, false, true, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors == 1) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 1, false, true, false >, hipFuncCachePreferShared);
                                img_acts_color < 2, 1, false, true, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                        }
                    }
                }
            }
        }
        else if (scale == true) {
            if (checkCaseBounds == false) {
                if (numFilterColors % 8 == 0) {
                    if (numFilterColors % 64 == 0) {
                        if (numFilters % 32 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_preloadfh_ty_8_tx_32_c_8_ff_32_fh_16_tex< 8), 32, 4, 8, 32, 16, true, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_preloadfh_ty_8_tx_32_c_8_ff_32_fh_16_tex< 8, 32, 4, 8, 32, 16, true, false, false ><<<blocks, threads, 0, stream>>>(tex_hidacts, tex_filters, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 2, 8, 32, 16, true, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 8, 32, 2, 8, 32, 16, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 1, 8, 32, 16, true, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 8, 32, 1, 8, 32, 16, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 1, 8, 32, 16, true, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 8, 32, 1, 8, 32, 16, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                        else if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 4, 8, 16, 16, true, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 8, 32, 4, 8, 16, 16, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 2, 8, 16, 16, true, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 8, 32, 2, 8, 16, 16, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 1, 8, 16, 16, true, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 8, 32, 1, 8, 16, 16, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 1, 8, 16, 16, true, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 8, 32, 1, 8, 16, 16, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors % 48 == 0) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_preloadfh_ty_4_tx_32_c_12_ff_16_fh_16< 4), 32, 4, 12, 16, 16, true, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_preloadfh_ty_4_tx_32_c_12_ff_16_fh_16< 4, 32, 4, 12, 16, 16, true, false, false ><<<blocks, threads, 0, stream>>>(tex_hidacts, tex_filters, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 2, 12, 16, 16, true, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 2, 12, 16, 16, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 12, 16, 16, true, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 12, 16, 16, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 12, 16, 16, true, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 12, 16, 16, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors % 32 == 0) {
                        if (numFilters % 32 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 4, 8, 32, 16, true, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 4, 8, 32, 16, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 2, 8, 32, 16, true, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 2, 8, 32, 16, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 8, 32, 16, true, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 8, 32, 16, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 8, 32, 16, true, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 8, 32, 16, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                        else if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 4, 8, 16, 16, true, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 4, 8, 16, 16, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 2, 8, 16, 16, true, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 2, 8, 16, 16, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 8, 16, 16, true, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 8, 16, 16, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 8, 16, 16, true, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 8, 16, 16, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors % 16 == 0) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 4, 4, 16, 16, true, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 4, 4, 16, 16, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 2, 4, 16, 16, true, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 2, 4, 16, 16, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 4, 16, 16, true, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 4, 16, 16, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 4, 16, 16, true, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 4, 16, 16, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors % 8 == 0) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 4, 2, 16, 16, true, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 4, 2, 16, 16, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 2, 2, 16, 16, true, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 2, 2, 16, 16, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 2, 16, 16, true, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 2, 16, 16, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 2, 16, 16, true, false, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 2, 16, 16, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                }
                else if (numFilterColors > 3) {
                    if (numFilterColors == 4) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor < 8), 4, true, false, false >, hipFuncCachePreferShared);
                                img_acts_mediumcolor < 8, 4, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor < 4), 4, true, false, false >, hipFuncCachePreferShared);
                                img_acts_mediumcolor < 4, 4, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor < 2), 4, true, false, false >, hipFuncCachePreferShared);
                                img_acts_mediumcolor < 2, 4, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor < 2), 4, true, false, false >, hipFuncCachePreferShared);
                                img_acts_mediumcolor < 2, 4, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors == 2) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 8), 2, true, false, false >, hipFuncCachePreferShared);
                                img_acts_color < 8, 2, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 4), 2, true, false, false >, hipFuncCachePreferShared);
                                img_acts_color < 4, 2, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 2, true, false, false >, hipFuncCachePreferShared);
                                img_acts_color < 2, 2, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 2, true, false, false >, hipFuncCachePreferShared);
                                img_acts_color < 2, 2, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                        }
                    }
                }
                else if (numFilterColors <= 3) {
                    if (numFilterColors == 3) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 8), 3, true, false, false >, hipFuncCachePreferShared);
                                img_acts_color < 8, 3, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 4), 3, true, false, false >, hipFuncCachePreferShared);
                                img_acts_color < 4, 3, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 3, true, false, false >, hipFuncCachePreferShared);
                                img_acts_color < 2, 3, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 3, true, false, false >, hipFuncCachePreferShared);
                                img_acts_color < 2, 3, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors == 2) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 8), 2, true, false, false >, hipFuncCachePreferShared);
                                img_acts_color < 8, 2, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 4), 2, true, false, false >, hipFuncCachePreferShared);
                                img_acts_color < 4, 2, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 2, true, false, false >, hipFuncCachePreferShared);
                                img_acts_color < 2, 2, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 2, true, false, false >, hipFuncCachePreferShared);
                                img_acts_color < 2, 2, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors == 1) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 128 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 8), 1, true, false, false >, hipFuncCachePreferShared);
                                img_acts_color < 8, 1, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 64 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 4), 1, true, false, false >, hipFuncCachePreferShared);
                                img_acts_color < 4, 1, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 32 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 1, true, false, false >, hipFuncCachePreferShared);
                                img_acts_color < 2, 1, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                            else if (numImages % 16 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 1, true, false, false >, hipFuncCachePreferShared);
                                img_acts_color < 2, 1, true, false, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                        }
                    }
                }
            }
            else if (checkCaseBounds == true) {
                if (numFilterColors % 8 == 0) {
                    if (numFilterColors % 64 == 0) {
                        if (numFilters % 32 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 1, 8, 32, 16, true, true, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 8, 32, 1, 8, 32, 16, true, true, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                        else if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 1, 8, 16, 16, true, true, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 8, 32, 1, 8, 16, 16, true, true, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors % 48 == 0) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 12, 16, 16, true, true, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 12, 16, 16, true, true, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors % 32 == 0) {
                        if (numFilters % 32 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 8, 32, 16, true, true, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 8, 32, 16, true, true, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                        else if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 8, 16, 16, true, true, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 8, 16, 16, true, true, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors % 16 == 0) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 4, 16, 16, true, true, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 4, 16, 16, true, true, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors % 8 == 0) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 2, 16, 16, true, true, false >, hipFuncCachePreferShared);
                                conv_img_acts_manycolor_kepler < 4, 32, 1, 2, 16, 16, true, true, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                }
                else if (numFilterColors > 3) {
                    if (numFilterColors == 4) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor < 2), 4, true, true, false >, hipFuncCachePreferShared);
                                img_acts_mediumcolor < 2, 4, true, true, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors == 2) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 2, true, true, false >, hipFuncCachePreferShared);
                                img_acts_color < 2, 2, true, true, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                        }
                    }
                }
                else if (numFilterColors <= 3) {
                    if (numFilterColors == 3) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 3, true, true, false >, hipFuncCachePreferShared);
                                img_acts_color < 2, 3, true, true, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors == 2) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 2, true, true, false >, hipFuncCachePreferShared);
                                img_acts_color < 2, 2, true, true, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                        }
                    }
                    else if (numFilterColors == 1) {
                        if (numFilters % 16 == 0) {
                            if (numImages % 1 == 0) {
                                hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 1, true, true, false >, hipFuncCachePreferShared);
                                img_acts_color < 2, 1, true, true, false ><<<blocks, threads, 0, stream>>>(hidacts_data, filters_data, targets_data, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                            }
                        }
                    }
                }
            }
        }
    }

    checkCudaErrors(hipDestroyTextureObject(tex_hidacts));
    checkCudaErrors(hipDestroyTextureObject(tex_filters));

    getLastCudaError("imgActs: kernel execution failed");
}


void convImgActs(caffe2::HIPContext* context, caffe2::TensorCUDA* hidActs, caffe2::TensorCUDA* filters, caffe2::TensorCUDA* targets,
                 int imgSizeY, int imgSizeX, int numModulesY, int paddingStart, int moduleStride, int numImgColors, int numGroups) {
    _imgActs(context, hidActs, filters, targets, imgSizeY, imgSizeX, numModulesY, paddingStart, moduleStride, numImgColors, numGroups, 0, 1, true);
}

void convImgActs(caffe2::HIPContext* context, caffe2::TensorCUDA* hidActs, caffe2::TensorCUDA* filters, caffe2::TensorCUDA* targets,
                 int imgSizeY, int imgSizeX, int numModulesY, int paddingStart, int moduleStride, int numImgColors, int numGroups,
                 float scaleTargets, float scaleOutput) {
    _imgActs(context, hidActs, filters, targets, imgSizeY, imgSizeX, numModulesY, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput, true);
}

void localImgActs(caffe2::HIPContext* context, caffe2::TensorCUDA* hidActs, caffe2::TensorCUDA* filters, caffe2::TensorCUDA* targets,
                  int imgSizeY, int imgSizeX, int numModulesY, int paddingStart, int moduleStride, int numImgColors, int numGroups) {
    _imgActs(context, hidActs, filters, targets, imgSizeY, imgSizeX, numModulesY, paddingStart, moduleStride, numImgColors, numGroups, 0, 1, false);
}

void localImgActs(caffe2::HIPContext* context, caffe2::TensorCUDA* hidActs, caffe2::TensorCUDA* filters, caffe2::TensorCUDA* targets,
                  int imgSizeY, int imgSizeX, int numModulesY, int paddingStart, int moduleStride, int numImgColors, int numGroups,
                  float scaleTargets, float scaleOutput) {
    _imgActs(context, hidActs, filters, targets, imgSizeY, imgSizeX, numModulesY, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput, false);
}

