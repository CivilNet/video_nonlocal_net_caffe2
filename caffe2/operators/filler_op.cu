#include "hip/hip_runtime.h"
#include <cmath>
#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/filler_op.h"

namespace caffe2 {

namespace {
__global__ void FillRangeKernel(const int n, float* data) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    data[index] = index;
  }
}

template <typename T>
__global__ void FillDiagonalKernel(
    const int num_diagonal_elements,
    const TIndex step_size,
    const T value,
    T* data) {
  CUDA_1D_KERNEL_LOOP(index, num_diagonal_elements) {
    data[index * step_size] = value;
  }
}
}

template <>
bool RangeFillOp<float, HIPContext>::Fill(TensorCUDA* output) {
  int N = output->size();
  FillRangeKernel<<<
      CAFFE_GET_BLOCKS(N),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(N, output->mutable_data<float>());
  return true;
}

template <>
template <typename T>
bool DiagonalFillOp<HIPContext>::FillWithType(TensorCUDA* output) {
  VerifyOutputShape(output);
  auto* data = output->template mutable_data<T>();
  int size = output->size();
  // first fill everything with 0
  math::Set<T, HIPContext>(size, T(0), data, &context_);

  T value = OperatorBase::GetSingleArgument<T>("value", 0);
  TIndex step_size = GetStepSize(output);
  int num_diagonal_elements = ceil((float)size / step_size);

  FillDiagonalKernel<<<
      CAFFE_GET_BLOCKS(num_diagonal_elements),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(num_diagonal_elements, step_size, value, data);
  return true;
}

REGISTER_HIP_OPERATOR(UniformFill, UniformFillOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(UniformIntFill, UniformFillOp<int, HIPContext>);
REGISTER_HIP_OPERATOR(ConstantFill, ConstantFillOp<HIPContext>);
REGISTER_HIP_OPERATOR(DiagonalFill, DiagonalFillOp<HIPContext>);
REGISTER_HIP_OPERATOR(GaussianFill, GaussianFillOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(XavierFill, XavierFillOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(MSRAFill, MSRAFillOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(RangeFill, RangeFillOp<float, HIPContext>);

} // namespace caffe2
