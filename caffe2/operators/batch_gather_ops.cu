#include "hip/hip_runtime.h"
#include <fstream>
#include "caffe2/core/common_gpu.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/batch_gather_ops.h"

namespace caffe2 {

template <typename T_INDEX, typename TData>
__global__ void BatchGatherKernel(
    const TData* src_base,
    TData* out,
    const T_INDEX* indices,
    const int M,
    const int N,
    const int data_batch_size,
    const int gathered_batch_size,
    const int block_size) {
  const int begin_idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int num_items = M * N * block_size;
  for (int s = begin_idx; s < num_items; s += blockDim.x * gridDim.x) {
    const int k = s % block_size;
    const int j = s / block_size % N;
    const int i = s / block_size / N;
    const T_INDEX idx = indices[j];
    const float* src_offset = src_base + i * data_batch_size + idx * block_size;
    float* dst_offset = out + i * gathered_batch_size + j * block_size;
    dst_offset[k] = src_offset[k];
  }
}

template <>
bool BatchGatherOp<HIPContext>::RunOnDevice() {
  return DispatchHelper<TensorTypes<int32_t, int64_t>>::call(
      this, OperatorBase::Input<TensorCUDA>(INDICES));
}

template <>
template <typename TInd>
bool BatchGatherOp<HIPContext>::DoRunWithType() {
  auto& data = Input(DATA);
  auto& indices = Input(INDICES);
  auto* output = Output(0);

  vector<TIndex> shape;
  shape.push_back(data.dim(0));
  shape.insert(shape.end(), indices.dims().begin(), indices.dims().end());
  shape.insert(shape.end(), data.dims().begin() + 2, data.dims().end());
  output->Resize(shape);

  const int block_size = data.size_from_dim(2);
  const int N = indices.size();
  const auto data_batch_size = data.size_from_dim(1);
  const auto gathered_batch_size = N * data.size_from_dim(2);
  const TInd* idxs = indices.template data<TInd>();
  auto src_base = static_cast<const float*>(data.raw_data());
  auto out = static_cast<float*>(output->raw_mutable_data(data.meta()));
  const int M = data.dim32(0);

  BatchGatherKernel<<<
      std::min(M, CAFFE_MAXIMUM_NUM_BLOCKS),
      std::min(N * block_size, CAFFE_CUDA_NUM_THREADS),
      0,
      context_.hip_stream()>>>(
      src_base,
      out,
      idxs,
      M,
      N,
      data_batch_size,
      gathered_batch_size,
      block_size);
  return true;
}

template <typename T_INDEX, typename TData>
__global__ void BatchGatherGradientKernel(
    const TData* grad_data,
    TData* out,
    const T_INDEX* indices,
    const int M,
    const int N,
    const int data_batch_size,
    const int gathered_batch_size,
    const int block_size) {
  int begin_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int num_items = M * N * block_size;
  for (int s = begin_idx; s < num_items; s += blockDim.x * gridDim.x) {
    const int k = s % block_size;
    const int j = s / block_size % N;
    const int i = s / block_size / N;
    const T_INDEX idx = indices[j];
    const float* src_offset =
        grad_data + i * gathered_batch_size + j * block_size;
    float* dst_offset = out + i * data_batch_size + idx * block_size;
    atomicAdd(dst_offset + k, src_offset[k]);
  }
}

template <>
bool BatchGatherGradientOp<HIPContext>::RunOnDevice() {
  return DispatchHelper<TensorTypes<int32_t, int64_t>>::call(
      this, OperatorBase::Input<TensorCUDA>(INDICES));
}

template <>
template <typename TInd>
bool BatchGatherGradientOp<HIPContext>::DoRunWithType() {
  return DispatchHelper<
      TensorTypes2<float, GenericTensorImplementation>,
      TInd>::call(this, OperatorBase::Input<TensorCUDA>(DATA));
}

template <>
template <typename TInd, typename TData>
bool BatchGatherGradientOp<HIPContext>::DoRunWithType2() {
  auto& data = Input(DATA);
  auto& indices = Input(INDICES);
  auto& grad = Input(GRAD);
  auto* output = Output(0);

  CAFFE_ENFORCE_EQ(data.dim(0), grad.dim(0), "batch sizes should be the same");

  output->ResizeLike(data);
  auto* out_data = output->template mutable_data<float>();
  math::Set<float, HIPContext>(output->size(), 0, out_data, &context_);

  const auto* grad_data = grad.template data<float>();

  const int M = grad.dim32(0);
  const int block_size = data.size_from_dim(2);
  const int N = indices.size();
  const auto data_batch_size = data.size_from_dim(1);
  const auto gathered_batch_size = N * data.size_from_dim(2);
  const TInd* idxs = indices.template data<TInd>();

  BatchGatherGradientKernel<<<
      std::min(M, CAFFE_MAXIMUM_NUM_BLOCKS),
      std::min(N * block_size, CAFFE_CUDA_NUM_THREADS),
      0,
      context_.hip_stream()>>>(
      grad_data,
      out_data,
      idxs,
      M,
      N,
      data_batch_size,
      gathered_batch_size,
      block_size);

  return true;
}

template <>
template <typename TInd>
bool BatchGatherGradientOp<HIPContext>::DoRunWithOtherType2() {
  CAFFE_THROW(
      "BatchGatherGradient is not implemented on tensor of type ",
      Input(DATA).meta().name(),
      "Consider adding it a type in the list DispatchHelper or implementing "
      "a generic version (which won't work for duplicated indices though)");
}

REGISTER_HIP_OPERATOR(BatchGather, BatchGatherOp<HIPContext>);
REGISTER_HIP_OPERATOR(BatchGatherGradient, BatchGatherGradientOp<HIPContext>);

} // namespace caffe2
