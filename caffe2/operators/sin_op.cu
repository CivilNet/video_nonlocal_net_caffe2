#include "hip/hip_runtime.h"
#include <cmath>

#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/elementwise_op.h"

namespace caffe2 {

template <typename T>
__global__ void SinKernel(const int N, const T* X, T* Y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    Y[i] = sin(X[i]);
  }
}

template <typename T>
__global__ void SinGradientKernel(const int N, const T* X, const T* dY, T* dX) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    dX[i] = dY[i] * cos(X[i]);
  }
}

struct SinCUDAFunctor {
  template <typename T>
  inline void
  operator()(const int n, const T* x, T* y, HIPContext* device_context) {
    SinKernel<T>
        <<<CAFFE_GET_BLOCKS(n),
           CAFFE_CUDA_NUM_THREADS,
           0,
           device_context->hip_stream()>>>(n, x, y);
    return;
  }
};

struct SinGradientCUDAFunctor {
  template <typename T>
  inline void Run(
      const int n,
      const T* x,
      const T* dy,
      T* dx,
      HIPContext* device_context) {
    SinGradientKernel<T>
        <<<CAFFE_GET_BLOCKS(n),
           CAFFE_CUDA_NUM_THREADS,
           0,
           device_context->hip_stream()>>>(n, x, dy, dx);
    return;
  }
};

REGISTER_HIP_OPERATOR(
    Sin,
    UnaryElementwiseOp<TensorTypes<float>, HIPContext, SinCUDAFunctor>);
REGISTER_HIP_OPERATOR(
    SinGradient,
    BinaryElementwiseOp<
        TensorTypes<float>,
        HIPContext,
        WithoutBroadcast<SinGradientCUDAFunctor>>);
} // namespace caffe2
