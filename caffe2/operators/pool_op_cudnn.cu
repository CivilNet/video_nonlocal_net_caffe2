#include "hip/hip_runtime.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/core/cudnn_wrappers.h"
#include "caffe2/operators/conv_pool_op_base.h"

#include <hipcub/hipcub.hpp>

namespace caffe2 {

namespace {

// Explicit fast paths for avg and max global pooling due to CuDNN global
// pooling performance bug which makes pooling extremely slow.
template <typename T>
__global__ void
global_avgpool_kernel_NCHW(const int NC, const int sz, const T* data, T* out) {
  typedef hipcub::BlockReduce<T, CAFFE_CUDA_NUM_THREADS> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  for (int j = blockIdx.x; j < NC; j += gridDim.x) {
    T sum(0);
    for (int k = threadIdx.x; k < sz; k += blockDim.x) {
      sum += data[j * sz + k];
    }
    float totalsum = BlockReduce(temp_storage).Sum(sum);
    if (threadIdx.x == 0) {
      out[j] = totalsum / sz;
    }
    __syncthreads();
  }
}

template <typename T>
__global__ void
global_avgpool_backward_NCHW(const int NC, const int sz, const T* dx, T* out) {
  CUDA_1D_KERNEL_LOOP(i, NC * sz) {
    out[i] = dx[i / sz] / sz;
  }
}

template <typename T>
__global__ void
global_maxpool_kernel_NCHW(const int NC, const int sz, const T* data, T* out) {
  typedef hipcub::BlockReduce<T, CAFFE_CUDA_NUM_THREADS> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  for (int j = blockIdx.x; j < NC; j += gridDim.x) {
    T max(-FLT_MAX);
    for (int k = threadIdx.x; k < sz; k += blockDim.x) {
      max = data[j * sz + k] > max ? data[j * sz + k] : max;
    }
    float totalmax = BlockReduce(temp_storage).Reduce(max, hipcub::Max());
    if (threadIdx.x == 0) {
      out[j] = totalmax;
    }
    __syncthreads();
  }
}

template <typename T>
__global__ void global_maxpool_backward_NCHW(
    const int NC,
    const int sz,
    const T* dx,
    T* out,
    const T* x,
    const T* in) {
  CUDA_1D_KERNEL_LOOP(i, NC * sz) {
    if (in[i] == x[i / sz]) {
      out[i] = dx[i / sz];
    } else {
      out[i] = 0.0;
    }
  }
}

template <typename T>
void setTensorDescriptor(
    const int size,
    const StorageOrder order,
    const int N,
    const int C,
    const int H,
    const int W,
    const int D,
    hipdnnTensorDescriptor_t& desc) {
  if (size == 4) {
    CUDNN_ENFORCE(hipdnnSetTensor4dDescriptor(
        desc,
        GetCudnnTensorFormat(order),
        cudnnTypeWrapper<T>::type,
        N,
        C,
        H,
        W));
  } else {
    vector<int> dims = {N, C, H, W, D};
    vector<int> strides;
    order == NCHW
        ? strides.insert(strides.end(), {C * H * W * D, H * W * D, W * D, D, 1})
        : strides.insert(
              strides.end(), {H * W * D * C, 1, W * D * C, D * C, C});
    CUDNN_ENFORCE(hipdnnSetTensorNdDescriptor(
        desc,
        cudnnTypeWrapper<T>::type,
        size > 3 ? size : 4,
        dims.data(),
        strides.data()));
  }
}

} // namespace

class CuDNNPoolOp : public ConvPoolOpBase<HIPContext> {
 public:
  CuDNNPoolOp(const OperatorDef& operator_def, Workspace* ws)
      : ConvPoolOpBase<HIPContext>(operator_def, ws),
        cudnn_wrapper_(&context_) {
    CUDNN_ENFORCE(hipdnnCreateTensorDescriptor(&bottom_desc_));
    CUDNN_ENFORCE(hipdnnCreateTensorDescriptor(&top_desc_));
    CUDNN_ENFORCE(hipdnnCreatePoolingDescriptor(&pooling_desc_));
    OPERATOR_NEEDS_FEATURE(kernel_.size() >=2 && kernel_.size() <=3,
        "Cudnn pooling only supports 4d and 5d tensor");
    if (legacy_pad_ != LegacyPadding::CAFFE_LEGACY_POOLING) {
      for (int i = 0; i < kernel_.size(); ++i) {
        OPERATOR_NEEDS_FEATURE(
            pads_[i] == pads_[kernel_.size() + i],
            "The current padding scheme leads to unequal padding on the left "
            "and right, which is not supported by cudnn.");
      }
    }
    // Figure out the pooling descriptor.
    if (operator_def.type().substr(0, 7) == "MaxPool") {
      bool deterministic =
          OperatorBase::GetSingleArgument<bool>("deterministic", false);
#if CUDNN_VERSION_MIN(6, 0, 0)
      mode_ =
          deterministic ? HIPDNN_POOLING_MAX_DETERMINISTIC : HIPDNN_POOLING_MAX;
#else
      mode_ = HIPDNN_POOLING_MAX;
#endif
    } else if (operator_def.type().substr(0, 11) == "AveragePool") {
      mode_ = HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING;
    } else {
      LOG(FATAL) << "Unsupported pooling method: " << operator_def.type();
    }
  }

  ~CuDNNPoolOp() {
    CUDNN_ENFORCE(hipdnnDestroyTensorDescriptor(bottom_desc_));
    CUDNN_ENFORCE(hipdnnDestroyTensorDescriptor(top_desc_));
    CUDNN_ENFORCE(hipdnnDestroyPoolingDescriptor(pooling_desc_));
  }

  template <typename T, typename M>
  bool DoRunWithType() {
    auto& X = Input(0);
    auto* Y = Output(0);
    int N = 0, C = 0, H = 0, W = 0, D = 0;
    int H_out = 0, W_out = 0, D_out = 0;

    // cuDNN pooling support only 2 and 3 spatial dimensions.
    CAFFE_ENFORCE(X.ndim() >= 4 && X.ndim() <= 5);

    switch (order_) {
      case StorageOrder::NHWC:
        N = X.dim32(0);
        H = X.dim32(1);
        W = X.ndim() > 3 ? X.dim32(2) : 1;
        D = X.ndim() > 4 ? X.dim32(3) : 1;
        C = X.dim32(X.ndim() - 1);
        ConvPoolOpBase::SetOutputSize(X, Y, C);
        H_out = Y->dim32(1);
        W_out = Y->ndim() > 3 ? Y->dim32(2) : 1;
        D_out = Y->ndim() > 4 ? Y->dim32(3) : 1;
        break;
      case StorageOrder::NCHW:
        N = X.dim32(0);
        C = X.dim32(1);
        H = X.dim32(2);
        W = X.ndim() > 3 ? X.dim32(3) : 1;
        D = X.ndim() > 4 ? X.dim32(4) : 1;
        ConvPoolOpBase::SetOutputSize(X, Y, C);
        H_out = Y->dim32(2);
        W_out = Y->ndim() > 3 ? Y->dim32(3) : 1;
        D_out = Y->ndim() > 4 ? Y->dim32(4) : 1;
        break;
      default:
        LOG(FATAL) << "Unknown storage order: " << order_;
    }

    // Fast path for global pooling, as cudnn is slow. But only
    // on float, because fp16 not supported for CUB.
    if (std::is_same<T, float>::value) {
      if (order_ == StorageOrder::NCHW && global_pooling_) {
        if (mode_ == HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING) {
          global_avgpool_kernel_NCHW<float>
              <<<std::min(N * C, CAFFE_MAXIMUM_NUM_BLOCKS),
                 CAFFE_CUDA_NUM_THREADS,
                 0,
                 context_.hip_stream()>>>(
                  N * C, H * W * D, X.data<float>(), Y->mutable_data<float>());
          return true;
        }
        if (mode_ == HIPDNN_POOLING_MAX) {
          global_maxpool_kernel_NCHW<float>
              <<<std::min(N * C, CAFFE_MAXIMUM_NUM_BLOCKS),
                 CAFFE_CUDA_NUM_THREADS,
                 0,
                 context_.hip_stream()>>>(
                  N * C, H * W * D, X.data<float>(), Y->mutable_data<float>());
          return true;
        }
      }
    }

    if (cudnn_input_dims_ != X.dims()) {
      // Dimensions changed; we will need to re-initialize things.
      VLOG(1) << "Changing the cudnn descriptor configurations.";
      cudnn_input_dims_ = X.dims();
      setTensorDescriptor<T>(X.ndim(), order_, N, C, H, W, D, bottom_desc_);
      setTensorDescriptor<T>(
          Y->ndim(), order_, N, C, H_out, W_out, D_out, top_desc_);
      for (int i = 0; i < kernel_.size(); ++i) {
        if (pads_[i] != pads_[kernel_.size() + i]) {
          CAFFE_ENFORCE(
              legacy_pad_ == LegacyPadding::CAFFE_LEGACY_POOLING,
              "Cudnn pooling only supports even padding on both sides, with "
              "the only exception of the caffe legacy pooling case where we "
              "try to preserve backward compatibility with Caffe.");
        }
      }
      if (kernel_.size() == 2) {
        CUDNN_ENFORCE(hipdnnSetPooling2dDescriptor(
            pooling_desc_,
            mode_,
            HIPDNN_NOT_PROPAGATE_NAN,
            kernel_h(),
            kernel_w(),
            pad_t(),
            pad_l(),
            stride_h(),
            stride_w()));
      } else {
        CUDNN_ENFORCE(hipdnnSetPoolingNdDescriptor(
            pooling_desc_,
            mode_,
            HIPDNN_NOT_PROPAGATE_NAN,
            kernel_.size(),
            kernel_.data(),
            pads_.data(),
            stride_.data()));
      }
    }
    // Carry out the pooling computation.
    const T* Xdata = X.template data<T>();
    T* Ydata = Y->template mutable_data<T>();
    CUDNN_ENFORCE(hipdnnPoolingForward(
        cudnn_wrapper_.inline_cudnn_handle(),
        pooling_desc_,
        cudnnTypeWrapper<T>::kOne(),
        bottom_desc_,
        Xdata,
        cudnnTypeWrapper<T>::kZero(),
        top_desc_,
        Ydata));
    return true;
  }

  bool RunOnDevice() final {
    auto& X = Input(0);
    auto* Y = Output(0);

    if (X.IsType<float>()) {
      return DoRunWithType<float, float>();
    } else if (X.IsType<float16>()) {
      return DoRunWithType<float16, float>();
    } else {
      LOG(FATAL) << "Unsupported input types";
    }
    return true;
  }

 protected:
  vector<TIndex> cudnn_input_dims_;

  CuDNNWrapper cudnn_wrapper_;
  hipdnnTensorDescriptor_t bottom_desc_;
  hipdnnTensorDescriptor_t top_desc_;
  hipdnnPoolingDescriptor_t pooling_desc_;
  hipdnnPoolingMode_t mode_;

 private:
};

class CuDNNPoolGradientOp : public ConvPoolOpBase<HIPContext> {
 public:
  CuDNNPoolGradientOp(const OperatorDef& operator_def, Workspace* ws)
      : ConvPoolOpBase<HIPContext>(operator_def, ws),
        cudnn_wrapper_(&context_) {
    CUDNN_ENFORCE(hipdnnCreateTensorDescriptor(&bottom_desc_));
    CUDNN_ENFORCE(hipdnnCreateTensorDescriptor(&top_desc_));
    CUDNN_ENFORCE(hipdnnCreatePoolingDescriptor(&pooling_desc_));
    // Figure out the pooling descriptor.
    if (operator_def.type() == "MaxPoolGradient" ||
        operator_def.type() == "MaxPool1DGradient" ||
        operator_def.type() == "MaxPool2DGradient" ||
        operator_def.type() == "MaxPool3DGradient") {
      bool deterministic =
          OperatorBase::GetSingleArgument<bool>("deterministic", false);
#if CUDNN_VERSION_MIN(6, 0, 0)
      mode_ =
          deterministic ? HIPDNN_POOLING_MAX_DETERMINISTIC : HIPDNN_POOLING_MAX;
#else
      mode_ = HIPDNN_POOLING_MAX;
#endif
    } else if (
        operator_def.type() == "AveragePoolGradient" ||
        operator_def.type() == "AveragePool1DGradient" ||
        operator_def.type() == "AveragePool2DGradient" ||
        operator_def.type() == "AveragePool3DGradient") {
      mode_ = HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING;
    } else {
      LOG(FATAL) << "Unsupported pooling method: " << operator_def.type();
    }
  }

  ~CuDNNPoolGradientOp() {
    CUDNN_ENFORCE(hipdnnDestroyTensorDescriptor(bottom_desc_));
    CUDNN_ENFORCE(hipdnnDestroyTensorDescriptor(top_desc_));
    CUDNN_ENFORCE(hipdnnDestroyPoolingDescriptor(pooling_desc_));
  }

  template <typename T, typename M>
  bool DoRunWithType() {
    auto& X = Input(0);
    auto& Y = Input(1);
    auto& dY = Input(2);
    auto* dX = Output(0);

    // cuDNN pooling support only 2 and 3 spatial dimensions.
    CAFFE_ENFORCE(X.ndim() >= 4 && X.ndim() <= 5);

    dX->ResizeLike(X);
    int N = 0, C = 0, H = 0, W = 0, D = 0;
    int H_out = 0, W_out = 0, D_out = 0;
    switch (order_) {
      case StorageOrder::NHWC:
        N = X.dim32(0);
        H = X.dim32(1);
        W = X.ndim() > 3 ? X.dim32(2) : 1;
        D = X.ndim() > 4 ? X.dim32(3) : 1;
        C = X.dim32(X.ndim() - 1);
        H_out = Y.dim32(1);
        W_out = Y.ndim() > 3 ? Y.dim32(2) : 1;
        D_out = Y.ndim() > 4 ? Y.dim32(3) : 1;
        break;
      case StorageOrder::NCHW:
        N = X.dim32(0);
        C = X.dim32(1);
        H = X.dim32(2);
        W = X.ndim() > 3 ? X.dim32(3) : 1;
        D = X.ndim() > 4 ? X.dim32(4) : 1;
        H_out = Y.dim32(2);
        W_out = Y.ndim() > 3 ? Y.dim32(3) : 1;
        D_out = Y.ndim() > 4 ? Y.dim32(4) : 1;
        break;
      default:
        LOG(FATAL) << "Unknown storage order: " << order_;
    }

    // Fast path for global pooling, as cudnn is slow. But only
    // on float, because fp16 not supported for CUB.
    if (std::is_same<T, float>::value) {
      if (order_ == StorageOrder::NCHW && global_pooling_) {
        if (mode_ == HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING) {
          global_avgpool_backward_NCHW<float>
              <<<CAFFE_GET_BLOCKS(dX->size()),
                 CAFFE_CUDA_NUM_THREADS,
                 0,
                 context_.hip_stream()>>>(
                  N * C,
                  H * W * D,
                  dY.data<float>(),
                  dX->mutable_data<float>());
          return true;
        }
#if CUDNN_VERSION_MIN(6, 0, 0)
        if (mode_ == HIPDNN_POOLING_MAX ||
            mode_ == HIPDNN_POOLING_MAX_DETERMINISTIC) {
#else
        if (mode_ == HIPDNN_POOLING_MAX) {
#endif
          global_maxpool_backward_NCHW<float>
              <<<CAFFE_GET_BLOCKS(dX->size()),
                 CAFFE_CUDA_NUM_THREADS,
                 0,
                 context_.hip_stream()>>>(
                  N * C,
                  H * W * D,
                  dY.data<float>(),
                  dX->mutable_data<float>(),
                  Y.data<float>(),
                  X.data<float>());
          return true;
        }
      }
    }

    if (kernel_.size() == 1) {
      ConvPoolOpBase<HIPContext>::ComputePads({H});
    } else if (kernel_.size() == 2) {
      ConvPoolOpBase<HIPContext>::ComputePads({H, W});
    } else if (kernel_.size() == 3) {
      ConvPoolOpBase<HIPContext>::ComputePads({H, W, D});
    } else {
      CAFFE_THROW("Unsupported kernel size :", kernel_.size());
    }

    if (cudnn_input_dims_ != X.dims()) {
      // Dimensions changed; we will need to re-initialize things.
      VLOG(1) << "Changing the cudnn descriptor configurations.";
      cudnn_input_dims_ = X.dims();
      setTensorDescriptor<T>(X.ndim(), order_, N, C, H, W, D, bottom_desc_);
      setTensorDescriptor<T>(
          Y.ndim(), order_, N, C, H_out, W_out, D_out, top_desc_);
      for (int i = 0; i < kernel_.size(); ++i) {
        if (pads_[i] != pads_[kernel_.size() + i]) {
          CAFFE_ENFORCE(
              legacy_pad_ == LegacyPadding::CAFFE_LEGACY_POOLING,
              "Cudnn pooling only supports even padding on both sides, with "
              "the only exception of the caffe legacy pooling case where we "
              "try to preserve backward compatibility with Caffe.");
        }
      }
      if (kernel_.size() == 2) {
        CUDNN_ENFORCE(hipdnnSetPooling2dDescriptor(
            pooling_desc_,
            mode_,
            HIPDNN_NOT_PROPAGATE_NAN,
            kernel_h(),
            kernel_w(),
            pad_t(),
            pad_l(),
            stride_h(),
            stride_w()));
      } else {
        CUDNN_ENFORCE(hipdnnSetPoolingNdDescriptor(
            pooling_desc_,
            mode_,
            HIPDNN_NOT_PROPAGATE_NAN,
            kernel_.size(),
            kernel_.data(),
            pads_.data(),
            stride_.data()));
      }
    }
    // Carry out the pooling computation.
    const T* Xdata = X.template data<T>();
    const T* Ydata = Y.template data<T>();
    const T* dYdata = dY.template data<T>();
    T* dXdata = dX->template mutable_data<T>();

    CUDNN_ENFORCE(hipdnnPoolingBackward(
        cudnn_wrapper_.inline_cudnn_handle(),
        pooling_desc_,
        cudnnTypeWrapper<T>::kOne(),
        top_desc_,
        Ydata,
        top_desc_,
        dYdata,
        bottom_desc_,
        Xdata,
        cudnnTypeWrapper<T>::kZero(),
        bottom_desc_,
        dXdata));
    return true;
  }

  bool RunOnDevice() final {
    auto& X = Input(0);
    auto& Y = Input(1);
    auto& dY = Input(2);
    auto* dX = Output(0);
    dX->ResizeLike(X);

    if (X.IsType<float>()) {
      return DoRunWithType<float, float>();
    } else if (X.IsType<float16>()) {
      return DoRunWithType<float16, float>();
    } else {
      LOG(FATAL) << "Unsupported input types";
    }
    return true;
  }

 protected:
  vector<TIndex> cudnn_input_dims_;

  CuDNNWrapper cudnn_wrapper_;
  hipdnnTensorDescriptor_t bottom_desc_;
  hipdnnTensorDescriptor_t top_desc_;
  hipdnnPoolingDescriptor_t pooling_desc_;
  hipdnnPoolingMode_t mode_;
};

namespace {
REGISTER_CUDNN_OPERATOR(AveragePool, CuDNNPoolOp);
REGISTER_CUDNN_OPERATOR(AveragePoolGradient, CuDNNPoolGradientOp);

REGISTER_CUDNN_OPERATOR(AveragePool1D, CuDNNPoolOp);
REGISTER_CUDNN_OPERATOR(AveragePool1DGradient, CuDNNPoolGradientOp);

REGISTER_CUDNN_OPERATOR(AveragePool2D, CuDNNPoolOp);
REGISTER_CUDNN_OPERATOR(AveragePool2DGradient, CuDNNPoolGradientOp);

REGISTER_CUDNN_OPERATOR(AveragePool3D, CuDNNPoolOp);
REGISTER_CUDNN_OPERATOR(AveragePool3DGradient, CuDNNPoolGradientOp);

REGISTER_CUDNN_OPERATOR(MaxPool, CuDNNPoolOp);
REGISTER_CUDNN_OPERATOR(MaxPoolGradient, CuDNNPoolGradientOp);

REGISTER_CUDNN_OPERATOR(MaxPool1D, CuDNNPoolOp);
REGISTER_CUDNN_OPERATOR(MaxPool1DGradient, CuDNNPoolGradientOp);

REGISTER_CUDNN_OPERATOR(MaxPool2D, CuDNNPoolOp);
REGISTER_CUDNN_OPERATOR(MaxPool2DGradient, CuDNNPoolGradientOp);

REGISTER_CUDNN_OPERATOR(MaxPool3D, CuDNNPoolOp);
REGISTER_CUDNN_OPERATOR(MaxPool3DGradient, CuDNNPoolGradientOp);
} // namespace
} // namespace caffe2
