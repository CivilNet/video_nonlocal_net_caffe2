#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/math_ops.h"

namespace caffe2 {

struct LogCUDAFunctor {
  template <typename T>
  inline void
  operator()(const int n, const T* x, T* y, HIPContext* device_context) {
    math::Log<T, HIPContext>(n, x, y, device_context);
  }
};

REGISTER_HIP_OPERATOR(
    Log,
    UnaryElementwiseOp<TensorTypes<float>, HIPContext, LogCUDAFunctor>);
}
