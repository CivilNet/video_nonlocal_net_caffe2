#include "hip/hip_runtime.h"
#include <cmath>

#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/elementwise_op.h"

namespace caffe2 {

template <typename T>
__global__ void SoftsignKernel(const int N, const T* X, T* Y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    Y[i] = X[i] / (1 + abs(X[i]));
  }
}

template <typename T>
__global__ void SoftsignGradientKernel(const int N, const T* x, const T* dy,
                              T* dx) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    dx[i] = dy[i] / pow(1 + abs(x[i]), 2);
  }
}

struct SoftsignCUDAFunctor {
  template <typename T>
  inline void
  operator()(const int n, const T* x, T* y, HIPContext* device_context) {
    SoftsignKernel<T><<<
        CAFFE_GET_BLOCKS(n),
        CAFFE_CUDA_NUM_THREADS,
        0,
        device_context->hip_stream()>>>(n, x, y);
    return;
  }
};

struct SoftsignGradientCUDAFunctor {
  template <typename T>
  inline void
  Run(const int n, const T* x, const T* dy, T* dx, HIPContext* device_context) {
    SoftsignGradientKernel<T><<<
        CAFFE_GET_BLOCKS(n),
        CAFFE_CUDA_NUM_THREADS,
        0,
        device_context->hip_stream()>>>(n, x, dy, dx);
    return;
  }
};

REGISTER_HIP_OPERATOR(
    Softsign,
    UnaryElementwiseOp<TensorTypes<float>, HIPContext, SoftsignCUDAFunctor>);
REGISTER_HIP_OPERATOR(
    SoftsignGradient,
    BinaryElementwiseOp<TensorTypes<float>, HIPContext, WithoutBroadcast<SoftsignGradientCUDAFunctor>>);
} // namespace caffe2
