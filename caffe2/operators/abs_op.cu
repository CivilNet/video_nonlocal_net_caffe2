#include "hip/hip_runtime.h"
#include <cmath>

#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/elementwise_op.h"

namespace caffe2 {

template <typename T>
__global__ void AbsKernel(const int N, const T* X, T* Y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    Y[i] = fabs(X[i]);
  }
}

template <typename T>
__global__ void AbsGradientKernel(const int N, const T* X, const T* dY, T* dX) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    dX[i] = X[i] == T(0) ? T(0) : (X[i] > T(0) ? dY[i] : -dY[i]);
  }
}

struct AbsCUDAFunctor {
  template <typename T>
  inline void
  operator()(const int n, const T* x, T* y, HIPContext* device_context) {
    AbsKernel<T>
        <<<CAFFE_GET_BLOCKS(n),
           CAFFE_CUDA_NUM_THREADS,
           0,
           device_context->hip_stream()>>>(n, x, y);
    return;
  }
};

struct AbsGradientCUDAFunctor {
  template <typename T>
  inline void Run(
      const int n,
      const T* x,
      const T* dy,
      T* dx,
      HIPContext* device_context) {
    AbsGradientKernel<T>
        <<<CAFFE_GET_BLOCKS(n),
           CAFFE_CUDA_NUM_THREADS,
           0,
           device_context->hip_stream()>>>(n, x, dy, dx);
    return;
  }
};

REGISTER_HIP_OPERATOR(
    Abs,
    UnaryElementwiseOp<TensorTypes<float>, HIPContext, AbsCUDAFunctor>);
REGISTER_HIP_OPERATOR(
    AbsGradient,
    BinaryElementwiseOp<
        TensorTypes<float>,
        HIPContext,
        WithoutBroadcast<AbsGradientCUDAFunctor>>);
} // namespace caffe2
