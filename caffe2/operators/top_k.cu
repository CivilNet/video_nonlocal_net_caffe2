#include "hip/hip_runtime.h"
#include "caffe2/operators/top_k.h"

#include <algorithm>
#include <array>
#include <functional>
#include <limits>
#include <numeric>
#include <vector>

#include <thrust/sort.h>
#include <thrust/system/cuda/execution_policy.h>

#include "caffe2/core/context.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/top_k_heap_selection.cuh"
#include "caffe2/operators/top_k_radix_selection.cuh"
#include "caffe2/utils/math.h"

namespace caffe2 {

namespace {

void MakeTransposeParams(
    const TIndex prev_size,
    const TIndex next_size,
    const TIndex n,
    TensorCUDA* x_dims,
    TensorCUDA* y_dims,
    TensorCUDA* axes,
    HIPContext* context) {
  const std::array<int, 3> x_dims_vec = {static_cast<int>(prev_size),
                                         static_cast<int>(n),
                                         static_cast<int>(next_size)};
  const std::array<int, 3> y_dims_vec = {static_cast<int>(prev_size),
                                         static_cast<int>(next_size),
                                         static_cast<int>(n)};
  const std::array<int, 3> axes_vec = {0, 2, 1};
  x_dims->Resize(3);
  context->Copy<int, CPUContext, HIPContext>(
      3, x_dims_vec.data(), x_dims->mutable_data<int>());
  y_dims->Resize(3);
  context->Copy<int, CPUContext, HIPContext>(
      3, y_dims_vec.data(), y_dims->mutable_data<int>());
  axes->Resize(3);
  context->Copy<int, CPUContext, HIPContext>(
      3, axes_vec.data(), axes->mutable_data<int>());
}

template <typename T, int kHeapSize, bool kSelectMax = true>
void RunHeapSelectionImpl(
    const T* input,
    const TIndex outer_size,
    const TIndex inner_size,
    const int k,
    T* values,
    TIndex* indices,
    HIPContext* context) {
  constexpr int kBlockSize = 256;
  constexpr int kNumWarps = kBlockSize / kWarpSize;
  constexpr int smem = kNumWarps * kHeapSize * (sizeof(T) + sizeof(TIndex));
  constexpr T kInitVal = kSelectMax ? std::numeric_limits<T>::lowest()
                                    : std::numeric_limits<T>::max();
  selectRowsViaHeap<T, TIndex, TIndex, kBlockSize, kHeapSize, kSelectMax>
      <<<outer_size, kBlockSize, smem, context->hip_stream()>>>(
          input,
          values,
          indices,
          kInitVal,
          std::numeric_limits<TIndex>::max(),
          outer_size,
          inner_size,
          k);
}

template <typename T, bool kSelectMax = true>
void RunRadixSelectionImpl(
    const T* input,
    const TIndex outer_size,
    const TIndex inner_size,
    const int k,
    T* values,
    TIndex* indices,
    HIPContext* context) {
  const int block = std::min(
      math::roundUp(static_cast<int>(inner_size), kWarpSize),
      CAFFE_CUDA_NUM_THREADS);
  gatherTopK<T, kSelectMax, TIndex>
      <<<outer_size, block, 0, context->hip_stream()>>>(
          input, inner_size, k, outer_size, values, indices);
  // Unfortunately the output is not currently sorted, and there is no batch
  // sorting utility available. Iterate over all of the slices and sort them
  // in-place using Thrust.
  for (int i = 0; i < outer_size; ++i) {
    thrust::sort_by_key(
        thrust::cuda::par.on(context->hip_stream()),
        values + i * k,
        values + i * k + k,
        indices + i * k,
        thrust::greater<T>());
  }
}

template <typename T>
void RunTopKOnLastDimCUDAImpl(
    const T* input,
    const TIndex outer_size,
    const TIndex inner_size,
    const int k,
    T* values,
    TIndex* indices,
    HIPContext* context) {
  // If k is small, uses heap selection, otherwise uses radix selection.
  if (k < 32) {
    RunHeapSelectionImpl<T, 32>(
        input, outer_size, inner_size, k, values, indices, context);
  } else if (k < 128) {
    RunHeapSelectionImpl<T, 128>(
        input, outer_size, inner_size, k, values, indices, context);
  } else if (k < 512) {
    RunHeapSelectionImpl<T, 512>(
        input, outer_size, inner_size, k, values, indices, context);
  } else {
    RunRadixSelectionImpl<T>(
        input, outer_size, inner_size, k, values, indices, context);
  }
}

__global__ void FlattenIndicesCUDA(
    const TIndex* src,
    const TIndex size,
    const TIndex stride,
    const TIndex n,
    const int k,
    TIndex* dst) {
  CUDA_1D_KERNEL_LOOP(i, size) {
    const TIndex x = i / stride / k;
    const TIndex y = i % stride;
#if __CUDA_ARCH__ >= 350
    dst[i] = __ldg(src + i) * stride + x * n * stride + y;
#else
    dst[i] = src[i] * stride + x * n * stride + y;
#endif
  }
}

template <typename T>
__global__ void SetTopKGradientCUDA(
    const T* values,
    const TIndex* indices,
    const TIndex size,
    const TIndex stride,
    const TIndex n,
    const int k,
    T* dst) {
  CUDA_1D_KERNEL_LOOP(i, size) {
    const TIndex x = i / stride / k;
    const TIndex y = i % stride;
#if __CUDA_ARCH__ >= 350
    dst[__ldg(indices + i) * stride + x * n * stride + y] = __ldg(values + i);
#else
    dst[indices[i] * stride + x * n * stride + y] = values[i];
#endif
  }
}

} // namespace

template <typename T>
class TopKOp<T, HIPContext> : public Operator<HIPContext> {
 public:
  USE_OPERATOR_FUNCTIONS(HIPContext);

  TopKOp(const OperatorDef& operator_def, Workspace* ws)
      : Operator<HIPContext>(operator_def, ws),
        OP_SINGLE_ARG(int, "k", k_, -1),
        OP_SINGLE_ARG(int, "axis", axis_, -1) {
    CAFFE_ENFORCE(k_ >= 1, "k argument must be >= 1");
  }

  ~TopKOp(){};

  bool RunOnDevice() override;

 private:
  const int k_;
  int axis_;

  // Buffers for HIPContext.
  TensorCUDA input_transposed_buffer_;
  TensorCUDA values_transposed_buffer_;
  TensorCUDA indices_transposed_buffer_;

  // Shape tensors on device for HIPContext.
  TensorCUDA input_dims_device_;
  TensorCUDA input_transposed_dims_device_;
  TensorCUDA input_axes_device_;

  TensorCUDA output_dims_device_;
  TensorCUDA output_transposed_dims_device_;
  TensorCUDA output_transposed_axes_device_;
};

template <typename T>
bool TopKOp<T, HIPContext>::RunOnDevice() {
  const auto& input = Input(0);
  auto* values = Output(0);
  auto* indices = Output(1);
  auto* flatten_indices = OutputSize() > 2 ? Output(2) : nullptr;

  const std::vector<TIndex>& input_dims = input.dims();
  if (axis_ == -1) {
    axis_ = input_dims.size() - 1;
  }
  CAFFE_ENFORCE_GE(axis_, 0);
  CAFFE_ENFORCE_LT(axis_, input_dims.size());
  CAFFE_ENFORCE_LE(
      k_,
      input_dims[axis_],
      "k argument should not be greater than the axis dim.");

  const bool need_transpose = axis_ < input_dims.size() - 1;
  std::vector<TIndex> output_dims = input_dims;
  output_dims[axis_] = k_;
  const TIndex prev_size = std::accumulate(
      input_dims.cbegin(),
      input_dims.cbegin() + axis_,
      TIndex(1),
      std::multiplies<TIndex>());
  const TIndex next_size = std::accumulate(
      input_dims.cbegin() + axis_ + 1,
      input_dims.cend(),
      TIndex(1),
      std::multiplies<TIndex>());
  const TIndex outer_size = input.size() / input_dims[axis_];
  const TIndex inner_size = input_dims[axis_];

  values->Resize(output_dims);
  indices->Resize(output_dims);
  if (flatten_indices != nullptr) {
    flatten_indices->Resize(indices->size());
  }
  const T* input_data = input.template data<T>();
  T* values_data = values->template mutable_data<T>();
  TIndex* indices_data = indices->template mutable_data<TIndex>();
  TIndex* flatten_indices_data = flatten_indices == nullptr
      ? nullptr
      : flatten_indices->template mutable_data<TIndex>();

  if (need_transpose) {
    MakeTransposeParams(
        prev_size,
        next_size,
        inner_size,
        &input_dims_device_,
        &input_transposed_dims_device_,
        &input_axes_device_,
        &context_);
    input_transposed_buffer_.Resize(
        std::vector<TIndex>{outer_size, inner_size});
    values_transposed_buffer_.Resize(std::vector<TIndex>{outer_size, k_});
    indices_transposed_buffer_.Resize(std::vector<TIndex>{outer_size, k_});
    math::Transpose(
        3,
        input_dims_device_.data<int>(),
        input_transposed_dims_device_.data<int>(),
        input_axes_device_.data<int>(),
        input.size(),
        input.template data<T>(),
        input_transposed_buffer_.mutable_data<T>(),
        &context_);
    input_data = input_transposed_buffer_.data<T>();
    values_data = values_transposed_buffer_.mutable_data<T>();
    indices_data = indices_transposed_buffer_.mutable_data<TIndex>();
  }
  RunTopKOnLastDimCUDAImpl<T>(
      input_data,
      outer_size,
      inner_size,
      k_,
      values_data,
      indices_data,
      &context_);
  if (need_transpose) {
    MakeTransposeParams(
        prev_size,
        next_size,
        k_,
        &output_dims_device_,
        &output_transposed_dims_device_,
        &output_transposed_axes_device_,
        &context_);
    math::Transpose(
        3,
        output_transposed_dims_device_.data<int>(),
        output_dims_device_.data<int>(),
        output_transposed_axes_device_.data<int>(),
        values_transposed_buffer_.size(),
        values_transposed_buffer_.data<T>(),
        values->template mutable_data<T>(),
        &context_);
    math::Transpose(
        3,
        output_transposed_dims_device_.data<int>(),
        output_dims_device_.data<int>(),
        output_transposed_axes_device_.data<int>(),
        indices_transposed_buffer_.size(),
        indices_transposed_buffer_.data<TIndex>(),
        indices->template mutable_data<TIndex>(),
        &context_);
  }

  // Flatten the indices if needed.
  if (flatten_indices != nullptr) {
    FlattenIndicesCUDA<<<
        CAFFE_GET_BLOCKS(indices->size()),
        CAFFE_CUDA_NUM_THREADS,
        0,
        context_.hip_stream()>>>(
        indices->template data<TIndex>(),
        indices->size(),
        next_size,
        inner_size,
        k_,
        flatten_indices->template mutable_data<TIndex>());
  }
  return true;
}

REGISTER_HIP_OPERATOR(TopK, TopKOp<float, HIPContext>);

template <typename T>
class TopKGradientOp<T, HIPContext> : public Operator<HIPContext> {
 public:
  USE_OPERATOR_FUNCTIONS(HIPContext);

  TopKGradientOp(const OperatorDef& operator_def, Workspace* ws)
      : Operator<HIPContext>(operator_def, ws),
        OP_SINGLE_ARG(int, "axis", axis_, -1) {}

  ~TopKGradientOp(){};

  bool RunOnDevice() override;

 private:
  int axis_;
};

template <typename T>
bool TopKGradientOp<T, HIPContext>::RunOnDevice() {
  const auto& values = Input(0);
  const auto& indices = Input(1);
  const auto& original_input = Input(2);
  auto* output = Output(0);
  const std::vector<TIndex>& values_dims = values.dims();
  const std::vector<TIndex>& origin_dims = original_input.dims();
  CAFFE_ENFORCE_EQ(values_dims.size(), origin_dims.size());
  output->Resize(origin_dims);
  T* output_data = output->template mutable_data<T>();
  if (axis_ == -1) {
    axis_ = values_dims.size() - 1;
  }
  const int k = values_dims[axis_];
  math::Set<T, HIPContext>(output->size(), T(0), output_data, &context_);
  const TIndex stride = std::accumulate(
      values_dims.cbegin() + axis_ + 1,
      values_dims.cend(),
      TIndex(1),
      std::multiplies<TIndex>());
  SetTopKGradientCUDA<<<
      CAFFE_GET_BLOCKS(indices.size()),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      values.template data<T>(),
      indices.template data<TIndex>(),
      values.size(),
      stride,
      origin_dims[axis_],
      k,
      output_data);
  return true;
}

REGISTER_HIP_OPERATOR(TopKGradient, TopKGradientOp<float, HIPContext>);

} // namespace caffe2
