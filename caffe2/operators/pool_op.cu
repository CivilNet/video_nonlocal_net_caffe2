#include "hip/hip_runtime.h"
// TODO(ataei): reduce the apparent redundancy of all the code below.
#include <cfloat>

#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/pool_op.h"

namespace caffe2 {
namespace {
class AveragePool {};
class MaxPool {};
}  // namespace

namespace {
template <typename T>
__global__ void Average1DPoolForwardNCHW(
    const int nthreads,
    const T* bottom_data,
    const int num,
    const int channels,
    const int height,
    const int pooled_height,
    const int kernel_h,
    const int stride_h,
    const int pad_t,
    T* top_data) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int n = index;
    int ph = n % pooled_height;
    n /= pooled_height;
    int c = n % channels;
    n /= channels;
    int hstart = ph * stride_h - pad_t;
    int hend = min(hstart + kernel_h, height);
    hstart = max(hstart, 0);
    top_data[index] = 0;
    int bottom_offset = (n * channels + c) * height;
    for (int h = hstart; h < hend; ++h) {
      top_data[index] += bottom_data[bottom_offset + h];
    }
    top_data[index] /= (hend - hstart);
  }
}

template <typename T>
__global__ void Average2DPoolForwardNCHW(
    const int nthreads,
    const T* bottom_data,
    const int num,
    const int channels,
    const int height,
    const int width,
    const int pooled_height,
    const int pooled_width,
    const int kernel_h,
    const int kernel_w,
    const int stride_h,
    const int stride_w,
    const int pad_t,
    const int pad_l,
    T* top_data) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int n = index;
    int pw = n % pooled_width;
    n /= pooled_width;
    int ph = n % pooled_height;
    n /= pooled_height;
    int c = n % channels;
    n /= channels;
    int hstart = ph * stride_h - pad_t;
    int wstart = pw * stride_w - pad_l;
    int hend = min(hstart + kernel_h, height);
    int wend = min(wstart + kernel_w, width);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    top_data[index] = 0;
    int bottom_offset = (n * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        top_data[index] += bottom_data[bottom_offset + h * width + w];
      }
    }
    top_data[index] /= (hend - hstart) * (wend - wstart);
  }
}

template <typename T>
__global__ void Average3DPoolForwardNCHW(
    const int nthreads,
    const T* bottom_data,
    const int num,
    const int channels,
    const int height,
    const int width,
    const int depth,
    const int pooled_height,
    const int pooled_width,
    const int pooled_depth,
    const int kernel_h,
    const int kernel_w,
    const int kernel_d,
    const int stride_h,
    const int stride_w,
    const int stride_d,
    const int pad_t,
    const int pad_l,
    const int pad_f,
    T* top_data) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int n = index;
    int pd = n % pooled_depth;
    n /= pooled_depth;
    int pw = n % pooled_width;
    n /= pooled_width;
    int ph = n % pooled_height;
    n /= pooled_height;
    int c = n % channels;
    n /= channels;
    int hstart = ph * stride_h - pad_t;
    int wstart = pw * stride_w - pad_l;
    int dstart = pd * stride_d - pad_f;
    int hend = min(hstart + kernel_h, height);
    int wend = min(wstart + kernel_w, width);
    int dend = min(dstart + kernel_d, depth);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    dstart = max(dstart, 0);
    top_data[index] = 0;
    int bottom_offset = (n * channels + c) * height * width * depth;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        for (int d = dstart; d < dend; ++d) {
          const int input_index =
              bottom_offset + h * width * depth + w * depth + d;
          top_data[index] += bottom_data[input_index];
        }
      }
    }
    top_data[index] /= (hend - hstart) * (wend - wstart) * (dend - dstart);
  }
}

template <typename T>
__global__ void Average1DPoolForwardNHWC(
    const int nthreads,
    const T* bottom_data,
    const int num,
    const int height,
    const int channels,
    const int pooled_height,
    const int kernel_h,
    const int stride_h,
    const int pad_t,
    T* top_data) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int c = index % channels;
    int ph = (index / channels) % pooled_height;
    int n = index / channels / pooled_height;
    int hstart = ph * stride_h - pad_t;
    int hend = min(hstart + kernel_h, height);
    hstart = max(hstart, 0);
    T output = 0;
    int bottom_offset = n * height * channels + c;
    for (int h = hstart; h < hend; ++h) {
      output += bottom_data[bottom_offset + h * channels];
    }
    int pool_size = (hend - hstart);
    top_data[index] = output / pool_size;
  }
}

template <typename T>
__global__ void Average2DPoolForwardNHWC(
    const int nthreads,
    const T* bottom_data,
    const int num,
    const int height,
    const int width,
    const int channels,
    const int pooled_height,
    const int pooled_width,
    const int kernel_h,
    const int kernel_w,
    const int stride_h,
    const int stride_w,
    const int pad_t,
    const int pad_l,
    T* top_data) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int c = index % channels;
    int pw = (index / channels) % pooled_width;
    int ph = (index / channels / pooled_width) % pooled_height;
    int n = index / channels / pooled_width / pooled_height;
    int hstart = ph * stride_h - pad_t;
    int wstart = pw * stride_w - pad_l;
    int hend = min(hstart + kernel_h, height);
    int wend = min(wstart + kernel_w, width);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    T output = 0;
    int bottom_offset = n * height * width * channels + c;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        output += bottom_data[bottom_offset + (h * width + w) * channels];
      }
    }
    int pool_size = (hend - hstart) * (wend - wstart);
    top_data[index] = output / pool_size;
  }
}

template <typename T>
__global__ void Average3DPoolForwardNHWC(
    const int nthreads,
    const T* bottom_data,
    const int num,
    const int height,
    const int width,
    const int depth,
    const int channels,
    const int pooled_height,
    const int pooled_width,
    const int pooled_depth,
    const int kernel_h,
    const int kernel_w,
    const int kernel_d,
    const int stride_h,
    const int stride_w,
    const int stride_d,
    const int pad_t,
    const int pad_l,
    const int pad_f,
    T* top_data) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int c = index % channels;
    int pd = (index / channels) % pooled_depth;
    int pw = (index / channels / pooled_depth) % pooled_width;
    int ph = (index / channels / pooled_depth / pooled_width) % pooled_height;
    int n = index / channels / pooled_depth / pooled_width / pooled_height;
    int hstart = ph * stride_h - pad_t;
    int wstart = pw * stride_w - pad_l;
    int dstart = pd * stride_d - pad_f;
    int hend = min(hstart + kernel_h, height);
    int wend = min(wstart + kernel_w, width);
    int dend = min(dstart + kernel_d, depth);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    dstart = max(dstart, 0);
    T output = 0;
    int bottom_offset = n * height * width * depth * channels + c;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        for (int d = dstart; d < dend; ++d) {
          const int bottom_index =
              bottom_offset + (h * depth * width + w * depth + d) * channels;
          output += bottom_data[bottom_index];
        }
      }
    }
    int pool_size = (hend - hstart) * (wend - wstart) * (dend - dstart);
    top_data[index] = output / pool_size;
  }
}

template <typename T>
__global__ void Ave1DPoolBackwardNCHW(
    const int nthreads,
    const T* const top_diff,
    const int num,
    const int channels,
    const int height,
    const int pooled_height,
    const int kernel_h,
    const int stride_h,
    const int pad_t,
    T* const bottom_diff) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int h = index % height + pad_t;
    const int c = (index / height) % channels;
    const int n = index / height / channels;
    const int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int phend = min(h / stride_h + 1, pooled_height);
    T gradient = 0;
    const T* const top_diff_slice =
        top_diff + (n * channels + c) * pooled_height;
    for (int ph = phstart; ph < phend; ++ph) {
      // figure out the pooling size
      int hstart = ph * stride_h - pad_t;
      int hend = min(hstart + kernel_h, height);
      hstart = max(hstart, 0);
      int pool_size = (hend - hstart);
      gradient += top_diff_slice[ph] / pool_size;
    }
    bottom_diff[index] = gradient;
  }
}

template <typename T>
__global__ void Ave2DPoolBackwardNCHW(
    const int nthreads,
    const T* const top_diff,
    const int num,
    const int channels,
    const int height,
    const int width,
    const int pooled_height,
    const int pooled_width,
    const int kernel_h,
    const int kernel_w,
    const int stride_h,
    const int stride_w,
    const int pad_t,
    const int pad_l,
    T* const bottom_diff) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int w = index % width + pad_l;
    const int h = (index / width) % height + pad_t;
    const int c = (index / width / height) % channels;
    const int n = index / width / height / channels;
    const int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int phend = min(h / stride_h + 1, pooled_height);
    const int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    const int pwend = min(w / stride_w + 1, pooled_width);
    T gradient = 0;
    const T* const top_diff_slice =
        top_diff + (n * channels + c) * pooled_height * pooled_width;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        // figure out the pooling size
        int hstart = ph * stride_h - pad_t;
        int wstart = pw * stride_w - pad_l;
        int hend = min(hstart + kernel_h, height);
        int wend = min(wstart + kernel_w, width);
        hstart = max(hstart, 0);
        wstart = max(wstart, 0);
        int pool_size = (hend - hstart) * (wend - wstart);
        gradient += top_diff_slice[ph * pooled_width + pw] / pool_size;
      }
    }
    bottom_diff[index] = gradient;
  }
}

template <typename T>
__global__ void Ave3DPoolBackwardNCHW(
    const int nthreads,
    const T* const top_diff,
    const int num,
    const int channels,
    const int height,
    const int width,
    const int depth,
    const int pooled_height,
    const int pooled_width,
    const int pooled_depth,
    const int kernel_h,
    const int kernel_w,
    const int kernel_d,
    const int stride_h,
    const int stride_w,
    const int stride_d,
    const int pad_t,
    const int pad_l,
    const int pad_f,
    T* const bottom_diff) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int d = index % depth + pad_f;
    const int w = (index / depth) % width + pad_l;
    const int h = (index / depth / width) % height + pad_t;
    const int c = (index / depth / width / height) % channels;
    const int n = index / depth / width / height / channels;
    const int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int phend = min(h / stride_h + 1, pooled_height);
    const int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    const int pwend = min(w / stride_w + 1, pooled_width);
    const int pdstart = (d < kernel_d) ? 0 : (d - kernel_d) / stride_d + 1;
    const int pdend = min(d / stride_d + 1, pooled_depth);
    T gradient = 0;
    const T* const top_diff_slice = top_diff +
        (n * channels + c) * pooled_height * pooled_width * pooled_depth;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        for (int pd = pdstart; pd < pdend; ++pd) {
          // figure out the pooling size
          int hstart = ph * stride_h - pad_t;
          int wstart = pw * stride_w - pad_l;
          int dstart = pd * stride_d - pad_f;
          int hend = min(hstart + kernel_h, height);
          int wend = min(wstart + kernel_w, width);
          int dend = min(dstart + kernel_d, depth);
          hstart = max(hstart, 0);
          wstart = max(wstart, 0);
          dstart = max(dstart, 0);
          int pool_size = (hend - hstart) * (wend - wstart) * (dend - dstart);
          const int pooled_index =
              ph * pooled_depth * pooled_width + pooled_depth * pw + pd;
          gradient += top_diff_slice[pooled_index] / pool_size;
        }
      }
    }
    bottom_diff[index] = gradient;
  }
}

template <typename T>
__global__ void Ave1DPoolBackwardNHWC(
    const int nthreads,
    const T* const top_diff,
    const int num,
    const int height,
    const int channels,
    const int pooled_height,
    const int kernel_h,
    const int stride_h,
    const int pad_t,
    T* const bottom_diff) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int c = index % channels;
    const int h = (index / channels) % height + pad_t;
    const int n = index / channels / height;
    const int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int phend = min(h / stride_h + 1, pooled_height);
    T gradient = 0;
    const T* const top_diff_slice = top_diff + n * pooled_height * channels + c;
    for (int ph = phstart; ph < phend; ++ph) {
      // figure out the pooling size
      int hstart = ph * stride_h - pad_t;
      int hend = min(hstart + kernel_h, height);
      hstart = max(hstart, 0);
      int pool_size = (hend - hstart);
      gradient += top_diff_slice[ph * channels] / pool_size;
    }
    bottom_diff[index] = gradient;
  }
}

template <typename T>
__global__ void Ave2DPoolBackwardNHWC(
    const int nthreads,
    const T* const top_diff,
    const int num,
    const int height,
    const int width,
    const int channels,
    const int pooled_height,
    const int pooled_width,
    const int kernel_h,
    const int kernel_w,
    const int stride_h,
    const int stride_w,
    const int pad_t,
    const int pad_l,
    T* const bottom_diff) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int c = index % channels;
    const int w = index / channels % width + pad_l;
    const int h = (index / channels / width) % height + pad_t;
    const int n = index / channels / width / height;
    const int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int phend = min(h / stride_h + 1, pooled_height);
    const int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    const int pwend = min(w / stride_w + 1, pooled_width);
    T gradient = 0;
    const T* const top_diff_slice =
        top_diff + n * pooled_height * pooled_width * channels + c;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        // figure out the pooling size
        int hstart = ph * stride_h - pad_t;
        int wstart = pw * stride_w - pad_l;
        int hend = min(hstart + kernel_h, height);
        int wend = min(wstart + kernel_w, width);
        hstart = max(hstart, 0);
        wstart = max(wstart, 0);
        int pool_size = (hend - hstart) * (wend - wstart);
        gradient +=
            top_diff_slice[(ph * pooled_width + pw) * channels] / pool_size;
      }
    }
    bottom_diff[index] = gradient;
  }
}

template <typename T>
__global__ void Ave3DPoolBackwardNHWC(
    const int nthreads,
    const T* const top_diff,
    const int num,
    const int height,
    const int width,
    const int depth,
    const int channels,
    const int pooled_height,
    const int pooled_width,
    const int pooled_depth,
    const int kernel_h,
    const int kernel_w,
    const int kernel_d,
    const int stride_h,
    const int stride_w,
    const int stride_d,
    const int pad_t,
    const int pad_l,
    const int pad_f,
    T* const bottom_diff) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int c = index % channels;
    const int d = index / channels % depth + pad_f;
    const int w = (index / channels / depth) % width + pad_l;
    const int h = (index / channels / depth / width) % height + pad_t;
    const int n = index / channels / depth / width / height;
    const int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int phend = min(h / stride_h + 1, pooled_height);
    const int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    const int pwend = min(w / stride_w + 1, pooled_width);
    const int pdstart = (d < kernel_d) ? 0 : (d - kernel_d) / stride_d + 1;
    const int pdend = min(d / stride_d + 1, pooled_depth);
    T gradient = 0;
    const T* const top_diff_slice = top_diff +
        n * pooled_height * pooled_width * pooled_depth * channels + c;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        for (int pd = pdstart; pd < pdend; ++pd) {
          // figure out the pooling size
          int hstart = ph * stride_h - pad_t;
          int wstart = pw * stride_w - pad_l;
          int dstart = pd * stride_d - pad_f;
          int hend = min(hstart + kernel_h, height);
          int wend = min(wstart + kernel_w, width);
          int dend = min(dstart + kernel_d, depth);
          hstart = max(hstart, 0);
          wstart = max(wstart, 0);
          dstart = max(dstart, 0);
          int pool_size = (hend - hstart) * (wend - wstart) * (dend - dstart);
          const int pooled_index =
              (ph * pooled_depth * pooled_width + pw * pooled_depth + pd) *
              channels;
          gradient += top_diff_slice[pooled_index] / pool_size;
        }
      }
    }
    bottom_diff[index] = gradient;
  }
}

}  // namespace

template <>
bool PoolOp<float, HIPContext, AveragePool>::RunOnDeviceWithOrderNCHW() {
  auto& X = Input(0);
  auto* Y = Output(0);
  ConvPoolOpBase<HIPContext>::SetOutputSize(X, Y, X.dim32(1));
  int output_size = Y->size();
  switch (kernel_.size()) {
    case 1:
      Average1DPoolForwardNCHW<float><<<
          CAFFE_GET_BLOCKS(output_size),
          CAFFE_CUDA_NUM_THREADS,
          0,
          context_.hip_stream()>>>(
          output_size,
          X.data<float>(),
          X.dim32(0),
          X.dim32(1),
          X.dim32(2),
          Y->dim32(2),
          kernel_h(),
          stride_h(),
          pad_t(),
          Y->mutable_data<float>());
      break;
    case 2:
      Average2DPoolForwardNCHW<float><<<
          CAFFE_GET_BLOCKS(output_size),
          CAFFE_CUDA_NUM_THREADS,
          0,
          context_.hip_stream()>>>(
          output_size,
          X.data<float>(),
          X.dim32(0),
          X.dim32(1),
          X.dim32(2),
          X.dim32(3),
          Y->dim32(2),
          Y->dim32(3),
          kernel_h(),
          kernel_w(),
          stride_h(),
          stride_w(),
          pad_t(),
          pad_l(),
          Y->mutable_data<float>());
      break;
    case 3:
      Average3DPoolForwardNCHW<float><<<
          CAFFE_GET_BLOCKS(output_size),
          CAFFE_CUDA_NUM_THREADS,
          0,
          context_.hip_stream()>>>(
          output_size,
          X.data<float>(),
          X.dim32(0),
          X.dim32(1),
          X.dim32(2),
          X.dim32(3),
          X.dim32(4),
          Y->dim32(2),
          Y->dim32(3),
          Y->dim32(4),
          kernel_h(),
          kernel_w(),
          kernel_[2],
          stride_h(),
          stride_w(),
          stride_[2],
          pad_t(),
          pad_l(),
          pads_[2],
          Y->mutable_data<float>());
      break;
    default:
      CAFFE_THROW("Unsupported pooling size : ", kernel_.size());
  }
  return true;
}

template <>
bool PoolOp<float, HIPContext, AveragePool>::RunOnDeviceWithOrderNHWC() {
  auto& X = Input(0);
  auto* Y = Output(0);
  ConvPoolOpBase<HIPContext>::SetOutputSize(X, Y, X.dim32(X.ndim() - 1));
  int output_size = Y->size();
  switch (kernel_.size()) {
    case 1:
      Average1DPoolForwardNHWC<float><<<
          CAFFE_GET_BLOCKS(output_size),
          CAFFE_CUDA_NUM_THREADS,
          0,
          context_.hip_stream()>>>(
          output_size,
          X.data<float>(),
          X.dim32(0),
          X.dim32(1),
          X.dim32(2),
          Y->dim32(1),
          kernel_h(),
          stride_h(),
          pad_t(),
          Y->mutable_data<float>());
      break;
    case 2:
      Average2DPoolForwardNHWC<float><<<
          CAFFE_GET_BLOCKS(output_size),
          CAFFE_CUDA_NUM_THREADS,
          0,
          context_.hip_stream()>>>(
          output_size,
          X.data<float>(),
          X.dim32(0),
          X.dim32(1),
          X.dim32(2),
          X.dim32(3),
          Y->dim32(1),
          Y->dim32(2),
          kernel_h(),
          kernel_w(),
          stride_h(),
          stride_w(),
          pad_t(),
          pad_l(),
          Y->mutable_data<float>());
      break;
    case 3:
      Average3DPoolForwardNHWC<float><<<
          CAFFE_GET_BLOCKS(output_size),
          CAFFE_CUDA_NUM_THREADS,
          0,
          context_.hip_stream()>>>(
          output_size,
          X.data<float>(),
          X.dim32(0),
          X.dim32(1),
          X.dim32(2),
          X.dim32(3),
          X.dim32(4),
          Y->dim32(1),
          Y->dim32(2),
          Y->dim32(3),
          kernel_h(),
          kernel_w(),
          kernel_[2],
          stride_h(),
          stride_w(),
          stride_[2],
          pad_t(),
          pad_l(),
          pads_[2],
          Y->mutable_data<float>());
      break;
    default:
      CAFFE_THROW("Unsupported pooling size : ", kernel_.size());
  }
  return true;
}

template <>
bool PoolGradientOp<float, HIPContext, AveragePool>::
    RunOnDeviceWithOrderNCHW() {
  auto& X = Input(0);
  auto& dY = Input(2);
  CAFFE_ENFORCE_EQ(dY.dim32(1), X.dim32(1));
  auto* dX = Output(0);
  dX->ResizeLike(X);
  vector<int> dims(X.dims().begin() + 2, X.dims().end());
  ConvPoolOpBase<HIPContext>::ComputePads(dims);
  switch (kernel_.size()) {
    case 1:
      Ave1DPoolBackwardNCHW<float><<<
          CAFFE_GET_BLOCKS(X.size()),
          CAFFE_CUDA_NUM_THREADS,
          0,
          context_.hip_stream()>>>(
          X.size(),
          dY.data<float>(),
          X.dim32(0),
          X.dim32(1),
          X.dim32(2),
          dY.dim32(2),
          kernel_h(),
          stride_h(),
          pad_t(),
          dX->mutable_data<float>());
      break;
    case 2:
      Ave2DPoolBackwardNCHW<float><<<
          CAFFE_GET_BLOCKS(X.size()),
          CAFFE_CUDA_NUM_THREADS,
          0,
          context_.hip_stream()>>>(
          X.size(),
          dY.data<float>(),
          X.dim32(0),
          X.dim32(1),
          X.dim32(2),
          X.dim32(3),
          dY.dim32(2),
          dY.dim32(3),
          kernel_h(),
          kernel_w(),
          stride_h(),
          stride_w(),
          pad_t(),
          pad_l(),
          dX->mutable_data<float>());
      break;
    case 3:
      Ave3DPoolBackwardNCHW<float><<<
          CAFFE_GET_BLOCKS(X.size()),
          CAFFE_CUDA_NUM_THREADS,
          0,
          context_.hip_stream()>>>(
          X.size(),
          dY.data<float>(),
          X.dim32(0),
          X.dim32(1),
          X.dim32(2),
          X.dim32(3),
          X.dim32(4),
          dY.dim32(2),
          dY.dim32(3),
          dY.dim32(4),
          kernel_h(),
          kernel_w(),
          kernel_[2],
          stride_h(),
          stride_w(),
          stride_[2],
          pad_t(),
          pad_l(),
          pads_[2],
          dX->mutable_data<float>());
      break;
    default:
      CAFFE_THROW("Unsupported pooling size : ", kernel_.size());
  }
  return true;
}

template <>
bool PoolGradientOp<float, HIPContext, AveragePool>::
    RunOnDeviceWithOrderNHWC() {
  auto& X = Input(0);
  auto& dY = Input(2);
  CAFFE_ENFORCE_EQ(X.ndim(), dY.ndim());
  CAFFE_ENFORCE_EQ(X.dim32(X.ndim() - 1), dY.dim32(dY.ndim() - 1));
  auto* dX = Output(0);
  dX->ResizeLike(X);
  vector<int> dims(X.dims().begin() + 1, X.dims().end() - 1);
  ConvPoolOpBase<HIPContext>::ComputePads(dims);
  switch (kernel_.size()) {
    case 1:
      Ave1DPoolBackwardNHWC<float><<<
          CAFFE_GET_BLOCKS(X.size()),
          CAFFE_CUDA_NUM_THREADS,
          0,
          context_.hip_stream()>>>(
          X.size(),
          dY.data<float>(),
          X.dim32(0),
          X.dim32(1),
          X.dim32(2),
          dY.dim32(1),
          kernel_h(),
          stride_h(),
          pad_t(),
          dX->mutable_data<float>());
      break;
    case 2:
      Ave2DPoolBackwardNHWC<float><<<
          CAFFE_GET_BLOCKS(X.size()),
          CAFFE_CUDA_NUM_THREADS,
          0,
          context_.hip_stream()>>>(
          X.size(),
          dY.data<float>(),
          X.dim32(0),
          X.dim32(1),
          X.dim32(2),
          X.dim32(3),
          dY.dim32(1),
          dY.dim32(2),
          kernel_h(),
          kernel_w(),
          stride_h(),
          stride_w(),
          pad_t(),
          pad_l(),
          dX->mutable_data<float>());
      break;
    case 3:
      Ave3DPoolBackwardNHWC<float><<<
          CAFFE_GET_BLOCKS(X.size()),
          CAFFE_CUDA_NUM_THREADS,
          0,
          context_.hip_stream()>>>(
          X.size(),
          dY.data<float>(),
          X.dim32(0),
          X.dim32(1),
          X.dim32(2),
          X.dim32(3),
          X.dim32(4),
          dY.dim32(1),
          dY.dim32(2),
          dY.dim32(3),
          kernel_h(),
          kernel_w(),
          kernel_[2],
          stride_h(),
          stride_w(),
          stride_[2],
          pad_t(),
          pad_l(),
          pads_[2],
          dX->mutable_data<float>());
      break;
    default:
      CAFFE_THROW("Unsupported pooling size : ", kernel_.size());
  }
  return true;
}


namespace {

template <typename T>
__global__ void MaxPool1DForwardNCHW(
    const int nthreads,
    const T* bottom_data,
    const int channels,
    const int height,
    const int pooled_height,
    const int kernel_h,
    const int stride_h,
    const int pad_t,
    T* top_data) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int ph = index % pooled_height;
    int c = (index / pooled_height) % channels;
    int n = index / pooled_height / channels;
    int hstart = ph * stride_h - pad_t;
    int hend = min(hstart + kernel_h, height);
    hstart = max(hstart, 0);
    T maxval = -FLT_MAX;
    const T* bdata_offset = bottom_data + n * channels * height;
    for (int h = hstart; h < hend; ++h) {
      int idx = c * height + h;
      if (bdata_offset[idx] > maxval) {
        maxval = bdata_offset[idx];
      }
    }
    top_data[index] = maxval;
  }
}

template <typename T>
__global__ void MaxPool2DForwardNCHW(
    const int nthreads,
    const T* bottom_data,
    const int channels,
    const int height,
    const int width,
    const int pooled_height,
    const int pooled_width,
    const int kernel_h,
    const int kernel_w,
    const int stride_h,
    const int stride_w,
    const int pad_t,
    const int pad_l,
    T* top_data) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h - pad_t;
    int wstart = pw * stride_w - pad_l;
    int hend = min(hstart + kernel_h, height);
    int wend = min(wstart + kernel_w, width);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    T maxval = -FLT_MAX;
    const T* bdata_offset = bottom_data + n * channels * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        int idx = c * height * width + h * width + w;
        if (bdata_offset[idx] > maxval) {
          maxval = bdata_offset[idx];
        }
      }
    }
    top_data[index] = maxval;
  }
}

template <typename T>
__global__ void MaxPool3DForwardNCHW(
    const int nthreads,
    const T* bottom_data,
    const int channels,
    const int height,
    const int width,
    const int depth,
    const int pooled_height,
    const int pooled_width,
    const int pooled_depth,
    const int kernel_h,
    const int kernel_w,
    const int kernel_d,
    const int stride_h,
    const int stride_w,
    const int stride_d,
    const int pad_t,
    const int pad_l,
    const int pad_f,
    T* top_data) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int pd = index % pooled_depth;
    int pw = (index / pooled_depth) % pooled_width;
    int ph = (index / pooled_depth / pooled_width) % pooled_height;
    int c = (index / pooled_depth / pooled_width / pooled_height) % channels;
    int n = index / pooled_depth / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h - pad_t;
    int wstart = pw * stride_w - pad_l;
    int hend = min(hstart + kernel_h, height);
    int wend = min(wstart + kernel_w, width);
    int dstart = pd * stride_d - pad_f;
    int dend = min(dstart + kernel_d, depth);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    dstart = max(dstart, 0);
    T maxval = -FLT_MAX;
    const T* bdata_offset = bottom_data + n * channels * height * width * depth;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        for (int d = dstart; d < dend; ++d) {
          int idx = ((c * height + h) * width + w) * depth + d;
          if (bdata_offset[idx] > maxval) {
            maxval = bdata_offset[idx];
          }
        }
      }
    }
    top_data[index] = maxval;
  }
}

template <typename T>
__global__ void MaxPool1DForwardNHWC(
    const int nthreads,
    const T* bottom_data,
    const int height,
    const int channels,
    const int pooled_height,
    const int kernel_h,
    const int stride_h,
    const int pad_t,
    T* top_data) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int n = index;
    int c = n % channels;
    n /= channels;
    int hstart = (n % pooled_height) * stride_h - pad_t;
    n /= pooled_height;
    int hend = min(hstart + kernel_h, height);
    hstart = max(hstart, 0);
    T maxval = -FLT_MAX;
    const T* bdata_offset = bottom_data + n * height * channels;
    for (int h = hstart; h < hend; ++h) {
      int idx = h * channels + c;
      if (bdata_offset[idx] > maxval) {
        maxval = bdata_offset[idx];
      }
    }
    top_data[index] = maxval;
  }
}

template <typename T>
__global__ void MaxPool2DForwardNHWC(
    const int nthreads,
    const T* bottom_data,
    const int height,
    const int width,
    const int channels,
    const int pooled_height,
    const int pooled_width,
    const int kernel_h,
    const int kernel_w,
    const int stride_h,
    const int stride_w,
    const int pad_t,
    const int pad_l,
    T* top_data) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int n = index;
    int c = n % channels;
    n /= channels;
    int wstart = (n % pooled_width) * stride_w - pad_l;
    n /= pooled_width;
    int hstart = (n % pooled_height) * stride_h - pad_t;
    n /= pooled_height;
    int hend = min(hstart + kernel_h, height);
    int wend = min(wstart + kernel_w, width);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    T maxval = -FLT_MAX;
    const T* bdata_offset = bottom_data + n * height * width * channels;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        int idx = (h * width + w) * channels + c;
        if (bdata_offset[idx] > maxval) {
          maxval = bdata_offset[idx];
        }
      }
    }
    top_data[index] = maxval;
  }
}

template <typename T>
__global__ void MaxPool3DForwardNHWC(
    const int nthreads,
    const T* bottom_data,
    const int height,
    const int width,
    const int depth,
    const int channels,
    const int pooled_height,
    const int pooled_width,
    const int pooled_depth,
    const int kernel_h,
    const int kernel_w,
    const int kernel_d,
    const int stride_h,
    const int stride_w,
    const int stride_d,
    const int pad_t,
    const int pad_l,
    const int pad_f,
    T* top_data) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int n = index;
    int c = n % channels;
    n /= channels;
    int dstart = (n % pooled_depth) * stride_d - pad_f;
    n /= pooled_depth;
    int wstart = (n % pooled_width) * stride_w - pad_l;
    n /= pooled_width;
    int hstart = (n % pooled_height) * stride_h - pad_t;
    n /= pooled_height;
    int hend = min(hstart + kernel_h, height);
    int wend = min(wstart + kernel_w, width);
    int dend = min(dstart + kernel_d, depth);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    dstart = max(dstart, 0);
    T maxval = -FLT_MAX;
    const T* bdata_offset = bottom_data + n * height * width * depth * channels;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        for (int d = dstart; d < dend; ++d) {
          int idx = ((h * width + w) * depth + d) * channels + c;
          if (bdata_offset[idx] > maxval) {
            maxval = bdata_offset[idx];
          }
        }
      }
    }
    top_data[index] = maxval;
  }
}

template <typename T>
__global__ void MaxPool1DBackwardNCHW(
    const int nthreads,
    const T* const bottom_data,
    const T* const top_data,
    const T* const top_diff,
    const int num,
    const int channels,
    const int height,
    const int pooled_height,
    const int kernel_h,
    const int stride_h,
    const int pad_t,
    T* const bottom_diff) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int h = index % height + pad_t;
    const int c = (index / height) % channels;
    const int n = index / height / channels;
    const int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int phend = min(h / stride_h + 1, pooled_height);
    const int top_offset = (n * channels + c) * pooled_height;
    bottom_diff[index] = 0;
    for (int ph = phstart; ph < phend; ++ph) {
      int top_local_offset = top_offset + ph;
      if (bottom_data[index] == top_data[top_local_offset]) {
        bottom_diff[index] += top_diff[top_local_offset];
      }
    }
  }
}

template <typename T>
__global__ void MaxPool2DBackwardNCHW(
    const int nthreads,
    const T* const bottom_data,
    const T* const top_data,
    const T* const top_diff,
    const int num,
    const int channels,
    const int height,
    const int width,
    const int pooled_height,
    const int pooled_width,
    const int kernel_h,
    const int kernel_w,
    const int stride_h,
    const int stride_w,
    const int pad_t,
    const int pad_l,
    T* const bottom_diff) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int w = index % width + pad_l;
    const int h = (index / width) % height + pad_t;
    const int c = (index / width / height) % channels;
    const int n = index / width / height / channels;
    const int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int phend = min(h / stride_h + 1, pooled_height);
    const int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    const int pwend = min(w / stride_w + 1, pooled_width);
    const int top_offset = (n * channels + c) * pooled_height * pooled_width;
    bottom_diff[index] = 0;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        int top_local_offset = top_offset + ph * pooled_width + pw;
        if (bottom_data[index] == top_data[top_local_offset]) {
          bottom_diff[index] += top_diff[top_local_offset];
        }
      }
    }
  }
}

template <typename T>
__global__ void MaxPool3DBackwardNCHW(
    const int nthreads,
    const T* const bottom_data,
    const T* const top_data,
    const T* const top_diff,
    const int num,
    const int channels,
    const int height,
    const int width,
    const int depth,
    const int pooled_height,
    const int pooled_width,
    const int pooled_depth,
    const int kernel_h,
    const int kernel_w,
    const int kernel_d,
    const int stride_h,
    const int stride_w,
    const int stride_d,
    const int pad_t,
    const int pad_l,
    const int pad_f,
    T* const bottom_diff) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int d = index % depth + pad_f;
    const int w = (index / depth) % width + pad_l;
    const int h = (index / depth / width) % height + pad_t;
    const int c = (index / depth / width / height) % channels;
    const int n = index / depth / width / height / channels;
    const int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int phend = min(h / stride_h + 1, pooled_height);
    const int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    const int pwend = min(w / stride_w + 1, pooled_width);
    const int pdstart = (d < kernel_d) ? 0 : (d - kernel_d) / stride_d + 1;
    const int pdend = min(d / stride_d + 1, pooled_depth);
    const int top_offset =
        (n * channels + c) * pooled_height * pooled_width * pooled_depth;
    bottom_diff[index] = 0;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        for (int pd = pdstart; pd < pdend; ++pd) {
          int top_local_offset =
              top_offset + (ph * pooled_width + pw) * pooled_depth + pd;
          if (bottom_data[index] == top_data[top_local_offset]) {
            bottom_diff[index] += top_diff[top_local_offset];
          }
        }
      }
    }
  }
}

template <typename T>
__global__ void MaxPool1DBackwardNHWC(
    const int nthreads,
    const T* const bottom_data,
    const T* const top_data,
    const T* const top_diff,
    const int num,
    const int height,
    const int channels,
    const int pooled_height,
    const int kernel_h,
    const int stride_h,
    const int pad_t,
    T* const bottom_diff) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int c = index % channels;
    const int h = (index / channels) % height + pad_t;
    const int n = index / channels / height;
    const int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int phend = min(h / stride_h + 1, pooled_height);
    const int top_offset = n * pooled_height * channels + c;
    bottom_diff[index] = 0;
    for (int ph = phstart; ph < phend; ++ph) {
      int top_local_offset = top_offset + ph * channels;
      if (bottom_data[index] == top_data[top_local_offset]) {
        bottom_diff[index] += top_diff[top_local_offset];
      }
    }
  }
}

template <typename T>
__global__ void MaxPool2DBackwardNHWC(
    const int nthreads,
    const T* const bottom_data,
    const T* const top_data,
    const T* const top_diff,
    const int num,
    const int height,
    const int width,
    const int channels,
    const int pooled_height,
    const int pooled_width,
    const int kernel_h,
    const int kernel_w,
    const int stride_h,
    const int stride_w,
    const int pad_t,
    const int pad_l,
    T* const bottom_diff) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int c = index % channels;
    const int w = index / channels % width + pad_l;
    const int h = (index / channels / width) % height + pad_t;
    const int n = index / channels / width / height;
    const int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int phend = min(h / stride_h + 1, pooled_height);
    const int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    const int pwend = min(w / stride_w + 1, pooled_width);
    const int top_offset =
        n * pooled_height * pooled_width * channels + c;
    bottom_diff[index] = 0;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        int top_local_offset = top_offset + (ph * pooled_width + pw) * channels;
        if (bottom_data[index] == top_data[top_local_offset]) {
          bottom_diff[index] += top_diff[top_local_offset];
        }
      }
    }
  }
}

template <typename T>
__global__ void MaxPool3DBackwardNHWC(
    const int nthreads,
    const T* const bottom_data,
    const T* const top_data,
    const T* const top_diff,
    const int num,
    const int height,
    const int width,
    const int depth,
    const int channels,
    const int pooled_height,
    const int pooled_width,
    const int pooled_depth,
    const int kernel_h,
    const int kernel_w,
    const int kernel_d,
    const int stride_h,
    const int stride_w,
    const int stride_d,
    const int pad_t,
    const int pad_l,
    const int pad_f,
    T* const bottom_diff) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int c = index % channels;
    const int d = index / channels % depth + pad_f;
    const int w = (index / depth / channels) % width + pad_l;
    const int h = (index / channels / depth / width) % height + pad_t;
    const int n = index / channels / depth / width / height;
    const int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int phend = min(h / stride_h + 1, pooled_height);
    const int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    const int pwend = min(w / stride_w + 1, pooled_width);
    const int pdstart = (d < kernel_d) ? 0 : (d - kernel_d) / stride_d + 1;
    const int pdend = min(d / stride_d + 1, pooled_depth);
    const int top_offset =
        n * pooled_height * pooled_width * pooled_depth * channels + c;
    bottom_diff[index] = 0;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        for (int pd = pdstart; pd < pdend; ++pd) {
          int top_local_offset = top_offset +
              ((ph * pooled_width + pw) * pooled_depth + d) * channels;
          if (bottom_data[index] == top_data[top_local_offset]) {
            bottom_diff[index] += top_diff[top_local_offset];
          }
        }
      }
    }
  }
}
}  // namespace

template <>
bool PoolOp<float, HIPContext, MaxPool>::RunOnDeviceWithOrderNCHW() {
  auto& X = Input(0);
  auto* Y = Output(0);
  ConvPoolOpBase<HIPContext>::SetOutputSize(X, Y, X.dim32(1));
  int output_size = Y->size();
  switch (kernel_.size()) {
    case 1:
      MaxPool1DForwardNCHW<float><<<
          CAFFE_GET_BLOCKS(output_size),
          CAFFE_CUDA_NUM_THREADS,
          0,
          context_.hip_stream()>>>(
          output_size,
          X.data<float>(),
          X.dim32(1),
          X.dim32(2),
          Y->dim32(2),
          kernel_h(),
          stride_h(),
          pad_t(),
          Y->mutable_data<float>());
      break;
    case 2:
      MaxPool2DForwardNCHW<float><<<
          CAFFE_GET_BLOCKS(output_size),
          CAFFE_CUDA_NUM_THREADS,
          0,
          context_.hip_stream()>>>(
          output_size,
          X.data<float>(),
          X.dim32(1),
          X.dim32(2),
          X.dim32(3),
          Y->dim32(2),
          Y->dim32(3),
          kernel_h(),
          kernel_w(),
          stride_h(),
          stride_w(),
          pad_t(),
          pad_l(),
          Y->mutable_data<float>());
      break;
    case 3:
      MaxPool3DForwardNCHW<float><<<
          CAFFE_GET_BLOCKS(output_size),
          CAFFE_CUDA_NUM_THREADS,
          0,
          context_.hip_stream()>>>(
          output_size,
          X.data<float>(),
          X.dim32(1),
          X.dim32(2),
          X.dim32(3),
          X.dim32(4),
          Y->dim32(2),
          Y->dim32(3),
          Y->dim32(4),
          kernel_h(),
          kernel_w(),
          kernel_[2],
          stride_h(),
          stride_w(),
          stride_[2],
          pad_t(),
          pad_l(),
          pads_[2],
          Y->mutable_data<float>());
      break;
    default:
      CAFFE_THROW("Unsupported pooling size : ", kernel_.size());
  }
  return true;
}

template <>
bool PoolOp<float, HIPContext, MaxPool>::RunOnDeviceWithOrderNHWC() {
  auto& X = Input(0);
  auto* Y = Output(0);
  ConvPoolOpBase<HIPContext>::SetOutputSize(X, Y, X.dim32(X.ndim() - 1));
  int output_size = Y->size();
  switch (kernel_.size()) {
    case 1:
      MaxPool1DForwardNHWC<float><<<
          CAFFE_GET_BLOCKS(output_size),
          CAFFE_CUDA_NUM_THREADS,
          0,
          context_.hip_stream()>>>(
          output_size,
          X.data<float>(),
          X.dim32(1),
          X.dim32(2),
          Y->dim32(1),
          kernel_h(),
          stride_h(),
          pad_t(),
          Y->mutable_data<float>());
      break;
    case 2:
      MaxPool2DForwardNHWC<float><<<
          CAFFE_GET_BLOCKS(output_size),
          CAFFE_CUDA_NUM_THREADS,
          0,
          context_.hip_stream()>>>(
          output_size,
          X.data<float>(),
          X.dim32(1),
          X.dim32(2),
          X.dim32(3),
          Y->dim32(1),
          Y->dim32(2),
          kernel_h(),
          kernel_w(),
          stride_h(),
          stride_w(),
          pad_t(),
          pad_l(),
          Y->mutable_data<float>());
      break;
    case 3:
      MaxPool3DForwardNHWC<float><<<
          CAFFE_GET_BLOCKS(output_size),
          CAFFE_CUDA_NUM_THREADS,
          0,
          context_.hip_stream()>>>(
          output_size,
          X.data<float>(),
          X.dim32(1),
          X.dim32(2),
          X.dim32(3),
          X.dim32(4),
          Y->dim32(1),
          Y->dim32(2),
          Y->dim32(3),
          kernel_h(),
          kernel_w(),
          kernel_[2],
          stride_h(),
          stride_w(),
          stride_[2],
          pad_t(),
          pad_l(),
          pads_[2],
          Y->mutable_data<float>());
      break;
    default:
      CAFFE_THROW("Unsupported pooling size : ", kernel_.size());
  }
  return true;
}

template <>
bool PoolGradientOp<float, HIPContext, MaxPool>::RunOnDeviceWithOrderNCHW() {
  auto& X = Input(0);
  auto& Y = Input(1);
  auto& dY = Input(2);
  CAFFE_ENFORCE_EQ(dY.ndim(), X.ndim());
  auto* dX = Output(0);
  dX->ResizeLike(X);
  vector<int> dims(X.dims().begin() + 2, X.dims().end());
  ConvPoolOpBase<HIPContext>::ComputePads(dims);
  switch (kernel_.size()) {
    case 1:
      MaxPool1DBackwardNCHW<float><<<
          CAFFE_GET_BLOCKS(X.size()),
          CAFFE_CUDA_NUM_THREADS,
          0,
          context_.hip_stream()>>>(
          X.size(),
          X.data<float>(),
          Y.data<float>(),
          dY.data<float>(),
          X.dim32(0),
          X.dim32(1),
          X.dim32(2),
          dY.dim32(2),
          kernel_h(),
          stride_h(),
          pad_t(),
          dX->mutable_data<float>());
      break;
    case 2:
      MaxPool2DBackwardNCHW<float><<<
          CAFFE_GET_BLOCKS(X.size()),
          CAFFE_CUDA_NUM_THREADS,
          0,
          context_.hip_stream()>>>(
          X.size(),
          X.data<float>(),
          Y.data<float>(),
          dY.data<float>(),
          X.dim32(0),
          X.dim32(1),
          X.dim32(2),
          X.dim32(3),
          dY.dim32(2),
          dY.dim32(3),
          kernel_h(),
          kernel_w(),
          stride_h(),
          stride_w(),
          pad_t(),
          pad_l(),
          dX->mutable_data<float>());
      break;
    case 3:
      MaxPool3DBackwardNCHW<float><<<
          CAFFE_GET_BLOCKS(X.size()),
          CAFFE_CUDA_NUM_THREADS,
          0,
          context_.hip_stream()>>>(
          X.size(),
          X.data<float>(),
          Y.data<float>(),
          dY.data<float>(),
          X.dim32(0),
          X.dim32(1),
          X.dim32(2),
          X.dim32(3),
          X.dim32(4),
          dY.dim32(2),
          dY.dim32(3),
          dY.dim32(4),
          kernel_h(),
          kernel_w(),
          kernel_[2],
          stride_h(),
          stride_w(),
          stride_[2],
          pad_t(),
          pad_l(),
          pads_[2],
          dX->mutable_data<float>());
      break;
    default:
      CAFFE_THROW("Unsupported pooling size : ", kernel_.size());
  }
  return true;
}

template <>
bool PoolGradientOp<float, HIPContext, MaxPool>::RunOnDeviceWithOrderNHWC() {
  auto& X = Input(0);
  auto& Y = Input(1);
  auto& dY = Input(2);
  CAFFE_ENFORCE_EQ(dY.ndim(), X.ndim());
  auto* dX = Output(0);
  dX->ResizeLike(X);
  vector<int> dims(X.dims().begin() + 1, X.dims().end() - 1);
  ConvPoolOpBase<HIPContext>::ComputePads(dims);
  switch (kernel_.size()) {
    case 1:
      MaxPool1DBackwardNHWC<float><<<
          CAFFE_GET_BLOCKS(X.size()),
          CAFFE_CUDA_NUM_THREADS,
          0,
          context_.hip_stream()>>>(
          X.size(),
          X.data<float>(),
          Y.data<float>(),
          dY.data<float>(),
          X.dim32(0),
          X.dim32(1),
          X.dim32(2),
          dY.dim32(1),
          kernel_h(),
          stride_h(),
          pad_t(),
          dX->mutable_data<float>());
    case 2:
      MaxPool2DBackwardNHWC<float><<<
          CAFFE_GET_BLOCKS(X.size()),
          CAFFE_CUDA_NUM_THREADS,
          0,
          context_.hip_stream()>>>(
          X.size(),
          X.data<float>(),
          Y.data<float>(),
          dY.data<float>(),
          X.dim32(0),
          X.dim32(1),
          X.dim32(2),
          X.dim32(3),
          dY.dim32(1),
          dY.dim32(2),
          kernel_h(),
          kernel_w(),
          stride_h(),
          stride_w(),
          pad_t(),
          pad_l(),
          dX->mutable_data<float>());
      break;
    case 3:
      MaxPool3DBackwardNHWC<float><<<
          CAFFE_GET_BLOCKS(X.size()),
          CAFFE_CUDA_NUM_THREADS,
          0,
          context_.hip_stream()>>>(
          X.size(),
          X.data<float>(),
          Y.data<float>(),
          dY.data<float>(),
          X.dim32(0),
          X.dim32(1),
          X.dim32(2),
          X.dim32(3),
          X.dim32(4),
          dY.dim32(1),
          dY.dim32(2),
          dY.dim32(3),
          kernel_h(),
          kernel_w(),
          kernel_[2],
          stride_h(),
          stride_w(),
          stride_[2],
          pad_t(),
          pad_l(),
          pads_[2],
          dX->mutable_data<float>());
      break;
    default:
      CAFFE_THROW("Unsupported pooling size : ", kernel_.size());
  }
  return true;
}

REGISTER_HIP_OPERATOR(AveragePool, PoolOp<float, HIPContext, AveragePool>);
REGISTER_HIP_OPERATOR(AveragePoolGradient,
                       PoolGradientOp<float, HIPContext, AveragePool>);

REGISTER_HIP_OPERATOR(AveragePool1D, PoolOp<float, HIPContext, AveragePool>);
REGISTER_HIP_OPERATOR(
    AveragePool1DGradient,
    PoolGradientOp<float, HIPContext, AveragePool>);

REGISTER_HIP_OPERATOR(AveragePool2D, PoolOp<float, HIPContext, AveragePool>);
REGISTER_HIP_OPERATOR(
    AveragePool2DGradient,
    PoolGradientOp<float, HIPContext, AveragePool>);

REGISTER_HIP_OPERATOR(AveragePool3D, PoolOp<float, HIPContext, AveragePool>);
REGISTER_HIP_OPERATOR(
    AveragePool3DGradient,
    PoolGradientOp<float, HIPContext, AveragePool>);

REGISTER_HIP_OPERATOR(MaxPool, PoolOp<float, HIPContext, MaxPool>);
REGISTER_HIP_OPERATOR(MaxPoolGradient,
                       PoolGradientOp<float, HIPContext, MaxPool>);

REGISTER_HIP_OPERATOR(MaxPool1D, PoolOp<float, HIPContext, MaxPool>);
REGISTER_HIP_OPERATOR(
    MaxPool1DGradient,
    PoolGradientOp<float, HIPContext, MaxPool>);

REGISTER_HIP_OPERATOR(MaxPool2D, PoolOp<float, HIPContext, MaxPool>);
REGISTER_HIP_OPERATOR(
    MaxPool2DGradient,
    PoolGradientOp<float, HIPContext, MaxPool>);

REGISTER_HIP_OPERATOR(MaxPool3D, PoolOp<float, HIPContext, MaxPool>);
REGISTER_HIP_OPERATOR(
    MaxPool3DGradient,
    PoolGradientOp<float, HIPContext, MaxPool>);
}  // namespace caffe2
