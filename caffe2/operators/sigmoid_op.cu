#include "hip/hip_runtime.h"
#include <cmath>

#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/elementwise_op.h"

namespace caffe2 {

template <typename T>
__global__ void SigmoidKernel(const int N, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    y[i] = 1. / (1. + exp(-x[i]));
  }
}

template <typename T>
__global__ void SigmoidGradientKernel(const int N, const T* y, const T* dy,
                              T* dx) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    dx[i] = dy[i] * y[i] * (1. - y[i]);
  }
}

struct SigmoidCUDAFunctor {
  template <typename T>
  inline void operator()(const int n, const T* x,
                         T* y, HIPContext* device_context) {
    SigmoidKernel<T><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS,
                    0, device_context->hip_stream()>>>(n, x, y);
    return;
  }
};

struct SigmoidGradientCUDAFunctor {
  template <typename T>
  inline void Run(const int n, const T* y, const T* dy,
                  T* dx, HIPContext* device_context) {
    SigmoidGradientKernel<T><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS,
                            0, device_context->hip_stream()>>>(n, y, dy, dx);
    return;
  }
};

REGISTER_HIP_OPERATOR(
    Sigmoid,
    UnaryElementwiseOp<TensorTypes<float>, HIPContext, SigmoidCUDAFunctor>);
REGISTER_HIP_OPERATOR(
    SigmoidGradient, BinaryElementwiseOp<
        TensorTypes<float>, HIPContext,
        WithoutBroadcast<SigmoidGradientCUDAFunctor>>);
}  // namespace caffe2
