#include "hip/hip_runtime.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/relu_op.h"

namespace caffe2 {
namespace {
template <typename T>
__global__ void ReluKernel(const int N, const T* X, T* Y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    Y[i] = X[i] > 0 ? X[i] : 0;
  }
}

template <typename T>
__global__ void ReluGradientKernel(const int N, const T* Y, const T* dY,
                              T* dX) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    dX[i] = Y[i] > 0 ? dY[i] : 0;
  }
}
}  // namespace

template <>
bool ReluOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);
  auto* Y = Output(0);
  CAFFE_ENFORCE_GT(X.size(), 0);
  Y->ResizeLike(X);
  ReluKernel<<<CAFFE_GET_BLOCKS(X.size()), CAFFE_CUDA_NUM_THREADS,
               0, context_.hip_stream()>>>(
      X.size(), X.data<float>(), Y->mutable_data<float>());
  return true;
}

template <>
bool ReluGradientOp<float, HIPContext>::RunOnDevice() {
  auto& Y = Input(0);
  auto& dY = Input(1);
  auto* dX = Output(0);
  CAFFE_ENFORCE_GT(Y.size(), 0);
  CAFFE_ENFORCE_EQ(dY.size(), Y.size());
  dX->ResizeLike(Y);
  ReluGradientKernel<<<CAFFE_GET_BLOCKS(Y.size()), CAFFE_CUDA_NUM_THREADS,
                       0, context_.hip_stream()>>>(
      Y.size(), Y.data<float>(), dY.data<float>(), dX->mutable_data<float>());
  return true;
}

REGISTER_HIP_OPERATOR(Relu, ReluOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(ReluGradient, ReluGradientOp<float, HIPContext>);
}  // namespace caffe2
