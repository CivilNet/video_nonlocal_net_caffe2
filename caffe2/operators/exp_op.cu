#include "hip/hip_runtime.h"
#include <cmath>

#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/elementwise_op.h"

namespace caffe2 {

template <typename T>
__global__ void ExpKernel(const int N, const T* X, T* Y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    Y[i] = __expf(X[i]);
  }
}

struct ExpCUDAFunctor {
  template <typename T>
  inline void operator()(const int n, const T* x,
                         T* y, HIPContext* device_context) {
    ExpKernel<T><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS,
                    0, device_context->hip_stream()>>>(n, x, y);
    return;
  }
  inline bool InplaceAllowed() {
    return true;
  }
};

REGISTER_HIP_OPERATOR(
    Exp, UnaryElementwiseOp<TensorTypes<float>, HIPContext, ExpCUDAFunctor>);
}  // namespace caffe2
