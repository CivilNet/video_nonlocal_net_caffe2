#include "hip/hip_runtime.h"
// Implements the math functions for CPU.

#include "caffe2/utils/math.h"

#include <cub/block/block_reduce.cuh>
#include <hipcub/hipcub.hpp>

#include "caffe2/core/context_gpu.h"
#include "caffe2/utils/conversions.h"

#if THRUST_VERSION >= 100800
#define THRUST_SUPPORTS_PER_THREAD
#endif  // THRUST_VERSION >= 100800

namespace caffe2 {
namespace math {

#define DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(T, Funcname, function)             \
__global__                                                                     \
void _Kernel_##T##_##Funcname(const int N, const T* x, T* y) {                 \
  CUDA_1D_KERNEL_LOOP(i, N) {                                                  \
    y[i] = function(x[i]);                                                     \
  }                                                                            \
}                                                                              \
template <>                                                                    \
void Funcname<T, HIPContext>(                                                 \
    const int N, const T* x, T* y,                                             \
    HIPContext* context) {                                                    \
  _Kernel_##T##_##Funcname<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS,      \
                                 0, context->hip_stream()>>>(                 \
      N, x, y);                                                                \
}

DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Exp, expf);
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Log, logf);
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Cos, cosf);
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Sin, sinf);
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Abs, fabsf);
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Sqrt, sqrtf);
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, InvSqrt, rsqrtf);

__device__ float cuda_sqrf(const float x) { return x * x; }

DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Sqr, cuda_sqrf);

#undef DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION

#define DELEGATE_SINCOS_CUDA_FUNCTION(T)                             \
  __global__ void _Kernel_##T##_##SinCos(                            \
      const int N, const T* x, T* ys, T* yc) {                       \
    CUDA_1D_KERNEL_LOOP(i, N) {                                      \
      sincos(x[i], ys + i, yc + i);                                  \
    }                                                                \
  }                                                                  \
  template <>                                                        \
  void SinCos<T, HIPContext>(                                       \
      const int N, const T* x, T* ys, T* yc, HIPContext* context) { \
    _Kernel_##T##_##SinCos<<<                                        \
        CAFFE_GET_BLOCKS(N),                                         \
        CAFFE_CUDA_NUM_THREADS,                                      \
        0,                                                           \
        context->hip_stream()>>>(N, x, ys, yc);                     \
  }

DELEGATE_SINCOS_CUDA_FUNCTION(float)
DELEGATE_SINCOS_CUDA_FUNCTION(double)

#undef DELEGATE_SINCOS_CUDA_FUNCTION

#define DELEGATE_SIMPLE_CUDA_BINARY_INFIX_FUNCTION(T, Funcname, expr)         \
  __global__ void _Kernel_##T##_##Funcname(                                   \
      const int N, const T* a, const T* b, T* y) {                            \
    CUDA_1D_KERNEL_LOOP(i, N) {                                               \
      float r = convert::To<T, float>(a[i]) expr convert::To<T, float>(b[i]); \
      y[i] = convert::To<float, T>(r);                                        \
    }                                                                         \
  }                                                                           \
  template <>                                                                 \
  void Funcname<T, HIPContext>(                                              \
      const int N, const T* a, const T* b, T* y, HIPContext* context) {      \
    _Kernel_##T##_##Funcname<<<                                               \
        CAFFE_GET_BLOCKS(N),                                                  \
        CAFFE_CUDA_NUM_THREADS,                                               \
        0,                                                                    \
        context->hip_stream()>>>(N, a, b, y);                                \
  }

DELEGATE_SIMPLE_CUDA_BINARY_INFIX_FUNCTION(float, Add, +);
DELEGATE_SIMPLE_CUDA_BINARY_INFIX_FUNCTION(int32_t, Add, +);
DELEGATE_SIMPLE_CUDA_BINARY_INFIX_FUNCTION(float, Sub, -);
DELEGATE_SIMPLE_CUDA_BINARY_INFIX_FUNCTION(float, Mul, *);
DELEGATE_SIMPLE_CUDA_BINARY_INFIX_FUNCTION(float, Div, /);

DELEGATE_SIMPLE_CUDA_BINARY_INFIX_FUNCTION(float16, Add, +);
DELEGATE_SIMPLE_CUDA_BINARY_INFIX_FUNCTION(float16, Sub, -);
DELEGATE_SIMPLE_CUDA_BINARY_INFIX_FUNCTION(float16, Mul, *);
DELEGATE_SIMPLE_CUDA_BINARY_INFIX_FUNCTION(float16, Div, /);

#undef DELEGATE_SIMPLE_CUDA_BINARY_INFIX_FUNCTION

#define DELEGATE_SIMPLE_CUDA_BINARY_PREFIX_FUNCTION(T, Funcname, func)    \
  __global__ void _Kernel_##T##_##Funcname(                               \
      const int N, const T* a, const T* b, T* y) {                        \
    CUDA_1D_KERNEL_LOOP(i, N) {                                           \
      float r =                                                           \
          func(convert::To<T, float>(a[i]), convert::To<T, float>(b[i])); \
      y[i] = convert::To<float, T>(r);                                    \
    }                                                                     \
  }                                                                       \
  template <>                                                             \
  void Funcname<T, HIPContext>(                                          \
      const int N, const T* a, const T* b, T* y, HIPContext* context) {  \
    _Kernel_##T##_##Funcname<<<                                           \
        CAFFE_GET_BLOCKS(N),                                              \
        CAFFE_CUDA_NUM_THREADS,                                           \
        0,                                                                \
        context->hip_stream()>>>(N, a, b, y);                            \
  }

DELEGATE_SIMPLE_CUDA_BINARY_PREFIX_FUNCTION(float, ElemwiseMax, fmaxf);

#undef DELEGATE_SIMPLE_CUDA_BINARY_INFIX_FUNCTION

#define DELEGATE_REDUCTION_FUNCTION(T, Funcname, func)                  \
  template <>                                                           \
  void Funcname<T, HIPContext>(                                        \
      const int N,                                                      \
      const T* src,                                                     \
      T* dst,                                                           \
      Tensor<HIPContext>* scratch_ptr,                                 \
      HIPContext* context) {                                           \
    size_t memRequired = 0;                                             \
    hipcub::DeviceReduce::func(                                            \
        nullptr, memRequired, src, dst, N, context->hip_stream());     \
    auto buffer_size =                                                  \
        static_cast<TIndex>((memRequired + sizeof(T) - 1) / sizeof(T)); \
    scratch_ptr->Resize(std::vector<TIndex>{buffer_size});              \
    hipcub::DeviceReduce::func(                                            \
        static_cast<void*>(scratch_ptr->mutable_data<T>()),             \
        memRequired,                                                    \
        src,                                                            \
        dst,                                                            \
        N,                                                              \
        context->hip_stream());                                        \
  }

DELEGATE_REDUCTION_FUNCTION(float, ReduceMin, Min)
DELEGATE_REDUCTION_FUNCTION(float, ReduceMax, Max)
DELEGATE_REDUCTION_FUNCTION(int32_t, ReduceMax, Max)
DELEGATE_REDUCTION_FUNCTION(int64_t, ReduceMax, Max)


#undef DELEGATE_REDUCTION_FUNCTION

// Caffe2 gemm provides a simpler interface to the gemm functions, with the
// limitation that the data has to be contiguous in memory.
template <>
void Gemm<float, HIPContext>(
    const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB,
    const int M,
    const int N,
    const int K,
    const float alpha,
    const float* A,
    const float* B,
    const float beta,
    float* C,
    HIPContext* context,
    TensorProto::DataType math_type) {
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_ENFORCE(hipblasSgemm(
      context->cublas_handle(),
      cuTransB,
      cuTransA,
      N,
      M,
      K,
      &alpha,
      B,
      ldb,
      A,
      lda,
      &beta,
      C,
      N));
}

template <>
void Gemm<float16, HIPContext>(
    const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB,
    const int M,
    const int N,
    const int K,
    const float alpha,
    const float16* A,
    const float16* B,
    const float beta,
    float16* C,
    HIPContext* context,
    TensorProto::DataType math_type) {
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  if (math_type == TensorProto_DataType_FLOAT) {
    CUBLAS_CHECK(cublasSgemmEx(
        context->cublas_handle(),
        cuTransB,
        cuTransA,
        N,
        M,
        K,
        &alpha,
        B,
        HIP_R_16F,
        ldb,
        A,
        HIP_R_16F,
        lda,
        &beta,
        C,
        HIP_R_16F,
        N));

  } else if (math_type == TensorProto_DataType_FLOAT16) {
    // convert alpha, beta from float -> __half
    auto alpha_fp16 = convert::floatToHalf(alpha);
    auto beta_fp16 = convert::floatToHalf(beta);

    // call hipblasHgemm
    CUBLAS_CHECK(hipblasHgemm(
        context->cublas_handle(),
        cuTransB,
        cuTransA,
        N,
        M,
        K,
        &alpha_fp16,
        (const __half*)B,
        ldb,
        (const __half*)A,
        lda,
        &beta_fp16,
        (__half*)C,
        N));
  } else {
    // fail
    CAFFE_THROW("Unsupported math type");
  }
}

template <>
void GemmBatched<float, HIPContext>(
    const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB,
    const int batch_size,
    const int M,
    const int N,
    const int K,
    const float alpha,
    const float* A,
    const float* B,
    const float beta,
    float* C,
    HIPContext* context,
    Tensor<HIPContext>* scratch,
    TensorProto::DataType math_type) {
  const int a_stride = M * K;
  const int b_stride = K * N;
  const int c_stride = M * N;
#if __CUDACC_VER_MAJOR__ < 8
  // loop over matrices in the batch
  for (int i = 0; i < batch_size; ++i) {
    math::Gemm<float, HIPContext>(
        TransA,
        TransB,
        M,
        N,
        K,
        alpha,
        A + a_stride * i,
        B + b_stride * i,
        beta,
        C + c_stride * i,
        context);
  }
#else
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  const int lda = (TransA == CblasNoTrans) ? K : M;
  const int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_ENFORCE(hipblasSgemmStridedBatched(
      context->cublas_handle(),
      cuTransB,
      cuTransA,
      N,
      M,
      K,
      &alpha,
      B,
      ldb,
      b_stride,
      A,
      lda,
      a_stride,
      &beta,
      C,
      N,
      c_stride,
      batch_size));
#endif
}

namespace {

__global__ void FloatToHalfKernel(const int N, const float* X, half* Y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    Y[i] = __float2half(X[i]);
  }
}

__global__ void HalfToFloatKernel(const int N, const half* X, float* Y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    Y[i] = __half2float(X[i]);
  }
}

};

template <>
void GemmBatched<float16, HIPContext>(
    const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB,
    const int batch_size,
    const int M,
    const int N,
    const int K,
    const float alpha,
    const float16* A,
    const float16* B,
    const float beta,
    float16* C,
    HIPContext* context,
    Tensor<HIPContext>* scratch,
    TensorProto::DataType math_type) {
  const int a_stride = M * K;
  const int b_stride = K * N;
  const int c_stride = M * N;
#if __CUDACC_VER_MAJOR__ < 8
  // loop over matrices in the batch
  for (int i = 0; i < batch_size; ++i) {
    math::Gemm<float16, HIPContext>(
        TransA,
        TransB,
        M,
        N,
        K,
        alpha,
        A + a_stride * i,
        B + b_stride * i,
        beta,
        C + c_stride * i,
        context);
  }
#else
  // 3 options:
  // 1) scratch != null = cast to fp32, SgemmStridedBatched, cast result to fp16
  // 2) math_type == FLOAT, scratch == nullptr = looped SgemmEx
  // 3) math_type == FLOAT16, scratch == nullptr = batched Hgemm

  if (scratch != nullptr) {
    const int A_size = a_stride * batch_size;
    const int B_size = b_stride * batch_size;
    // cast, hipblasSgemmStridedBatched, cast
    size_t in_elems = A_size + B_size;
    size_t out_elems = c_stride * batch_size;

    scratch->Resize(in_elems + out_elems);
    float* scratch_ptr = scratch->mutable_data<float>();

    float* A_fp32 = scratch_ptr;
    float* B_fp32 = scratch_ptr + A_size;
    float* C_fp32 = scratch_ptr + A_size + B_size;

    // cast A, B into fp32
    HalfToFloatKernel<<<CAFFE_GET_BLOCKS(A_size),
                        CAFFE_CUDA_NUM_THREADS,
                        0,
                        context->hip_stream()>>>(A_size, (half*)A, A_fp32);
    HalfToFloatKernel<<<CAFFE_GET_BLOCKS(B_size),
                        CAFFE_CUDA_NUM_THREADS,
                        0,
                        context->hip_stream()>>>(B_size, (half*)B, B_fp32);

    // run fp32 batched Gemm
    GemmBatched<float, HIPContext>(
        TransA,
        TransB,
        batch_size,
        M,
        N,
        K,
        alpha,
        A_fp32,
        B_fp32,
        beta,
        C_fp32,
        context);

    // cast result back to fp16
    FloatToHalfKernel<<<
        CAFFE_GET_BLOCKS(batch_size * M * N),
        CAFFE_CUDA_NUM_THREADS,
        0,
        context->hip_stream()>>>(batch_size * M * N, C_fp32, (half*)C);
  } else {
    if (math_type == TensorProto_DataType_FLOAT) {
      // loop over matrices in the batch
      for (int i = 0; i < batch_size; ++i) {
        math::Gemm<float16, HIPContext>(
            TransA,
            TransB,
            M,
            N,
            K,
            alpha,
            A + a_stride * i,
            B + b_stride * i,
            beta,
            C + c_stride * i,
            context);
      }
    } else if (math_type == TensorProto_DataType_FLOAT16) {
      // Note that cublas follows fortran order, so the order is different from
      // the cblas convention.
      const int lda = (TransA == CblasNoTrans) ? K : M;
      const int ldb = (TransB == CblasNoTrans) ? N : K;
      hipblasOperation_t cuTransA =
          (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
      hipblasOperation_t cuTransB =
          (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;

      // convert alpha, beta from float -> __half
      auto alpha_fp16 = convert::floatToHalf(alpha);
      auto beta_fp16 = convert::floatToHalf(beta);
      CUBLAS_ENFORCE(hipblasHgemmStridedBatched(
          context->cublas_handle(),
          cuTransB,
          cuTransA,
          N,
          M,
          K,
          &alpha_fp16,
          (const __half*)B,
          ldb,
          b_stride,
          (const __half*)A,
          lda,
          a_stride,
          &beta_fp16,
          (__half*)C,
          N,
          c_stride,
          batch_size));
    }
  }
#endif
}

#if CUDA_VERSION >= 9000

// No change, but required. Defer to default CUDA engine
template <>
void Gemm<float, HIPContext, TensorCoreEngine>(
    const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB,
    const int M,
    const int N,
    const int K,
    const float alpha,
    const float* A,
    const float* B,
    const float beta,
    float* C,
    HIPContext* context,
    TensorProto::DataType math_type) {
  return Gemm<float,HIPContext>(TransA,
                                 TransB,
                                 M,
                                 N,
                                 K,
                                 alpha,
                                 A,
                                 B,
                                 beta,
                                 C,
                                 context,
                                 math_type);
}

template <>
void Gemm<float16, HIPContext, TensorCoreEngine>(
    const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB,
    const int M,
    const int N,
    const int K,
    const float alpha,
    const float16* A,
    const float16* B,
    const float beta,
    float16* C,
    HIPContext* context,
    TensorProto::DataType math_type) {
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;

  // enable TensorCore for this call on this handle
  if (TensorCoreAvailable()) {
    CUBLAS_ENFORCE(hipblasSetMathMode(
        context->cublas_handle(),
        HIPBLAS_TENSOR_OP_MATH));
  }

  CUBLAS_CHECK(hipblasGemmEx(
      context->cublas_handle(),
      cuTransB,
      cuTransA,
      N,
      M,
      K,
      &alpha,
      B,
      HIP_R_16F,
      ldb,
      A,
      HIP_R_16F,
      lda,
      &beta,
      C,
      HIP_R_16F,
      N,
      HIP_R_32F,
      CUBLAS_GEMM_DFALT_TENSOR_OP));

  // Now disable TensorCore math for subsequent calls to this handle
  if (TensorCoreAvailable()) {
    CUBLAS_ENFORCE(hipblasSetMathMode(
        context->cublas_handle(),
        HIPBLAS_DEFAULT_MATH));
  }
}

template <>
void GemmBatched<float, HIPContext, TensorCoreEngine>(
    const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB,
    const int batch_size,
    const int M,
    const int N,
    const int K,
    const float alpha,
    const float* A,
    const float* B,
    const float beta,
    float* C,
    HIPContext* context,
    Tensor<HIPContext>* scratch,
    TensorProto::DataType math_type) {
  return GemmBatched<float, HIPContext, DefaultEngine>(
      TransA,
      TransB,
      batch_size,
      M,
      N,
      K,
      alpha,
      A,
      B,
      beta,
      C,
      context,
      scratch,
      math_type);
}

template <>
void GemmBatched<float16, HIPContext, TensorCoreEngine>(
    const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB,
    const int batch_size,
    const int M,
    const int N,
    const int K,
    const float alpha,
    const float16* A,
    const float16* B,
    const float beta,
    float16* C,
    HIPContext* context,
    Tensor<HIPContext>* scratch,
    TensorProto::DataType math_type) {
  return GemmBatched<float16, HIPContext, DefaultEngine>(
      TransA,
      TransB,
      batch_size,
      M,
      N,
      K,
      alpha,
      A,
      B,
      beta,
      C,
      context,
      scratch,
      math_type);
}

#endif // CUDA_VERSION >= 9000

template <>
void GemmEx<float, HIPContext>(
    const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB,
    const int M,
    const int N,
    const int K,
    const float alpha,
    const float* A,
    const int lda,
    const float* B,
    const int ldb,
    const float beta,
    float* C,
    const int ldc,
    HIPContext* context) {
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_ENFORCE(hipblasSgemm(
      context->cublas_handle(),
      cuTransB,
      cuTransA,
      N,
      M,
      K,
      &alpha,
      B,
      ldb,
      A,
      lda,
      &beta,
      C,
      ldc));
}

template <>
void Gemv<float, HIPContext>(
    const CBLAS_TRANSPOSE TransA,
    const int M,
    const int N,
    const float alpha,
    const float* A,
    const float* x,
    const float beta,
    float* y,
    HIPContext* context,
    TensorProto::DataType math_type) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_ENFORCE(hipblasSgemv(
      context->cublas_handle(),
      cuTransA,
      N,
      M,
      &alpha,
      A,
      N,
      x,
      1,
      &beta,
      y,
      1));
}

// Batched Add variants
namespace {

template <typename T>
__global__ void AddStripedBatchKernel(
    const int N,
    const T* first,
    T* Y,
    const int stripe,
    const int batch) {
  for (int j = 0; j < batch; j++) {
    const T* x = first + j * stripe;
    CUDA_1D_KERNEL_LOOP(i, N) {
      float tmpY = convert::To<T, float>(Y[i]);
      tmpY += convert::To<T,float>(x[i]);
      Y[i] = convert::To<float,T>(tmpY);
    }
  }
}
} // namespace

#define CAFFE2_SPECIALIZED_CUDA_ADD_STRIPED_BATCH(T)           \
  template <>                                                  \
  void AddStripedBatch<T, HIPContext>(                        \
      const int N,                                             \
      const T* first,                                          \
      T* Y,                                                    \
      const int stripe,                                        \
      const int batch,                                         \
      HIPContext* context) {                                  \
    AddStripedBatchKernel<T><<<                                \
        CAFFE_GET_BLOCKS(N),                                   \
        CAFFE_CUDA_NUM_THREADS,                                \
        0,                                                     \
        context->hip_stream()>>>(N, first, Y, stripe, batch); \
  }

CAFFE2_SPECIALIZED_CUDA_ADD_STRIPED_BATCH(float);
CAFFE2_SPECIALIZED_CUDA_ADD_STRIPED_BATCH(float16);
#undef CAFFE2_SPECIALIZED_CUDA_ADD_STRIPED_BATCH

template <>
void Gemv<float16, HIPContext>(
    const CBLAS_TRANSPOSE TransA,
    const int M,
    const int N,
    const float alpha,
    const float16* A,
    const float16* x,
    const float beta,
    float16* y,
    HIPContext* context,
    TensorProto::DataType math_type) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;

  // sort out what we need to call cublasSgemmEx / hipblasHgemm
  int m = (cuTransA == HIPBLAS_OP_N) ? N : M;
  int k = (cuTransA == HIPBLAS_OP_N) ? M : N;
  int LDA = (cuTransA == HIPBLAS_OP_N) ? m : k;
  int LDC = m;

  if (math_type == TensorProto_DataType_FLOAT) {
    CUBLAS_CHECK(cublasSgemmEx(
        context->cublas_handle(),
        cuTransA,
        HIPBLAS_OP_N,
        m,
        1,
        k,
        &alpha,
        A,
        HIP_R_16F,
        LDA,
        x,
        HIP_R_16F,
        k,
        &beta,
        y,
        HIP_R_16F,
        LDC));
  } else if (math_type == TensorProto_DataType_FLOAT16) {
    auto alpha_fp16 = convert::floatToHalf(alpha);
    auto beta_fp16 = convert::floatToHalf(beta);

    CUBLAS_CHECK(hipblasHgemm(
        context->cublas_handle(),
        cuTransA,
        HIPBLAS_OP_N,
        m,
        1,
        k,
        &alpha_fp16,
        (const __half*)A,
        LDA,
        (const __half*)x,
        k,
        &beta_fp16,
        (__half*)y,
        LDC));
  } else {
    // fail
    CAFFE_THROW("Unsupported math type");
  }
}

namespace {
template <typename T>
__global__ void SetKernel(const int N, const T alpha, T* Y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    Y[i] = alpha;
  }
}
} // namespace

#define CAFFE2_SPECIALIZED_CUDA_SET(T)                             \
  template <>                                                      \
  void Set<T, HIPContext>(                                        \
      const size_t N, const T alpha, T* Y, HIPContext* context) { \
    SetKernel<<<                                                   \
        CAFFE_GET_BLOCKS(N),                                       \
        CAFFE_CUDA_NUM_THREADS,                                    \
        0,                                                         \
        context->hip_stream()>>>(N, alpha, Y);                    \
  }

CAFFE2_SPECIALIZED_CUDA_SET(float);
CAFFE2_SPECIALIZED_CUDA_SET(double);
CAFFE2_SPECIALIZED_CUDA_SET(bool);
CAFFE2_SPECIALIZED_CUDA_SET(int8_t);
CAFFE2_SPECIALIZED_CUDA_SET(int16_t);
CAFFE2_SPECIALIZED_CUDA_SET(float16);
CAFFE2_SPECIALIZED_CUDA_SET(int);
CAFFE2_SPECIALIZED_CUDA_SET(int64_t);
CAFFE2_SPECIALIZED_CUDA_SET(char);
CAFFE2_SPECIALIZED_CUDA_SET(uint8_t);
CAFFE2_SPECIALIZED_CUDA_SET(uint16_t);
#undef CAFFE2_SPECIALIZED_CUDA_SET

namespace {
template <typename T>
__global__ void
UniformShift(const size_t N, const float min, const float max, T* x) {
  float scale = max - min;
  CUDA_1D_KERNEL_LOOP(i, N) {
    x[i] = convert::To<float, T>(convert::To<T, float>(x[i]) * scale + min);
  }
}

__global__ void
UniformIntFit(const size_t N, const int min, const int max, unsigned int* x) {
  int* x_int = reinterpret_cast<int*>(x);
  int range = (max - min + 1);
  CUDA_1D_KERNEL_LOOP(i, N) {
    x_int[i] = min + static_cast<int>(x[i] % range);
  }
}
} // namespace

template <>
void RandUniform<float, HIPContext>(
    const size_t n,
    const float min,
    const float max,
    float* r,
    HIPContext* context) {
  CURAND_ENFORCE(hiprandGenerateUniform(context->curand_generator(), r, n));
  UniformShift<float>
      <<<CAFFE_GET_BLOCKS(n),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(n, min, max, r);
}

template <>
void RandUniform<double, HIPContext>(
    const size_t n,
    const double min,
    const double max,
    double* r,
    HIPContext* context) {
  CURAND_ENFORCE(
      hiprandGenerateUniformDouble(context->curand_generator(), r, n));
  UniformShift<double>
      <<<CAFFE_GET_BLOCKS(n),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(n, min, max, r);
}

template <>
void RandUniform<int, HIPContext>(
    const size_t n,
    const int min,
    const int max,
    int* r,
    HIPContext* context) {
  CURAND_ENFORCE(hiprandGenerate(
      context->curand_generator(), reinterpret_cast<unsigned int*>(r), n));
  UniformIntFit<<<
      CAFFE_GET_BLOCKS(n),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context->hip_stream()>>>(
      n, min, max, reinterpret_cast<unsigned int*>(r));
}

template <typename T>
size_t HandleOddLengthRandGaussian(
    const size_t n,
    const T mean,
    const T std,
    T* r,
    HIPContext* context) {
  if (n % 2 == 1) {
    std::default_random_engine generator;
    std::normal_distribution<T> distribution(mean, std);
    const T random_value = distribution(generator);
    math::Set<T, HIPContext>(1, random_value, r + (n - 1), context);
    return n - 1;
  }
  return n;
}

template <>
void RandGaussian<float, HIPContext>(
    const size_t n,
    const float mean,
    const float std,
    float* r,
    HIPContext* context) {
  // If n is odd, we add a random Gaussian value at the end manually
  // and generate n-1 random values using hiprandGenerateNormal.
  // hiprandGenerateNormal requires n to be even.
  const size_t even_n =
      HandleOddLengthRandGaussian<float>(n, mean, std, r, context);
  CURAND_ENFORCE(
      hiprandGenerateNormal(context->curand_generator(), r, even_n, mean, std));
}

template <>
void RandGaussian<double, HIPContext>(
    const size_t n,
    const double mean,
    const double std,
    double* r,
    HIPContext* context) {
  const size_t even_n =
      HandleOddLengthRandGaussian<double>(n, mean, std, r, context);
  CURAND_ENFORCE(hiprandGenerateNormalDouble(
      context->curand_generator(), r, even_n, mean, std));
}

template <>
void Dot<float, HIPContext>(
    const int n,
    const float* a,
    const float* b,
    float* y,
    HIPContext* context) {
  float result;
  CUBLAS_ENFORCE(hipblasSdot(context->cublas_handle(), n, a, 1, b, 1, &result));
  context->Copy<float, CPUContext, HIPContext>(1, &result, y);
}

template <>
void Dot<float16, HIPContext>(
    const int n,
    const float16* a,
    const float16* b,
    float16* y,
    HIPContext* context) {
  float16 result;
  // execute with 32-bit math
  CUBLAS_CHECK(hipblasDotEx(
      context->cublas_handle(),
      n,
      a,
      HIP_R_16F,
      1,
      b,
      HIP_R_16F,
      1,
      &result,
      HIP_R_16F,
      HIP_R_32F));
  context->Copy<float16, CPUContext, HIPContext>(1, &result, y);
}

// A previous version of caffe2 used Thrust but it turns out that thrust
// reduction has an implicit scratch space allocation and deallocation, which
// may interfere with NCCL and create a deadlock. Hence we are using a custom
// reduction here.
#define SUM_KERNEL_NTHREADS 128
template <typename T>
__global__ void SumKernel(const int N, const T* X, T* Y, bool square) {
  const int idx = threadIdx.x;
  __shared__ float reduction_buffer[SUM_KERNEL_NTHREADS];

  reduction_buffer[idx] = 0;

  // A multilevel reduction.
  // N -> 128
  if (!square) {
    for (int i = idx; i < N; i += SUM_KERNEL_NTHREADS) {
      reduction_buffer[idx] += convert::To<T, float>(X[i]);
    }
  } else {
    for (int i = idx; i < N; i += SUM_KERNEL_NTHREADS) {
      float Xi = convert::To<T, float>(X[i]);
      reduction_buffer[idx] += Xi * Xi;
    }
  }
  __syncthreads();
  // 128 -> 32
  if (idx < 32) {
    reduction_buffer[idx] +=
        reduction_buffer[idx + 32] +
        reduction_buffer[idx + 64] +
        reduction_buffer[idx + 96];
  }
  __syncthreads();
  // 32 -> 1
  if (idx == 0) {
    float tmp = 0;
    for (int i = 0; i < 32; ++i) {
      tmp += reduction_buffer[i];
    }
    *Y = convert::To<float, T>(tmp);
  }
}

// According to the benchmarks script
// caffe2/caffe2/experiments/python/device_reduce_sum_bench.py,
// device reduce is slower for N <= 10000.
#define DEVICE_REDUCE_SIZE_THRESHOLD 10000

namespace {

template <typename T>
__global__ void SumConvertKernel(float* sum, T* dest) {
  *dest = convert::To<float, T>(*sum);
}

template <typename T, typename IterT>
void SumGenericIter(
    const int N,
    IterT it,
    T*& dest,
    HIPContext* context,
    Tensor<HIPContext>* scratch_ptr) {
  size_t memRequired = 0;
  hipcub::DeviceReduce::Sum(
      nullptr, memRequired, it, dest, N, context->hip_stream());
  auto buffer_size =
      static_cast<TIndex>((memRequired + sizeof(T) - 1) / sizeof(T));
  if (!dest) {
    // allocate one more T at the end of scratch for dest
    scratch_ptr->Resize(std::vector<TIndex>{buffer_size + 1});
    dest = scratch_ptr->template mutable_data<T>() + buffer_size;
  } else {
    scratch_ptr->Resize(std::vector<TIndex>{buffer_size});
  }
  hipcub::DeviceReduce::Sum(
      static_cast<void*>(scratch_ptr->template mutable_data<T>()),
      memRequired,
      it,
      dest,
      N,
      context->hip_stream());
}
} // namespace

template <>
void Sum<float, HIPContext>(
    const int N,
    const float* x,
    float* y,
    HIPContext* context,
    Tensor<HIPContext>* scratch_ptr) {
  if (scratch_ptr && N > DEVICE_REDUCE_SIZE_THRESHOLD) {
    SumGenericIter<float>(N, x, y, context, scratch_ptr);
  } else {
    SumKernel<<<1, SUM_KERNEL_NTHREADS, 0, context->hip_stream()>>>(
      N, x, y, false);
  }
}

template <>
void Sum<int32_t, HIPContext>(
    const int N,
    const int32_t* x,
    int32_t* y,
    HIPContext* context,
    Tensor<HIPContext>* scratch_ptr) {
  if (scratch_ptr && N > DEVICE_REDUCE_SIZE_THRESHOLD) {
    SumGenericIter<int32_t>(N, x, y, context, scratch_ptr);
  } else {
    SumKernel<<<1, SUM_KERNEL_NTHREADS, 0, context->hip_stream()>>>(
        N, x, y, false);
  }
}

namespace {
template <typename T>
struct FloatTransform {
  inline __host__ __device__ float operator()(const T v) const {
    return convert::To<T, float>(v);
  }
};
} // namespace

#define CAFFE2_MATH_SUM_FUNC(T)                                           \
  template <>                                                             \
  void Sum<T, HIPContext>(                                               \
      const int N,                                                        \
      const T* x,                                                         \
      T* y,                                                               \
      HIPContext* context,                                               \
      Tensor<HIPContext>* scratch_ptr) {                                 \
    if (scratch_ptr && N > DEVICE_REDUCE_SIZE_THRESHOLD) {                \
      FloatTransform<T> transform;                                        \
      hipcub::TransformInputIterator<float, FloatTransform<T>, const T*> it( \
          x, transform);                                                  \
      float* sum = nullptr;                                               \
      SumGenericIter<float>(N, it, sum, context, scratch_ptr);            \
      SumConvertKernel<<<1, 1, 0, context->hip_stream()>>>(sum, y);      \
    } else {                                                              \
      SumKernel<<<1, SUM_KERNEL_NTHREADS, 0, context->hip_stream()>>>(   \
          N, x, y, false);                                                \
    }                                                                     \
  }

CAFFE2_MATH_SUM_FUNC(float16)
#undef CAFFE2_MATH_SUM_FUNC

namespace {
template <typename T>
struct SqrTransform {
  inline __host__ __device__ T operator()(const T v) const {
    return v * v;
  }
};
} //  namespace

template <>
void SumSqr<float, HIPContext>(
    const int N,
    const float* x,
    float* y,
    HIPContext* context,
    Tensor<HIPContext>* scratch_ptr) {
  if (scratch_ptr && N > DEVICE_REDUCE_SIZE_THRESHOLD) {
    SqrTransform<float> transform;
    hipcub::TransformInputIterator<float, SqrTransform<float>, const float*> it(
        x, transform);
    SumGenericIter<float>(N, it, y, context, scratch_ptr);
  } else {
    SumKernel<<<1, SUM_KERNEL_NTHREADS, 0, context->hip_stream()>>>(
        N, x, y, true);
  }
}

#define CAFFE2_MATH_SUMSQR_FUNC(T)                                      \
  template <>                                                           \
  void SumSqr<T, HIPContext>(                                          \
      const int N,                                                      \
      const T* x,                                                       \
      T* y,                                                             \
      HIPContext* context,                                             \
      Tensor<HIPContext>* scratch_ptr) {                               \
    if (scratch_ptr && N > DEVICE_REDUCE_SIZE_THRESHOLD) {              \
      FloatTransform<T> float_transform;                                \
      hipcub::TransformInputIterator<float, FloatTransform<T>, const T*>   \
          float_it(x, float_transform);                                 \
      SqrTransform<float> sqr_transform;                                \
      hipcub::TransformInputIterator<                                      \
          float,                                                        \
          SqrTransform<float>,                                          \
          decltype(float_it)>                                           \
          it(float_it, sqr_transform);                                  \
      float* sum = nullptr;                                             \
      SumGenericIter<float>(N, it, sum, context, scratch_ptr);          \
      SumConvertKernel<<<1, 1, 0, context->hip_stream()>>>(sum, y);    \
    } else {                                                            \
      SumKernel<<<1, SUM_KERNEL_NTHREADS, 0, context->hip_stream()>>>( \
          N, x, y, true);                                               \
    }                                                                   \
  }

CAFFE2_MATH_SUMSQR_FUNC(float16)
#undef CAFFE2_MATH_SUMSQR_FUNC
#undef DEVICE_REDUCE_SIZE_THRESHOLD

namespace {
template <typename T>
__global__ void SelectKernel(
    const int N, const int D, const T* x, const int* idx, T* y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    y[i] = x[i * D + idx[i]];
  }
}
}  // namespace

template <>
void Select<float, HIPContext>(
      const int N, const int D, const float* x, const int* idx, float* y,
      HIPContext* context) {
  SelectKernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS,
                        0, context->hip_stream()>>>(N, D, x, idx, y);
}

template <>
void Select<float16, HIPContext>(
    const int N,
    const int D,
    const float16* x,
    const int* idx,
    float16* y,
    HIPContext* context) {
  SelectKernel<float16><<<
      CAFFE_GET_BLOCKS(N),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context->hip_stream()>>>(N, D, x, idx, y);
}

namespace {
template <typename T>
__global__ void ScaleKernel(const int n, const float alpha, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    // y[i] = convert::To<float,T>(convert::To<T, float>(x[i]) * alpha);
    y[i] = convert::Get<T>(convert::Get<float>(x[i]) * alpha);
  }
}

template <typename T>
__global__ void
ScaleKernelDeviceAlpha(const int n, const float* alpha, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    y[i] = x[i] * (*alpha);
  }
}

template <typename T>
__global__ void PowKernel(const int n, const T* x, const T exponent, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    y[i] = powf(x[i], exponent);
  }
}

// fp16 specialization
template <>
__global__ void ScaleKernelDeviceAlpha(
    const int n,
    const float* alpha,
    const float16* x,
    float16* y) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    y[i] = convert::To<float, float16>(
        convert::To<float16, float>(x[i]) * (*alpha));
  }
}

}  // namespace

template <>
void Powx<float, HIPContext>(
    const int N,
    const float* a,
    const float b,
    float* y,
    HIPContext* context) {
  PowKernel<<<
      CAFFE_GET_BLOCKS(N),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context->hip_stream()>>>(N, a, b, y);
}

template <>
void Scale<float, HIPContext>(
    const int n,
    const float alpha,
    const float* x,
    float* y,
    HIPContext* context) {
  ScaleKernel<float><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS,
                       0, context->hip_stream()>>>(n, alpha, x, y);
}

template <>
void Scale<float16, HIPContext>(
    const int n,
    const float alpha,
    const float16* x,
    float16* y,
    HIPContext* context) {
  ScaleKernel<float16><<<
      CAFFE_GET_BLOCKS(n),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context->hip_stream()>>>(n, alpha, x, y);
}

template <>
void Scale<float, HIPContext>(
    const int n, const float* alpha, const float *x, float* y,
    HIPContext* context) {
  ScaleKernelDeviceAlpha<float><<<
      CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
          n, alpha, x, y);
}

template <>
void Scale<float16, HIPContext>(
    const int n,
    const float* alpha,
    const float16* x,
    float16* y,
    HIPContext* context) {
  ScaleKernelDeviceAlpha<float16><<<
      CAFFE_GET_BLOCKS(n),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context->hip_stream()>>>(n, alpha, x, y);
}

template <>
void Axpy<float, HIPContext>(
    const int N,
    const float alpha,
    const float* X,
    float* Y,
    HIPContext* context) {
  CUBLAS_ENFORCE(hipblasSaxpy(context->cublas_handle(), N, &alpha, X, 1, Y, 1));
}

template <>
void Axpy<double, HIPContext>(
    const int N,
    const float alpha,
    const double* X,
    double* Y,
    HIPContext* context) {
  double alpha_d{alpha};
  CUBLAS_ENFORCE(
      hipblasDaxpy(context->cublas_handle(), N, &alpha_d, X, 1, Y, 1));
}

template <>
void Axpy<float16, HIPContext>(
    const int N,
    const float alpha,
    const float16* X,
    float16* Y,
    HIPContext* context) {
  CUBLAS_CHECK(hipblasAxpyEx(
      context->cublas_handle(),
      N,
      &alpha,
      HIP_R_16F,
      X,
      HIP_R_16F,
      1,
      Y,
      HIP_R_16F,
      1,
      HIP_R_32F));
}

namespace {
template <typename T>
__global__ void AxpyKernel(const int n, const float* a, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    y[index] = convert::Get<T>(
        convert::Get<float>(x[index]) * (*a) + convert::Get<float>(y[index]));
  }
}
}  // namespace

template <>
void Axpy<float, HIPContext>(
    const int n, const float* alpha, const float* X,
    float* Y, HIPContext* context) {
  AxpyKernel<float><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS,
                       0, context->hip_stream()>>>(n, alpha, X, Y);
}

template <>
void Axpy<float16, HIPContext>(
    const int n,
    const float* alpha,
    const float16* X,
    float16* Y,
    HIPContext* context) {
  AxpyKernel<float16><<<
      CAFFE_GET_BLOCKS(n),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context->hip_stream()>>>(n, alpha, X, Y);
}

namespace {
template <typename T>
__global__ void AxpbyKernel(const int n, const T a, const T* x,
                             const T b, T* y) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    y[index] = x[index] * a + y[index] * b;
  }
}
}  // namespace

template <>
void Axpby<float, HIPContext>(
    const int n, const float a, const float* x, const float b, float* y,
    HIPContext* context) {
  AxpbyKernel<float><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS,
                       0, context->hip_stream()>>>(n, a, x, b, y);
}

namespace {

template <typename T>
__global__ void im2col_gpu_kernel_nchw(const int n, const T* data_im,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int dilation_h, const int dilation_w,
    const int pad_t, const int pad_l,
    const int stride_h, const int stride_w,
    const int height_col, const int width_col,
    T* data_col) {

  CUDA_1D_KERNEL_LOOP(index, n) {
    int w_out = index % width_col;
    int h_index = index / width_col;
    int h_out = h_index % height_col;
    int channel_in = h_index / height_col;
    int channel_out = channel_in * kernel_h * kernel_w;
    int h_in = h_out * stride_h - pad_t;
    int w_in = w_out * stride_w - pad_l;
    T* data_col_ptr = data_col;
    data_col_ptr += (channel_out * height_col + h_out) * width_col + w_out;
    const T* data_im_ptr = data_im;
    data_im_ptr += (channel_in * height + h_in) * width + w_in;
    for (int i = 0; i < kernel_h; ++i) {
      for (int j = 0; j < kernel_w; ++j) {
        int h = h_in + i * dilation_h;
        int w = w_in + j * dilation_w;
        *data_col_ptr = (h >= 0 && w >= 0 && h < height && w < width) ?
            data_im_ptr[i * dilation_h * width + j * dilation_w] : 0;
        data_col_ptr += height_col * width_col;
      }
    }
  }
}

template <typename T>
__global__ void im2col_gpu_kernel_nhwc(const int n, const T* data_im,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int dilation_h, const int dilation_w,
    const int pad_t, const int pad_l,
    const int stride_h, const int stride_w,
    const int width_col, const int channels,
    T* data_col) {

  const int dkernel_h = dilation_h * (kernel_h - 1) + 1;
  const int dkernel_w = dilation_w * (kernel_w - 1) + 1;

  CUDA_1D_KERNEL_LOOP(index, n) {
    int channel_in = index % channels;
    int w_out = index / channels % width_col;
    int h_out = index / channels / width_col;
    int h_in = h_out * stride_h - pad_t;
    int w_in = w_out * stride_w - pad_l;
    T* local_data_col = data_col +
        ((h_out * width_col) + w_out) * channels * kernel_h * kernel_w
        + channel_in;
    for (int i = 0; i < dkernel_h; i += dilation_h) {
      int h = h_in + i;
      for (int j = 0; j < dkernel_w; j += dilation_w) {
        int w = w_in + j;
        *local_data_col = (h >= 0 && w >= 0 && h < height && w < width) ?
            data_im[(h * width + w) * channels + channel_in] : 0;
        local_data_col += channels;
      }
    }
  }
}

template <typename T>
__global__ void col2im_gpu_kernel_nchw(const int n, const T* data_col,
    const int height, const int width,
    const int patch_h, const int patch_w,
    const int dilation_h, const int dilation_w,
    const int pad_t, const int pad_l,
    const int stride_h, const int stride_w,
    const int height_col, const int width_col,
    T* data_im) {

  const int dpatch_h = dilation_h * (patch_h - 1) + 1;
  const int dpatch_w = dilation_w * (patch_w - 1) + 1;

  CUDA_1D_KERNEL_LOOP(index, n) {
    T val = 0;
    int w = index % width + pad_l;
    int h = (index / width) % height + pad_t;
    int c = index / (width * height);

    // compute the start and end of the output
    int w_col_start = (w < dpatch_w) ? 0 : (w - dpatch_w) / stride_w + 1;
    int w_col_end = min(w / stride_w + 1, width_col);
    int h_col_start = (h < dpatch_h) ? 0 : (h - dpatch_h) / stride_h + 1;
    int h_col_end = min(h / stride_h + 1, height_col);

    for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
      for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
        int h_k = (h - h_col * stride_h);
        int w_k = (w - w_col * stride_w);
        if (h_k % dilation_h == 0 && w_k % dilation_w == 0) {
          h_k /= dilation_h;
          w_k /= dilation_w;
          int data_col_index =
              (((c * patch_h + h_k) * patch_w + w_k) * height_col + h_col) *
                  width_col +
              w_col;
          val += data_col[data_col_index];
        }
      }
    }
    data_im[index] = val;
  }
}

template <typename T>
__global__ void col2im_gpu_kernel_nhwc(const int n, const T* data_col,
    const int width, const int channels,
    const int patch_h, const int patch_w,
    const int dilation_h, const int dilation_w,
    const int pad_t, const int pad_l,
    const int stride_h, const int stride_w,
    const int height_col, const int width_col,
    T* data_im) {

  const int dpatch_h = dilation_h * (patch_h - 1) + 1;
  const int dpatch_w = dilation_w * (patch_w - 1) + 1;

  CUDA_1D_KERNEL_LOOP(index, n) {
    T val = 0;
    int c = index % channels;
    int w = index / channels % width + pad_l;
    int h = index / channels / width + pad_t;
    // compute the start and end of the output
    int w_col_start = (w < dpatch_w) ? 0 : (w - dpatch_w) / stride_w + 1;
    int w_col_end = min(w / stride_w + 1, width_col);
    int h_col_start = (h < dpatch_h) ? 0 : (h - dpatch_h) / stride_h + 1;
    int h_col_end = min(h / stride_h + 1, height_col);
    int channels_col = patch_h * patch_w * channels;

    for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
      for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
        int h_k = h - h_col * stride_h;
        int w_k = w - w_col * stride_w;
        if (h_k % dilation_h == 0 && w_k % dilation_w == 0) {
          h_k /= dilation_h;
          w_k /= dilation_w;
          int c_col = (h_k * patch_w + w_k) * channels + c;
          val += data_col[(h_col * width_col + w_col) * channels_col + c_col];
        }
      }
    }
    data_im[index] = val;
  }
}

// Ported from caffe1
template <typename T, int num_axes>
__global__ void im2col_nd_gpu_kernel(
    const int n,
    const T* data_im,
    const int* im_shape,
    const int* col_shape,
    const int* kernel_shape,
    const int* pad,
    const int* stride,
    const int* dilation,
    T* data_col) {
  int d_offset[num_axes]; // NOLINT(runtime/arrays)
  int d_iter[num_axes]; // NOLINT(runtime/arrays)

  __shared__ int shared_dilation[num_axes];
  __shared__ int shared_kernel_shape[num_axes];
  __shared__ int shared_pad[num_axes];
  __shared__ int shared_stride[num_axes];
  __shared__ int shared_col_shape[num_axes + 1];
  __shared__ int shared_im_shape[num_axes + 1];

  if (threadIdx.x < num_axes) {
    shared_dilation[threadIdx.x] = dilation[threadIdx.x];
    shared_kernel_shape[threadIdx.x] = kernel_shape[threadIdx.x];
    shared_pad[threadIdx.x] = pad[threadIdx.x];
    shared_stride[threadIdx.x] = stride[threadIdx.x];
  }
  if (threadIdx.x < num_axes + 1) {
    shared_col_shape[threadIdx.x] = col_shape[threadIdx.x];
    shared_im_shape[threadIdx.x] = im_shape[threadIdx.x];
  }
  __syncthreads();

  int i;
  int kernel_size = 1;
  for (i = 0; i < num_axes; ++i) {
    kernel_size *= shared_kernel_shape[i];
  }
  CUDA_1D_KERNEL_LOOP(index, n) {
    if (index >= col_shape[0]) {
      break;
    }
    // Initialize offset, computed in the loop below, with intermediate
    // computations used to compute the spatial indices.
    int offset = index;
    for (i = num_axes - 1; i >= 0; --i) {
      if (i < num_axes - 1) {
        offset /= shared_kernel_shape[i + 1];
      }
      d_offset[i] = offset % shared_kernel_shape[i];
    }
    for (i = 0; i < num_axes; ++i) {
      d_iter[i] = 0;
    }
    bool incremented;
    do {
      int index_col = index;
      int index_im = index / kernel_size;
      bool in_range = true;
      for (i = 0; i < num_axes; ++i) {
        const int d = d_iter[i];
        const int d_im = d * shared_stride[i] - shared_pad[i] +
            d_offset[i] * shared_dilation[i];
        in_range &= (d_im >= 0 && d_im < shared_im_shape[i + 1]);

        index_col *= shared_col_shape[i + 1];
        index_col += d;
        index_im *= shared_im_shape[i + 1];
        index_im += d_im;
      }
      if (in_range) {
        // data_col[index_col] = 0;
        data_col[index_col] = data_im[index_im];
        // T temp = data_im[index_im];
      } else {
        data_col[index_col] = 0;
      }

      incremented = false;
      for (i = num_axes - 1; i >= 0; --i) {
        // const int d_max = shared_kernel_shape[i];
        const int d_max = shared_col_shape[i + 1];
        if (d_iter[i] == d_max - 1) {
          d_iter[i] = 0;
        } else { // d_iter[i] < d_max - 1
          ++d_iter[i];
          incremented = true;
          break;
        }
      } // for (int i = num_axes - 1; i >= 0; --i)
    } while (incremented); // do
  } // CUDA_KERNEL_LOOP(index, n)
}

template <typename T, int num_axes>
__global__ void col2im_nd_gpu_kernel(
    const int n,
    const T* data_col,
    const int* im_shape,
    const int* col_shape,
    const int* kernel_shape,
    const int* pad,
    const int* stride,
    const int* dilation,
    T* data_im) {
  int d_im[num_axes]; // NOLINT(runtime/arrays)
  int d_col_iter[num_axes]; // NOLINT(runtime/arrays)
  int d_col_start[num_axes]; // NOLINT(runtime/arrays)
  int d_col_end[num_axes]; // NOLINT(runtime/arrays)

  __shared__ int shared_dilation[num_axes];
  __shared__ int shared_kernel_shape[num_axes];
  __shared__ int shared_pad[num_axes];
  __shared__ int shared_stride[num_axes];
  __shared__ int shared_col_shape[num_axes + 1];
  __shared__ int shared_im_shape[num_axes + 1];

  if (threadIdx.x < num_axes) {
    shared_dilation[threadIdx.x] = dilation[threadIdx.x];
    shared_kernel_shape[threadIdx.x] = kernel_shape[threadIdx.x];
    shared_pad[threadIdx.x] = pad[threadIdx.x];
    shared_stride[threadIdx.x] = stride[threadIdx.x];
  }

  if (threadIdx.x < num_axes + 1) {
    shared_col_shape[threadIdx.x] = col_shape[threadIdx.x];
    shared_im_shape[threadIdx.x] = im_shape[threadIdx.x];
  }
  __syncthreads();

  CUDA_1D_KERNEL_LOOP(index, n) {
    // Initialize channel_in, computed in the loop below, with intermediate
    // computations used to compute the spatial indices.
    int c_im = index;
    // Calculate d_im (image dimensions).
    for (int i = num_axes - 1; i >= 0; --i) {
      d_im[i] = c_im % shared_im_shape[i + 1] + shared_pad[i];
      c_im /= shared_im_shape[i + 1];
    }
    // Calculate col start/end indices.
    bool done = false;
    for (int i = 0; i < num_axes; ++i) {
      const int kernel_extent =
          shared_dilation[i] * (shared_kernel_shape[i] - 1) + 1;
      d_col_start[i] = d_col_iter[i] = (d_im[i] < kernel_extent)
          ? 0
          : (d_im[i] - kernel_extent) / shared_stride[i] + 1;
      d_col_end[i] =
          min(d_im[i] / shared_stride[i] + 1, shared_col_shape[i + 1]);
      if (d_col_start[i] >= d_col_end[i]) {
        // Skip computation if the dimension is 0 at any spatial axis --
        // final val will be 0.
        data_im[index] = 0;
        done = true;
        break; // for (int i = 0; i < num_axes; ++i)
      }
    }
    if (done) {
      continue; // CUDA_KERNEL_LOOP(index, n)
    }
    // Loop over the col to compute the output val.
    T val = 0;
    bool incremented = true;
    bool skip = false;
    do {
      // Compute the final offset.
      int final_offset = 0;
      int kernel_shape_prod = 1;
      int kernel_index;
      for (int i = num_axes - 1; i >= 0; --i) {
        kernel_index = d_im[i] - d_col_iter[i] * shared_stride[i];
        if (kernel_index % shared_dilation[i]) {
          skip = true;
          break;
        } else {
          kernel_index /= shared_dilation[i];
          final_offset += kernel_index * kernel_shape_prod;
          kernel_shape_prod *= shared_kernel_shape[i];
        }
      }
      if (!skip) {
        final_offset += kernel_shape_prod * c_im;
        for (int i = 0; i < num_axes; ++i) {
          final_offset *= shared_col_shape[i + 1];
          final_offset += d_col_iter[i];
        }
        val += data_col[final_offset];
      }
      skip = false;
      incremented = false;
      for (int i = num_axes - 1; i >= 0; --i) {
        const int d_max = d_col_end[i];
        if (d_col_iter[i] == d_max - 1) {
          d_col_iter[i] = d_col_start[i];
        } else { // d_col_iter[i] < d_max - 1
          ++d_col_iter[i];
          incremented = true;
          break; // for (int i = num_axes - 1; i >= 0; --i)
        }
      } // for (int i = num_axes - 1; i >= 0; --i)
    } while (incremented);
    data_im[index] = val;
  } // CUDA_KERNEL_LOOP(index, n)
}

}  // namespace

template <>
void Im2col<float, HIPContext, StorageOrder::NCHW>(
    const float* data_im, const int channels,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int dilation_h, const int dilation_w,
    const int pad_t, const int pad_l, const int pad_b, const int pad_r,
    const int stride_h,
    const int stride_w, float* data_col, HIPContext* context) {

  const int dkernel_h = dilation_h * (kernel_h - 1) + 1;
  const int dkernel_w = dilation_w * (kernel_w - 1) + 1;

  // We are going to launch channels * height_col * width_col kernels, each
  // kernel responsible for copying a single-channel grid.
  int height_col = (height + pad_t + pad_b - dkernel_h) / stride_h + 1;
  int width_col = (width + pad_l + pad_r - dkernel_w) / stride_w + 1;
  int num_kernels = channels * height_col * width_col;
  // NOLINT_NEXT_LINE(whitespace/operators)
  im2col_gpu_kernel_nchw<float><<<CAFFE_GET_BLOCKS(num_kernels),
                                  CAFFE_CUDA_NUM_THREADS, 0,
                                  context->hip_stream()>>>(
      num_kernels, data_im, height, width, kernel_h, kernel_w,
      dilation_h, dilation_w, pad_t, pad_l, stride_h, stride_w,
      height_col, width_col, data_col);
}

template <>
void Im2col<float, HIPContext, StorageOrder::NHWC>(
    const float* data_im, const int channels,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int dilation_h, const int dilation_w,
    const int pad_t, const int pad_l, const int pad_b, const int pad_r,
    const int stride_h,
    const int stride_w, float* data_col, HIPContext* context) {

  const int dkernel_h = dilation_h * (kernel_h - 1) + 1;
  const int dkernel_w = dilation_w * (kernel_w - 1) + 1;

  // We are going to launch height_col * width_col * channels kernels, each
  // kernel responsible for copying a single-channel grid.
  int height_col = (height + pad_t + pad_b - dkernel_h) / stride_h + 1;
  int width_col = (width + pad_l + pad_r - dkernel_w) / stride_w + 1;
  int num_kernels = height_col * width_col * channels;
  // NOLINT_NEXT_LINE(whitespace/operators)
  im2col_gpu_kernel_nhwc<float><<<CAFFE_GET_BLOCKS(num_kernels),
                                  CAFFE_CUDA_NUM_THREADS, 0,
                                  context->hip_stream()>>>(
      num_kernels, data_im, height, width, kernel_h, kernel_w,
      dilation_h, dilation_w, pad_t, pad_l, stride_h, stride_w,
      width_col, channels, data_col);
}


template <>
void Col2im<float, HIPContext, StorageOrder::NCHW>(
    const float* data_col, const int channels,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int dilation_h, const int dilation_w,
    const int pad_t, const int pad_l, const int pad_b, const int pad_r,
    const int stride_h,
    const int stride_w, float* data_im, HIPContext* context) {

  const int dkernel_h = dilation_h * (kernel_h - 1) + 1;
  const int dkernel_w = dilation_w * (kernel_w - 1) + 1;

  int height_col = (height + pad_t + pad_b - dkernel_h) / stride_h + 1;
  int width_col = (width + pad_l + pad_r - dkernel_w) / stride_w + 1;
  int num_kernels = channels * height * width;
  // To avoid involving atomic operations, we will launch one kernel per
  // bottom dimension, and then in the kernel add up the top dimensions.
  col2im_gpu_kernel_nchw<float><<<CAFFE_GET_BLOCKS(num_kernels),
                                  CAFFE_CUDA_NUM_THREADS, 0,
                                  context->hip_stream()>>>(
      num_kernels, data_col, height, width, kernel_h, kernel_w,
      dilation_h, dilation_w,
      pad_t, pad_l, stride_h, stride_w,
      height_col, width_col, data_im);
}

template <>
void Col2im<float, HIPContext, StorageOrder::NHWC>(
    const float* data_col, const int channels,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int dilation_h, const int dilation_w,
    const int pad_t, const int pad_l, const int pad_b, const int pad_r,
    const int stride_h,
    const int stride_w, float* data_im, HIPContext* context) {

  const int dkernel_h = dilation_h * (kernel_h - 1) + 1;
  const int dkernel_w = dilation_w * (kernel_w - 1) + 1;

  int height_col = (height + pad_t + pad_b - dkernel_h) / stride_h + 1;
  int width_col = (width + pad_l + pad_r - dkernel_w) / stride_w + 1;
  int num_kernels = height * width * channels;
  // To avoid involving atomic operations, we will launch one kernel per
  // bottom dimension, and then in the kernel add up the top dimensions.
  col2im_gpu_kernel_nhwc<float><<<CAFFE_GET_BLOCKS(num_kernels),
                                  CAFFE_CUDA_NUM_THREADS, 0,
                                  context->hip_stream()>>>(
      num_kernels, data_col, width, channels, kernel_h, kernel_w,
      dilation_h, dilation_w,
      pad_t, pad_l, stride_h, stride_w, height_col, width_col, data_im);
}

template <>
void Col2imNd<float, HIPContext, StorageOrder::NCHW>(
    const float* data_col,
    const int* img_shape,
    const int* col_shape,
    const int img_size,
    const int col_size,
    const int* kernel_shape,
    const int* stride,
    const int* dilation,
    const int* pad,
    const int N,
    float* data_img,
    HIPContext* context) {
  CAFFE_ENFORCE_LT(
      N, CAFFE_CUDA_NUM_THREADS, "num_axes should be smaller than block size.");

#define COL2IM_ND_KERNEL(n)                                                   \
  col2im_nd_gpu_kernel<float, n> /* NOLINT_NEXT_LINE(whitespace/operators) */ \
      <<<CAFFE_GET_BLOCKS(img_size),                                          \
         CAFFE_CUDA_NUM_THREADS,                                              \
         0,                                                                   \
         context->hip_stream()>>>(                                           \
          img_size,                                                           \
          data_col,                                                           \
          img_shape,                                                          \
          col_shape,                                                          \
          kernel_shape,                                                       \
          pad,                                                                \
          stride,                                                             \
          dilation,                                                           \
          data_img)

  switch (N) {
    case 1:
      COL2IM_ND_KERNEL(1);
      break;
    case 2:
      COL2IM_ND_KERNEL(2);
      break;
    case 3:
      COL2IM_ND_KERNEL(3);
      break;
    case 4:
      COL2IM_ND_KERNEL(4);
      break;
    case 5:
      COL2IM_ND_KERNEL(5);
      break;
    default:
      CAFFE_THROW(
          "Col2imNd does not support computation with ", N, " spatial axes");
  }
}

template <>
void Im2colNd<float, HIPContext, StorageOrder::NCHW>(
    const float* data_img,
    const int* img_shape,
    const int* col_shape,
    const int img_size,
    const int col_size,
    const int* kernel_shape,
    const int* stride,
    const int* dilation,
    const int* pad,
    const int N,
    float* data_col,
    HIPContext* context,
    bool /*accumlate_output*/) {
  CAFFE_ENFORCE_LT(
      N, CAFFE_CUDA_NUM_THREADS, "num_axes should be smaller than block size.");

#define IM2COL_ND_KERNEL(n)                                                   \
  im2col_nd_gpu_kernel<float, n> /* NOLINT_NEXT_LINE(whitespace/operators) */ \
      <<<CAFFE_GET_BLOCKS(col_size),                                          \
         CAFFE_CUDA_NUM_THREADS,                                              \
         0,                                                                   \
         context->hip_stream()>>>(                                           \
          col_size,                                                           \
          data_img,                                                           \
          img_shape,                                                          \
          col_shape,                                                          \
          kernel_shape,                                                       \
          pad,                                                                \
          stride,                                                             \
          dilation,                                                           \
          data_col)

  switch (N) {
    case 1:
      IM2COL_ND_KERNEL(1);
      break;
    case 2:
      IM2COL_ND_KERNEL(2);
      break;
    case 3:
      IM2COL_ND_KERNEL(3);
      break;
    case 4:
      IM2COL_ND_KERNEL(4);
    case 5:
      IM2COL_ND_KERNEL(5);
      break;
    default:
      CAFFE_THROW(
          "Im2colNd does not support computation with ", N, " spatial axes");
  }
}

template <>
void CopyMatrix<HIPContext>(
    const size_t itemsize,
    const int M,
    const int N,
    const void* A,
    const int lda,
    void* B,
    const int ldb,
    HIPContext* context,
    TypeMeta::TypedCopy copy) {
  CAFFE_ENFORCE(!copy, "Copy constructor is not supported in CUDA context");
  hipMemcpy2DAsync(B, ldb * itemsize, A, lda * itemsize, N * itemsize, M,
                    hipMemcpyDeviceToDevice, context->hip_stream());
}

template <>
void CopyVector<float, HIPContext>(
    const int N,
    const float* src,
    float* dst,
    HIPContext* context) {
  if (src != dst && N > 0) {
    hipMemcpyAsync(
        dst,
        src,
        sizeof(float) * N,
        hipMemcpyDeviceToDevice,
        context->hip_stream());
  }
}

namespace {
__global__ void rowwise_max_kernel(
    const int rows,
    const int cols,
    const float* data,
    float* out) {
  typedef hipcub::BlockReduce<float, CAFFE_CUDA_NUM_THREADS> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  for (int rowIndex = blockIdx.x; rowIndex < rows; rowIndex += gridDim.x) {
    float maxval = -FLT_MAX;
    // NB: The memory accesses here are sequentialized; without unrolling
    // the loop, there will not be any ILP.  However, because we are running
    // this kernel with a lot of threads, this should not be a big problem.
    // However, if we reduce the number of threads to take advantage of
    // warp-wide synchronization, this may become a problem again.
    for (int colIndex = threadIdx.x; colIndex < cols; colIndex += blockDim.x) {
      maxval = max(data[rowIndex * cols + colIndex], maxval);
    }
    maxval = BlockReduce(temp_storage).Reduce(maxval, hipcub::Max());
    if (threadIdx.x == 0) {
      out[rowIndex] = maxval;
    }
    __syncthreads();
  }
}

__global__ void colwise_max_kernel(
    const int rows,
    const int cols,
    const float* data,
    float* out) {
  typedef hipcub::BlockReduce<float, CAFFE_CUDA_NUM_THREADS> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  for (int colIndex = blockIdx.x; colIndex < cols; colIndex += gridDim.x) {
    float maxval = -FLT_MAX;
    for (int rowIndex = threadIdx.x; rowIndex < rows; rowIndex += blockDim.x) {
      maxval = max(data[rowIndex * cols + colIndex], maxval);
    }
    maxval = BlockReduce(temp_storage).Reduce(maxval, hipcub::Max());
    if (threadIdx.x == 0) {
      out[colIndex] = maxval;
    }
    __syncthreads();
  }
}

} // namespace

template <>
void RowwiseMax(
    const int N,
    const int D,
    const float* x,
    float* y,
    HIPContext* context) {
  rowwise_max_kernel<<<
      std::min(N, CAFFE_MAXIMUM_NUM_BLOCKS),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context->hip_stream()>>>(N, D, x, y);
}

template <>
void ColwiseMax(
    const int N,
    const int D,
    const float* x,
    float* y,
    HIPContext* context) {
  colwise_max_kernel<<<
      std::min(D, CAFFE_MAXIMUM_NUM_BLOCKS),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context->hip_stream()>>>(N, D, x, y);
}

namespace {
__global__ void
maximum_kernel(const int N, const float alpha, const float* x, float* y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    y[i] = fmaxf(x[i], alpha);
  }
}
} // namespace

template <>
void Maximum(
    const int N,
    const float alpha,
    const float* x,
    float* y,
    HIPContext* context) {
  maximum_kernel<<<
      std::min(N, CAFFE_MAXIMUM_NUM_BLOCKS),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context->hip_stream()>>>(N, alpha, x, y);
}

namespace {

constexpr int kCompileTimeCUDAMaxTransposeDims = 8;

__device__ void ComputeXStride(
    const int num_axes,
    const int* x_dims,
    const int* axes,
    int* x_strides) {
  int buff[kCompileTimeCUDAMaxTransposeDims];
  int cur_stride = 1;
#pragma unroll
  for (int i = num_axes - 1; i >= 0; --i) {
    buff[i] = cur_stride;
#if __CUDA_ARCH__ >= 350
    cur_stride *= __ldg(x_dims + i);
#else
    cur_stride *= x_dims[i];
#endif
  }
#pragma unroll
  for (int i = 0; i < num_axes; ++i) {
#if __CUDA_ARCH__ >= 350
    x_strides[i] = buff[__ldg(axes + i)];
#else
    x_strides[i] = buff[axes[i]];
#endif
  }
}

__device__ int GetXIndex(
    const int num_axes,
    const int* y_dims,
    const int* x_strides,
    int y_index) {
  int x_index = 0;
#pragma unroll
  for (int i = num_axes - 1; i >= 0 && y_index > 0; --i) {
    x_index += (y_index % y_dims[i]) * x_strides[i];
    y_index /= y_dims[i];
  }
  return x_index;
}

template <typename T>
__global__ void TransposeCUDA(
    const int num_axes,
    const int* x_dims,
    const int* y_dims,
    const int* axes,
    const int data_size,
    const T* X,
    T* Y) {
  __shared__ int x_strides[kCompileTimeCUDAMaxTransposeDims];
  __shared__ int y_dims_shared[kCompileTimeCUDAMaxTransposeDims];
  const int tid = threadIdx.x;
  if (tid == 0) {
    ComputeXStride(num_axes, x_dims, axes, x_strides);
  }
  if (tid < num_axes) {
    y_dims_shared[tid] = y_dims[tid];
  }
  __syncthreads();
  CUDA_1D_KERNEL_LOOP(y_index, data_size) {
    const int x_index = GetXIndex(num_axes, y_dims_shared, x_strides, y_index);
#if __CUDA_ARCH__ >= 350
    Y[y_index] = __ldg(X + x_index);
#else
    Y[y_index] = X[x_index];
#endif
  }
}

} // namespace

#define CAFFE2_SPECIALIZED_CUDA_TRANSPOSE(T)                  \
  template <>                                                 \
  void Transpose<T, HIPContext>(                             \
      const int num_axes,                                     \
      const int* x_dims,                                      \
      const int* y_dims,                                      \
      const int* axes,                                        \
      const int data_size,                                    \
      const T* X,                                             \
      T* Y,                                                   \
      HIPContext* context) {                                 \
    CAFFE_ENFORCE(                                            \
        num_axes <= kCompileTimeCUDAMaxTransposeDims,         \
        "num_axes exceeds compile time max.");                \
    TransposeCUDA<T>                                          \
        <<<CAFFE_GET_BLOCKS(data_size),                       \
           CAFFE_CUDA_NUM_THREADS,                            \
           0,                                                 \
           context->hip_stream()>>>(                         \
            num_axes, x_dims, y_dims, axes, data_size, X, Y); \
  }
CAFFE2_SPECIALIZED_CUDA_TRANSPOSE(float)
CAFFE2_SPECIALIZED_CUDA_TRANSPOSE(double)
CAFFE2_SPECIALIZED_CUDA_TRANSPOSE(int)
CAFFE2_SPECIALIZED_CUDA_TRANSPOSE(long)
#undef CAFFE2_SPECIALIZED_CUDA_TRANSPOSE

} // namespace math
} // namespace caffe2
