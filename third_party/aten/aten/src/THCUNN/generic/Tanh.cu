
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/Tanh.cu"
#else

#include "../common.h"

void THNN_(Tanh_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output)
{
  THCUNN_assertSameGPU(state, 2, input, output);
  THCTensor_(resizeAs)(state, output, input);
  THCTensor_(tanh)(state, output, input);
}

void THNN_(Tanh_updateGradInput)(
           THCState *state,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCTensor *output)
{
  THCUNN_check_shape(state, output, gradOutput);
  THCUNN_assertSameGPU(state, 3, output, gradOutput, gradInput);
  THCTensor_(resizeAs)(state, gradInput, output);
  THC_pointwiseApply3(state, gradInput, output, gradOutput, tanh_updateGradInput_functor<real>());
}

#endif
