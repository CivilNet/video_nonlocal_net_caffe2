
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/LeakyReLU.cu"
#else

#include "../common.h"

void THNN_(LeakyReLU_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           accreal negval_,
           bool inplace)
{
  real negval = ScalarConvert<accreal, real>::to(negval_);

  THCUNN_assertSameGPU(state, 2, input, output);

  if (inplace)
  {
    THC_pointwiseApply1(state, input, LeakyReLUUpdateOutputIP<real>(negval));
    THCTensor_(set)(state, output, input);
  }
  else
  {
    THCTensor_(resizeAs)(state, output, input);
    THC_pointwiseApply2(state, output, input, LeakyReLUUpdateOutput<real>(negval));
  }

  THCudaCheck(cudaGetLastError());
}

void THNN_(LeakyReLU_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           accreal negval_,
           bool inplace)
{
  real negval = ScalarConvert<accreal, real>::to(negval_);

  THCUNN_check_nElement(state, input, gradOutput);
  THCUNN_assertSameGPU(state, 3, input, gradInput, gradOutput);

  if (inplace)
  {
    THC_pointwiseApply2(state, gradOutput, input, LeakyReLUUpdateGradInputIP<real>(negval));
    THCTensor_(set)(state, gradInput, gradOutput);
  }
  else
  {
    THCTensor_(resizeAs)(state, gradInput, input);
    THC_pointwiseApply3(state, gradInput, input, gradOutput, LeakyReLUUpdateGradInput<real>(negval));
  }

  THCudaCheck(cudaGetLastError());
}

#endif
