
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/MultiLabelMarginCriterion.cu"
#else

// TODO: improve error messages
void THNN_(MultiLabelMarginCriterion_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCIndexTensor *target,
           THCTensor *output,
           THCTensor *istarget,
           bool sizeaverage,
           bool reduce)
{
  input = THCTensor_(newContiguous)(state, input);
  target = THCIndexTensor_(newContiguous)(state, target);
  istarget = THCTensor_(newContiguous)(state, istarget);
  THCTensor_(resizeAs)(state, istarget, input);

  if(input->nDimension == 1)
  {
    int dim = input->size[0];
    THArgCheck((target->nDimension == 1) && (target->size[0] == dim), 3,
        "inconsistent target size");
    THCTensor_(resize1d)(state, output, 1);

    dim3 blocks(1);
    dim3 threads(MULTILABELMARGIN_THREADS);

    cunn_MultiLabelMarginCriterion_updateOutput_kernel<real, accreal>
      <<<blocks, threads, 0, THCState_getCurrentStream(state)>>>(
        THCTensor_(data)(state, output),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        THCTensor_(data)(state, istarget),
        1, dim,
        sizeaverage
        );
    THCudaCheck(cudaGetLastError());
  }
  else if(input->nDimension == 2)
  {
    int nframe = input->size[0];
    int dim = input->size[1];
    THArgCheck((target->nDimension == 2) && (target->size[0] == nframe)
               && (target->size[1] == dim), 3, "inconsistent target size");

    dim3 blocks(input->size[0]);
    dim3 threads(MULTILABELMARGIN_THREADS);

    if (reduce)
    {
      THCTensor *output_tmp = THCTensor_(newWithSize1d)(state, input->size[0]);
      THCTensor_(resize1d)(state, output, 1);

      cunn_MultiLabelMarginCriterion_updateOutput_kernel<real, accreal>
        <<<blocks, threads, 0, THCState_getCurrentStream(state)>>>(
          THCTensor_(data)(state, output_tmp),
          THCTensor_(data)(state, input),
          THCIndexTensor_(data)(state, target),
          THCTensor_(data)(state, istarget),
          nframe, dim,
          sizeaverage
          );
      THCudaCheck(cudaGetLastError());
      THCTensor_(set1d)(state, output, 0, ScalarConvert<accreal, real>::to(THCTensor_(sumall)(state, output_tmp)));
      THCTensor_(free)(state, output_tmp);
    }
    else
    {
    THCTensor_(resize1d)(state, output, input->size[0]);

    cunn_MultiLabelMarginCriterion_updateOutput_kernel<real, accreal>
      <<<blocks, threads, 0, THCState_getCurrentStream(state)>>>(
        THCTensor_(data)(state, output),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        THCTensor_(data)(state, istarget),
        nframe, dim,
        false
        );
    THCudaCheck(cudaGetLastError());
    }
  }
  else
    THError("vector or matrix expected");

  THCTensor_(free)(state, input);
  THCIndexTensor_(free)(state, target);
}

void THNN_(MultiLabelMarginCriterion_updateGradInput)(
            THCState *state,
            THCTensor *input,
            THCIndexTensor *target,
            THCTensor *gradOutput,
            THCTensor *gradInput,
            THCTensor *istarget,
            bool sizeaverage,
            bool reduce)
{
  input = THCTensor_(newContiguous)(state, input);
  target = THCIndexTensor_(newContiguous)(state, target);
  istarget = THCTensor_(newContiguous)(state, istarget);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);
  THCTensor_(resizeAs)(state, gradInput, input);

  if(gradInput->nDimension == 1)
  {
    int dim = gradInput->size[0];
    THArgCheck((target->nDimension == 1) && (target->size[0] == dim), 3,
               "inconsistent target size");
    THArgCheck((istarget->nDimension == 1) && (istarget->size[0] == dim), 3,
               "inconsistent isTarget size");
    dim3 blocks(1);
    dim3 threads(MULTILABELMARGIN_THREADS);

    cunn_MultiLabelMarginCriterion_updateGradInput_kernel<real, accreal>
      <<<blocks, threads, 0, THCState_getCurrentStream(state)>>>(
        THCTensor_(data)(state, gradInput),
        THCTensor_(data)(state, gradOutput),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        THCTensor_(data)(state, istarget),
        1, gradInput->size[0],
        sizeaverage,
        reduce);

  }
  else if(gradInput->nDimension == 2)
  {
    int nframe = gradInput->size[0];
    int dim = gradInput->size[1];
    THArgCheck((target->nDimension == 2) && (target->size[0] == nframe)
               && (target->size[1] == dim), 3, "inconsistent target size");
    THArgCheck((istarget->nDimension == 2) && (istarget->size[0] == nframe)
               && (istarget->size[1] == dim), 3, "inconsistent isTarget size");
    dim3 blocks(gradInput->size[0]);
    dim3 threads(MULTILABELMARGIN_THREADS);

    cunn_MultiLabelMarginCriterion_updateGradInput_kernel<real, accreal>
      <<<blocks, threads, 0, THCState_getCurrentStream(state)>>>(
        THCTensor_(data)(state, gradInput),
        THCTensor_(data)(state, gradOutput),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        THCTensor_(data)(state, istarget),
        gradInput->size[0], gradInput->size[1],
        sizeaverage,
        reduce);
  }
  else
    THError("vector or matrix expected");

  THCudaCheck(cudaGetLastError());

  THCTensor_(free)(state, input);
  THCIndexTensor_(free)(state, target);
  THCTensor_(free)(state, istarget);
  THCTensor_(free)(state, gradOutput);
}

#endif
