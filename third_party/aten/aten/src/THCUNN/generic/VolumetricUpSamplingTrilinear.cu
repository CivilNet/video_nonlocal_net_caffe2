
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/VolumetricUpSamplingTrilinear.cu"
#else

#include "../linear_upsampling.h"

static inline void THNN_(VolumetricUpSamplingTrilinear_shapeCheck)
                        (THCState *state,
                         THCTensor *input, THCTensor *gradOutput,
                         int nBatch, int nChannels,
                         int inputDepth, int inputHeight, int inputWidth,
                         int outputDepth, int outputHeight, int outputWidth) {
  THArgCheck(inputDepth > 0 && inputHeight > 0 && inputWidth > 0
             && outputDepth && outputHeight > 0 && outputWidth > 0, 2,
             "input and output sizes should be greater than 0,"
             " but got input (D: %d, H: %d, W: %d) output (D: %d, H: %d, W: %d)",
             inputDepth, inputHeight, inputWidth, outputDepth, outputHeight, outputWidth);
  if (input != NULL) {
     THCUNN_argCheck(state, input->nDimension == 5, 2, input,
                     "5D input tensor expected but got: %s");
  }

  if (gradOutput != NULL) {
    THCUNN_check_dim_size(state, gradOutput, 5, 0, nBatch);
    THCUNN_check_dim_size(state, gradOutput, 5, 1, nChannels);
    THCUNN_check_dim_size(state, gradOutput, 5, 2, outputDepth);
    THCUNN_check_dim_size(state, gradOutput, 5, 3, outputHeight);
    THCUNN_check_dim_size(state, gradOutput, 5, 4, outputWidth);
  }
}

void THNN_(VolumetricUpSamplingTrilinear_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           int outputDepth,
           int outputHeight,
           int outputWidth,
           bool align_corners)
{
  int nbatch = THCTensor_(size)(state, input, 0);
  int channels = THCTensor_(size)(state, input, 1);
  int inputDepth = THCTensor_(size)(state, input, 2);
  int inputHeight = THCTensor_(size)(state, input, 3);
  int inputWidth = THCTensor_(size)(state, input, 4);
  THNN_(VolumetricUpSamplingTrilinear_shapeCheck)
       (state, input, NULL,
        nbatch, channels,
        inputDepth, inputHeight, inputWidth,
        outputDepth, outputHeight, outputWidth);
  input = THCTensor_(newContiguous)(state, input);
  THCUNN_assertSameGPU(state, 2, input, output);
  THCTensor_(resize5d)(state, output,
                       THCTensor_(size)(state, input, 0),
                       THCTensor_(size)(state, input, 1),
                       outputDepth, outputHeight, outputWidth);
  THCTensor_(zero)(state, output);
  THCDeviceTensor<real, 5> idata = toDeviceTensor<real, 5>(state, input);
  THCDeviceTensor<real, 5> odata = toDeviceTensor<real, 5>(state, output);
  THAssert(inputDepth > 0 && inputHeight > 0 && inputWidth > 0 && outputDepth > 0 && outputHeight > 0 && outputWidth > 0);
  const accreal rdepth = linear_upsampling_compute_scale<accreal>(inputDepth, outputDepth, align_corners);
  const accreal rheight = linear_upsampling_compute_scale<accreal>(inputHeight, outputHeight, align_corners);
  const accreal rwidth = linear_upsampling_compute_scale<accreal>(inputWidth, outputWidth, align_corners);
  const int num_kernels = outputDepth * outputHeight * outputWidth;
  const int num_threads =
    THCState_getCurrentDeviceProperties(state)->maxThreadsPerBlock;
  cudaStream_t stream = THCState_getCurrentStream(state);
  caffe_gpu_interp2_kernel<real, accreal> <<<THCCeilDiv(num_kernels, num_threads), num_threads ,
   0 , stream>>>(num_kernels, rdepth, rheight, rwidth, align_corners, idata, odata);
  THCudaCheck(cudaGetLastError());
  THCTensor_(free)(state, input);
}


void THNN_(VolumetricUpSamplingTrilinear_updateGradInput)(
           THCState *state,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           int nbatch,
           int nchannels,
           int inputDepth,
           int inputHeight,
           int inputWidth,
           int outputDepth,
           int outputHeight,
           int outputWidth,
           bool align_corners)
{
  THNN_(VolumetricUpSamplingTrilinear_shapeCheck)
       (state, NULL, gradOutput,
        nbatch, nchannels,
        inputDepth, inputHeight, inputWidth,
        outputDepth, outputHeight, outputWidth);
  gradInput = THCTensor_(newContiguous)(state, gradInput);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);
  THCUNN_assertSameGPU(state, 2, gradOutput, gradInput);
  THCTensor_(resize5d)(state, gradInput, nbatch, nchannels, inputDepth, inputHeight, inputWidth);
  THCTensor_(zero)(state, gradInput);
  THCDeviceTensor<real, 5> data1 = toDeviceTensor<real, 5>(state, gradInput);
  THCDeviceTensor<real, 5> data2 = toDeviceTensor<real, 5>(state, gradOutput);
  const accreal rdepth = linear_upsampling_compute_scale<accreal>(inputDepth, outputDepth, align_corners);
  const accreal rheight = linear_upsampling_compute_scale<accreal>(inputHeight, outputHeight, align_corners);
  const accreal rwidth = linear_upsampling_compute_scale<accreal>(inputWidth, outputWidth, align_corners);
  const int num_kernels = outputDepth * outputHeight * outputWidth;
  const int num_threads =
    THCState_getCurrentDeviceProperties(state)->maxThreadsPerBlock;
  cudaStream_t stream = THCState_getCurrentStream(state);
  caffe_gpu_interp2_kernel_backward<real ,accreal> <<<THCCeilDiv(num_kernels, num_threads),
  num_threads, 0, stream>>>(num_kernels, rdepth, rheight, rwidth, align_corners, data1, data2);
  THCudaCheck(cudaGetLastError());
  THCTensor_(free)(state, gradInput);
  THCTensor_(free)(state, gradOutput);
}

#endif
