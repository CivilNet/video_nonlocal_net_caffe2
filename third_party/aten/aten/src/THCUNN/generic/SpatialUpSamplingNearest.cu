
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/SpatialUpSamplingNearest.cu"
#else

#include "../common.h"

static inline void THNN_(SpatialUpSamplingNearest_shapeCheck)
                        (THCState *state,THCTensor *input, THCTensor *gradOutput,
                         int scale_factor) {
  THArgCheck(input != NULL, 2, "4D input tensor expected but got NULL");
  THArgCheck(scale_factor > 1, 4,
             "scale_factor must be greater than 1, but got: %d", scale_factor);
  THCUNN_argCheck(state, input->nDimension == 3 || input->nDimension == 4, 2, input,
                  "3D or 4D input tensor expected but got: %s");
  if (input->nDimension == 3) {
    int nChannels    = THCTensor_(size)(state, input, 0);
    int inputHeight  = THCTensor_(size)(state, input, 1);
    int inputWidth   = THCTensor_(size)(state, input, 2);
    int outputHeight = inputHeight * scale_factor;
    int outputWidth  = inputWidth  * scale_factor;
    if (gradOutput != NULL) {
      THCUNN_check_dim_size(state, gradOutput, 3, 0, nChannels);
      THCUNN_check_dim_size(state, gradOutput, 3, 1, outputHeight);
      THCUNN_check_dim_size(state, gradOutput, 3, 2, outputWidth);
    }
  } else {
    int nBatch       = THCTensor_(size)(state, input, 0);
    int nChannels    = THCTensor_(size)(state, input, 1);
    int inputHeight  = THCTensor_(size)(state, input, 2);
    int inputWidth   = THCTensor_(size)(state, input, 3);
    int outputHeight = inputHeight * scale_factor;
    int outputWidth  = inputWidth  * scale_factor;
    if (gradOutput != NULL) {
      THCUNN_check_dim_size(state, gradOutput, 4, 0, nBatch);
      THCUNN_check_dim_size(state, gradOutput, 4, 1, nChannels);
      THCUNN_check_dim_size(state, gradOutput, 4, 2, outputHeight);
      THCUNN_check_dim_size(state, gradOutput, 4, 3, outputWidth);
    }
  }
}

void THNN_(SpatialUpSamplingNearest_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           int scale_factor)
{
  THCTensor_(zero)(state, output);

  THCUNN_assertSameGPU(state, 2, input, output);
  THNN_(SpatialUpSamplingNearest_shapeCheck)(state, input, NULL, scale_factor);
  int inputHeight = THCTensor_(size)(state, input, input->nDimension-2);
  int inputWidth  = THCTensor_(size)(state, input,  input->nDimension-1);
  int outputHeight = inputHeight * scale_factor;
  int outputWidth = inputWidth * scale_factor;

   if (input->nDimension == 3) {
     THCTensor_(resize3d)(state, output,
                          THCTensor_(size)(state, input, 0),
                          outputHeight, outputWidth);
   } else {
     THCTensor_(resize4d)(state, output,
                          THCTensor_(size)(state, input, 0),
                          THCTensor_(size)(state, input, 1),
                          outputHeight, outputWidth);
  }

  input = THCTensor_(newContiguous)(state, input);
  // This is for allocating output Tensor
  int64_t no_elements = 1;
  for(int i = 0; i < input->nDimension; i++){
    no_elements *= input->size[i];
  }
  no_elements *= scale_factor * scale_factor;

  int d1;
  int d2;
  int d3;

  if (input->nDimension == 3) {
    d1 = output->size[0];
    d2 = output->size[1];
    d3 = output->size[2];
  } else {
    d1 = output->size[1];
    d2 = output->size[2];
    d3 = output->size[3];
  }

  real *input_data = THCTensor_(data)(state, input);
  real *output_data = THCTensor_(data)(state, output);

  // cuda blocks & threads:
  int64_t nthreads = 256;
  // Max number of blocks: http://en.wikipedia.org/wiki/CUDA
  // 65535 for SM 2.x, 2^32 -1 for >= 3.0
  // TODO: When we move to SM 3.5 we should update this
  int64_t n_xblocks = min(max((int)ceil((float)no_elements / nthreads), 1), 65535);
  int64_t n_yblocks = (int64_t)ceil((float)no_elements / (float)(n_xblocks * nthreads));
  if (n_yblocks > 65535) {
    THError("Input size is too large!  aborting");
  }
  dim3 blocks(n_xblocks, n_yblocks);
  dim3 threads(nthreads);

  // kernel:
  upscale<<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (input_data, output_data, no_elements, scale_factor, d1, d2, d3);
  THCudaCheck(cudaGetLastError());

  // final cut:
  THCTensor_(free)(state, input);
}

void THNN_(SpatialUpSamplingNearest_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           int scale_factor)
{

  THCUNN_assertSameGPU(state, 2, gradOutput, gradInput);
  THNN_(SpatialUpSamplingNearest_shapeCheck)(state, input, gradOutput, scale_factor);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);
  THCTensor_(resizeAs)(state, gradInput, input);

  THCTensor_(zero)(state, gradInput);

  real *gradInput_data = THCTensor_(data)(state, gradInput);
  real *gradOutput_data = THCTensor_(data)(state, gradOutput);

  int64_t no_elements = 1;
  for(int i = 0; i < gradInput->nDimension; i++){
    no_elements *= gradInput->size[i];
  }

  int d1;
  int d2;
  int d3;

  if (gradInput->nDimension == 3) {
    d1 = gradInput->size[0];
    d2 = gradInput->size[1];
    d3 = gradInput->size[2];
  } else {
    d1 = gradInput->size[1];
    d2 = gradInput->size[2];
    d3 = gradInput->size[3];
  }

  // cuda blocks & threads:
  int64_t nthreads = 256;
  // Max number of blocks: http://en.wikipedia.org/wiki/CUDA
  // 65535 for SM 2.x, 2^32 -1 for >= 3.0
  // TODO: When we move to SM 3.5 we should update this
  int64_t n_xblocks = min(max((int)ceil((float)no_elements / nthreads), 1), 65535);
  int64_t n_yblocks = (int64_t)ceil((float)no_elements / (float)(n_xblocks * nthreads));
  if (n_yblocks > 65535) {
    THError("Input size is too large!  aborting");
  }
  dim3 blocks(n_xblocks, n_yblocks);
  dim3 threads(nthreads);

  // kernel:
  downscale<real ,accreal> <<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (gradInput_data, gradOutput_data, no_elements,
    scale_factor, d1, d2, d3);
  THCudaCheck(cudaGetLastError());
  THCTensor_(free)(state, gradOutput);
}

#endif
