
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/Sqrt.cu"
#else

#include "../common.h"

void THNN_(Sqrt_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           accreal eps_)
{
  real eps = ScalarConvert<accreal, real>::to(eps_);
  THCUNN_assertSameGPU(state, 2, input, output);
  THCTensor_(resizeAs)(state, output, input);
  THC_pointwiseApply2(state, output, input, sqrtupdateOutput_functor<real>(eps));
}

void THNN_(Sqrt_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCTensor *output)
{
  THCUNN_check_shape(state, output, gradOutput);
  THCUNN_assertSameGPU(state, 3, output, gradOutput, gradInput);
  THCTensor_(resizeAs)(state, gradInput, output);
  THC_pointwiseApply3(state, gradInput, output, gradOutput, sqrtupdateGradInput_functor<real>());
}

#endif
