
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/SpatialGridSamplerBilinear.cu"
#else

static inline void THNN_(SpatialGridSamplerBilinear_shapeCheck)(
    THCState *state,
    THCTensor *input,
    THCTensor *grid,
    THCTensor *gradOutput) {
  THCUNN_argCheck(state, THCTensor_(nDimension)(state, input) == 4, 2, input,
      "4D input tensor expected but got: %s");
  THCUNN_argCheck(state, THCTensor_(nDimension)(state, grid) == 4, 2, grid,
      "4D grid tensor expected but got: %s");

  int64_t nbatch   = THCTensor_(size)(state, input, 0);
  int64_t channels = THCTensor_(size)(state, input, 1);
  int64_t iheight   = THCTensor_(size)(state, input, 2);
  int64_t iwidth    = THCTensor_(size)(state, input, 3);
  int64_t oheight   = THCTensor_(size)(state, grid, 1);
  int64_t owidth    = THCTensor_(size)(state, grid, 2);

  THCUNN_check_dim_size(state, grid, 4, 0, nbatch);
  THCUNN_check_dim_size(state, grid, 4, 3, 2);

  if (gradOutput != NULL) {
    THCUNN_check_dim_size(state, gradOutput, 4, 0, nbatch);
    THCUNN_check_dim_size(state, gradOutput, 4, 1, channels);
    THCUNN_check_dim_size(state, gradOutput, 4, 2, oheight);
    THCUNN_check_dim_size(state, gradOutput, 4, 3, owidth);
  }
}

TH_API void THNN_(SpatialGridSamplerBilinear_updateOutput)(
    THCState *state,
    THCTensor *input,
    THCTensor *grid,
    THCTensor *output,
    int padding_mode) {

  THCUNN_assertSameGPU(state, 3, input, grid, output);
  THNN_(SpatialGridSamplerBilinear_shapeCheck)(state, input, grid, NULL);
  int64_t N = THCTensor_(size)(state, input, 0);
  int64_t C = THCTensor_(size)(state, input, 1);
  int64_t IH = THCTensor_(size)(state, input, 2);
  int64_t IW = THCTensor_(size)(state, input, 3);
  int64_t H = THCTensor_(size)(state,grid, 1);
  int64_t W = THCTensor_(size)(state, grid, 2);

  // resize output to the same shape as input
  THCTensor_(resize4d)(state, output, N, C, H, W);

  THCDeviceTensor<real, 4> devInput = toDeviceTensor<real, 4>(state, input);
  THCDeviceTensor<real, 4> devGrid = toDeviceTensor<real, 4>(state, grid);
  THCDeviceTensor<real, 4> devOutput = toDeviceTensor<real, 4>(state, output);

  int count = static_cast<int>(N*H*W);
  SpatialGridSamplerBilinear_updateOutput_kernel
    <<<GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
      count, devInput, devGrid, devOutput, padding_mode);
  THCudaCheck(cudaGetLastError());
}

TH_API void THNN_(SpatialGridSamplerBilinear_updateGradInput)(
    THCState *state,
    THCTensor *input, THCTensor *gradInput,
    THCTensor *grid, THCTensor *gradGrid,
    THCTensor *gradOutput,
    int padding_mode) {

  THCUNN_assertSameGPU(state, 5, input, gradInput, grid, gradGrid, gradOutput);
  THNN_(SpatialGridSamplerBilinear_shapeCheck)(state, input, grid, gradOutput);
  int64_t N = THCTensor_(size)(state, input, 0);
  int64_t C = THCTensor_(size)(state, input, 1);
  int64_t IH = THCTensor_(size)(state, input, 2);
  int64_t IW = THCTensor_(size)(state, input, 3);
  int64_t H = THCTensor_(size)(state, grid, 1);
  int64_t W = THCTensor_(size)(state, grid, 2);

  THCTensor_(resize4d)(state, gradInput, N, C, IH, IW);
  THCTensor_(resize4d)(state, gradGrid, N, H, W, 2);
  THCTensor_(zero)(state, gradInput);
  THCTensor_(zero)(state, gradGrid);

  THCDeviceTensor<real, 4> devInput = toDeviceTensor<real, 4>(state, input);
  THCDeviceTensor<real, 4> devGradInput = toDeviceTensor<real, 4>(state, gradInput);
  THCDeviceTensor<real, 4> devGrid = toDeviceTensor<real, 4>(state, grid);
  THCDeviceTensor<real, 4> devGradGrid = toDeviceTensor<real, 4>(state, gradGrid);
  THCDeviceTensor<real, 4> devGradOutput = toDeviceTensor<real, 4>(state, gradOutput);

  int count = static_cast<int>(N*H*W);
  SpatialGridSamplerBilinear_updateGradInput_kernel
    <<<GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
      count, devInput, devGradInput, devGrid, devGradGrid, devGradOutput, padding_mode);
  THCudaCheck(cudaGetLastError());
}

#endif
