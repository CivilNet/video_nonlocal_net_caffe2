
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/SoftMarginCriterion.cu"
#else

#include "THCApply.cuh"

void THNN_(SoftMarginCriterion_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *output,
           bool sizeAverage,
           bool reduce)
{
  THCUNN_check_shape(state, input, target);
  THCUNN_assertSameGPU(state, 3, input, target, output);

  if (!reduce) {
    THCTensor_(resizeAs)(state, output, input);
    THC_pointwiseApply3(state, input, target, output,
        softmargin_no_reduce_functor<real, accreal>());
    return;
  }

  accreal sum;
  ptrdiff_t size = THCTensor_(nElement)(state, input);

  input = THCTensor_(newContiguous)(state, input);
  target = THCTensor_(newContiguous)(state, target);
  THCTensor_(resize1d)(state, output, 1);

  thrust::device_ptr<real> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<real> target_data(THCTensor_(data)(state, target));
  sum = thrust::inner_product(input_data, input_data+size, target_data, (accreal) 0, thrust::plus<accreal>(), softmargin_functor<real, accreal>());

  if(sizeAverage)
    sum /= size;

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, target);

  THCTensor_(set1d)(state, output, 0, ScalarConvert<accreal, real>::to(sum));
}

void THNN_(SoftMarginCriterion_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           bool sizeAverage,
           bool reduce)
{
  THCUNN_check_shape(state, input, target);
  THCUNN_assertSameGPU(state, 4, input, target, gradInput, gradOutput);

  THCTensor_(resizeAs)(state, gradInput, input);

  if (!reduce) {
    THCUNN_check_shape(state, gradOutput, input);
    THC_pointwiseApply3(state, input, target, gradInput,
        softmargin_updateGradInput_no_reduce_functor<real, accreal>());
    THCTensor_(cmul)(state, gradInput, gradInput, gradOutput);
    return;
  }

  ptrdiff_t size = THCTensor_(nElement)(state, input);
  accreal norm = (sizeAverage ? 1./size : 1.);

  input = THCTensor_(newContiguous)(state, input);
  target = THCTensor_(newContiguous)(state, target);


  thrust::device_ptr<real> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<real> target_data(THCTensor_(data)(state, target));
  thrust::device_ptr<real> gradInput_data(THCTensor_(data)(state, gradInput));

  thrust::transform(input_data, input_data+size, target_data, gradInput_data,
                    softmargin_updateGradInput_functor<real, accreal>(norm, THCTensor_(get1d)(state, gradOutput, 0)));

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, target);
}

#endif
