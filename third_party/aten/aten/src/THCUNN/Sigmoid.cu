#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "THCHalf.h"
#include "THCHalfAutoNumerics.cuh"
#include <THC/THCApply.cuh>

template <typename T>
struct sigmoid_updateGradInput_functor {
  __device__ __forceinline__ void operator()(T* gradInput, const T *output, const T *gradOutput) const {
    *gradInput = *gradOutput * (1.f - *output) * (*output);
  }
};

#ifdef CUDA_HALF_TENSOR
template <>
struct sigmoid_updateGradInput_functor<half> {
  __device__ __forceinline__ void operator()(half* gradInput, const half *output, const half *gradOutput) const {
#ifdef CUDA_HALF_INSTRUCTIONS
    const half one = __float2half(1.f);
    *gradInput = __hmul(*gradOutput, __hmul(__hadd(one, __hneg(*output)), *output));
#else
    const float out = __half2float(*output);
    const float go = __half2float(*gradOutput);
    *gradInput = __float2half(go * (1.f - out) * out);
#endif
  }
};
#endif

#include "generic/Sigmoid.cu"
#include "THCGenerateFloatTypes.h"
