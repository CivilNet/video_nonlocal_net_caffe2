#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "THCHalf.h"
#include "THCHalfAutoNumerics.cuh"
#include "THCAtomics.cuh"

#define CUDA_MAX_THREADS 1024   // this is safe, in reality 256 is our limit

#define START_IND(a,b,c) (int)floor((float)(a * c) / b)
#define END_IND(a,b,c) (int)ceil((float)((a + 1) * c) / b)
// #define START_IND(a,b,c) a * c / b
// #define END_IND(a,b,c)  (a + 1) * c / b + ((a + 1) * c % b > 0)?1:0

// 5d tensor B x D x T x H x W

/*
 * Description:
 *    this function adaptively maxpools an input 4D tensor along dimensions 2 and 3
 *    4D input, 4D output, 4D argmax x and y
 */
 template <typename T>
__global__ void cunn_VolumetricAdaptiveMaxPooling_updateOutput_kernel(
                        T *input, T *output, THCIndex_t *indices,
                        int isizeT, int isizeH, int isizeW,
                        int osizeT, int osizeH, int osizeW,
                        int64_t istrideD,
                        int64_t istrideT, int64_t istrideH, int64_t istrideW,
                        int64_t offsetZ)
{
  // iterators on output pixels
  int ot, oh, ow;

  // compute offsets based on thread/block ID
  int ostartH = blockIdx.y * blockDim.y + threadIdx.y;
  int oendH   = osizeH;
  int ostepH  = gridDim.y * blockDim.y;
  int ostartW = threadIdx.x;
  int oendW   = osizeW;
  int ostepW  = blockDim.x;

  // select output plane
  int64_t o_plane = blockIdx.x + offsetZ;
  ot = o_plane % osizeT;     // output frame/time
  int d = o_plane / osizeT;  // slice/feature

  // input frame/time ramge is fixed.
  int istartT = START_IND(ot, osizeT, isizeT);
  int iendT = END_IND(ot, osizeT, isizeT);
  int kT = iendT - istartT;

  // input offset by slice/feature and earliest relevant frame/time
  T *input_dt = input + d*istrideD + istartT*istrideT;
  // output offset by slice/feature and frame/time
  T *output_dt = output + o_plane*osizeH*osizeW;
  // indices offset by slice/feature and frame/time
  THCIndex_t *indices_dt = indices + o_plane*osizeH*osizeW;

  // For all output pixels...
  for(oh = ostartH; oh < oendH; oh += ostepH) {

    int istartH = START_IND(oh, osizeH, isizeH);
    int iendH   = END_IND(oh, osizeH, isizeH);
    int kH = iendH - istartH;

    for(ow = ostartW; ow < oendW; ow += ostepW) {

      int istartW = START_IND(ow, osizeW, isizeW);
      int iendW   = END_IND(ow, osizeW, isizeW);
      int kW = iendW - istartW;

      // Compute the average pooling from corresponding input pixels
      T *ptr_input = input_dt + istartH*istrideH + istartW*istrideW;
      T *ptr_output = output_dt + oh*osizeW + ow;
      THCIndex_t *ptr_ind = indices_dt + oh*osizeW + ow;
      int64_t argmax = -1;
      T max = THCNumerics<T>::min();

      int it, ih, iw;
      for(it = 0; it < kT; ++it) {
        for(ih = 0; ih < kH; ++ih) {
          for(iw = 0; iw < kW; ++iw) {
            T val = ptr_input[ih*istrideH + iw*istrideW];
            if (val > max) {
              max = val;
              argmax = (it+istartT)*isizeH*isizeW + (ih+istartH)*isizeW + iw+istartW;
            }
          }
        }
        ptr_input += istrideT;   // next input frame
      }
      // Update output and argmax
      *ptr_output = max;
      *ptr_ind = argmax + TH_INDEX_BASE;
    }
  }
}

/*
 * Description:
 *    This function computes the gradInput from gradOutput.
 *
 *    gridDim.y blocks work together on a single 2D output plane specified by
 *    (blockIdx.x + offsetZ).
 *
 *    Assumes that input size can be perfectly divided by output size, i.e.
 *    each input pixel can only be argmax of one output pixel.
 */
 template <typename T>
__global__ void cunn_VolumetricAdaptiveMaxPooling_updateGradInput_kernel(
  T *gradInput, T *gradOutput, THCIndex_t *indices,
  int isizeT, int isizeH, int isizeW,
  int osizeT, int osizeH, int osizeW,
  int64_t offsetZ
)
{
  // iterators on output pixels
  int oh, ow;

  // compute offsets based on thread/block ID
  int ostartH = blockIdx.y * blockDim.y + threadIdx.y;
  int oendH   = osizeH;
  int ostepH  = gridDim.y * blockDim.y;
  int ostartW = threadIdx.x;
  int oendW   = osizeW;
  int ostepW  = blockDim.x;

  // select output plane
  int64_t o_plane = blockIdx.x + offsetZ;
  int d = o_plane / osizeT;     // output slice/feature

  // gradInput offset by slice/feature
  T *gradInput_d = gradInput + d*isizeT*isizeH*isizeW;
  // gradOutput offset by slice/feature and frame/otme
  T *gradOutput_dt = gradOutput + o_plane*osizeH*osizeW;
  // indices offset by slice/feature and frame/otme
  THCIndex_t *indices_dt = indices + o_plane*osizeH*osizeW;

  // For all output pixels...
  for(oh = ostartH; oh < oendH; oh += ostepH) {
    for(ow = ostartW; ow < oendW; ow += ostepW) {
      // Compute the gradients for the argmax input pixel
      T *ptr_gradOutput = gradOutput_dt + oh*osizeW + ow;
      THCIndex_t *ptr_ind = indices_dt + oh*osizeW + ow;
      T grad_delta = *ptr_gradOutput;
      int argmax = (*ptr_ind) - TH_INDEX_BASE;
      gradInput_d[argmax] += grad_delta;
    }
  }
}


/*
 * Description:
 *    This function computes the gradInput from gradOutput.
 *
 *    gridDim.y blocks work together on a single 2D output plane specified by
 *    (blockIdx.x + offsetZ).
 *
 *    Uses atomic add.
 */
 template <typename T>
__global__ void cunn_atomic_VolumetricAdaptiveMaxPooling_updateGradInput_kernel(
  T *gradInput, T *gradOutput, THCIndex_t *indices,
  int isizeT, int isizeH, int isizeW,
  int osizeT, int osizeH, int osizeW,
  int64_t offsetZ
)
{
  // iterators on output pixels
  int oh, ow;

  // compute offsets based on thread/block ID
  int ostartH = blockIdx.y * blockDim.y + threadIdx.y;
  int oendH   = osizeH;
  int ostepH  = gridDim.y * blockDim.y;
  int ostartW = threadIdx.x;
  int oendW   = osizeW;
  int ostepW  = blockDim.x;

  // select output plane
  int64_t o_plane = blockIdx.x + offsetZ;
  int d = o_plane / osizeT;     // output slice/feature

  // gradInput offset by slice/feature
  T *gradInput_d = gradInput + d*isizeT*isizeH*isizeW;
  // gradOutput offset by slice/feature and frame/otme
  T *gradOutput_dt = gradOutput + o_plane*osizeH*osizeW;
  // indices offset by slice/feature and frame/otme
  THCIndex_t *indices_dt = indices + o_plane*osizeH*osizeW;

  // For all output pixels...
  for(oh = ostartH; oh < oendH; oh += ostepH) {
    for(ow = ostartW; ow < oendW; ow += ostepW) {
      // Compute the gradients for the argmax input pixel
      T *ptr_gradOutput = gradOutput_dt + oh*osizeW + ow;
      THCIndex_t *ptr_ind = indices_dt + oh*osizeW + ow;
      T grad_delta = *ptr_gradOutput;
      int64_t argmax = (*ptr_ind) - TH_INDEX_BASE;
      atomicAdd(&(gradInput_d[argmax]), grad_delta);
    }
  }
}

#include "generic/VolumetricAdaptiveMaxPooling.cu"
#include "THCGenerateFloatTypes.h"

#undef CUDA_MAX_THREADS
