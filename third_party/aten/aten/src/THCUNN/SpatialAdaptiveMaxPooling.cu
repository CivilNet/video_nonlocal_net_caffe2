#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "THCHalf.h"
#include "THCHalfAutoNumerics.cuh"
#include "THCAtomics.cuh"

#define CUDA_MAX_THREADS 1024   // this is safe, in reality 256 is our limit

#define START_IND(a,b,c) (int)floor((float)(a * c) / b)
#define END_IND(a,b,c) (int)ceil((float)((a + 1) * c) / b)
// #define START_IND(a,b,c) a * c / b
// #define END_IND(a,b,c)  (a + 1) * c / b + ((a + 1) * c % b > 0)?1:0

// 4d tensor B x D x H x W

/*
 * Description:
 *    this function adaptively maxpools an input 4D tensor along dimensions 2 and 3
 *    4D input, 4D output, 4D argmax x and y
 */
 template <typename T>
__global__ void adaptivemaxpool(T *input, T *output, THCIndex_t *indices,
                        int isizeH, int isizeW,
                        int osizeH, int osizeW,
                        int64_t istrideD, int64_t istrideH, int64_t istrideW)
{
  // iterators
  int oh, ow;

  // compute offsets based on thread/block ID
  int o_plane = blockIdx.x;
  int i_plane = o_plane;

  int ostartW = threadIdx.x;
  int oendW = osizeW;
  const int ostepW = blockDim.x;

  int ostartH = blockDim.y*blockIdx.y + threadIdx.y;
  int oendH = osizeH;
  const int ostepH = blockDim.y*gridDim.y;
  // select input/output plane
  output = output + o_plane*osizeH*osizeW;
  input = input + i_plane*istrideD;
  indices = indices + o_plane*osizeH*osizeW;

  // For all output pixels...
  for(oh = ostartH; oh < oendH; oh += ostepH) {

    int istartH = START_IND(oh, osizeH, isizeH);
    int iendH   = END_IND(oh, osizeH, isizeH);
    int kH = iendH - istartH;

    for(ow = ostartW; ow < oendW; ow += ostepW) {
      int istartW = START_IND(ow, osizeW, isizeW);
      int iendW   = END_IND(ow, osizeW, isizeW);

      int kW = iendW - istartW;

      // Compute the mean of the input image...
      T *ptr_input = input + istartH*istrideH + istartW*istrideW;
      T *ptr_output = output + oh*osizeW + ow;
      THCIndex_t *ptr_ind = indices + oh*osizeW + ow;
      int argmax = -1;
      T max = THCNumerics<T>::min();
      int ih, iw;
      for(ih = 0; ih < kH; ih++) {
        for(iw = 0; iw < kW; iw++) {
          T val = ptr_input[iw*istrideW];
          if (val > max) {
            max = val;
            argmax = (ih+istartH)*isizeW + iw+istartW;
          }
        }
        ptr_input += istrideH; // next input line
      }
      // Update output and argmax
      *ptr_output = max;
      *ptr_ind = argmax + TH_INDEX_BASE;
    }
  }
}

/*
 * Description:
 *    this function computes the gradInput from weight and gradOutput
 */
 template <typename T>
__global__ void adaptivemaxgradinput(T *gradInput, T *gradOutput, THCIndex_t *indices,
                             int isizeH, int isizeW,
                             int osizeH, int osizeW)
{
  // iterators
  int oh, ow;

  // compute offsets based on thread/block ID
  int o_plane = blockIdx.x;
  int i_plane = o_plane;
  //int k = blockIdx.x % sizeD;

  int ostartW = threadIdx.x;
  int oendW = osizeW;
  int ostepW = blockDim.x;

  int ostartH = blockDim.y*blockIdx.y + threadIdx.y;
  int oendH = osizeH;
  int ostepH = blockDim.y*gridDim.y;

  // select input/output plane
  gradOutput = gradOutput + o_plane*osizeH*osizeW;
  gradInput = gradInput + i_plane*isizeH*isizeW;
  indices = indices + o_plane*osizeH*osizeW;

  // compute gradInput
  for(oh = ostartH; oh < oendH; oh += ostepH) {

    for(ow = ostartW; ow < oendW; ow += ostepW) {

      T *ptr_gradOutput = gradOutput + oh*osizeW + ow;
      THCIndex_t *ptr_ind = indices + oh*osizeW + ow;
      T z = *ptr_gradOutput;

      int argmax = (*ptr_ind) - TH_INDEX_BASE;

      gradInput[argmax] += z;
    }
  }
}

/*
 * Description:
 *    this function computes the gradInput from weight and gradOutput
 *    when kH != dH or kW != dW (uses atomic add)
 */
 template <typename T>
__global__ void atomicadaptivemaxgradinput(
  T *gradInput, T *gradOutput, THCIndex_t *indices,
  int isizeH, int isizeW, int osizeH, int osizeW
)
{
  // iterators
  int oh, ow;

  // compute offsets based on thread/block ID
  int o_plane = blockIdx.x;
  int i_plane = o_plane;

  int ostartW = threadIdx.x;
  int oendW = osizeW;
  int ostepW = blockDim.x;

  int ostartH = blockDim.y*blockIdx.y + threadIdx.y;
  int oendH = osizeH;
  int ostepH = blockDim.y*gridDim.y;

  // select input/output plane
  gradOutput = gradOutput + o_plane*osizeH*osizeW;
  gradInput = gradInput + i_plane*isizeH*isizeW;
  indices = indices + o_plane*osizeH*osizeW;

  // compute gradInput
  for(oh = ostartH; oh < oendH; oh += ostepH) {

    for(ow = ostartW; ow < oendW; ow += ostepW) {

      T *ptr_gradOutput = gradOutput + oh*osizeW + ow;
      THCIndex_t *ptr_ind = indices + oh*osizeW + ow;
      T z = *ptr_gradOutput;

      int argmax = (*ptr_ind) - TH_INDEX_BASE;

      // atomic add since different threads could update same variable
      atomicAdd(&(gradInput[argmax]), z);
    }
  }
}

#include "generic/SpatialAdaptiveMaxPooling.cu"
#include "THCGenerateFloatTypes.h"

#undef CUDA_MAX_THREADS
