#include "hip/hip_runtime.h"
#include "ATen/NativeFunctions.h"
#include <cfloat>

namespace at {
namespace native {

__host__ __device__ __forceinline__ float fmin(float a, float b) {
  return a > b ? b : a;
}

__host__ __device__ __forceinline__ float fmax(float a, float b) {
  return a > b ? a : b;
}

template <typename T>
__global__ void RoiPooling2d_forward_kernel(
  const int outputElements,
  const T *input,
  const T *rois,
  const T spatialScale,
  const int inputChannels,
  const int inputHeight,
  const int inputWidth,
  const int pooledHeight,
  const int pooledWidth,
  T *output,
  int *argmaxes)
{
  for (int linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
       linearIndex < outputElements;
       linearIndex += blockDim.x * gridDim.x)
  {
    // Calculate position in output Tensor, i.e. a specific combination
    // of proposal, channel, pool height and pool width
    // TODO: write to improve performance by minimize computation
    int pw = linearIndex % pooledWidth;
    int ph = (linearIndex / pooledWidth) % pooledHeight;
    int ch = (linearIndex / pooledWidth / pooledHeight) % inputChannels;
    int proposal = linearIndex / pooledWidth / pooledHeight / inputChannels;

    // Get particular proposal data
    const T *roisOffset = rois + (proposal * 5);
    int n = roisOffset[0];
    int startWidth = llrintf(roisOffset[1] * spatialScale);
    int startHeight = llrintf(roisOffset[2] * spatialScale);
    int endWidth = llrintf(roisOffset[3] * spatialScale);
    int endHeight = llrintf(roisOffset[4] * spatialScale);

    // TODO: fix malformed RoIs to be 1x1

    int roiHeight = endHeight - startHeight;
    int roiWidth = endWidth - startWidth;

    // Calculate size of tile based on the size of this particular RoI and the
    // output size
    T tileHeight = static_cast<T>(roiHeight) / static_cast<T>(pooledHeight);
    T tileWidth = static_cast<T>(roiWidth) / static_cast<T>(pooledWidth);

    // Calculate offset into the pooled region
    int tileHStart = static_cast<int>(floorf(static_cast<T>(ph) * tileHeight));
    int tileWStart = static_cast<int>(floorf(static_cast<T>(pw) * tileWidth));
    int tileHEnd = static_cast<int>(ceilf(static_cast<T>(ph + 1) * tileHeight));
    int tileWEnd = static_cast<int>(ceilf(static_cast<T>(pw + 1) * tileWidth));

    // Calculate offset into the image itself, based on RoI + pooled offsets,
    // and ensure it falls within image boundaries
    tileHStart = fmin(fmax(tileHStart + startHeight, 0), inputHeight);
    tileWStart = fmin(fmax(tileWStart + startWidth, 0), inputWidth);
    tileHEnd = fmin(fmax(tileHEnd + startHeight, 0), inputHeight);
    tileWEnd = fmin(fmax(tileWEnd + startWidth, 0), inputWidth);

    // If our pooling region is empty, we set the output to 0, otherwise to
    // the min float so we can calculate the max properly
    bool isEmpty = (tileHStart >= tileHEnd) || (tileWStart >= tileWEnd);
    T max = isEmpty ? 0 : FLT_MIN;
    // If nothing is pooled, argmax = -1 causes nothing to be backprop'd
    int maxIdx = -1;

    const T *inputOffset = input + ((n * inputChannels + ch) * inputHeight * inputWidth);
    for (int th = tileHStart; th < tileHEnd; ++th) {
      for (int tw = tileWStart; tw < tileWEnd; ++tw) {
        int index = (th * inputWidth) + tw;
	if (inputOffset[index] > max) {
          max = inputOffset[index];
	  maxIdx = index;
	}
      }
    }
    output[linearIndex] = max;

    // TODO optional argmax
    argmaxes[linearIndex] = maxIdx;
  }
}

std::tuple<Tensor, Tensor> RoiPooling2d_forward_cuda(
  const Tensor& input,
  const Tensor& rois,
  int64_t pooledHeight,
  int64_t pooledWidth,
  double spatialScale)
{

  // Input is the output of the last convolutional layer in the Backbone network, so
  // it should be in the format of NCHW
  AT_ASSERT(input.ndimension() == 4, "Input to RoI Pooling should be a NCHW Tensor");

  // ROIs is the set of region proposals to process. It is a 2D Tensor where the first
  // dim is the # of proposals, and the second dim is the proposal itself in the form
  // [batch_index startW startH endW endH]
  AT_ASSERT(rois.ndimension() == 2, "RoI Proposals should be a 2D Tensor, (batch_sz x proposals)");
  AT_ASSERT(rois.size(1) == 5, "Proposals should be of the form [batch_index startW startH endW enH]");

  auto proposals = rois.size(0);
  auto inputChannels = input.size(1);
  auto inputHeight = input.size(2);
  auto inputWidth = input.size(3);

  // Output Tensor is (num_rois, C, pooledHeight, pooledWidth)
  auto output = input.type().tensor({proposals, inputChannels, pooledHeight, pooledWidth});

  // TODO: need some mechanism for determining train vs. test

  // During training, we need to store the argmaxes for the pooling operation, so
  // the argmaxes Tensor should be the same size as the output Tensor
  auto argmaxes = input.type().toScalarType(kInt).tensor({proposals, inputChannels, pooledHeight, pooledWidth});

  AT_ASSERT(input.is_contiguous(), "input must be contiguous");
  AT_ASSERT(rois.is_contiguous(), "rois must be contiguous");

  dim3 block(512);
  dim3 grid((output.numel() + 512 - 1) / 512);
  RoiPooling2d_forward_kernel<<<grid, block, 0, globalContext().getCurrentCUDAStream()>>>(
    output.numel(), input.data<float>(), rois.data<float>(), static_cast<float>(spatialScale), inputChannels,
    inputHeight, inputWidth, pooledHeight, pooledWidth, output.data<float>(), argmaxes.data<int>());
  AT_ASSERT(hipGetLastError() == hipSuccess, "RoiPooling2d_forward_kernel failed");

  return std::make_tuple(output, argmaxes);
}

template <typename T>
__global__ void RoiPooling2d_backward_kernel(
  const int outputElements,
  const T *gradOutput,
  const int *argmaxes,
  const int proposals,
  const T spatialScale,
  const int inputChannels,
  const int inputHeight,
  const int inputWidth,
  const int pooledHeight,
  const int pooledWidth,
  T *gradInput,
  const T *rois)
{
  for (int linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
       linearIndex < outputElements;
       linearIndex += blockDim.x * gridDim.x)
  {
    int pw = linearIndex % pooledWidth;
    int ph = (linearIndex / pooledWidth) / pooledHeight;
    int ch = (linearIndex / pooledWidth / pooledHeight) % inputChannels;
    int proposal = linearIndex / pooledWidth / pooledHeight / inputChannels;

    const T *roisOffset = rois + (proposal * 5);
    int n = roisOffset[0];
    int gradInputOffset = (n * inputChannels + ch) * inputHeight * inputWidth;
    int gradOutputOffset = (n * inputChannels + ch) * pooledHeight * pooledWidth;
    const T* gradOutputShifted = gradOutput + gradOutputOffset;
    T *gradInputShifted = gradInput + gradInputOffset;
    const int *argmaxesShifted = argmaxes + gradOutputOffset;

    int argmax = argmaxesShifted[ph * pooledWidth + pw];
    if (argmax != -1) {
      atomicAdd(gradInputShifted + argmax, gradOutputShifted[ph * pooledWidth + pw]);
    }
  }
}

Tensor RoiPooling2d_backward_cuda(
  const Tensor& input,
  const Tensor& rois,
  int64_t pooledHeight,
  int64_t pooledWidth,
  double spatialScale,
  const Tensor& gradOutput,
  const Tensor& argmaxes)
{
  // TODO: assertions?

  auto proposals = rois.size(0);
  auto inputChannels = input.size(1);
  auto inputHeight = input.size(2);
  auto inputWidth = input.size(3);

  auto gradInput = input.type().tensor(input.sizes());

  dim3 block(512);
  dim3 grid((gradInput.numel() + 512 - 1) / 512);
  RoiPooling2d_backward_kernel<<<grid, block, 0, globalContext().getCurrentCUDAStream()>>>(
    gradOutput.numel(), gradOutput.data<float>(), argmaxes.data<int>(), proposals,
    static_cast<float>(spatialScale), inputChannels, inputHeight, inputWidth,
    pooledHeight, pooledWidth, gradInput.data<float>(), rois.data<float>());
  AT_ASSERT(hipGetLastError() == hipSuccess, "RoiPooling2d_forward_kernel failed");

  return gradInput;
}

} // at::native
} // at
