#include "hip/hip_runtime.h"
#include "ATen/ATen.h"
#include "ATen/TensorUtils.h"
#include "ATen/NativeFunctions.h"

#include "ATen/cuda/AccumulateType.cuh"
#include "ATen/cuda/CUDATensorMethods.cuh"
#include "ATen/cuda/CUDATypeConversion.cuh"

#include <THC/THCDeviceUtils.cuh>
#include <THC/THCNumerics.cuh>
#include <THC/THCTensorMathReduce.cuh>
#include <THC/THCTensorSort.cuh>
#include <THC/THCThrustAllocator.cuh>
#include <THCUNN/THCHalfAutoNumerics.cuh>

#include <thrust/execution_policy.h>
#include <thrust/unique.h>

const int WARP_SIZE = 32;
const int MODE_SUM = 0;
const int MODE_MEAN = 1;

namespace at {
namespace native {

namespace {

template <typename scalar_t>
__global__ void EmbeddingBag_updateOutputKernel(
    int64_t *input, int64_t *offsets, scalar_t *weight, scalar_t *output,
    int64_t *offset2bag, int64_t numIndices, int64_t numBags, int64_t stride,
    int mode, int64_t *bag_size) {

  // the strategy here is that each bag x feature is handled by a single thread

  using accscalar_t = cuda::acc_type<scalar_t>;
  int64_t chunksPerBag = THCCeilDiv(stride, (int64_t)blockDim.x);
  int64_t numChunks = numBags * chunksPerBag;
  int64_t chunkOffset = blockIdx.x * blockDim.y + threadIdx.y;
  int64_t chunkStride = gridDim.x * blockDim.y;

  for (int64_t chunk = chunkOffset; chunk < numChunks; chunk += chunkStride) {
    int64_t featureDim = (chunk % chunksPerBag) * blockDim.x + threadIdx.x;
    if (featureDim < stride) {
      int64_t bag = chunk / chunksPerBag;
      scalar_t *weightFeat = weight + featureDim;
      int64_t begin = offsets[bag];
      int64_t end = (bag < numBags - 1) ? (offsets[bag + 1]) : numIndices;
      assert(end >= begin);
      accscalar_t weightFeatSum = scalar_cast<accscalar_t>(0);
      int64_t bag_size_ = 0;
      for (int64_t emb = begin; emb < end; emb++) {
        const int weightRow = ((int)input[emb]) * stride;
        weightFeatSum += scalar_cast<accscalar_t>(weightFeat[weightRow]);
        bag_size_++;
        if (featureDim == 0) {
          offset2bag[emb] = bag;
        }
      }
      if (mode == MODE_MEAN) {
        weightFeatSum = weightFeatSum / scalar_cast<accscalar_t>(bag_size_);
        bag_size[bag] = bag_size_;
      }
      (void)MODE_SUM; // silence warnings about unused MODE_SUM;
      output[bag * stride + featureDim] = scalar_cast<scalar_t>(weightFeatSum);
    }
  }
}

// FIXME: removed the accGradParametersKernelByFeature case present in
// LookupTable. That kernel is faster at small sizes (<768 indices), which
// does not need EmbeddingBag (LookupTable + Sum works fine), but would
// still be nice to not be slow in that case.

template <typename scalar_t>
__global__ void EmbeddingBag_accGradParametersKernel(
    int64_t *input, int64_t *indices, scalar_t *gradOutput,
    scalar_t *gradWeight, int64_t *offset2bag, int64_t *count, ptrdiff_t numel,
    int64_t stride, int mode, int64_t *bag_size) {

  using accscalar_t = cuda::acc_type<scalar_t>;
  int idx = blockIdx.x * 4 + threadIdx.y;

  // Each warp is responsible for an input into the LookupTable.
  // If the preceding input has the same as this input, then the warp
  // exits immediately. The warp also processes subsequent inputs with the
  // same value.  //
  // Input Warp
  // 1     <warp 1>
  // 1     <warp 1> (<warp 2> exits without doing any work)
  // 5     <warp 3>
  // 8     <warp 4>

  // Number of values proceessed by each thread (grain size)
  const int SZ = 4;

  if (idx < numel && (idx == 0 || input[idx] != input[idx - 1])) {
    do {
      const int startFeature = threadIdx.x + blockIdx.y * blockDim.x * SZ;
      const int weightRow = ((int)input[idx]) * stride;

      // Note: only this line changes from LookupTable_accgradParametersKernel
      const int origRow = ((int)indices[idx]);
      const int seq_number = offset2bag[origRow];
      const int gradOutputRow = ((int)seq_number) * stride;

      const accscalar_t scale = count ? (accscalar_t)1.0 / count[idx] : 1.0;

      accscalar_t gradient[SZ];
      accscalar_t weight[SZ];

#pragma unroll
      for (int ii = 0; ii < SZ; ii++) {
        int featureDim = startFeature + ii * WARP_SIZE;
        if (featureDim < stride) {
          gradient[ii] =
              scalar_cast<accscalar_t>(gradOutput[gradOutputRow + featureDim]);
          if (mode == MODE_MEAN) {
            gradient[ii] /= bag_size[seq_number];
          }
          weight[ii] =
              scalar_cast<accscalar_t>(gradWeight[weightRow + featureDim]);
        }
      }

#pragma unroll
      for (int ii = 0; ii < SZ; ii++) {
        weight[ii] += gradient[ii] * scale;
      }

#pragma unroll
      for (int ii = 0; ii < SZ; ii++) {
        int featureDim = startFeature + ii * WARP_SIZE;
        if (featureDim < stride) {
          gradWeight[weightRow + featureDim] =
              scalar_cast<scalar_t>(weight[ii]);
        }
      }

      idx++;
    } while (idx < numel && input[idx] == input[idx - 1]);
  }
}
}

std::tuple<Tensor, Tensor, Tensor>
embedding_bag_cuda(const Tensor &weight, const Tensor &indices,
                   const Tensor &offsets, const bool scale_grad_by_freq,
                   const int64_t mode, bool sparse) {
  auto indices_arg = TensorArg(indices, "indices", 1);
  checkScalarType("embedding_bag_cuda", indices_arg, kLong);
  checkContiguous("embedding_bag_cuda", indices_arg);
  auto offsets_arg = TensorArg(offsets, "offsets", 1);
  checkScalarType("embedding_bag_cuda", offsets_arg, kLong);
  checkContiguous("embedding_bag_cuda", offsets_arg);
  auto weight_arg = TensorArg(weight, "weight", 1);
  checkContiguous("embedding_bag_cuda", weight_arg);
  checkSameGPU("embedding_bag_cuda", weight_arg, indices_arg);
  checkSameGPU("embedding_bag_cuda", weight_arg, offsets_arg);

  int64_t numIndices = indices.sizes()[0];
  int64_t numBags = offsets.sizes()[0];
  int64_t stride = weight.sizes()[1];

  auto bag_size = at::zeros(indices.type(), offsets.sizes());
  auto offset2bag =
      at::zeros(indices.type(), {indices.sizes()[0]}); // offset2bag = [0 0 0 0 0]

  hipStream_t stream = globalContext().getCurrentCUDAStream();

  auto output = at::zeros(weight.type(), {offsets.sizes()[0], weight.sizes()[1]});

  dim3 block = dim3(32, 8);
  int grid = 1024;
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(weight.type(), "embedding_bag_cuda", [&] {
    using cuda_scalar_t = cuda::type<scalar_t>;
    EmbeddingBag_updateOutputKernel<cuda_scalar_t><<<grid, block, 0, stream>>>(
        indices.data<int64_t>(), offsets.data<int64_t>(),
        weight.data<cuda_scalar_t>(), output.data<cuda_scalar_t>(),
        offset2bag.data<int64_t>(), numIndices, numBags, stride, mode,
        bag_size.data<int64_t>());
  });

  THCudaCheck(hipGetLastError());
  return std::tuple<Tensor, Tensor, Tensor>(output, offset2bag, bag_size);
}

Tensor embedding_bag_backward_cuda(const Tensor &grad_, const Tensor &indices,
                                   const Tensor &offsets,
                                   const Tensor &offset2bag,
                                   const Tensor &bag_size_, int64_t num_weights,
                                   bool scale_grad_by_freq, int64_t mode) {
  Tensor grad = grad_.contiguous();
  auto indices_arg = TensorArg(indices, "indices", 1);
  checkScalarType("embedding_bag_cuda", indices_arg, kLong);
  checkContiguous("embedding_bag_cuda", indices_arg);
  auto offsets_arg = TensorArg(offsets, "offsets", 1);
  checkScalarType("embedding_bag_cuda", offsets_arg, kLong);
  checkContiguous("embedding_bag_cuda", offsets_arg);
  auto grad_arg = TensorArg(grad, "grad", 1);
  checkContiguous("embedding_bag_cuda", grad_arg);
  checkSameGPU("embedding_bag_cuda", grad_arg, offsets_arg);
  checkSameGPU("embedding_bag_cuda", grad_arg, indices_arg);

  Tensor &bag_size = const_cast<Tensor &>(bag_size_);

  auto grad_weight = at::zeros(grad_.type(), {num_weights, grad.sizes()[1]});

  int nDim = indices.ndimension();

  ptrdiff_t numel = indices.numel();
  int64_t stride = grad_weight.stride(0);

  hipStream_t stream = globalContext().getCurrentCUDAStream();

  auto sorted_indices = indices.type().tensor(indices.sizes());
  auto orig_indices = indices.type().tensor(indices.sizes());
  using device_ptr = thrust::device_ptr<int64_t>;

  // Sort the inputs into sorted with the corresponding indices; we
  // don't need a stable or multidimensional sort, so just use Thrust
  // directly
  {
    sorted_indices.copy_(indices);

    auto allocator = THCThrustAllocator(globalContext().lazyInitCUDA());
    auto policy = thrust::cuda::par(allocator).on(stream);

    // Fill sortedOrigIndices with sequential indices
    auto count_iter = thrust::counting_iterator<int64_t>(0);
    auto orig_data = device_ptr(orig_indices.data<int64_t>());
    thrust::copy(policy, count_iter, count_iter + numel, orig_data);

    // Sort; a stable sort is not required
    auto sorted_data = device_ptr(sorted_indices.data<int64_t>());
    thrust::sort_by_key(policy, sorted_data, sorted_data + numel, orig_data,
                        ThrustLTOp<int64_t>());
  }

  Tensor count;
  if (scale_grad_by_freq) {
    count = indices.type().tensor(indices.sizes());

    auto allocator = THCThrustAllocator(globalContext().lazyInitCUDA());
    auto policy = thrust::cuda::par(allocator).on(stream);

    // Compute an increasing sequence per unique item in sortedIndices:
    // sorted: 2 5 5 5 7 7 8 9 9
    //  count: 1 1 2 3 1 2 1 1 2
    auto sorted_data = device_ptr(sorted_indices.data<int64_t>());
    auto count_data = device_ptr(count.data<int64_t>());
    thrust::inclusive_scan_by_key(policy, sorted_data, sorted_data + numel,
                                  thrust::make_constant_iterator(1),
                                  count_data);

    // Take the maximum of each count per unique key in reverse:
    // sorted: 2 5 5 5 7 7 8 9 9
    //  count: 1 3 3 3 2 2 1 2 2
    thrust::inclusive_scan_by_key(
        policy, thrust::make_reverse_iterator(sorted_data + numel),
        thrust::make_reverse_iterator(sorted_data),
        thrust::make_reverse_iterator(count_data + numel),
        thrust::make_reverse_iterator(count_data + numel),
        thrust::equal_to<int64_t>(), thrust::maximum<int64_t>());
  }

  dim3 grid(THCCeilDiv(numel, (ptrdiff_t)4), THCCeilDiv(stride, (int64_t)128));
  dim3 block(32, 4);
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      grad.type(), "embedding_bag_backward_cuda", [&] {
        using cuda_scalar_t = cuda::type<scalar_t>;
        EmbeddingBag_accGradParametersKernel<
            cuda_scalar_t><<<grid, block, 0, stream>>>(
            sorted_indices.data<int64_t>(), orig_indices.data<int64_t>(),
            grad.data<cuda_scalar_t>(), grad_weight.data<cuda_scalar_t>(),
            offset2bag.data<int64_t>(),
            count.defined() ? count.data<int64_t>() : nullptr, numel, stride,
            mode, bag_size.data<int64_t>());
      });

  THCudaCheck(hipGetLastError());
  return grad_weight;
}
}
}
