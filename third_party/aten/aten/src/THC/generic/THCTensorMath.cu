
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCTensorMath.cu"
#else

THC_API void
THCTensor_(fill)(THCState* state, THCTensor *self_, real value)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self_));

  if (!THC_pointwiseApply1(
        state, self_, TensorFillOp<real>(value))) {
    THArgCheck(false, 1, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(zero)(THCState *state, THCTensor *self_)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self_));
  if (THCTensor_(isContiguous)(state, self_)) {
    THCudaCheck(cudaMemsetAsync(THCTensor_(data)(state, self_),
                                0,
                                sizeof(real) * THCTensor_(nElement)(state, self_),
                                THCState_getCurrentStream(state)));
  } else {
    if (!THC_pointwiseApply1(
          state, self_,
          TensorFillOp<real>(ScalarConvert<int, real>::to(0)))) {
      THArgCheck(false, 1, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(zeros)(THCState *state, THCTensor *r_, THLongStorage *size)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, r_));
  THCTensor_(resize)(state, r_, size, NULL);
  THCTensor_(zero)(state, r_);
}

THC_API void
THCTensor_(zerosLike)(THCState *state, THCTensor *r_, THCTensor *input)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, r_, input));
  THCTensor_(resizeAs)(state, r_, input);
  THCTensor_(zero)(state, r_);
}

THC_API void
THCTensor_(ones)(THCState *state, THCTensor *r_, THLongStorage *size)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, r_));
  THCTensor_(resize)(state, r_, size, NULL);
  THCTensor_(fill)(state, r_, ScalarConvert<int, real>::to(1));
}

THC_API void
THCTensor_(onesLike)(THCState *state, THCTensor *r_, THCTensor *input)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, r_, input));
  THCTensor_(resizeAs)(state, r_, input);
  THCTensor_(fill)(state, r_, ScalarConvert<int, real>::to(1));
}

THC_API void
THCTensor_(reshape)(THCState *state, THCTensor *r_, THCTensor *t, THLongStorage *size)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, r_, t));
  THCTensor_(resize)(state, r_, size, NULL);
  THCTensor_(copy)(state, r_, t);
}

ptrdiff_t
THCTensor_(numel)(THCState *state, THCTensor *t)
{
  return THCTensor_(nElement)(state, t);
}

void THCTensor_(cat)(THCState *state, THCTensor *result,
		     THCTensor *ta, THCTensor *tb, int dimension)
{
  THCTensor* inputs[2];
  inputs[0] = ta;
  inputs[1] = tb;
  THCTensor_(catArray)(state, result, inputs, 2, dimension);
}

void THCTensor_(check_shape_except_dim)(THCState *state, 
    THCTensor *first, THCTensor *second, int dimension);
inline void THCTensor_(check_shape_except_dim)(THCState *state, 
    THCTensor *first, THCTensor *second, int dimension)
{
  int first_dims = THCTensor_(nDimension)(state, first);
  int second_dims = THCTensor_(nDimension)(state, second);
  THArgCheck(first_dims == second_dims, 0,
      "Tensors must have same number of dimensions: got %d and %d",
      first_dims, second_dims);
  for (int dim = 0; dim < first_dims; dim++) {
    if (dim == dimension) {
      continue;
    }
    int64_t first_dim_size = THCTensor_(size)(state, first, dim);
    int64_t second_dim_size = THCTensor_(size)(state, second, dim);
    THArgCheck(first_dim_size == second_dim_size, 0,
        "Sizes of tensors must match except in dimension %d. Got %lld and %lld in dimension %d",
        dimension, (long long)first_dim_size, (long long)second_dim_size, dim);
  }
}

void THCTensor_(catArray)(THCState *state, THCTensor *result,
			  THCTensor **inputs, int numInputs, int dimension)
{
  THLongStorage *size;
  int i, j, cohortMax;
  int64_t offset;
  bool hasEmptyInput = false;
  THCTensor *notEmptyTensor = NULL;

  // Even in the case where dimension is negative (i.e. when we want
  // to cat along the last dimension), this logic still works, as the
  // loop below will overwrite the value
  int nDims = dimension + 1;

  // cat_dimension is the actual dimension we cat along
  int cat_dimension = dimension;

  for (i = 0; i < numInputs; i++)
  {
    int inputDim = THCTensor_(nDimension)(state, inputs[i]);
    hasEmptyInput |= !inputDim;
    if (inputDim > 0) {
      nDims = inputDim;
      notEmptyTensor = inputs[i];
    }
  }

  // If all inputs are empty tensors, return an empty tensor
  if (notEmptyTensor == NULL) {
    return;
  }

  // In the event that the user specified -1 as the concat dimension, then
  // we want to pick the nDims as dimension to cat along (and thus nDims - 1 as the
  // value due to 0-based indexing). If the nDims is // 0 (i.e. we are catting all
  // empty tensors), then we set cat_dimension to be 0
  if (dimension + TH_INDEX_BASE == -1) {
    cat_dimension = nDims ? (nDims - 1) : 0;
  }

  THArgCheck(numInputs > 0, 3, "invalid number of inputs %d", numInputs);
  THArgCheck(cat_dimension >= 0, 4, "invalid dimension %d", dimension + TH_INDEX_BASE);
  
  size = THLongStorage_newWithSize(nDims);
  
  // Compute size of the result in the cat dimension
  int64_t cat_dim_size = 0;
  for (int i = 0; i < numInputs; i++) {
    THCTensor *tensor = inputs[i];
    if (THCTensor_(nDimension)(state, tensor) == 0) {
      continue;
    }
    THCTensor_(check_shape_except_dim)(state, notEmptyTensor, tensor, cat_dimension);
    cat_dim_size += THCTensor_(size)(state, tensor, cat_dimension);
  }

  // Compute the size of the result
  for (int dim = 0; dim < nDims; dim++) {
    int64_t result_dim_size = THCTensor_(size)(state, notEmptyTensor, dim);
    if (dim == cat_dimension) {
      result_dim_size = cat_dim_size;
    }
    size->data[dim] = result_dim_size;
  }
  THCTensor_(resize)(state, result, size, NULL);
  THLongStorage_free(size);

  // We parallelize the copy if all 6 conditions pass:
  //
  // 1. There is more than one input tensor
  // 2. No empty inputs
  // 3. The result tensor is 32-bit indexable
  // 4. The number of dimensions is <= 4
  // 5. All input tensors are contiguous (output tensor may be non-contig)
  // 6. All input tensors can use 32-bit indexing
  // 7. All input tensors are on the same device

  if (numInputs > 1 &&
      !hasEmptyInput &&
      THCTensor_(nDimension)(state, result) <= CAT_ARRAY_MAX_INPUT_DIMS &&
      TensorUtils<THCTensor>::canUse32BitIndexMath(state, result) &&
      TensorUtils<THCTensor>::allContiguous(state, inputs, numInputs) &&
      TensorUtils<THCTensor>::all32BitIndexable(state, inputs, numInputs) &&
      TensorUtils<THCTensor>::allSameDevice(state, inputs, numInputs)) {

    // First, let's set up our kernel parameters. We start with a raw pointer to the storage
    // for the output Tensor.
    real *data = THCTensor_(data)(state, result);

    // Kernel Parameter
    size_t tensorMetadataSize = sizeof(CatArrInputTensor<real, unsigned int>) * CAT_ARRAY_BATCH_SIZE;
    CatArrInputTensor<real, unsigned int> *d_inputs;
    THCudaCheck(THCudaMalloc(state, (void**) &d_inputs, tensorMetadataSize));

    OutputTensorSizeStride<unsigned int, CAT_ARRAY_MAX_INPUT_DIMS> param;

    // Next, let's initialize the size, stride arrays for the output Tensor.
    for (i = 0; i < nDims; ++i) {
      param.outputSize[i] = THCTensor_(size)(state, result, i);
      param.outputStride[i] = THCTensor_(stride)(state, result, i);
    }

    THCStream* stream = THCState_getStream(state);

    // Template Declarations for dim = 1, 2, 3, 4
#define HANDLE_CASE(DIMS) \
  CatArrayBatchedCopy<real, unsigned int, DIMS><<<catGrid, applyBlock, 0, stream->stream>>>(data, d_inputs, param, cat_dimension, param.outputStride[cat_dimension]);

    // Now we loop
    offset = 0;
    for (i = 0; i < numInputs; i += CAT_ARRAY_BATCH_SIZE) {
      // Re-allocate stackInputs every iteration to avoid read-after-write hazard
      CatArrInputTensor<real, unsigned int>* stackInputs = (CatArrInputTensor<real, unsigned int>*) THCudaHostAlloc(state, tensorMetadataSize);
      cohortMax = 0;
      for (j = 0; j < CAT_ARRAY_BATCH_SIZE && (i+j) < numInputs; ++j) {
        int64_t dimSize = cat_dimension < THCTensor_(nDimension)(state, inputs[i+j])
          ? THCTensor_(size)(state, inputs[i+j], cat_dimension)
          : 1;

        stackInputs[j].input = THCTensor_(data)(state, inputs[i+j]);
        stackInputs[j].offset = offset;
        stackInputs[j].dimSize = dimSize;
        stackInputs[j].nElements = THCTensor_(nElement)(state, inputs[i+j]);
        cohortMax = cohortMax > (int) stackInputs[j].nElements ? cohortMax : (int) stackInputs[j].nElements;

        // update offset
        offset += dimSize;
      }
      THCudaCheck(cudaMemcpyAsync(
          d_inputs,
          stackInputs,
          j * sizeof(CatArrInputTensor<real, unsigned int>),
          cudaMemcpyHostToDevice,
          stream->stream));
      THCudaHostRecord(state, stackInputs);
      THCudaHostFree(state, stackInputs);

      // Next, let's consider how we set our kernel launch parameters.
      // We borrow from THCApply, which the kernel's internal indexing
      // is based on.
      dim3 applyBlock = getApplyBlock();

      //Get grid where x dim fills half gpu and y dim is number of tensors.
      //This will have cating two tensors fill the entire grid, but prevent
      //many threads from needlessly load meta data if their sizes is small.
      dim3 catGrid;
      getCatGrid(state, j, catGrid);


      switch (nDims) {
        case 1:
          HANDLE_CASE(1);
          break;
        case 2:
          HANDLE_CASE(2);
          break;
        case 3:
          HANDLE_CASE(3);
          break;
        case 4:
          HANDLE_CASE(4);
          break;
      }
      THCudaCheck(cudaGetLastError());
    }
    THCudaCheck(THCudaFree(state, d_inputs));
#undef HANDLE_CASE
  } else {
    offset = 0;
    for (j = 0; j < numInputs; j++)
    {
      // No reason to copy when input is empty
      if (!THCTensor_(nDimension)(state, inputs[j])) continue;

      int64_t dimSize = cat_dimension < THCTensor_(nDimension)(state, inputs[j])
               ? THCTensor_(size)(state, inputs[j], cat_dimension)
               : 1;

      THCTensor *nt = THCTensor_(newWithTensor)(state, result);
      THCTensor_(narrow)(state, nt, NULL, cat_dimension, offset, dimSize);
      THCTensor_(copy)(state, nt, inputs[j]);
      THCTensor_(free)(state, nt);
      offset += dimSize;
    }
  }
}

void THCTensor_(nonzero)(THCState* state, THCudaLongTensor *tensor,
                          THCTensor *self)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self  ));
  THCAssertSameGPU(THCudaLongTensor_checkGPU(state, 1, tensor));


  using namespace thrust::placeholders;
  THCThrustAllocator thrustAlloc(state);
  self = THCTensor_(newContiguous)(state, self);
  thrust::device_ptr<real> self_data(THCTensor_(data)(state, self));

  int num_dim = THCTensor_(nDimension)(state, self);
  int64_t N = THCTensor_(nElement)(state, self);

  THCudaLongTensor_resize2d(state, tensor, N, num_dim);
  tensor = THCudaLongTensor_newContiguous(state, tensor);
  thrust::device_ptr<int64_t> tensor_data(THCudaLongTensor_data(state, tensor));

  thrust::counting_iterator<int64_t> idxfirst(0);
  thrust::counting_iterator<int64_t> idxlast = idxfirst + N;

  typedef thrust::device_ptr<int64_t> Iter;
  strided_range<Iter> strided_tensor(tensor_data,
                                     tensor_data+N*num_dim, num_dim);

#if CUDA_VERSION >= 7000
  cudaStream_t stream = THCState_getCurrentStream(state);
#endif

  strided_range<Iter>::iterator dend = thrust::copy_if(
#if CUDA_VERSION >= 7000
    thrust::cuda::par(thrustAlloc).on(stream),
#endif
    idxfirst,
    idxlast,
    self_data,
    strided_tensor.begin(),
    NonZeroOp<real>()
  );

  int64_t num_nonzeros = thrust::distance(strided_tensor.begin(), dend);

  int64_t div = 1;
  for (int dim = num_dim-1; dim >= 0; dim--) {
    strided_range<Iter> stride_dim(tensor_data+dim,
                                   tensor_data+N*num_dim, num_dim);
    thrust::transform(
#if CUDA_VERSION >= 7000
      thrust::cuda::par(thrustAlloc).on(stream),
#endif
      strided_tensor.begin(),
      strided_tensor.end(),
      stride_dim.begin(),
      idx_functor(div, self->size[dim])
    );
    div *= self->size[dim];
  }

  THCudaLongTensor_resize2d(state, tensor, num_nonzeros, num_dim);

  THCTensor_(free)(state, self);
  THCudaLongTensor_free(state, tensor);

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(diag)(THCState *state, THCTensor *self_, THCTensor *src_, int64_t k){
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src_));
  int nDimension = THCTensor_(nDimension)(state, src_);
  THArgCheck((nDimension == 2) || (nDimension == 1), 1, "expected a matrix or a vector");
  if (nDimension == 2) {
    int64_t stride0 = THCTensor_(stride)(state, src_, 0);
    int64_t stride1 = THCTensor_(stride)(state, src_, 1);
    int64_t size0 = THCTensor_(size)(state, src_, 0);
    int64_t size1 = THCTensor_(size)(state, src_, 1);
    int64_t size = (k > 0) ? min((int64_t)size0, (int64_t)size1 - k) : min((int64_t)size0 + k, (int64_t)size1);
    THCTensor_(resize1d)(state, self_, size);
    int64_t strideSelf = THCTensor_(stride)(state, self_, 0);
    const dim3 threads(min((int64_t)THCState_getCurrentDeviceProperties(state)->maxThreadsPerBlock, (int64_t)size));
    dim3 grid(min((int64_t)1024, (int64_t)THCCeilDiv(size, (int64_t)threads.x)));
    int64_t start = (k >= 0 ? k * stride1 : -k * stride0);
    THCTensor_copyFromDiagonal<real><<<grid, threads, 0, THCState_getCurrentStream(state)>>>
    (THCTensor_(data)(state, self_), THCTensor_(data)(state, src_), start, size, stride0 + stride1, strideSelf);
  } else {
    ptrdiff_t totalElements = THCTensor_(nElement)(state, src_);
    ptrdiff_t size = (k > 0) ? totalElements + k : totalElements - k;
    int64_t strideSrc = THCTensor_(stride)(state, src_, 0);
    THCTensor_(resize2d)(state, self_, size, size);
    THCTensor_(zero)(state, self_);
    int64_t stride0 = THCTensor_(stride)(state, self_, 0);
    int64_t stride1 = THCTensor_(stride)(state, self_, 1);
    const dim3 threads(min((int64_t)THCState_getCurrentDeviceProperties(state)->maxThreadsPerBlock, (int64_t)size));
    dim3 grid(min((int64_t)1024, (int64_t)THCCeilDiv(size, (ptrdiff_t)threads.x)));
    ptrdiff_t start = (k >= 0 ? k * stride1 : -k * stride0);
    THCTensor_copyToDiagonal<real><<<grid, threads, 0, THCState_getCurrentStream(state)>>>
    (THCTensor_(data)(state, self_), THCTensor_(data)(state, src_), start, totalElements, stride0 + stride1, strideSrc);
  }
  THCudaCheck(cudaGetLastError());
}

void THCTensor_(eye)(THCState *state, THCTensor *self_, int64_t n, int64_t m)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self_));
  THArgCheck(n > 0, 1, "invalid argument");

  if(m <= 0)
    m = n;

  THCTensor_(resize2d)(state, self_, n, m);
  THCTensor_(zero)(state, self_);

  int64_t sz = THMin(n, m);
  int64_t stride = THCTensor_(stride)(state, self_, 0) +
                   THCTensor_(stride)(state, self_, 1);

  THCTensor *diag = THCTensor_(newWithStorage1d)(state, self_->storage,
      self_->storageOffset,  sz, stride);

  THCTensor_(fill)(state, diag, ScalarConvert<int, real>::to(1));
  THCTensor_(free)(state, diag);
}

accreal THCTensor_(trace)(THCState *state, THCTensor *src_) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, src_));
  THArgCheck((src_->nDimension == 2), 1, "expected a matrix");
  THCTensor *diag = THCTensor_(new)(state);
  THCTensor_(diag)(state, diag, src_, 0);
  accreal trace = THCTensor_(sumall)(state, diag);
  THCTensor_(free)(state, diag);
  return trace;
}

#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_HALF)

void THCTensor_(linspace)(THCState *state, THCTensor *r_, real a, real b, int64_t n) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, r_));
  THArgCheck(n > 1 || (n == 1 && (a == b)), 3, "invalid number of points");
  if (THCTensor_(nElement)(state, r_) != n) THCTensor_(resize1d)(state, r_, n);
  if (n == 1) THCTensor_(fill)(state, r_, a);
  else {
    THCTensor *r = THCTensor_(isContiguous)(state, r_)
                   ? r_ // if r_ is contiguous we can direct work on it
                   : THCTensor_(newContiguous)(state, r_);
    real step = THCNumerics<real>::div(THCNumerics<real>::sub(b, a),
                                       ScalarConvert<int64_t,real>::to(n - 1));
    LinspaceOp<real> linspace_method(a, step);
    thrust::device_ptr<real> data_(THCTensor_(data)(state, r));
    thrust::tabulate(data_, data_ + n, linspace_method);
    if (!THCTensor_(isContiguous)(state, r_)) { // We need to move data back to r_
      THCTensor_(freeCopyTo)(state, r, r_);
    }
  }
  THCudaCheck(cudaGetLastError());
}

void THCTensor_(logspace)(THCState *state, THCTensor *r_, real a, real b, int64_t n) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, r_));
  THArgCheck(n > 1 || (n == 1 && (a == b)), 3, "invalid number of points");
  if (THCTensor_(nElement)(state, r_) != n) THCTensor_(resize1d)(state, r_, n);
  if (n == 1) THCTensor_(fill)(state, r_, THCNumerics<real>::exp10(a));
  else {
    THCTensor *r = THCTensor_(isContiguous)(state, r_)
                   ? r_
                   : THCTensor_(newContiguous)(state, r_);
    real step = THCNumerics<real>::div(THCNumerics<real>::sub(b, a),
                                       ScalarConvert<int64_t,real>::to(n - 1));
    LogspaceOp<real> logspace_method(a, step);
    thrust::device_ptr<real> data_(THCTensor_(data)(state, r));
    thrust::tabulate(data_, data_ + n, logspace_method);
    if (!THCTensor_(isContiguous)(state, r_)) {
      THCTensor_(freeCopyTo)(state, r, r_);
    }
  }
  THCudaCheck(cudaGetLastError());
}

#endif

void THCTensor_(range)(THCState *state, THCTensor *r_, accreal xmin, accreal xmax, accreal step) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, r_));
  THArgCheck(step > 0 || step < 0, 3, "step must be nonzero");
  THArgCheck(((step > 0) && (xmax >= xmin)) || ((step < 0) && (xmax <= xmin))
              , 2, "upper bound and larger bound inconsistent with step sign");
  ptrdiff_t size = (ptrdiff_t) (((xmax - xmin) / step) + 1);
  if (THCTensor_(nElement)(state, r_) != size) THCTensor_(resize1d)(state, r_, size);
  THCTensor *r = THCTensor_(newContiguous)(state, r_);
  LinspaceOp<real,accreal> linspace_method(xmin, step);
  thrust::device_ptr<real> data_(THCTensor_(data)(state, r));
  thrust::tabulate(data_, data_ + size, linspace_method);
  THCTensor_(freeCopyTo)(state, r, r_);
  THCudaCheck(cudaGetLastError());
}

void THCTensor_(arange)(THCState* state, THCTensor *r_, accreal xmin, accreal xmax, accreal step) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, r_));
  THArgCheck(step > 0 || step < 0, 3, "step must be nonzero");
  THArgCheck(((step > 0) && (xmax >= xmin)) || ((step < 0) && (xmax <= xmin))
              , 2, "upper bound and larger bound inconsistent with step sign");
  ptrdiff_t size = (ptrdiff_t) ceil(ScalarConvert<accreal, double>::to(xmax - xmin) / step);
  if (THCTensor_(nElement)(state, r_) != size) THCTensor_(resize1d)(state, r_, size);
  THCTensor *r = THCTensor_(newContiguous)(state, r_);
  LinspaceOp<real,accreal> linspace_method(xmin, step);
  thrust::device_ptr<real> data_(THCTensor_(data)(state, r));
  thrust::tabulate(data_, data_ + size, linspace_method);
  THCTensor_(freeCopyTo)(state, r, r_);
  THCudaCheck(cudaGetLastError());
}

#endif
