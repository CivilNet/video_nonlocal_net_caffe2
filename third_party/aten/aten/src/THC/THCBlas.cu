#include "THCBlas.h"
#include "THCGeneral.h"
#include "THCHalf.h"

float THCudaBlas_Sdot(THCState *state, int64_t n, float *x, int64_t incx, float *y, int64_t incy)
{
  if (n == 1) {
    incx = 1;
    incy = 1;
  }

  if ((n <= INT_MAX) && (incx <= INT_MAX) && (incy <= INT_MAX)) {
    int i_n = (int)n;
    int i_incx = (int)incx;
    int i_incy = (int)incy;
    float result;
    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    THCublasCheck(hipblasSdot(handle, i_n, x, i_incx, y, i_incy, &result));
    return result;
  }

  THError("Cublas_Sdot only supports n, incx and incy "
          "up to signed integer limits: %d", INT_MAX);
  return 0;
}

double THCudaBlas_Ddot(THCState *state, int64_t n, double *x, int64_t incx, double *y, int64_t incy)
{
  if (n == 1) {
    incx = 1;
    incy = 1;
  }

  if ((n <= INT_MAX) && (incx <= INT_MAX) && (incy <= INT_MAX)) {
    int i_n = (int)n;
    int i_incx = (int)incx;
    int i_incy = (int)incy;
    double result;
    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    THCublasCheck(hipblasDdot(handle, i_n, x, i_incx, y, i_incy, &result));
    return result;
  }

  THError("Cublas_Ddot only supports n, incx and incy "
          "up to signed integer limits: %d", INT_MAX);
  return 0;
}

#ifdef CUDA_HALF_TENSOR
half THCudaBlas_Hdot(THCState *state, int64_t n, half *x, int64_t incx, half *y, int64_t incy)
{
#if CUDA_VERSION >= 8000
  if (n == 1) {
    incx = 1;
    incy = 1;
  }

  if ((n <= INT_MAX) && (incx <= INT_MAX) && (incy <= INT_MAX)) {
    half result;
    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    THCublasCheck(hipblasDotEx(handle, n,
                              x, HIP_R_16F, incx,
                              y, HIP_R_16F, incy,
                              &result, HIP_R_16F,
                              HIP_R_32F));
    return result;
  }

  THError("Cublas_Hdot only supports n, incx and incy "
          "up to signed integer limits: %d", INT_MAX);
  return THC_float2half(0);
#else
  THError("Cublas_Hdot requires CUDA 8.0+");
  return THC_float2half(0);
#endif
}
#endif

/* Level 2 */
void THCudaBlas_Sgemv(THCState *state, char trans, int64_t m, int64_t n, float alpha, float *a, int64_t lda, float *x, int64_t incx, float beta, float *y, int64_t incy)
{
  if(n == 1)
    lda = m;

  hipblasOperation_t op;
  if (trans == 't') op = HIPBLAS_OP_T;
  else if (trans == 'n') op = HIPBLAS_OP_N;
  else if (trans == 'c') op = HIPBLAS_OP_C;
  else THError("Cublas_Sgemv parameter trans should be 't', 'n' or 'c'.");

  if( (m <= INT_MAX) && (n <= INT_MAX) &&
      (lda > 0) && (lda <= INT_MAX) &&
      (incx > 0) && (incx <= INT_MAX) &&
      (incy > 0) && (incy <= INT_MAX) )
  {
    int i_m = (int)m;
    int i_n = (int)n;
    int i_lda = (int)lda;
    int i_incx = (int)incx;
    int i_incy = (int)incy;

    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    THCublasCheck(hipblasSgemv(handle, op, i_m, i_n, &alpha, a, i_lda, x, i_incx, &beta, y, i_incy));
    return;
  }
  THError("Cublas_Sgemv only supports m, n, lda, incx, incy"
          "in the range 0 < [val] <= %d", INT_MAX);
}

void THCudaBlas_Dgemv(THCState *state, char trans, int64_t m, int64_t n, double alpha, double *a, int64_t lda, double *x, int64_t incx, double beta, double *y, int64_t incy)
{
  if(n == 1)
    lda = m;

  hipblasOperation_t op;
  if (trans == 't') op = HIPBLAS_OP_T;
  else if (trans == 'n') op = HIPBLAS_OP_N;
  else if (trans == 'c') op = HIPBLAS_OP_C;
  else THError("Cublas_Sgemv parameter trans should be 't', 'n' or 'c'.");

  if( (m <= INT_MAX) && (n <= INT_MAX) &&
      (lda > 0) && (lda <= INT_MAX) &&
      (incx > 0) && (incx <= INT_MAX) &&
      (incy > 0) && (incy <= INT_MAX) )
  {
    int i_m = (int)m;
    int i_n = (int)n;
    int i_lda = (int)lda;
    int i_incx = (int)incx;
    int i_incy = (int)incy;

    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    THCublasCheck(hipblasDgemv(handle, op, i_m, i_n, &alpha, a, i_lda, x, i_incx, &beta, y, i_incy));
    return;
  }
  THError("Cublas_Dgemv only supports m, n, lda, incx, incy"
          "in the range 0 < [val] <= %d", INT_MAX);
}

void THCudaBlas_Sger(THCState *state, int64_t m, int64_t n, float alpha, float *x, int64_t incx, float *y, int64_t incy, float *a, int64_t lda)
{
  if(n == 1)
    lda = m;

  if( (m <= INT_MAX) && (n <= INT_MAX) && (lda <= INT_MAX)  && (incx <= INT_MAX) && (incy <= INT_MAX) )
    {
      int i_m = (int)m;
      int i_n = (int)n;
      int i_lda = (int)lda;
      int i_incx = (int)incx;
      int i_incy = (int)incy;

      hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
      hipblasSetStream(handle, THCState_getCurrentStream(state));
      THCublasCheck(hipblasSger(handle, i_m, i_n, &alpha, x, i_incx, y, i_incy, a, i_lda));
      return;
    }
  THError("Cublas_Sger only supports m, n, lda, incx, incy"
          "with the bound [val] <= %d", INT_MAX);
}

void THCudaBlas_Dger(THCState *state, int64_t m, int64_t n, double alpha, double *x, int64_t incx, double *y, int64_t incy, double *a, int64_t lda)
{
  if(n == 1)
    lda = m;

  if( (m <= INT_MAX) && (n <= INT_MAX) && (lda <= INT_MAX)  && (incx <= INT_MAX) && (incy <= INT_MAX) )
    {
      int i_m = (int)m;
      int i_n = (int)n;
      int i_lda = (int)lda;
      int i_incx = (int)incx;
      int i_incy = (int)incy;

      hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
      hipblasSetStream(handle, THCState_getCurrentStream(state));
      THCublasCheck(hipblasDger(handle, i_m, i_n, &alpha, x, i_incx, y, i_incy, a, i_lda));
      return;
    }
  THError("Cublas_Dger only supports m, n, lda, incx, incy"
          "with the bound [val] <= %d", INT_MAX);
}


hipblasOperation_t convertTransToCublasOperation(char trans) {
  if (trans == 't') return HIPBLAS_OP_T;
  else if (trans == 'n') return HIPBLAS_OP_N;
  else if (trans == 'c') return HIPBLAS_OP_C;
  else {
    THError("trans must be one of: t, n, c");
    return HIPBLAS_OP_T;
  }
}

void adjustLd(char transa, char transb, int64_t m, int64_t n, int64_t k, int64_t *lda, int64_t *ldb, int64_t *ldc)
{
  int transa_ = ((transa == 't') || (transa == 'T'));
  int transb_ = ((transb == 't') || (transb == 'T'));

  if(n == 1)
    *ldc = m;

  if(transa_)
  {
    if(m == 1)
      *lda = k;
  }
  else
  {
    if(k == 1)
      *lda = m;
  }

  if(transb_)
  {
    if(k == 1)
      *ldb = n;
  }
  else
  {
    if(n == 1)
      *ldb = k;
  }
}

/* Level 3 */
void THCudaBlas_Sgemm(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k, float alpha, float *a, int64_t lda, float *b, int64_t ldb, float beta, float *c, int64_t ldc)
{
  adjustLd(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  if( (m <= INT_MAX) && (n <= INT_MAX) && (k <= INT_MAX) && (lda <= INT_MAX)  && (ldb <= INT_MAX) && (ldc <= INT_MAX) )
  {
    int i_m = (int)m;
    int i_n = (int)n;
    int i_k = (int)k;
    int i_lda = (int)lda;
    int i_ldb = (int)ldb;
    int i_ldc = (int)ldc;

    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    THCublasCheck(hipblasSgemm(handle, opa, opb, i_m, i_n, i_k, &alpha, a, i_lda, b, i_ldb, &beta, c, i_ldc));
    return;
  }
  THError("Cublas_Sgemm only supports m, n, k, lda, ldb, ldc"
          "with the bound [val] <= %d", INT_MAX);
}

#ifdef CUDA_HALF_TENSOR
// In CUDA 8.0, definition of data types for sgemmex changed
#if CUDA_VERSION < 8000
#  define HIP_R_16F CUBLAS_DATA_HALF
#endif

void THCudaBlas_Hgemm(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k, half alpha, half *a, int64_t lda, half *b, int64_t ldb, half beta, half *c, int64_t ldc)
{
  adjustLd(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  if( (m <= INT_MAX) && (n <= INT_MAX) && (k <= INT_MAX) && (lda <= INT_MAX)  && (ldb <= INT_MAX) && (ldc <= INT_MAX) )
    {
      int i_m = (int)m;
      int i_n = (int)n;
      int i_k = (int)k;
      int i_lda = (int)lda;
      int i_ldb = (int)ldb;
      int i_ldc = (int)ldc;

      hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
      hipblasSetStream(handle, THCState_getCurrentStream(state));

      // Simulated Hgemm
      float fAlpha = THC_half2float(alpha);
      float fBeta = THC_half2float(beta);

#if CUDA_VERSION < 9000
      THCublasCheck(cublasSgemmEx(handle, opa, opb,
                                  i_m, i_n, i_k, &fAlpha,
                                  a, HIP_R_16F, i_lda, b, HIP_R_16F,
                                  i_ldb, &fBeta, c, HIP_R_16F, i_ldc));
#else
      hipDeviceProp_t* prop = THCState_getCurrentDeviceProperties(state);
      if (prop->major >= 5){
        THCublasCheck(hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH));
        THCublasCheck(hipblasGemmEx(handle, opa, opb,
                                   i_m, i_n, i_k, &fAlpha,
                                   a, HIP_R_16F, i_lda, b, HIP_R_16F,
                                   i_ldb, &fBeta, c, HIP_R_16F, i_ldc,
                                   HIP_R_32F, CUBLAS_GEMM_DFALT_TENSOR_OP));
        THCublasCheck(hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH));
      }else{
        THCublasCheck(cublasSgemmEx(handle, opa, opb,
                                    i_m, i_n, i_k, &fAlpha,
                                    a, HIP_R_16F, i_lda, b, HIP_R_16F,
                                    i_ldb, &fBeta, c, HIP_R_16F, i_ldc));
      }
#endif
      return;
    }
  THError("Cublas_Hgemm only supports m, n, k, lda, ldb, ldc"
          "with th bound [val] <= %d", INT_MAX);
}
#endif

void THCudaBlas_Dgemm(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k, double alpha, double *a, int64_t lda, double *b, int64_t ldb, double beta, double *c, int64_t ldc)
{
  adjustLd(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  if( (m <= INT_MAX) && (n <= INT_MAX) && (k <= INT_MAX) && (lda <= INT_MAX)  && (ldb <= INT_MAX) && (ldc <= INT_MAX) )
  {
    int i_m = (int)m;
    int i_n = (int)n;
    int i_k = (int)k;
    int i_lda = (int)lda;
    int i_ldb = (int)ldb;
    int i_ldc = (int)ldc;

    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    THCublasCheck(hipblasDgemm(handle, opa, opb, i_m, i_n, i_k, &alpha, a, i_lda, b, i_ldb, &beta, c, i_ldc));
    return;
  }
  THError("Cublas_Dgemm only supports m, n, k, lda, ldb, ldc"
          "with the bound [val] <= %d", INT_MAX);
}

#if CUDA_VERSION >= 9010
void THCudaBlas_HgemmStridedBatched(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k,
                             half alpha, const half *a, int64_t lda, int64_t strideA, const half *b, int64_t ldb, int64_t strideB,
                             half beta, half *c, int64_t ldc, int64_t strideC, int64_t batchCount)
{
  if( (m >= INT_MAX) || (n >= INT_MAX) || (k >= INT_MAX) || (lda >= INT_MAX)  || (ldb >= INT_MAX) || (ldc >= INT_MAX) || (batchCount >= INT_MAX) )

  {
    THError("Cublas_SgemmStridedBatched only supports m, n, k, lda, ldb, ldc, batchCount"
            "with the bound [val] <= %d", INT_MAX);
  }

  adjustLd(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  float fAlpha = THC_half2float(alpha);
  float fBeta = THC_half2float(beta);
  THCublasCheck(hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH));
  THCublasCheck(hipblasGemmStridedBatchedEx(handle,
                                   opa, opb, (int)m, (int)n, (int)k,
                                   (void*)&fAlpha, a, HIP_R_16F, (int)lda, strideA,
                                   b, HIP_R_16F, (int)ldb, strideB,
                                   (void*)&fBeta, c, HIP_R_16F, (int)ldc, strideC,
                                   (int)batchCount, HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));
  THCublasCheck(hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH));
}
#endif

void THCudaBlas_SgemmBatched(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k,
                             float alpha, const float *a[], int64_t lda, const float *b[], int64_t ldb,
                             float beta, float *c[], int64_t ldc, int64_t batchCount)
{
  if( (m >= INT_MAX) || (n >= INT_MAX) || (k >= INT_MAX) || (lda >= INT_MAX)  || (ldb >= INT_MAX) || (ldc >= INT_MAX) || (batchCount >= INT_MAX) )
  {
    THError("Cublas_SgemmBatched only supports m, n, k, lda, ldb, ldc, batchCount"
            "with the bound [val] <= %d", INT_MAX);
  }

  adjustLd(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasSgemmBatched(handle,
                                   opa, opb, (int)m, (int)n, (int)k,
                                   &alpha, a, (int)lda, b, (int)ldb, &beta, c, (int)ldc,
                                   (int)batchCount));
}

#if CUDA_VERSION >= 8000
void THCudaBlas_SgemmStridedBatched(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k,
                             float alpha, const float *a, int64_t lda, int64_t strideA, const float *b, int64_t ldb, int64_t strideB,
                             float beta, float *c, int64_t ldc, int64_t strideC, int64_t batchCount)
{
  if( (m >= INT_MAX) || (n >= INT_MAX) || (k >= INT_MAX) || (lda >= INT_MAX)  || (ldb >= INT_MAX) || (ldc >= INT_MAX) || (batchCount >= INT_MAX) )

  {
    THError("Cublas_SgemmStridedBatched only supports m, n, k, lda, ldb, ldc, batchCount"
            "with the bound [val] <= %d", INT_MAX);
  }

  adjustLd(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasSgemmStridedBatched(handle,
                                   opa, opb, (int)m, (int)n, (int)k,
                                   &alpha, a, (int)lda, strideA, b, (int)ldb, strideB, &beta, c, (int)ldc, strideC,
                                   (int)batchCount));
}
#endif

void THCudaBlas_DgemmBatched(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k,
                             double alpha, const double *a[], int64_t lda, const double *b[], int64_t ldb,
                             double beta, double *c[], int64_t ldc, int64_t batchCount)
{
  if( (m >= INT_MAX) || (n >= INT_MAX) || (k >= INT_MAX) || (lda >= INT_MAX)  || (ldb >= INT_MAX) || (ldc >= INT_MAX) || (batchCount >= INT_MAX) )
  {
    THError("Cublas_DgemmBatched only supports m, n, k, lda, ldb, ldc, batchCount"
            "with the bound [val] <= %d", INT_MAX);
  }

  adjustLd(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasDgemmBatched(handle,
                                   opa, opb, (int)m, (int)n, (int)k,
                                   &alpha, a, (int)lda, b, (int)ldb, &beta, c, (int)ldc,
                                   (int)batchCount));
}

#if CUDA_VERSION >= 8000
void THCudaBlas_DgemmStridedBatched(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k,
                             double alpha, const double *a, int64_t lda, int64_t strideA, const double *b, int64_t ldb, int64_t strideB,
                             double beta, double *c, int64_t ldc, int64_t strideC, int64_t batchCount)
{
  if( (m >= INT_MAX) || (n >= INT_MAX) || (k >= INT_MAX) || (lda >= INT_MAX)  || (ldb >= INT_MAX) || (ldc >= INT_MAX) || (batchCount >= INT_MAX) )
  {
    THError("Cublas_DgemmBatched only supports m, n, k, lda, ldb, ldc, batchCount"
            "with the bound [val] <= %d", INT_MAX);
  }

  adjustLd(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasDgemmStridedBatched(handle,
                                   opa, opb, (int)m, (int)n, (int)k,
                                   &alpha, a, (int)lda, strideA, b, (int)ldb, strideB, &beta, c, (int)ldc, strideC,
                                   (int)batchCount));
}
#endif

/* Inverse */
void THCudaBlas_Sgetrf(THCState *state, int n, float **a, int lda, int *pivot, int *info, int batchSize) {
  if( (n >= INT_MAX) || (lda >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Sgetrf only supports n, lda, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }
  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasSgetrfBatched(handle, n, a, lda, pivot, info, batchSize));
}

void THCudaBlas_Dgetrf(THCState *state, int n, double **a, int lda, int *pivot, int *info, int batchSize) {
  if( (n >= INT_MAX) || (lda >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Dgetrf only supports n, lda, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }
  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasDgetrfBatched(handle, n, a, lda, pivot, info, batchSize));
}

THC_API void THCudaBlas_Sgetrs(THCState *state, char transa, int n, int nrhs, const float **a, int lda, int *pivot, float **b, int ldb, int *info, int batchSize)
{
  if( (n >= INT_MAX) || (nrhs >= INT_MAX) || (lda >= INT_MAX) || (ldb >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Dgetrs only supports n, nrhs, lda, ldb, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }

  // no need to adjust leading dimensions, since matrices are square
  hipblasOperation_t opa = convertTransToCublasOperation(transa);

  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasSgetrsBatched(handle, opa, n, nrhs, a, lda, pivot, b, ldb, info, batchSize));
}


THC_API void THCudaBlas_Dgetrs(THCState *state, char transa, int n, int nrhs, const double **a, int lda, int *pivot, double **b, int ldb, int *info, int batchSize)
{
  if( (n >= INT_MAX) || (nrhs >= INT_MAX) || (lda >= INT_MAX) || (ldb >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Dgetrs only supports n, nrhs, lda, ldb, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }

  // no need to adjust leading dimensions, since matrices are square
  hipblasOperation_t opa = convertTransToCublasOperation(transa);

  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasDgetrsBatched(handle, opa, n, nrhs, a, lda, pivot, b, ldb, info, batchSize));
}

void THCudaBlas_Sgetri(THCState *state, int n, const float **a, int lda, int *pivot, float **c, int ldc, int *info, int batchSize) {

  if( (n >= INT_MAX) || (lda >= INT_MAX)|| (ldc >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Sgetri only supports n, lda, ldc, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }
  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasSgetriBatched(handle, n, a, lda, pivot, c, ldc, info, batchSize));
}

void THCudaBlas_Dgetri(THCState *state, int n, const double **a, int lda, int *pivot, double **c, int ldc, int *info, int batchSize) {

  if( (n >= INT_MAX) || (lda >= INT_MAX)|| (ldc >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Dgetri only supports n, lda, ldc, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }
  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasDgetriBatched(handle, n, a, lda, pivot, c, ldc, info, batchSize));
}
