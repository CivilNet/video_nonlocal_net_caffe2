
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCTensorMathPointwise.cu"
#else

#define IMPLEMENT_CUDA_TENSOR_BASIC_FUNC_(NAME, CFUNC, REAL)             \
  struct Tensor_##NAME##_##REAL##_Op {                                  \
    __device__ __forceinline__ void operator()(real* out, real* in) const { \
      *out = CFUNC(*in);                                                \
    }                                                                   \
                                                                        \
    __device__ __forceinline__ void operator()(real* v) const {         \
      *v = CFUNC(*v);                                                   \
    }                                                                   \
  };                                                                    \
                                                                        \
  void THCTensor_(NAME)(THCState* state, THCTensor* self_, THCTensor* src) { \
    THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src));               \
    if (self_ == src) {                                                 \
      if (!THC_pointwiseApply1(state, self_, Tensor_##NAME##_##REAL##_Op())) { \
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);                      \
      }                                                                 \
    } else {                                                            \
      THCTensor_(resizeAs)(state, self_, src);                          \
                                                                        \
      if (!THC_pointwiseApply2(state, self_, src, Tensor_##NAME##_##REAL##_Op())) { \
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);                      \
      }                                                                 \
    }                                                                   \
                                                                        \
    THCudaCheck(cudaGetLastError());                                    \
  }

#define IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(NAME, CFUNC, REAL) \
  IMPLEMENT_CUDA_TENSOR_BASIC_FUNC_(NAME, CFUNC, REAL)

#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_HALF)

IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  log, THCNumerics<real>::log,   Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(lgamma, THCNumerics<real>::lgamma, Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(log10, THCNumerics<real>::log10, Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(log1p, THCNumerics<real>::log1p, Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC( log2, THCNumerics<real>::log2,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  exp, THCNumerics<real>::exp,   Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(expm1, THCNumerics<real>::expm1, Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  cos, THCNumerics<real>::cos,   Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  sin, THCNumerics<real>::sin,   Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC( sqrt, THCNumerics<real>::sqrt,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(rsqrt, THCNumerics<real>::rsqrt, Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC( ceil, THCNumerics<real>::ceil,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(floor, THCNumerics<real>::floor, Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(trunc, THCNumerics<real>::trunc, Real)

IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  acos, THCNumerics<real>::acos,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  cosh, THCNumerics<real>::cosh,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  asin, THCNumerics<real>::asin,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  sinh, THCNumerics<real>::sinh,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(   tan, THCNumerics<real>::tan,   Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  atan, THCNumerics<real>::atan,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  tanh, THCNumerics<real>::tanh,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(   erf, THCNumerics<real>::erf,   Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(erfinv, THCNumerics<real>::erfinv,Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC( round, THCNumerics<real>::round, Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  frac, THCNumerics<real>::frac,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  cinv, THCNumerics<real>::cinv,  Real)

#endif

IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  neg, THCNumerics<real>::neg,   Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  abs, THCNumerics<real>::abs,   Real)

#undef IMPLEMENT_CUDA_TENSOR_BASIC_FUNC_
#undef IMPLEMENT_CUDA_TENSOR_BASIC_FUNC

void THCTensor_(sign)(THCState* state, THCTensor* self_, THCTensor* src) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src));
  if (self_ == src) {
    if (!THC_pointwiseApply1(state, self_, TensorSignOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src);

    if (!THC_pointwiseApply2(state, self_, src, TensorSignOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(clamp)(THCState *state, THCTensor *self_, THCTensor *src, real min_value,
  real max_value)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src));
  if (self_ == src) {
    if (!THC_pointwiseApply1(state, self_, TensorClampOp<real>(min_value, max_value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src);

    if (!THC_pointwiseApply2(state, self_, src, TensorClampOp<real>(min_value, max_value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(cross)(THCState *state, THCTensor *self, THCTensor *x, THCTensor *y, int dimension)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self, x, y));

  int i;
  int nd = THCTensor_(nDimension)(state, x);
  ptrdiff_t nelem = THCTensor_(nElement)(state, x);
  THArgCheck(nd == THCTensor_(nDimension)(state, y), 1, "tensors must have same number of dimensions");
  for (i = 0; i < nd; i++) {
    THArgCheck(THCTensor_(size)(state, x, i) == THCTensor_(size)(state, y, i), 1, "dimension %i of x and y does not match", i);
    if (dimension < 0 && THCTensor_(size)(state, x, i) == 3) {
      dimension = i;
    }
  }

  THArgCheck(dimension >= 0 && dimension < nd, 3, "dimension %d out of range", dimension+1);
  THArgCheck(THCTensor_(size)(state, x, dimension) == 3, 3,
      "dimension %d does not have size 3", dimension+1);
  THCTensor_(resizeAs)(state, self, x);

  int64_t sx = THCTensor_(stride)(state, x, dimension);
  int64_t sy = THCTensor_(stride)(state, y, dimension);
  int64_t so = THCTensor_(stride)(state, self, dimension);
  THCTensor *nx = THCTensor_(newNarrow)(state, x, dimension, 0, 1);
  THCTensor *ny = THCTensor_(newNarrow)(state, y, dimension, 0, 1);
  THCTensor *nself = THCTensor_(newNarrow)(state, self, dimension, 0, 1);
  if (!THC_pointwiseApply3(state, nself, nx, ny, TensorCrossOp<real>(sx, sy, so))) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }
  THCTensor_(free)(state, nx);
  THCTensor_(free)(state, ny);
  THCTensor_(free)(state, nself);
}

#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_HALF)

void THCTensor_(atan2)(THCState *state, THCTensor *self_, THCTensor *tx, THCTensor *ty)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self_, tx, ty));
  THArgCheck(THCTensor_(nElement)(state, tx) ==
             THCTensor_(nElement)(state, ty), 3, "sizes do not match");
  THCTensor_(resizeAs)(state, self_, tx);

  if (!THC_pointwiseApply3(state, self_, tx, ty, TensorATan2Op<real>())) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(sigmoid)(THCState* state, THCTensor* self_, THCTensor* src) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src));
  if (self_ == src) {
    if (!THC_pointwiseApply1(state, self_, TensorSigmoidOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src);

    if (!THC_pointwiseApply2(state, self_, src, TensorSigmoidOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(digamma)(THCState* state, THCTensor* self_, THCTensor* src) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src));
  if (self_ != src) {
    THCTensor_(resizeAs)(state, self_, src);
  }
  if (!THC_pointwiseApply2(state, self_, src, TensorDigammaOp<real, accreal>())) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(polygamma)(THCState* state, THCTensor* self_, int64_t n, THCTensor* src) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src));
  if (self_ != src) {
    THCTensor_(resizeAs)(state, self_, src);
  }
  switch (n) {
    case 0:
      if (!THC_pointwiseApply2(state, self_, src, TensorDigammaOp<real, accreal>())) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
      break;
    case 1:
      if (!THC_pointwiseApply2(state, self_, src, TensorTrigammaOp<real, accreal>())) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
      break;
    default:
      THError("polygamma(n,x) is not implemented for n>=2");
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(lerp)(THCState *state, THCTensor *result, THCTensor *a, THCTensor *b, real w)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, result, a, b));
  THArgCheck(THCTensor_(nElement)(state, a) ==
             THCTensor_(nElement)(state, b), 3, "sizes do not match");
  THCTensor_(resizeAs)(state, result, a);

  if (!THC_pointwiseApply3(state, result, a, b, TensorLerpOp<real>(w))) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
}

#endif

THC_API void
THCTensor_(cadd)(THCState *state, THCTensor *self_, THCTensor* src1, real value, THCTensor *src2)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    if (value == ScalarConvert<int, real>::to(1)) {
      // self += src2
      if (!THC_pointwiseApply2(state, self_, src2, TensorAddOp<real>())) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else {
      // self += value * src2
      if (!THC_pointwiseApply2(state, self_, src2, TensorCAddOp<real>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    if (value == ScalarConvert<int, real>::to(1)) {
      // self = src1 + src2
      if (!THC_pointwiseApply3(state, self_, src1, src2, TensorAddOp<real>())) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else {
      // self = src1 + value * src2
      if (!THC_pointwiseApply3(state, self_, src1, src2, TensorCAddOp<real>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    }
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(csub)(THCState *state, THCTensor *self_, THCTensor* src1, real value, THCTensor *src2)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    if (value == ScalarConvert<int, real>::to(1)) {
      // self -= src2
      if (!THC_pointwiseApply2(state, self_, src2, TensorSubOp<real>())) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else {
      // self += -value * src2
      if (!THC_pointwiseApply2(state, self_, src2,
                                   TensorCAddOp<real>(
                                     ScalarNegate<real>::to(value)))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    if (value == ScalarConvert<int, real>::to(1)) {
      // self = src1 - src2
      if (!THC_pointwiseApply3(state, self_, src1, src2, TensorSubOp<real>())) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else {
      // self = src1 - value * src2
      if (!THC_pointwiseApply3(state, self_, src1, src2,
                                   TensorCAddOp<real>(
                                     ScalarNegate<real>::to(value)))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    }
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(cmul)(THCState *state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    // self *= src2
    if (!THC_pointwiseApply2(state, self_, src2, TensorMulOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    // self = src1 * src2
    if (!THC_pointwiseApply3(state, self_, src1, src2, TensorMulOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(cpow)(THCState *state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    // self = pow(self, src2)
    if (!THC_pointwiseApply2(state, self_, src2, TensorCPowOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    // self = pow(src1, src2)
    if (!THC_pointwiseApply3(state, self_, src1, src2, TensorCPowOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(pow)(THCState *state, THCTensor *self_, THCTensor *src, real value) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src));
  if (self_ == src) {
    if (THCNumerics<real>::eq(value, ScalarConvert<int, real>::to(1))) {
      if (!THC_pointwiseApply1(state, self_, TensorPowOp<real, 1>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else if (THCNumerics<real>::eq(value, ScalarConvert<int, real>::to(2))) {
      if (!THC_pointwiseApply1(state, self_, TensorPowOp<real, 2>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else if (THCNumerics<real>::eq(value, ScalarConvert<int, real>::to(3))) {
      if (!THC_pointwiseApply1(state, self_, TensorPowOp<real, 3>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
#if defined(THC_REAL_IS_HALF) || defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)
    } else if (THCNumerics<real>::eq(value, ScalarConvert<int, real>::to(-1))) {
      if (!THC_pointwiseApply1(state, self_, TensorPowOp<real, -1>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else if (THCNumerics<real>::eq(value, ScalarConvert<int, real>::to(-2))) {
      if (!THC_pointwiseApply1(state, self_, TensorPowOp<real, -2>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
#endif
    } else {
      // fallback implementation using pow
      if (!THC_pointwiseApply1(state, self_, TensorPowOp<real, -3>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src);

    if (THCNumerics<real>::eq(value, ScalarConvert<int, real>::to(1))) {
      if (!THC_pointwiseApply2(state, self_, src, TensorPowOp<real, 1>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else if (THCNumerics<real>::eq(value, ScalarConvert<int, real>::to(2))) {
      if (!THC_pointwiseApply2(state, self_, src, TensorPowOp<real, 2>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else if (THCNumerics<real>::eq(value, ScalarConvert<int, real>::to(3))) {
      if (!THC_pointwiseApply2(state, self_, src, TensorPowOp<real, 3>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
#if defined(THC_REAL_IS_HALF) || defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)
    } else if (THCNumerics<real>::eq(value, ScalarConvert<int, real>::to(-1))) {
      if (!THC_pointwiseApply2(state, self_, src, TensorPowOp<real, -1>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else if (THCNumerics<real>::eq(value, ScalarConvert<int, real>::to(-2))) {
      if (!THC_pointwiseApply2(state, self_, src, TensorPowOp<real, -2>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
#endif
    } else {
      // fallback implementation using pow
      if (!THC_pointwiseApply2(state, self_, src, TensorPowOp<real, -3>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    }
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(tpow)(THCState *state, THCTensor *self_, real value, THCTensor *src)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src));
  if (self_ == src) {
    if (!THC_pointwiseApply1(state, self_, TensorTPowOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src);

    if (!THC_pointwiseApply2(state, self_, src, TensorTPowOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}
THC_API void
THCTensor_(cdiv)(THCState* state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    // self /= src2
    if (!THC_pointwiseApply2(state, self_, src2, TensorDivOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    // self = src1 / src2
    if (!THC_pointwiseApply3(state, self_, src1, src2, TensorDivOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(clshift)(THCState* state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
#if defined(THC_REAL_IS_HALF)
  return THError("clshift not supported for torch.CudaHalfTensor");
#else
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    // self /= src2
    if (!THC_pointwiseApply2(state, self_, src2, TensorLShiftOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    // self = src1 / src2
    if (!THC_pointwiseApply3(state, self_, src1, src2, TensorLShiftOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
#endif
}

THC_API void
THCTensor_(crshift)(THCState* state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
#if defined(THC_REAL_IS_HALF)
  return THError("crshift not supported for torch.CudaHalfTensor");
#else
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    // self /= src2
    if (!THC_pointwiseApply2(state, self_, src2, TensorRShiftOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    // self = src1 / src2
    if (!THC_pointwiseApply3(state, self_, src1, src2, TensorRShiftOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
#endif
}

THC_API void
THCTensor_(cmax)(THCState *state, THCTensor *self, THCTensor *src1, THCTensor *src2)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 2, "sizes do not match");

  if (self == src1) {
    if (!THC_pointwiseApply2(state, self, src2, TensorMaxOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self, src1);
    if (!THC_pointwiseApply3(state, self, src1, src2, TensorMaxOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }
}

THC_API void
THCTensor_(cmin)(THCState *state, THCTensor *self, THCTensor *src1, THCTensor *src2)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 2, "sizes do not match");

  if (self == src1) {
    if (!THC_pointwiseApply2(state, self, src2, TensorMinOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self, src1);
    if (!THC_pointwiseApply3(state, self, src1, src2, TensorMinOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }
}

THC_API void
THCTensor_(cremainder)(THCState *state, THCTensor *self, THCTensor *src1, THCTensor *src2)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 2, "sizes do not match");

  if (self == src1) {
    if (!THC_pointwiseApply2(state, self, src2, TensorCRemainderOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self, src1);
    if (!THC_pointwiseApply3(state, self, src1, src2, TensorCRemainderOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }
}

THC_API void
THCTensor_(cfmod)(THCState *state, THCTensor *self, THCTensor *src1, THCTensor *src2)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 2, "sizes do not match");

  if (self == src1) {
    if (!THC_pointwiseApply2(state, self, src2, TensorCFmodOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self, src1);
    if (!THC_pointwiseApply3(state, self, src1, src2, TensorCFmodOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }
}

THC_API void
THCTensor_(cmaxValue)(THCState *state, THCTensor *self, THCTensor *src, real value)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self, src));

  if (self == src) {
    if (!THC_pointwiseApply1(state, self, TensorMaxValueOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self, src);
    if (!THC_pointwiseApply2(state, self, src, TensorMaxValueOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }
}

THC_API void
THCTensor_(cminValue)(THCState *state, THCTensor *self, THCTensor *src, real value)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self, src));

  if (self == src) {
    if (!THC_pointwiseApply1(state, self, TensorMinValueOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self, src);
    if (!THC_pointwiseApply2(state, self, src, TensorMinValueOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }
}

THC_API void
THCTensor_(addcmul)(THCState *state, THCTensor *self_, THCTensor *t, real value, THCTensor *src1, THCTensor *src2)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 4, self_, t, src1, src2));
  if(self_ != t)
  {
    THCTensor_(resizeAs)(state, self_, t);
    THCTensor_(copy)(state, self_, t);
  }
  else
  {
    THArgCheck(THCTensor_(nElement)(state, self_) == THCTensor_(nElement)(state, src1),
               1, "sizes do not match");
  }

  THArgCheck(THCTensor_(nElement)(state, src1) == THCTensor_(nElement)(state, src2),
             3, "sizes do not match");

  if (!THC_pointwiseApply3(state, self_, src1, src2, TensorAddCMulOp<real>(value))) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(addcdiv)(THCState *state, THCTensor *self_, THCTensor *t, real value, THCTensor *src1, THCTensor *src2)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 4, self_, t, src1, src2));
  if(self_ != t)
  {
    THCTensor_(resizeAs)(state, self_, t);
    THCTensor_(copy)(state, self_, t);
  }
  else
  {
    THArgCheck(THCTensor_(nElement)(state, self_) == THCTensor_(nElement)(state, src1),
               1, "sizes do not match");
  }
  THArgCheck(THCTensor_(nElement)(state, src1) == THCTensor_(nElement)(state, src2),
             3, "sizes do not match");

  if (!THC_pointwiseApply3(state, self_, src1, src2, TensorAddCDivOp<real>(value))) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(cbitand)(THCState* state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
#if defined(THC_REAL_IS_HALF) || defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)
  return THError("cbitand is only supported for integer type tensors");
#else
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    // self /= src2
    if (!THC_pointwiseApply2(state, self_, src2, TensorBitAndOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    // self = src1 / src2
    if (!THC_pointwiseApply3(state, self_, src1, src2, TensorBitAndOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
#endif
}

THC_API void
THCTensor_(cbitor)(THCState* state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
#if defined(THC_REAL_IS_HALF) || defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)
  return THError("cbitor is only supported for integer type tensors");
#else
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    // self /= src2
    if (!THC_pointwiseApply2(state, self_, src2, TensorBitOrOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    // self = src1 / src2
    if (!THC_pointwiseApply3(state, self_, src1, src2, TensorBitOrOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
#endif
}

THC_API void
THCTensor_(cbitxor)(THCState* state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
#if defined(THC_REAL_IS_HALF) || defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)
  return THError("cbitor is only supported for integer type tensors");
#else
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    // self /= src2
    if (!THC_pointwiseApply2(state, self_, src2, TensorBitXorOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    // self = src1 / src2
    if (!THC_pointwiseApply3(state, self_, src1, src2, TensorBitXorOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
#endif
}
#endif
