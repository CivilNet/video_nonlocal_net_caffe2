
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCTensorRandom.cu"
#else

#define NUM_BLOCKS min((int)THCCeilDiv(size, (ptrdiff_t) BLOCK_SIZE), MAX_NUM_BLOCKS)

#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_HALF)

THC_API void THCTensor_(uniform)(THCState* state, THCTensor *self_, double a, double b)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self_));
  ptrdiff_t size = THCTensor_(nElement)(state, self_);
  if (size == 0) return;
  THCGenerator* gen = THCRandom_getGenerator(state);
  THCTensor *self = THCTensor_(newContiguous)(state, self_);
  real *data = THCTensor_(data)(state, self);

  generate_uniform<<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
      gen->state.gen_states, size, data, a, b);

  THCTensor_(freeCopyTo)(state, self, self_);
};

THC_API void THCTensor_(normal)(THCState* state, THCTensor *self_, double mean, double stdv)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self_));
  ptrdiff_t size = THCTensor_(nElement)(state, self_);
  if (size == 0) return;
  THCGenerator* gen = THCRandom_getGenerator(state);
  THCTensor *self = THCTensor_(newContiguous)(state, self_);
  real *data = THCTensor_(data)(state, self);

  generate_normal<<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
      gen->state.gen_states, size, data, mean, stdv);

  THCTensor_(freeCopyTo)(state, self, self_);
};

THC_API void THCTensor_(normal_means)(THCState *state, THCTensor *self, THCTensor *means, double stddev) {
  THCTensor_(resizeAs)(state, self, means);
  THCTensor_(normal)(state, self, 0, stddev);
  THCTensor_(cadd)(state, self, self, ScalarConvert<int, real>::to(1), means);
}

THC_API void THCTensor_(normal_stddevs)(THCState *state, THCTensor *self, double mean, THCTensor *stddevs)
{
  THCTensor_(resizeAs)(state, self, stddevs);
  THCTensor_(normal)(state, self, 0, 1);
  THCTensor_(cmul)(state, self, self, stddevs);
  THCTensor_(add)(state, self, self, ScalarConvert<double, real>::to(mean));
}

THC_API void THCTensor_(normal_means_stddevs)(THCState *state, THCTensor *self, THCTensor *means, THCTensor *stddevs)
{
  THCTensor_(resizeAs)(state, self, means);
  THCTensor_(normal)(state, self, 0, 1);
  THCTensor_(cmul)(state, self, self, stddevs);
  THCTensor_(cadd)(state, self, self, ScalarConvert<int, real>::to(1), means);
}

THC_API void THCTensor_(logNormal)(THCState* state, THCTensor *self_, double mean, double stdv)
{

  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self_));
  ptrdiff_t size = THCTensor_(nElement)(state, self_);
  if (size == 0) return;
  THCGenerator* gen = THCRandom_getGenerator(state);

  THCTensor *self = THCTensor_(newContiguous)(state, self_);
  real *data = THCTensor_(data)(state, self);

  generateLogNormal<real><<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
      gen->state.gen_states, size, data, mean, stdv);

  THCTensor_(freeCopyTo)(state, self, self_);
};

THC_API void THCTensor_(exponential)(THCState* state, THCTensor *self_, double lambda)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self_));
  ptrdiff_t size = THCTensor_(nElement)(state, self_);
  if (size == 0) return;
  THCGenerator* gen = THCRandom_getGenerator(state);

  THCTensor *self = THCTensor_(newContiguous)(state, self_);
  real *data = THCTensor_(data)(state, self);

  generate_exponential<<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
      gen->state.gen_states, size, data, lambda);

  THCTensor_(freeCopyTo)(state, self, self_);
};

THC_API void THCTensor_(cauchy)(THCState* state, THCTensor *self_, double median, double sigma)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self_));
  ptrdiff_t size = THCTensor_(nElement)(state, self_);
  if (size == 0) return;
  THCGenerator* gen = THCRandom_getGenerator(state);

  THCTensor *self = THCTensor_(newContiguous)(state, self_);
  real *data = THCTensor_(data)(state, self);

  generate_cauchy<<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
      gen->state.gen_states, size, data, median, sigma);

  THCTensor_(freeCopyTo)(state, self, self_);
};

void THCTensor_(renormRows)(struct THCState* state,
                             THCTensor* t) {
  THAssert(THCTensor_(nDimension)(state, t) == 2);
  int64_t rows = THCTensor_(size)(state, t, 0);
  int64_t cols = THCTensor_(size)(state, t, 1);

  cudaDeviceProp* props = THCState_getCurrentDeviceProperties(state);
  THAssert(props != NULL);

  int numSM = props->multiProcessorCount;
  int maxThreads = props->maxThreadsPerBlock;

  dim3 grid(rows < numSM * 4 ? rows : numSM * 4);
  dim3 block(cols < maxThreads ? cols : maxThreads);

  renormRowsL1<real>
    <<<grid, block, block.x * sizeof(real),
    THCState_getCurrentStream(state)>>>(THCTensor_(data)(state, t),
                                        rows, cols);
}

THC_API void THCTensor_(multinomial)(struct THCState *state,
                                      THCudaLongTensor *self,
                                      THCTensor *prob_dist,
                                      int n_sample,
                                      int with_replacement)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self, prob_dist));
  THCGenerator* gen = THCRandom_getGenerator(state);

  int inputSize = THCTensor_(nDimension)(state, prob_dist);
  THArgCheck(inputSize > 0 && inputSize <= 2, 2,
             "prob_dist must be 1 or 2 dim");

  // Categories are in the innermost dimension
  int64_t numDist =
    inputSize == 1 ? 1 : THCTensor_(size)(state, prob_dist, 0);
  int64_t numCategoriesLong =
    inputSize == 1 ? THCTensor_(size)(state, prob_dist, 0) :
    THCTensor_(size)(state, prob_dist, 1);

  // Since the index tensor is float, numCategories cannot exceed max
  // float integer precision
  THArgCheck(numCategoriesLong <= FLOAT32_MAX_CONSECUTIVE_INT, 2,
             "number of categories cannot exceed 2^24");
  int numCategories = (int) numCategoriesLong;

  THArgCheck(n_sample > 0, 3, "cannot sample <= 0 samples");

  if (!with_replacement) {
    THArgCheck(n_sample <= numCategories, 2,
               "cannot sample n_sample > prob_dist:size(1) samples without "
               "replacement");
  }

  int free_prob_dist = 0;

  // Restructure data for 2d
  if (inputSize == 1) {
    THCTensor *temp = THCTensor_(new)(state);
    THCTensor_(unsqueeze1d)(state, temp, prob_dist, 0);
    prob_dist = temp;
    free_prob_dist = 1;
  }

  THCudaLongTensor_resize2d(state, self, numDist, n_sample);

  // get current device properties
  cudaDeviceProp* props = THCState_getCurrentDeviceProperties(state);
  THAssert(props != NULL);
  int numSM = props->multiProcessorCount;
  int maxThreads = props->maxThreadsPerBlock;
  int maxShared = props->sharedMemPerBlock;
  int requiredShared = (numCategories < maxThreads ? numCategories : maxThreads)
                                * (sizeof(real) * sizeof(accreal));

  if (n_sample == 1 && maxShared >= requiredShared) {
    // Optimized allocation-free implementation
    // To exploit greater parallelism for the sampling, generate the
    // Uniform random samples in a separate kernel launch, into
    // temporarily allocated memory. The device RNG is thread-limited
    THCTensor *sampled = THCTensor_(newWithSize2d)(state, numDist, n_sample);
    THCTensor_(uniform)(state, sampled, 0.0, 1.0);

    dim3 block(numCategories < maxThreads ? numCategories : maxThreads);
    dim3 grid(numDist < numSM * 4 ? numDist : numSM * 4);

    sampleMultinomialOnce<real, accreal>
      <<<grid, block,
         requiredShared,
         THCState_getCurrentStream(state)>>>(
      THCudaLongTensor_data(state, self),
      numDist,
      numCategories,
      THCTensor_(data)(state, sampled),
      THCTensor_(data)(state, prob_dist),
      THCTensor_(stride)(state, prob_dist, 0),
      THCTensor_(stride)(state, prob_dist, 1)
      );
    THCTensor_(free)(state, sampled);
  } else {
    // Generic, slow implementation with memory allocations

    // For sampling without replacement, we modify the distribution
    // for subsequent samples in this space
    THCTensor* origDist = THCTensor_(new)(state);
    THCTensor_(resizeAs)(state, origDist, prob_dist);
    THCTensor_(copy)(state, origDist, prob_dist);

    THCTensor* normDist = THCTensor_(new)(state);
    THCTensor_(resizeAs)(state, normDist, prob_dist);

    THCTensor* prefixSum = THCTensor_(new)(state);

    // Renorm along rows
    THCTensor_(copy)(state, normDist, origDist);
    THCTensor_(renormRows)(state, normDist);

    // Prefix sum along rows
    THCTensor_(cumsum)(state, prefixSum, normDist, 1);

    if (with_replacement) {
      // Sample with replacement

      // Binary search is warp divergent (so effectively we're running
      // with just a single thread), but for better utilization,
      // we need each block to have at least 4 warps.
      dim3 block(32, 4);

      // Each warp in a block will generate a sample from one
      // distribution concurrently.
      dim3 grid(numDist < MAX_NUM_BLOCKS ? numDist : MAX_NUM_BLOCKS);

      sampleMultinomialWithReplacement
        <<<grid, block, 0, THCState_getCurrentStream(state)>>>(
          gen->state.gen_states,
          n_sample,
          THCudaLongTensor_data(state, self),
          numDist, numCategories,
          THCTensor_(data)(state, prefixSum));
    } else {
      // Sample without replacement

      // Binary search is warp divergent (so effectively we're running
      // with just a single thread), but for better utilization,
      // we need each block to have at least 4 warps.
      dim3 block(32, 4);

      // Each warp in a block will generate a sample from a different
      // distribution concurrently.
      ptrdiff_t numBlocks = THCCeilDiv(numDist, (int64_t) 4);
      dim3 grid(numBlocks < MAX_NUM_BLOCKS ? numBlocks : MAX_NUM_BLOCKS);

      for (int sample = 0; sample < n_sample; ++sample) {
        if (sample > 0) {
          // Update probabilities
          // Renorm along rows
          THCTensor_(copy)(state, normDist, origDist);
          THCTensor_(renormRows)(state, normDist);

          // Prefix sum along rows
          THCTensor_(cumsum)(state, prefixSum, normDist, 1);
        }

        // The kernel can only draw one sample before we have to
        // recalculate our distribution
        sampleMultinomialWithoutReplacement
          <<<grid, block, 0, THCState_getCurrentStream(state)>>>(
            gen->state.gen_states,
            n_sample,
            sample,
            THCudaLongTensor_data(state, self),
            numDist, numCategories,
            THCTensor_(data)(state, origDist),
            THCTensor_(data)(state, prefixSum));
      }
    }

    THCTensor_(free)(state, prefixSum);
    THCTensor_(free)(state, normDist);
    THCTensor_(free)(state, origDist);
  }

  // Revert data restructuring based on input sizes
  if (inputSize == 1) {
    THCudaLongTensor_resize1d(state, self, n_sample);
  }
  if (free_prob_dist) {
    THCTensor_(free)(state, prob_dist);
  }
}

THC_API void THCTensor_(multinomialAliasSetup)(THCState *state, THCTensor *_probs, THCudaLongTensor *_J, THCTensor *_q){
  THAssert(THCTensor_(isContiguous)(state, _q));
  THAssert(THCudaLongTensor_isContiguous(state, _J));
  THAssert(THCTensor_(isContiguous)(state, _probs));
  int64_t inputsize = THCTensor_(nElement)(state, _probs);
  THCudaLongTensor *smaller = THCudaLongTensor_newWithSize1d(state, inputsize);
  THCudaLongTensor *larger = THCudaLongTensor_newWithSize1d(state, inputsize);
  THCudaLongTensor *smaller_short = THCudaLongTensor_newWithSize1d(state, inputsize);
  THCudaLongTensor *larger_short = THCudaLongTensor_newWithSize1d(state, inputsize);

  THCudaLongTensor_resize1d(state, _J, inputsize);
  THCTensor_(resize1d)(state, _q, inputsize);

  real one = ScalarConvert<int64_t, real>::to(1);
  int inputBlockDim = THCCeilDiv((int)inputsize + BLOCK_SIZE - 1, BLOCK_SIZE);
  aliasMultinomialFilter
    <<<inputBlockDim, BLOCK_SIZE, 0, THCState_getCurrentStream(state) >>>(
                     THCTensor_(data)(state, _q),
                     THCTensor_(data)(state, _probs),
                     THCudaLongTensor_data(state, smaller),
                     THCudaLongTensor_data(state, larger),
                     THCudaLongTensor_data(state, _J),
                     THCudaLongTensor_data(state, smaller_short),
                     THCudaLongTensor_data(state, larger_short),
                     one, inputsize
                     );

  THCudaLongTensor_nonzero(state, smaller_short, smaller);
  THCudaLongTensor_nonzero(state, larger_short, larger);
  int h_large_c = THCudaLongTensor_nElement(state, larger_short);
  THCudaLongTensor_resize1d(state, smaller_short, inputsize);
  THCudaLongTensor_resize1d(state, larger_short, inputsize);
  aliasMultinomialSetup
    <<<1, 1, 0, THCState_getCurrentStream(state)>>>(
                THCudaLongTensor_data(state, _J),
                THCTensor_(data)(state, _q),
                inputsize,
                THCudaLongTensor_data(state, smaller_short),
                THCudaLongTensor_data(state, larger_short),
                inputsize - h_large_c, h_large_c
                );
  real q_max = THCTensor_(maxall)(state, _q);
  condDiv<<<
    inputBlockDim, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
                      THCTensor_(data)(state, _q),
                      THCudaLongTensor_data(state, _J),
                      inputsize, q_max
                      );

  THCudaLongTensor_free(state, smaller);
  THCudaLongTensor_free(state, larger);
  THCudaLongTensor_free(state, smaller_short);
  THCudaLongTensor_free(state, larger_short);
}

THC_API void THCTensor_(multinomialAliasDraw)(THCState *state, THCudaLongTensor *self, THCudaLongTensor *_J, THCTensor *_q){
  THAssert(THCTensor_(isContiguous)(state, _q));
  THAssert(THCudaLongTensor_isContiguous(state, _J));
  THCGenerator* gen = THCRandom_getGenerator(state);
  int64_t K = THCudaLongTensor_nElement(state, _J);
  int64_t output_nelem = THCudaLongTensor_nElement(state, self);
  ptrdiff_t size = THCudaLongTensor_nElement(state, self);

  THCTensor *uniform = THCTensor_(newWithSize1d)(state, output_nelem);
  THCTensor *bernoulli = THCTensor_(newWithSize1d)(state, output_nelem);

  THCTensor_(uniform)(state, uniform, 0, K);
  THCTensor_(uniform)(state, bernoulli, 0, 1);

  multinomialAliasDrawKernel
    <<<THCCeilDiv((int)output_nelem+BLOCK_SIZE-1, BLOCK_SIZE), BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
          size,
          THCudaLongTensor_data(state, self),
          THCudaLongTensor_data(state, _J),
          THCTensor_(data)(state, _q),
          K,
          THCTensor_(data)(state, uniform),
          THCTensor_(data)(state, bernoulli)
          );
}

THC_API void THCTensor_(rand)(THCState *state, THCTensor *r_, THLongStorage *size)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, r_));
  THCTensor_(resize)(state, r_, size, NULL);
  THCTensor_(uniform)(state, r_, 0, 1);
}

void THCTensor_(randn)(THCState *state, THCTensor *r_, THLongStorage *size)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, r_));
  THCTensor_(resize)(state, r_, size, NULL);
  THCTensor_(normal)(state, r_, 0, 1);
}

#endif

#if defined(THC_REAL_IS_DOUBLE)
GENERATE_KERNEL1(generate_bernoulli, double, double p, double, curand_uniform_double, x <= p)
#else
GENERATE_KERNEL1(generate_bernoulli, real, double p, float, curand_uniform, (ScalarConvert<bool, real>::to(x <= p)))
#endif

THC_API void THCTensor_(bernoulli)(THCState* state, THCTensor *self_, double p)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self_));
  ptrdiff_t size = THCTensor_(nElement)(state, self_);
  if (size == 0) return;
  THCGenerator* gen = THCRandom_getGenerator(state);
  THCTensor *self = THCTensor_(newContiguous)(state, self_);
  real *data = THCTensor_(data)(state, self);

  generate_bernoulli<<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
      gen->state.gen_states, size, data, p);

  THCTensor_(freeCopyTo)(state, self, self_);
};

void THCTensor_(bernoulli_Tensor)(THCState *state, THCTensor *self, THCTensor* p)
{
#if defined(THC_REAL_IS_FLOAT)
  THCTensor_(bernoulli_FloatTensor)(state, self, p);
#elif defined(THC_REAL_IS_DOUBLE)
  THCTensor_(bernoulli_DoubleTensor)(state, self, p);
#endif
}

#define DEFINE_BERNOULLI_TENSOR(NAME, PROB_TYPE, PROB_DATA_TYPE)               \
THC_API void THCTensor_(NAME)(THCState* state,                                 \
        THCTensor *self_, PROB_TYPE *probs_)                                   \
{                                                                              \
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, probs_));             \
  ptrdiff_t size = THCTensor_(nElement)(state, self_);                         \
  if (size == 0) return;                                                       \
  THCGenerator* gen = THCRandom_getGenerator(state);                           \
  THCTensor *self = THCTensor_(newContiguous)(state, self_);                   \
  PROB_TYPE *probs = PROB_TYPE##_newContiguous(state, probs_);                 \
  ptrdiff_t prob_size = PROB_TYPE##_nElement(state, probs);                    \
  real *result_data = THCTensor_(data)(state, self);                           \
  PROB_DATA_TYPE *probs_data = PROB_TYPE##_data(state, probs);                 \
                                                                               \
  THArgCheck(size == prob_size, 3, "inconsistent tensor size");                \
                                                                               \
  generate_bernoulli_tensor<<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>( \
      gen->state.gen_states, size, result_data, probs_data);                         \
                                                                               \
  PROB_TYPE##_free(state, probs);                                              \
  THCTensor_(freeCopyTo)(state, self, self_);                                  \
}

DEFINE_BERNOULLI_TENSOR(bernoulli_FloatTensor, THCudaTensor, float)
DEFINE_BERNOULLI_TENSOR(bernoulli_DoubleTensor, THCudaDoubleTensor, double)

#if defined(THC_REAL_IS_DOUBLE)
GENERATE_KERNEL1(generate_geometric, double, double p, double, curand_uniform_double, ceil(log(x) / log(1-p)))
#else
GENERATE_KERNEL1(generate_geometric, real, double p, float, curand_uniform, (ScalarConvert<float, real>::to(ceilf(logf(x) / log(1-p)))))
#endif

#if defined(THC_REAL_IS_LONG) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_FLOAT)
#define CURAND64(STATE) (((uint64_t)curand(STATE)) << 32) | (uint64_t)curand(STATE)
GENERATE_KERNEL2(generate_random, real, int32_t base, uint32_t range, uint32_t, curand, \
    static_cast<real>(static_cast<int32_t>((x % range) + base)))
GENERATE_KERNEL2(generate_random_64, real, int64_t base, uint64_t range, uint64_t, CURAND64, \
    static_cast<real>(static_cast<int64_t>((x % range) + base)))
#elif defined(THC_REAL_IS_HALF)
GENERATE_KERNEL2(generate_random, real, int32_t base, uint32_t range, uint32_t, curand,
    (ScalarConvert<int32_t, real>::to(static_cast<int32_t>(x % range + base))))
#else
GENERATE_KERNEL2(generate_random, real, int32_t base, uint32_t range, uint32_t, curand,
    static_cast<real>(static_cast<int32_t>(x % range + base)))
#endif

THC_API void THCTensor_(geometric)(THCState* state, THCTensor *self_, double p)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self_));
  ptrdiff_t size = THCTensor_(nElement)(state, self_);
  if (size == 0) return;
  THCGenerator* gen = THCRandom_getGenerator(state);

  THCTensor *self = THCTensor_(newContiguous)(state, self_);
  real *data = THCTensor_(data)(state, self);

  generate_geometric<<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
      gen->state.gen_states, size, data, p);

  THCTensor_(freeCopyTo)(state, self, self_);
};

THC_API void THCTensor_(clampedRandom)(THCState* state, THCTensor *self_, int64_t min_val, int64_t max_val)
{
  THArgCheck(min_val < max_val, 2,
             "max must be greater than min, but got: min = %lld, max = %lld", min_val, max_val);
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self_));
  ptrdiff_t size = THCTensor_(nElement)(state, self_);
  if (size == 0) return;
  THCGenerator* gen = THCRandom_getGenerator(state);
  THCTensor *self = THCTensor_(newContiguous)(state, self_);
  real *data = THCTensor_(data)(state, self);

  uint64_t range = max_val - min_val;

#if defined(THC_REAL_IS_LONG) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_FLOAT)
  if (range > 1ULL << 32) {
    generate_random_64<<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
        gen->state.gen_states, size, data, min_val, range);
  } else {
#endif
    generate_random<<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
        gen->state.gen_states, size, data, min_val, range);
#if defined(THC_REAL_IS_LONG) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_FLOAT)
  }
#endif

  THCTensor_(freeCopyTo)(state, self, self_);
};

THC_API void THCTensor_(cappedRandom)(THCState* state, THCTensor *self_, int64_t max_val)
{
  THCTensor_(clampedRandom)(state, self_, 0LL, max_val);
};

#define HLF_MANT_DIG 11

THC_API void THCTensor_(random)(THCState* state, THCTensor *self_)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self_));
  ptrdiff_t size = THCTensor_(nElement)(state, self_);
  if (size == 0) return;
  THCGenerator* gen = THCRandom_getGenerator(state);
  THCTensor *self = THCTensor_(newContiguous)(state, self_);
  real *data = THCTensor_(data)(state, self);

#if defined(THC_REAL_IS_HALF)
  generate_random<<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
      gen->state.gen_states, size, data, 0UL, (1UL << HLF_MANT_DIG) + 1);
#elif defined(THC_REAL_IS_FLOAT)
  generate_random<<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
      gen->state.gen_states, size, data, 0UL, (1UL << FLT_MANT_DIG) + 1);
#elif defined(THC_REAL_IS_DOUBLE)
  generate_random_64<<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
      gen->state.gen_states, size, data, 0ULL, (1ULL << DBL_MANT_DIG) + 1);
#elif defined(THC_REAL_IS_LONG)
  generate_random_64<<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
      gen->state.gen_states, size, data, 0ULL, static_cast<uint64_t>(std::numeric_limits<real>::max()) + 1);
#else
  generate_random<<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
      gen->state.gen_states, size, data, 0UL, static_cast<uint32_t>(std::numeric_limits<real>::max()) + 1);
#endif

  THCTensor_(freeCopyTo)(state, self, self_);
};

#undef HLF_MANT_DIG
#undef CURAND64
#undef NUM_BLOCKS

#endif
