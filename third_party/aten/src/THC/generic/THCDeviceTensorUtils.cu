
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCDeviceTensorUtils.cu"
#else

/// Constructs a THCDeviceTensor initialized from a THCudaTensor. Will
/// error if the dimensionality does not match exactly.
template <typename T, int Dim,
          typename IndexT, template <typename U> class PtrTraits>
THCDeviceTensor<T, Dim, IndexT, PtrTraits>
toDeviceTensor(THCState* state, THCTensor* t);

template <typename T, int Dim, typename IndexT>
THCDeviceTensor<T, Dim, IndexT, DefaultPtrTraits>
toDeviceTensor(THCState* state, THCTensor* t) {
  return toDeviceTensor<T, Dim, IndexT, DefaultPtrTraits>(state, t);
}

template <typename T, int Dim>
THCDeviceTensor<T, Dim, int, DefaultPtrTraits>
toDeviceTensor(THCState* state, THCTensor* t) {
  return toDeviceTensor<T, Dim, int, DefaultPtrTraits>(state, t);
}

template <typename T, int Dim,
          typename IndexT, template <typename U> class PtrTraits>
THCDeviceTensor<T, Dim, IndexT, PtrTraits>
toDeviceTensor(THCState* state, THCTensor* t) {
  if (Dim != THCTensor_(nDimension)(state, t)) {
    THError("THCudaTensor dimension mismatch");
  }
  // Determine the maximum offset into the tensor achievable; `IndexT`
  // must be smaller than this type in order to use it.
  ptrdiff_t maxOffset = 0;
  IndexT sizes[Dim];
  IndexT strides[Dim];

  for (int i = 0; i < Dim; ++i) {
    int64_t size = THCTensor_(size)(state, t, i);
    int64_t stride = THCTensor_(stride)(state, t, i);

    maxOffset += (size - 1) * stride;

    sizes[i] = (IndexT) size;
    strides[i] = (IndexT) stride;
  }

  if (maxOffset > std::numeric_limits<IndexT>::max()) {
    THError("THCudaTensor sizes too large for THCDeviceTensor conversion");
  }

  return THCDeviceTensor<T, Dim, IndexT, PtrTraits>(
    THCTensor_(data)(state, t), sizes, strides);
}

#endif
