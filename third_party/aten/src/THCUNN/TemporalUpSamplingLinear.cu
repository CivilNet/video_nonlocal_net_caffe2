#include "hip/hip_runtime.h"
// Adapted from interp.cpp from Caffe util by Pauline Luc
// Originally developed by George Papandreou
#include "THCUNN.h"
#include "common.h"
#include "linear_upsampling.h"
#include "THCDeviceTensor.cuh"
#include "THCDeviceTensorUtils.cuh"
#include "THCDeviceUtils.cuh"
#include "THCHalf.h"
#include "THCHalfAutoNumerics.cuh"
#include "THCAtomics.cuh"

template<typename Dtype, typename Acctype>
__global__ void caffe_gpu_interp2_kernel(const int n,
    const Acctype rwidth, const bool align_corners,
    const THCDeviceTensor<Dtype, 3> data1, THCDeviceTensor<Dtype, 3> data2) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  const int batchsize = data1.getSize(0);
  const int channels = data1.getSize(1);
  const int width1 = data1.getSize(2);
  const int width2 = data2.getSize(2);

  if (index < n) {
    const int w2 = index % width2;
    // special case: just copy
    if (width1 == width2) {
      const int w1 = w2;
      for (int n = 0; n < batchsize ; n++){
        for (int c = 0; c < channels; ++c) {
          const Dtype val = data1[n][c][w1];
          data2[n][c][w2] = val;
        }
      }
      return;
    }
    //
    const Acctype w1r = linear_upsampling_compute_source_index<Acctype>(rwidth, w2, align_corners);
    const int w1 = w1r;
    const int w1p = (w1 < width1 - 1) ? 1 : 0;
    const Acctype w1lambda = w1r - w1;
    const Acctype w0lambda = Acctype(1) - w1lambda;
    //
    for (int n = 0; n < batchsize ; n++){
        for (int c = 0; c < channels; ++c) {
        const Acctype val = w0lambda * data1[n][c][w1]
                            + w1lambda * data1[n][c][w1+w1p];
        data2[n][c][w2] = ScalarConvert<Acctype, Dtype>::to(val);
      }
    }
  }
}

// Backward (adjoint) operation 1 <- 2 (accumulates)
template <typename Dtype, typename Acctype>
__global__ void caffe_gpu_interp2_kernel_backward(const int n,
    const Acctype rwidth, const bool align_corners,
    THCDeviceTensor<Dtype, 3> data1, const THCDeviceTensor<Dtype, 3> data2){
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  const int batchsize = data1.getSize(0);
  const int channels = data1.getSize(1);
  const int width1 = data1.getSize(2);
  const int width2 = data2.getSize(2);
  if (index < n) {
    const int w2 = index % width2;
    // special case: just copy
    if (width1 == width2) {
      const int w1 = w2;
      for (int n = 0; n < batchsize ; n++){
        for (int c = 0; c < channels; ++c) {
          const Dtype val = data2[n][c][w1];
          data1[n][c][w2] += val;
        }
      }
      return;
    }
    //
    const Acctype w1r = linear_upsampling_compute_source_index<Acctype>(rwidth, w2, align_corners);
    const int w1 = w1r;
    const int w1p = (w1 < width1 - 1) ? 1 : 0;
    const Acctype w1lambda = w1r - w1;
    const Acctype w0lambda = Acctype(1) - w1lambda;
    //
    for (int n = 0; n < batchsize ; n++){
      for (int c = 0; c < channels; ++c) {
        const Dtype d2val = data2[n][c][w2];
        atomicAdd(data1[n][c][w1].data(),
                  ScalarConvert<Acctype, Dtype>::to(w0lambda * d2val));
        atomicAdd(data1[n][c][w1+w1p].data(),
                  ScalarConvert<Acctype, Dtype>::to(w1lambda * d2val));
      }
    }
  }
}


#include "generic/TemporalUpSamplingLinear.cu"
#include "THCGenerateFloatTypes.h"
