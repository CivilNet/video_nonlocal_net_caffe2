#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "THCHalf.h"
#include "THCHalfAutoNumerics.cuh"
#include "THCNumerics.cuh"
#include <THC/THCApply.cuh>

template <typename T>
struct TensorSigmoidOp {
  __device__ __forceinline__ void operator()(T* out, T* in) const {
    T one = (T) 1.0;
    *out = one / (one + THCNumerics<T>::exp(- *in));
  }

  __device__ __forceinline__ void operator()(T* v) const {
    T one = (T) 1.0;
    *v = one / (one + THCNumerics<T>::exp(- *v));
  }
};

#ifdef CUDA_HALF_TENSOR
template <>
struct TensorSigmoidOp<half> {
  __device__ __forceinline__ void operator()(half* out, half* in) const {
#ifdef CUDA_HALF_INSTRUCTIONS
    half one = ScalarConvert<int, half>::to(1);
    *out = hdiv(one, __hadd(one, hexp(__hneg(*in))));
#else
    float fin = ScalarConvert<half, float>::to(*in);
    *out = ScalarConvert<float, half>::to(1.0f / (1.0f + expf(- fin)));
#endif
  }

  __device__ __forceinline__ void operator()(half* v) const {
#ifdef CUDA_HALF_INSTRUCTIONS
    half one = ScalarConvert<int, half>::to(1);
    *v = hdiv(one, __hadd(one, hexp(__hneg(*v))));
#else
    float fv = ScalarConvert<half, float>::to(*v);
    *v = ScalarConvert<float, half>::to(1.0f / (1.0f + expf(- fv)));
#endif
  }
};
#endif

#include "generic/FusedRNNKernel.cu"
#include "THCGenerateFloatTypes.h"
