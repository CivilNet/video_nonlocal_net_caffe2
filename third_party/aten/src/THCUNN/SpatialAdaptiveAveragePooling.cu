#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "THCHalf.h"
#include "THCHalfAutoNumerics.cuh"
#include "THCAtomics.cuh"

#define START_IND(a,b,c) (int)floor((float)(a * c) / b)
#define END_IND(a,b,c) (int)ceil((float)((a + 1) * c) / b)
// #define START_IND(a,b,c) a * c / b
// #define END_IND(a,b,c)  (a + 1) * c / b + ((a + 1) * c % b > 0)?1:0


#define CUDA_MAX_THREADS 1024   // this is safe, in reality 256 is our limit

// 4d tensor B x D x H x W
// All kernels view batch dim B and feature dim D as collapsed.

/*
 * Description:
 *    this function adaptively average pools an input 4D tensor along dimensions 2 and 3
 *    4D input, 4D output
 */
 template <typename T>
__global__ void adaptiveaveragepool(T *input, T *output,
                        int isizeH, int isizeW,
                        int osizeH, int osizeW,
                        int64_t istrideD, int64_t istrideH, int64_t istrideW)
{
  // iterators on output pixels
  int oh, ow;

  // select input/output plane based on thread/block ID
  int o_plane = blockIdx.x;
  int i_plane = o_plane;

  output = output + o_plane*osizeH*osizeW;
  input = input + i_plane*istrideD;

  int ostartH = blockDim.y*blockIdx.y + threadIdx.y;
  int oendH = osizeH;
  const int ostepH = blockDim.y*gridDim.y;

  int ostartW = threadIdx.x;
  int oendW = osizeW;
  const int ostepW = blockDim.x;

  // For all output pixels...
  for(oh = ostartH; oh < oendH; oh += ostepH) {

    int istartH = START_IND(oh, osizeH, isizeH);
    int iendH   = END_IND(oh, osizeH, isizeH);
    int kH = iendH - istartH;

    for(ow = ostartW; ow < oendW; ow += ostepW) {

      int istartW = START_IND(ow, osizeW, isizeW);
      int iendW   = END_IND(ow, osizeW, isizeW);
      int kW = iendW - istartW;

      // Compute the average pooling over corresponding input pixels
      T *ptr_input = input + istartH*istrideH + istartW*istrideW;
      T *ptr_output = output + oh*osizeW + ow;
      T sum = ScalarConvert<int, T>::to(0);
      int ih, iw;
      for(ih = 0; ih < kH; ++ih) {
        for(iw = 0; iw < kW; ++iw) {
          T val = ptr_input[iw*istrideW];
          sum += val;
        }
        ptr_input += istrideH; // next input line
      }
      // Update output
      *ptr_output = sum / kH / kW;
    }
  }
}

/*
 * Description:
 *    this function computes the gradInput from gradOutput
 */
 template <typename T>
__global__ void adaptiveaveragegradinput(
  T *gradInput, T *gradOutput,
  int isizeH, int isizeW, int osizeH, int osizeW
)
{
  // iterators on input pixels
  int ih, iw;

  // select input/output plane based on thread/block ID
  int i_plane = blockIdx.x;
  int o_plane = i_plane;

  gradOutput = gradOutput + o_plane*osizeH*osizeW;
  gradInput = gradInput + i_plane*isizeH*isizeW;

  int istartH = blockDim.y*blockIdx.y + threadIdx.y;
  int iendH = isizeH;
  int istepH = blockDim.y*gridDim.y;

  int istartW = threadIdx.x;
  int iendW = isizeW;
  int istepW = blockDim.x;

  // compute gradInput
  for(ih = istartH; ih < iendH; ih += istepH) {

    int ostartH = START_IND(ih, isizeH, osizeH);
    int oendH   = END_IND(ih, isizeH, osizeH);

    for(iw = istartW; iw < iendW; iw += istepW) {

      int ostartW = START_IND(iw, isizeW, osizeW);
      int oendW   = END_IND(iw, isizeW, osizeW);

      // Compute the gradients over corresponding output pixels
      T *ptr_gradInput = gradInput + ih*isizeW + iw;

      int oh, ow;
      for(oh = ostartH; oh < oendH; ++oh) {
        int kH = START_IND(oh, osizeH, isizeH) - END_IND(oh, osizeH, isizeH);
        for(ow = ostartW; ow < oendW; ++ow) {
          int kW = START_IND(ow, osizeW, isizeW) - END_IND(ow, osizeW, isizeW);
          T grad_delta = gradOutput[ow + oh*osizeW] / kH / kW;
          *ptr_gradInput += grad_delta;
        }
      }
    }
  }
}

/*
 * Description:
 *    this function computes the gradInput from gradOutput
 *    (uses atomic add)
 */
 template <typename T>
__global__ void atomicadaptiveaveragegradinput(
  T *gradInput, T *gradOutput,
  int isizeH, int isizeW, int osizeH, int osizeW
)
{
  // iterators on output indices
  int oh, ow;

  // select input/output plane based on thread/block ID
  int o_plane = blockIdx.x;
  int i_plane = o_plane;

  gradOutput = gradOutput + o_plane*osizeW*osizeH;
  gradInput = gradInput + i_plane*isizeW*isizeH;

  int ostartH = blockDim.y*blockIdx.y + threadIdx.y;
  int oendH = osizeH;
  int ostepH = blockDim.y*gridDim.y;

  int ostartW = threadIdx.x;
  int oendW = osizeW;
  int ostepW = blockDim.x;

  // For all output pixels...
  for(oh = ostartH; oh < oendH; oh += ostepH) {

    int istartH = START_IND(oh, osizeH, isizeH);
    int iendH   = END_IND(oh, osizeH, isizeH);
    int kH = iendH - istartH;

    for(ow = ostartW; ow < oendW; ow += ostepW) {

      int istartW = START_IND(ow, osizeW, isizeW);
      int iendW   = END_IND(ow, osizeW, isizeW);
      int kW = iendW - istartW;

      // Compute the gradients for over corresponding input pixels
      T *ptr_gradInput = gradInput + istartH*isizeW + istartW;
      T *ptr_gradOutput = gradOutput + oh*osizeW + ow;
      T grad_delta = *ptr_gradOutput / kW / kH;

      int ih, iw;
      for(ih = 0; ih < kH; ++ih) {
        for(iw = 0; iw < kW; ++iw) {
          // atomic add since different threads could update same variable
          atomicAdd(&(ptr_gradInput[iw]), grad_delta);
        }
        ptr_gradInput += isizeW; // next input line
      }
    }
  }
}

#include "generic/SpatialAdaptiveAveragePooling.cu"
#include "THCGenerateFloatTypes.h"

#undef CUDA_MAX_THREADS
#undef START_IND
#undef END_IND
