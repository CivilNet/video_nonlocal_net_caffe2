
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/MultiMarginCriterion.cu"
#else

// TODO: improve error messages
void THNN_(MultiMarginCriterion_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCIndexTensor *target,
           THCTensor *output,
           bool sizeAverage,
           int p,
           THCTensor *weights,
           accreal margin_,
           bool reduce)
{
  real margin = ScalarConvert<accreal, real>::to(margin_);
  THCUNN_assertSameGPU(state, 2, input, target);
  input = THCTensor_(newContiguous)(state, input);
  if(weights)
    weights = THCTensor_(newContiguous)(state, weights);
  if (input->nDimension == 1)
  {
    dim3 blocks(1);
    dim3 threads(MULTIMARGIN_THREADS);
    THCTensor_(resize1d)(state, output, 1);
    if (p == 1)
    {
      cunn_MultiMarginCriterion_updateOutput_kernel<1, real, accreal> <<<blocks,threads, 0, THCState_getCurrentStream(state)>>>(
        THCTensor_(data)(state, output),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        weights ? THCTensor_(data)(state, weights) : NULL,
        1, input->size[0],
        sizeAverage,
        margin
      );
    }
    else if (p == 2)
    {
      cunn_MultiMarginCriterion_updateOutput_kernel<2, real, accreal> <<<blocks,threads, 0, THCState_getCurrentStream(state)>>>(
        THCTensor_(data)(state, output),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        weights ? THCTensor_(data)(state, weights) : NULL,
        1, input->size[0],
        sizeAverage,
        margin
      );
    }
    THCudaCheck(cudaGetLastError());
  }
  else if (input->nDimension == 2)
  {
    int nframe = input->size[0];
    THArgCheck((target->nDimension == 1) && (target->size[0] == nframe), 3,
               "inconsistent target size");
    dim3 blocks(input->size[0]);
    dim3 threads(MULTIMARGIN_THREADS);

    if (!reduce)
    {
      THCTensor_(resize1d)(state, output, input->size[0]);
      if (p == 1)
      {
        cunn_MultiMarginCriterion_updateOutput_kernel<1, real, accreal> <<<blocks,threads, 0, THCState_getCurrentStream(state)>>>(
          THCTensor_(data)(state, output),
          THCTensor_(data)(state, input),
          THCIndexTensor_(data)(state, target),
          weights ? THCTensor_(data)(state, weights) : NULL,
          nframe, input->size[1],
          false,
          margin
        );
      }
      else if (p == 2)
      {
        cunn_MultiMarginCriterion_updateOutput_kernel<2, real, accreal> <<<blocks,threads, 0, THCState_getCurrentStream(state)>>>(
          THCTensor_(data)(state, output),
          THCTensor_(data)(state, input),
          THCIndexTensor_(data)(state, target),
          weights ? THCTensor_(data)(state, weights) : NULL,
          nframe, input->size[1],
          false,
          margin
        );
      }
      THCudaCheck(cudaGetLastError());
    }
    else
    {
      THCTensor_(resize1d)(state, output, 1);
      THCTensor *output_ = THCTensor_(newWithSize1d)(state, input->size[0]);  // tmp output buffer
      if (p == 1)
      {
        cunn_MultiMarginCriterion_updateOutput_kernel<1, real, accreal> <<<blocks,threads, 0, THCState_getCurrentStream(state)>>>(
          THCTensor_(data)(state, output_),
          THCTensor_(data)(state, input),
          THCIndexTensor_(data)(state, target),
          weights ? THCTensor_(data)(state, weights) : NULL,
          nframe, input->size[1],
          sizeAverage,
          margin
        );
      }
      else if (p == 2)
      {
        cunn_MultiMarginCriterion_updateOutput_kernel<2, real, accreal> <<<blocks,threads, 0, THCState_getCurrentStream(state)>>>(
          THCTensor_(data)(state, output_),
          THCTensor_(data)(state, input),
          THCIndexTensor_(data)(state, target),
          weights ? THCTensor_(data)(state, weights) : NULL,
          input->size[0], input->size[1],
          sizeAverage,
          margin
        );
      }
      THCudaCheck(cudaGetLastError());
      float sum = THCTensor_(sumall)(state, output_);
      THCTensor_(set1d)(state, output, 0, ScalarConvert<accreal, real>::to(sum));
      THCTensor_(free)(state, output_);
    }
  }
  else
  {
    THError("vector or matrix expected");
  }

  THCTensor_(free)(state, input);
  if(weights)
    THCTensor_(free)(state, weights);
}

void THNN_(MultiMarginCriterion_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCIndexTensor *target,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           bool sizeAverage,
           int p,
           THCTensor *weights,
           accreal margin_,
           bool reduce)
{
  real margin = ScalarConvert<accreal, real>::to(margin_);
  THCUNN_assertSameGPU(state, 3, input, gradInput, target);
  input = THCTensor_(newContiguous)(state, input);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);
  THCTensor_(resizeAs)(state, gradInput, input);
  if(weights)
    weights = THCTensor_(newContiguous)(state, weights);

  if (input->nDimension == 1)
  {
    dim3 blocks(1);
    dim3 threads(MULTIMARGIN_THREADS);

    if (p == 1)
    {
      cunn_MultiMarginCriterion_updateGradInput_kernel<1, real, accreal> <<<blocks,threads, 0, THCState_getCurrentStream(state)>>>(
        THCTensor_(data)(state, gradInput),
        THCTensor_(data)(state, gradOutput),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        weights ? THCTensor_(data)(state, weights) : NULL,
        1, gradInput->size[0],
        sizeAverage,
        margin,
        reduce
      );
    }
    else if (p == 2)
    {
      cunn_MultiMarginCriterion_updateGradInput_kernel<2, real, accreal> <<<blocks,threads, 0, THCState_getCurrentStream(state)>>>(
        THCTensor_(data)(state, gradInput),
        THCTensor_(data)(state, gradOutput),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        weights ? THCTensor_(data)(state, weights) : NULL,
        1, gradInput->size[0],
        sizeAverage,
        margin,
        reduce
      );
    }
    THCudaCheck(cudaGetLastError());
  }
  else if (input->nDimension == 2)
  {
    int nframe = gradInput->size[0];
    THArgCheck((target->nDimension == 1) && (target->size[0] == nframe), 3,
               "inconsistent target size");
    dim3 blocks(gradInput->size[0]);
    dim3 threads(MULTIMARGIN_THREADS);

    if (p == 1)
    {
      cunn_MultiMarginCriterion_updateGradInput_kernel<1, real, accreal> <<<blocks,threads, 0, THCState_getCurrentStream(state)>>>(
        THCTensor_(data)(state, gradInput),
        THCTensor_(data)(state, gradOutput),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        weights ? THCTensor_(data)(state, weights) : NULL,
        nframe, gradInput->size[1],
        sizeAverage,
        margin,
        reduce
      );
    }
    else if (p == 2)
    {
      cunn_MultiMarginCriterion_updateGradInput_kernel<2, real, accreal> <<<blocks,threads, 0, THCState_getCurrentStream(state)>>>(
        THCTensor_(data)(state, gradInput),
        THCTensor_(data)(state, gradOutput),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        weights ? THCTensor_(data)(state, weights) : NULL,
        nframe, gradInput->size[1],
        sizeAverage,
        margin,
        reduce
      );
    }
    THCudaCheck(cudaGetLastError());
  }
  else
  {
    THError("vector or matrix expected");
  }

  THCTensor_(free)(state, input);
  if(weights)
    THCTensor_(free)(state, weights);
}

#endif
