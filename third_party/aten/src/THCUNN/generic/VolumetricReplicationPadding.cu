
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/VolumetricReplicationPadding.cu"
#else

static inline void THNN_(VolumetricReplicationPadding_shapeCheck)(
                         THCState *state,
                         THCTensor *input,
                         THCTensor *gradOutput,
                         int pleft, int pright,
                         int ptop, int pbottom,
                         int pfront, int pback) {
  THArgCheck(TensorUtils<THCTensor>::canUse32BitIndexMath(state, input), 2,
             "input tensor must fit into 32-bit index math");
  int numInputDims = THCTensor_(nDimension)(state, input);

  THCUNN_argCheck(state, numInputDims == 4 || numInputDims == 5, 2, input,
    "4D or 5D (batch mode) tensor expected for input, but got: %s");

  int planeDim = 0;
  int dimd = 1;
  int dimh = 2;
  int dimw = 3;
  if (numInputDims == 5) {
    planeDim++;
    dimd++;
    dimh++;
    dimw++;
    }

  int numPlanes = THCTensor_(size)(state, input, planeDim);
  int idepth = input->size[dimd];
  int iheight = input->size[dimh];
  int iwidth = input->size[dimw];
  int odepth = idepth + pfront + pback;
  int oheight = iheight + ptop + pbottom;
  int owidth  = iwidth + pleft + pright;
  THArgCheck(owidth >= 1 || oheight >= 1 || odepth >= 1, 2,
             "input (D: %d H: %d, W: %d) is too small."
             " Calculated output D: %d H: %d W: %d",
             idepth, iheight, iwidth, odepth, oheight, owidth);

  if (gradOutput != NULL) {
    THArgCheck(TensorUtils<THCTensor>::canUse32BitIndexMath(state, gradOutput),
               3, "output gradient tensor must fit into 32-bit index math");

    THArgCheck(numPlanes == THCTensor_(size)(state, gradOutput, planeDim), 3,
               "gradOutput width unexpected. Expected: %d, Got: %d",
               numPlanes, THCTensor_(size)(state, gradOutput, planeDim));
    THArgCheck(owidth == THCTensor_(size)(state, gradOutput, dimw), 3,
               "gradOutput width unexpected. Expected: %d, Got: %d",
               owidth, THCTensor_(size)(state, gradOutput, dimw));
    THArgCheck(oheight == THCTensor_(size)(state, gradOutput, dimh), 3,
               "gradOutput height unexpected. Expected: %d, Got: %d",
               oheight, THCTensor_(size)(state, gradOutput, dimh));
    THArgCheck(odepth == THCTensor_(size)(state, gradOutput, dimd), 3,
               "gradOutput depth unexpected. Expected: %d, Got: %d",
               odepth, THCTensor_(size)(state, gradOutput, dimd));
  }
}

void THNN_(VolumetricReplicationPadding_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           int pleft, int pright,
           int ptop, int pbottom,
           int pfront, int pback) {
  THNN_(VolumetricReplicationPadding_shapeCheck)(
        state, input, NULL, pleft, pright, ptop,
        pbottom, pfront, pback);

  int planeDim = 0;
  int dimd = 1;
  int dimh = 2;
  int dimw = 3;
  int numBatch = 1;

  int numInputDims = THCTensor_(nDimension)(state, input);

  if (numInputDims == 5) {
    numBatch = THCTensor_(size)(state, input, 0);
    planeDim++;
    dimd++;
    dimh++;
    dimw++;
  }

  int numPlanes = THCTensor_(size)(state, input, planeDim);
  int inputD = THCTensor_(size)(state, input, dimd);
  int inputH = THCTensor_(size)(state, input, dimh);
  int inputW = THCTensor_(size)(state, input, dimw);
  int outputD = inputD + pfront + pback;
  int outputH = inputH + ptop + pbottom;
  int outputW  = inputW + pleft + pright;

  THCDeviceTensor<real, 5> devInput;
  THCDeviceTensor<real, 5> devOutput;

  if (numInputDims == 4) {
    THCTensor_(resize4d)(state, output, numPlanes, outputD, outputH, outputW);

    devInput = toDeviceTensor<real, 4>(state, input).upcastOuter<5>();
    devOutput = toDeviceTensor<real, 4>(state, output).upcastOuter<5>();
  } else {
    THCTensor_(resize5d)(state, output, numBatch, numPlanes, outputD, outputH,
                          outputW);

    devInput = toDeviceTensor<real, 5>(state, input);
    devOutput = toDeviceTensor<real, 5>(state, output);
  }

  int outputPlaneSize = devOutput.getSize(2) * devOutput.getSize(3) *
      devOutput.getSize(4);
  dim3 gridSize(THCCeilDiv(outputPlaneSize, 256),
            devOutput.getSize(1),
            devOutput.getSize(0));
  dim3 blockSize(outputPlaneSize > 256 ? 256 : outputPlaneSize);

  VolumetricReplicationPadding_updateOutput<real><<<gridSize, blockSize, 0, THCState_getCurrentStream(state)>>>(
    devInput, devOutput, pfront, pback, ptop, pbottom, pleft, pright);
}

void THNN_(VolumetricReplicationPadding_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           int pleft, int pright,
           int ptop, int pbottom,
           int pfront, int pback) {
  THNN_(VolumetricReplicationPadding_shapeCheck)(
        state, input, gradOutput, pleft, pright, ptop,
        pbottom, pfront, pback);

  int planeDim = 0;
  int dimd = 1;
  int dimh = 2;
  int dimw = 3;

  int numInputDims = THCTensor_(nDimension)(state, input);
  if (numInputDims == 5) {
    planeDim++;
    dimd++;
    dimh++;
    dimw++;
  }

  THCTensor_(resizeAs)(state, gradInput, input);
  THCTensor_(zero)(state, gradInput);

  THCDeviceTensor<real, 5> devGradInput;
  THCDeviceTensor<real, 5> devGradOutput;

  if (numInputDims == 4) {
    devGradInput = toDeviceTensor<real, 4>(state, gradInput).upcastOuter<5>();
    devGradOutput =
        toDeviceTensor<real, 4>(state, gradOutput).upcastOuter<5>();
  } else {
    devGradInput = toDeviceTensor<real, 5>(state, gradInput);
    devGradOutput = toDeviceTensor<real, 5>(state, gradOutput);
  }

  int outputPlaneSize = devGradOutput.getSize(2) * devGradOutput.getSize(3) *
      devGradOutput.getSize(4);
  dim3 gridSize(THCCeilDiv(outputPlaneSize, 256),
            devGradOutput.getSize(1),
            devGradOutput.getSize(0));
  dim3 blockSize(outputPlaneSize > 256 ? 256 : outputPlaneSize);

  VolumetricReplicationPadding_updateGradInput<<<gridSize, blockSize, 0, THCState_getCurrentStream(state)>>>(
    devGradInput, devGradOutput, pfront, pback, ptop, pbottom, pleft, pright);
}

#endif
