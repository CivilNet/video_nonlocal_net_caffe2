
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/SpatialUpSamplingBilinear.cu"
#else

#include "../linear_upsampling.h"

static inline void THNN_(SpatialUpSamplingBilinear_shapeCheck)
                        (THCState *state,
                         THCTensor *input, THCTensor *gradOutput,
                         int nBatch, int nChannels,
                         int inputHeight, int inputWidth,
                         int outputHeight, int outputWidth) {
  THArgCheck(inputHeight > 0 && inputWidth > 0
             && outputHeight > 0 && outputWidth > 0, 2,
             "input and output sizes should be greater than 0,"
             " but got input (H: %d, W: %d) output (H: %d, W: %d)",
             inputHeight, inputWidth, outputHeight, outputWidth);
  if (input != NULL) {
     THCUNN_argCheck(state, input->nDimension == 4, 2, input,
                     "4D input tensor expected but got: %s");
  }

  if (gradOutput != NULL) {
    THCUNN_check_dim_size(state, gradOutput, 4, 0, nBatch);
    THCUNN_check_dim_size(state, gradOutput, 4, 1, nChannels);
    THCUNN_check_dim_size(state, gradOutput, 4, 2, outputHeight);
    THCUNN_check_dim_size(state, gradOutput, 4, 3, outputWidth);
  }
}

void THNN_(SpatialUpSamplingBilinear_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           int outputHeight,
           int outputWidth,
           bool align_corners)
{
  int nbatch = THCTensor_(size)(state, input, 0);
  int channels = THCTensor_(size)(state, input, 1);
  int inputHeight = THCTensor_(size)(state, input, 2);
  int inputWidth = THCTensor_(size)(state, input, 3);
  THNN_(SpatialUpSamplingBilinear_shapeCheck)
       (state, input, NULL,
        nbatch, channels,
        inputHeight, inputWidth,
        outputHeight, outputWidth);
  input = THCTensor_(newContiguous)(state, input);
  THCUNN_assertSameGPU(state, 2, input, output);
  THCTensor_(resize4d)(state, output,
                       THCTensor_(size)(state, input, 0),
                       THCTensor_(size)(state, input, 1),
                       outputHeight, outputWidth);
  THCTensor_(zero)(state, output);
  THCDeviceTensor<real, 4> idata = toDeviceTensor<real, 4>(state, input);
  THCDeviceTensor<real, 4> odata = toDeviceTensor<real, 4>(state, output);
  THAssert(inputHeight > 0 && inputWidth > 0 && outputHeight > 0 && outputWidth > 0);
  const accreal rheight = linear_upsampling_compute_scale<accreal>(inputHeight, outputHeight, align_corners);
  const accreal rwidth = linear_upsampling_compute_scale<accreal>(inputWidth, outputWidth, align_corners);
  const int num_kernels = outputHeight * outputWidth;
  const int num_threads =
    THCState_getCurrentDeviceProperties(state)->maxThreadsPerBlock;
  cudaStream_t stream = THCState_getCurrentStream(state);
  caffe_gpu_interp2_kernel<real, accreal> <<<THCCeilDiv(num_kernels, num_threads), num_threads ,
   0 , stream>>>(num_kernels, rheight, rwidth, align_corners, idata, odata);
  THCudaCheck(cudaGetLastError());
  THCTensor_(free)(state, input);
}


void THNN_(SpatialUpSamplingBilinear_updateGradInput)(
           THCState *state,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           int nbatch,
           int nchannels,
           int inputHeight,
           int inputWidth,
           int outputHeight,
           int outputWidth,
           bool align_corners)
{
  THNN_(SpatialUpSamplingBilinear_shapeCheck)
       (state, NULL, gradOutput,
        nbatch, nchannels,
        inputHeight, inputWidth,
        outputHeight, outputWidth);
  gradInput = THCTensor_(newContiguous)(state, gradInput);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);
  THCUNN_assertSameGPU(state, 2, gradOutput, gradInput);
  THCTensor_(resize4d)(state, gradInput, nbatch, nchannels, inputHeight, inputWidth);
  THCTensor_(zero)(state, gradInput);
  THCDeviceTensor<real, 4> data1 = toDeviceTensor<real, 4>(state, gradInput);
  THCDeviceTensor<real, 4> data2 = toDeviceTensor<real, 4>(state, gradOutput);
  const accreal rheight = linear_upsampling_compute_scale<accreal>(inputHeight, outputHeight, align_corners);
  const accreal rwidth = linear_upsampling_compute_scale<accreal>(inputWidth, outputWidth, align_corners);
  const int num_kernels = outputHeight * outputWidth;
  const int num_threads =
    THCState_getCurrentDeviceProperties(state)->maxThreadsPerBlock;
  cudaStream_t stream = THCState_getCurrentStream(state);
  caffe_gpu_interp2_kernel_backward<real ,accreal> <<<THCCeilDiv(num_kernels, num_threads),
  num_threads, 0, stream>>>(num_kernels, rheight, rwidth, align_corners, data1, data2);
  THCudaCheck(cudaGetLastError());
  THCTensor_(free)(state, gradInput);
  THCTensor_(free)(state, gradOutput);
}

#endif
