
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/GatedLinearUnit.cu"
#else

void THNN_(GatedLinear_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           int dim)
{
  THCUNN_assertSameGPU(state, 2, input, output);

  // size output to half of input
  dim = dim - TH_INDEX_BASE;
  const int64_t nIn = THCTensor_(size)(state, input, dim);
  THArgCheck(nIn % 2 == 0, 2, "Halving dimension must be even. Dim %d is size %ld",
      dim + TH_INDEX_BASE, nIn);
  const int64_t inputSize = THCTensor_(size)(state, input, dim) / 2;
  THLongStorage *newSizes = THCTensor_(newSizeOf)(state, input);
  THLongStorage_set(newSizes, dim, inputSize);
  THCTensor_(resize)(state, output, newSizes, NULL);

  // halve tensor
  THCTensor *firstHalf = THCTensor_(newNarrow)(state, input, dim, 0, inputSize);
  THCTensor *secondHalf = THCTensor_(newNarrow)(state, input, dim, inputSize, inputSize);

  // x = x1:cmul( sigmoid(x2) )
  THC_pointwiseApply3(state, output, secondHalf, firstHalf, gatedLinearCSigMul_functor<real, accreal>());

  THLongStorage_free(newSizes);
  THCTensor_(free)(state, firstHalf);
  THCTensor_(free)(state, secondHalf);
}

void THNN_(GatedLinear_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           int dim)
{
  THCUNN_assertSameGPU(state, 2, gradOutput, gradInput);
  dim = dim - TH_INDEX_BASE;
  const int64_t nIn = THCTensor_(size)(state, input, dim);
  THArgCheck(nIn % 2 == 0, 2, "Halving dimension must be even. Dim %d is size %ld",
      dim + TH_INDEX_BASE, nIn);

  THCTensor_(resizeAs)(state, gradInput, input);
  const int64_t inputSize = THCTensor_(size)(state, input, dim) / 2;
  THCTensor *firstHalf = THCTensor_(newNarrow)(state, input, dim, 0, inputSize);
  THCTensor *gradInputfirstHalf = THCTensor_(newNarrow)(state, gradInput, dim, 0, inputSize);
  const int64_t stride_i = THCTensor_(stride)(state, input, dim) * inputSize;
  const int64_t stride_gI = THCTensor_(stride)(state, gradInput, dim) * inputSize;
  THC_pointwiseApply3(state, gradInputfirstHalf, gradOutput, firstHalf, gatedLinearDerivative<real,accreal>(stride_i, stride_gI)); 
  THCTensor_(free)(state, firstHalf);
  THCTensor_(free)(state, gradInputfirstHalf);
}

#endif
