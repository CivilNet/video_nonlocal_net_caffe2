
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/Col2Im.cu"
#else

static inline void THNN_(Col2Im_shapeCheck)(
                         THCState *state,
                         THCTensor *input,
                         THCTensor *gradOutput,
                         int outputHeight, int outputWidth,
                         int kH, int kW, int dH, int dW,
                         int padH, int padW, int sH, int sW) {

  THArgCheck(kW > 0 && kH > 0, 6,
             "kernel size should be greater than zero, but got kH: %d kW: %d", kH, kW);
  THArgCheck(sW > 0 && sH > 0, 12,
             "stride should be greater than zero, but got sH: %d sW: %d", sH, sW);
  THArgCheck(dW > 0 && dH > 0, 8,
             "dilation should be greater than zero, but got dH: %d dW: %d", dH, dW);

  int ndim = THCTensor_(nDimension)(state, input);
  THCUNN_argCheck(state, ndim == 2 || ndim == 3, 2, input,
                  "2D or 3D input tensor expected but got %s");

  int batch_dim = (ndim == 4) ? 0 : -1;
  long nInputPlane  = input->size[batch_dim + 1];
  long inputLength  = input->size[batch_dim + 2];

  long nOutputPlane = nInputPlane / (kW * kH);

  if (outputWidth < 1 || outputHeight < 1) {
    THError("Given input size: (%lld x %lld). "
            "Calculated output size: (%lld x %d x %d). Output size is too small",
            (long long)nInputPlane, (long long)inputLength, (long long)nOutputPlane, outputHeight, outputWidth);
  }
}

void THNN_(Col2Im_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           int outputHeight, int outputWidth,
           int kH, int kW,
           int dH, int dW,
           int padH, int padW,
           int sH, int sW) {

  THCUNN_assertSameGPU(state, 2, input, output);

  THNN_(Col2Im_shapeCheck)(state, input, NULL, outputHeight, outputWidth,
                           kH, kW, dH, dW, padH, padW, sH, sW);

  bool batched_input = true;
  if (input->nDimension == 2) {
      // Force batch
      batched_input = false;
      THCTensor_(resize3d)(state, input, 1, input->size[0], input->size[1]);
  }

  long batchSize = input->size[0];
  long nInputPlane = input->size[1];
  long nOutputPlane = nInputPlane / (kW * kH);

  input = THCTensor_(newContiguous)(state, input);

  THCTensor_(resize4d)(state, output, batchSize, nOutputPlane, outputHeight, outputWidth);
  THCTensor_(zero)(state, output);

  THCTensor *input_n = THCTensor_(new)(state);
  THCTensor *output_n = THCTensor_(new)(state);

  int height_col = (outputHeight + 2 * padH - (dH * (kH - 1) + 1)) / sH + 1;
  int width_col = (outputWidth + 2 * padW - (dW * (kW - 1) + 1)) / sW + 1;

  for (int elt = 0; elt < batchSize; elt++) {
    THCTensor_(select)(state, input_n, input, 0, elt);
    THCTensor_(select)(state, output_n, output, 0, elt);

    col2im<real, accreal>(
      THCState_getCurrentStream(state),
      THCTensor_(data)(state, input_n),
      nOutputPlane,
      outputHeight, outputWidth,
      height_col, width_col,
      kH, kW,
      padH, padW,
      sH, sW,
      dH, dW, THCTensor_(data)(state, output_n));
  }

  THCTensor_(free)(state, input_n);
  THCTensor_(free)(state, output_n);

  if (!batched_input) {
      THCTensor_(resize3d)(state, output, nOutputPlane, outputHeight, outputWidth);
  }
  THCTensor_(free)(state, input);
}

void THNN_(Col2Im_updateGradInput)(
           THCState *state,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           int kH, int kW,
           int dH, int dW,
           int padH, int padW,
           int sH, int sW) {

  THNN_(Im2Col_updateOutput)(state, gradOutput, gradInput,
                             kH, kW, dH, dW, padH, padW, sH, sW);

}

#endif
