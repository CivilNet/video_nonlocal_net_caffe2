
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/IndexLinear.cu"
#else

static bool THCUNN_checkKeysValues(THCState *state, THCudaLongTensor* keys,
                                   THCTensor* values)
{
    return THCudaLongTensor_size(state, keys, 0) == THCTensor_(nElement)(state, values)
        && THCTensor_(nDimension)(state, values) == 1
        && THCudaLongTensor_nDimension(state, keys) == 1;
}

void THNN_(IndexLinear_updateOutput)(
    THCState *state,
    THCudaLongTensor *keys,
    int64_t keysOffset,
    THCTensor *values,
    THCudaLongTensor *sizes,
    THCudaLongTensor *cumSumSizes,
    THCTensor *output,
    THCTensor *weight,
    THCTensor *bias,
    THCTensor *normalizedValues,
    int   train)
{
    // Make sure these inputs are contiguous to accelerate computations
    THArgCheck(THCudaLongTensor_isContiguous(state, keys), 1,
               "keys vector must be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, values), 3,
               "values vector must be contiguous");
    THArgCheck(THCudaLongTensor_isContiguous(state, sizes), 4,
               "sizes vector must be contiguous");
    THArgCheck(THCudaLongTensor_isContiguous(state, cumSumSizes), 5,
               "cumSumSizes vector must be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, output), 6,
               "output vector must be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, weight), 7,
               "weight matrix must be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, bias), 8,
               "bias vector must be contiguous");
    THArgCheck(THCUNN_checkKeysValues(state, keys, values), 1,
               "Keys and values should have the same number of elements");

    int64_t batchSize = sizes->size[0];
    int64_t outDim = bias->size[0];
    int64_t wDim = weight->size[1];
    int64_t weightStride = weight->stride[0];
    int maxNormalize = wDim - outDim;
    int64_t keysSize = keys->size[0];
    int64_t nnzPerRow = divup(keysSize, batchSize);

    THCTensor_(resize2d)(state, output, batchSize, outDim);
    int64_t *keysData        = THCudaLongTensor_data (state, keys);
    real *valuesData      = THCTensor_(data)      (state, values);
    int64_t *cumSumSizesData = THCudaLongTensor_data (state, cumSumSizes);
    real *biasData        = THCTensor_(data)      (state, bias);
    real *weightData      = THCTensor_(data)      (state, weight);
    real *outData         = THCTensor_(data)      (state, output);

    cudaStream_t stream = THCState_getCurrentStream(state);
    dim3 threads(THREADS_X, THREADS_Y);
    int blocks_x = divup(outDim, threads.x);
    int blocks_y = batchSize;
    int nnzPerBlock = ((outDim == 1 || batchSize == 1) ? THREADS_X : NNZ_PER_BLOCK_MAX);
    int blocks_z = divup(nnzPerRow, nnzPerBlock);

    dim3 blocks(blocks_x, blocks_y, blocks_z);

    if (blocks_z > 1) {
        THCudaCheck(cudaMemsetAsync(outData, 0, outDim * batchSize * sizeof(real), stream));
    }

    real *normalizedValuesData = NULL;
    if (maxNormalize && train) {
        THCTensor_(resize1d)(state, normalizedValues, keysSize);
        normalizedValuesData = THCTensor_(data)(state, normalizedValues);
        updateOutput<real, true><<<blocks, threads, 0, stream>>>
            (outData, normalizedValuesData, valuesData, cumSumSizesData, keysData,
             batchSize, outDim, weightData, biasData, weightStride, keysOffset, maxNormalize, nnzPerBlock);
    } else {
        updateOutput<real, false><<<blocks, threads, 0, stream>>>
            (outData, normalizedValuesData, valuesData, cumSumSizesData, keysData,
             batchSize, outDim, weightData, biasData, weightStride, keysOffset, maxNormalize, nnzPerBlock);
    }
}

void THNN_(IndexLinear_accGradParameters)(
    THCState *state,
    THCudaLongTensor *keys,
    int64_t keysOffset,
    THCTensor *values,
    THCudaLongTensor *sizes,
    THCudaLongTensor *cumSumSizes,
    THCTensor *gradOutput,
    THCTensor *gradWeight,
    THCTensor *gradBias,
    THCTensor *weight,
    THCTensor *bias,
    THCTensor* valuesBuffer,
    accreal weightDecay,
    accreal scale)
{
    int64_t keysSize = keys->size[0];
    int64_t batchSize = sizes->size[0];
    int64_t outDim = bias->size[0];
    int64_t wDim = weight->size[1];
    int maxNormalize = wDim - outDim;

    // Make sure these inputs are contiguous to accelerate computations
    THArgCheck(THCudaLongTensor_isContiguous(state, keys), 1,
               "keys vector must be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, values), 3,
               "values vector must be contiguous");
    THArgCheck(THCudaLongTensor_isContiguous(state, sizes), 4,
               "sizes vector must be contiguous");
    THArgCheck(THCudaLongTensor_isContiguous(state, cumSumSizes), 5,
               "cumSumSizes vector must be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, gradOutput), 6,
               "gradOutput vector must be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, gradWeight), 7,
               "gradWeight matrix must be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, gradBias), 8,
               "gradBias vector must be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, weight), 9,
               "weight matrix must be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, bias), 10,
               "bias vector must be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, valuesBuffer), 11,
               "valuesBuffer vector must be contiguous");
    THArgCheck(THCUNN_checkKeysValues(state, keys, values), 1,
               "Keys and values should have the same number of elements");

    THCTensor_(resize2d)(state, gradWeight, keysSize, outDim * (maxNormalize > 0 ? 2 : 1));

    real *valuesData      = THCTensor_(data)      (state, values);
    int64_t *cumSumSizesData = THCudaLongTensor_data (state, cumSumSizes);
    real *gradOutputData  = THCTensor_(data)      (state, gradOutput);
    real *gradBiasData    = THCTensor_(data)      (state, gradBias);
    real *gradWeightData  = THCTensor_(data)      (state, gradWeight);
    int64_t gradWeightStride = gradWeight->stride[0];

    cudaStream_t stream = THCState_getCurrentStream(state);
    dim3 threads(THREADS_X, THREADS_Y);
    int blocks_x = divup(outDim, threads.x);
    accGradBias<real, false><<<blocks_x, threads, 0, stream>>>
        (gradBiasData, gradOutputData, outDim, batchSize, scale, weightDecay);

    dim3 blocks(blocks_x, batchSize);
    accGradWeight<real><<<blocks, threads, 0, stream>>>
        (gradWeightData, gradOutputData, valuesData, cumSumSizesData, outDim,
         gradWeightStride, scale, weightDecay, maxNormalize);
}

void THNN_(IndexLinear_accUpdateGradParameters)(
    THCState *state,
    THCudaLongTensor *keys,
    int64_t keysOffset,
    THCTensor *values,
    THCudaLongTensor *sizes,
    THCudaLongTensor *cumSumSizes,
    THCTensor *gradOutput,
    THCTensor *weight,
    THCTensor *bias,
    accreal weightDecay,
    accreal scale)
{
    // Make sure these inputs are contiguous to accelerate computations
    THArgCheck(THCudaLongTensor_isContiguous(state, keys), 1,
               "keys vector must be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, values), 3,
               "values vector must be contiguous");
    THArgCheck(THCudaLongTensor_isContiguous(state, sizes), 4,
               "sizes vector must be contiguous");
    THArgCheck(THCudaLongTensor_isContiguous(state, cumSumSizes), 5,
               "cumSumSizes vector must be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, gradOutput), 6,
               "gradOutput vector must be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, weight), 7,
               "weight matrix must be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, bias), 8,
               "bias vector must be contiguous");
    THArgCheck(THCUNN_checkKeysValues(state, keys, values), 1,
               "Keys and values should have the same number of elements");

    int64_t batchSize = sizes->size[0];
    int64_t outDim = bias->size[0];
    int64_t keysSize = keys->size[0];
    int64_t wDim = weight->size[1];
    int maxNormalize = wDim - outDim;

    real *biasData         = THCTensor_(data)      (state, bias);
    real *weightData       = THCTensor_(data)      (state, weight);
    real *gradOutputData   = THCTensor_(data)      (state, gradOutput);
    real *valuesData       = THCTensor_(data)      (state, values);
    int64_t *keysData         = THCudaLongTensor_data (state, keys);
    int64_t *cumSumSizesData  = THCudaLongTensor_data (state, cumSumSizes);
    int64_t weightStride = weight->stride[0];

    cudaStream_t stream = THCState_getCurrentStream(state);
    dim3 threads(THREADS_X, THREADS_Y);
    int blocks_x = divup(outDim, threads.x);

    accGradBias<real, true><<<blocks_x, threads, 0, stream>>>
        (biasData, gradOutputData, outDim, batchSize, scale, weightDecay);

    int64_t nnzPerRow = divup(keysSize, batchSize);
    int blocks_y = divup(nnzPerRow, REPEAT * threads.y);
    dim3 blocks(blocks_x, blocks_y);

    for (int64_t batchId = 0; batchId < batchSize; batchId++) {
        accUpdateWeight<real><<<blocks, threads, 0, stream>>>
            (weightData, weightStride, gradOutputData, outDim, valuesData,
             cumSumSizesData, keysData, keysOffset, scale, weightDecay, maxNormalize,
             batchId);
    }
}

void THNN_(IndexLinear_updateParameters)(
    THCState *state,
    THCTensor *gradWeight,
    THCTensor *gradBias,
    THCTensor *weight,
    THCTensor *bias,
    THCudaLongTensor *runningKeys,
    THCudaLongTensor *cumSumSizes,
    int64_t keysOffset,
    accreal weightDecay,
    accreal learningRate)
{
    // Make sure these inputs are contiguous to accelerate computations
    THArgCheck(THCTensor_(isContiguous)(state, gradWeight), 1,
               "gradWeight matrix must be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, gradBias), 2,
               "gradBias vector must be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, weight), 3,
               "weight matrix must be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, bias), 4,
               "bias vector must be contiguous");
    THArgCheck(THCudaLongTensor_isContiguous(state, runningKeys), 5,
               "runningKeys vector must be contiguous");
    THArgCheck(THCudaLongTensor_isContiguous(state, cumSumSizes), 6,
               "cumSumSizes vector must be contiguous");

    int64_t outDim = bias->size[0];
    int64_t wDim = weight->size[1];
    int maxNormalize = wDim - outDim;
    int64_t keysSize = runningKeys->size[0];
    int64_t batchSize = cumSumSizes->size[0];

    THCTensor_(cadd)(state, bias, bias, -learningRate, gradBias);
    int64_t gradWeightStride = gradWeight->stride[0];
    int64_t weightStride = weight->stride[0];

    int64_t *keysData        = THCudaLongTensor_data (state, runningKeys);
    int64_t *cumSumSizesData = THCudaLongTensor_data (state, cumSumSizes);
    real *gradWeightData  = THCTensor_(data)      (state, gradWeight);
    real *weightData      = THCTensor_(data)      (state, weight);

    dim3 threads(THREADS_X, THREADS_Y);
    int64_t nnzPerRow = divup(keysSize, batchSize);
    int blocks_x = divup(outDim, threads.x);
    int blocks_y = divup(nnzPerRow, REPEAT * threads.y);
    dim3 blocks(blocks_x, blocks_y);
    cudaStream_t stream = THCState_getCurrentStream(state);

    for (int64_t batchId = 0; batchId < batchSize; batchId++) {
        updateWeight<real><<<blocks, threads, 0, stream>>>
            (weightData, gradWeightData, keysData, cumSumSizesData, outDim,
             gradWeightStride, weightStride, keysOffset, learningRate, weightDecay,
             maxNormalize, batchId);
    }
}
#endif
