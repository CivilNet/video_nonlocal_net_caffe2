#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "common.h"
#include "THCDeviceTensor.cuh"
#include "THCDeviceTensorUtils.cuh"
#include "THCDeviceUtils.cuh"
#include "THCReduceApplyUtils.cuh"
#include <THC/THCApply.cuh>

#include "THCHalf.h"
#include "THCHalfAutoNumerics.cuh"
#include "THCAtomics.cuh"

template <typename Dtype>
__global__ void TemporalReplicationPadding_updateOutput(
  THCDeviceTensor<Dtype, 3> input,
  THCDeviceTensor<Dtype, 3> output,
  int padL, int padR) {

  int outputPointId = threadIdx.x + blockIdx.x * blockDim.x;
  int plane = blockIdx.y;
  int batch = blockIdx.z;
  if (outputPointId >= output.getSize(2)) {
    return;
  }
  int outputPointX = outputPointId % output.getSize(2);

  int iStartX = max(0, -padL);
  int oStartX = max(0, padL);

  int inputPointX = min(max(padL, outputPointX), input.getSize(2) + padL - 1) - oStartX + iStartX;

  Dtype valueToCopy = input[batch][plane][inputPointX];
  output[batch][plane][outputPointX] = valueToCopy;
}

template <typename Dtype>
__global__ void TemporalReplicationPadding_updateGradInput(
  THCDeviceTensor<Dtype, 3> gradInput,
  THCDeviceTensor<Dtype, 3> gradOutput,
  int padL, int padR) {

  int outputPointId = threadIdx.x + blockIdx.x * blockDim.x;
  int plane = blockIdx.y;
  int batch = blockIdx.z;
  if (outputPointId >= gradOutput.getSize(2)) {
    return;
  }
  int outputPointX = outputPointId % gradOutput.getSize(2);

  int iStartX = max(0, -padL);
  int oStartX = max(0, padL);

  int inputPointX = min(max(padL, outputPointX), gradInput.getSize(2) + padL - 1) - oStartX + iStartX;

  Dtype valueToCopy = gradOutput[batch][plane][outputPointX];
  atomicAdd(&gradInput[batch][plane][inputPointX], valueToCopy);
}


#include "generic/TemporalReplicationPadding.cu"
#include "THCGenerateFloatTypes.h"
