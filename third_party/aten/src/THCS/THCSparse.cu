#include "THCSparse.h"

void THCudaSparse_Xcoo2csr(THCState *state, const int *coorowind, int64_t nnz, int64_t m, int *csrrowptr) {
  THAssertMsg((m <= INT_MAX) && (nnz <= INT_MAX),
    "hipsparseXcoo2csr only supports m, nnz with the bound [val] <= %d",
    INT_MAX);
  hipsparseHandle_t handle = THCState_getCurrentSparseHandle(state);
  hipsparseSetStream(handle, THCState_getCurrentStream(state));
  THCusparseCheck(hipsparseXcoo2csr(handle, coorowind, nnz, m, csrrowptr,
    TH_INDEX_BASE ? HIPSPARSE_INDEX_BASE_ONE : HIPSPARSE_INDEX_BASE_ZERO
  ));
}

hipsparseOperation_t convertTransToCusparseOperation(char trans) {
  if (trans == 't') return HIPSPARSE_OPERATION_TRANSPOSE;
  else if (trans == 'n') return HIPSPARSE_OPERATION_NON_TRANSPOSE;
  else if (trans == 'c') return HIPSPARSE_OPERATION_CONJUGATE_TRANSPOSE;
  else {
    THError("trans must be one of: t, n, c");
    return HIPSPARSE_OPERATION_TRANSPOSE;
  }
}

void adjustLd(char transb, int64_t m, int64_t n, int64_t k, int64_t *ldb, int64_t *ldc)
{
  int transb_ = ((transb == 't') || (transb == 'T'));

  if(n == 1)
    *ldc = m;

  if(transb_)
  {
    if(k == 1)
      *ldb = n;
  }
  else
  {
    if(n == 1)
      *ldb = k;
  }
}

/* Level 3 */
void THCudaSparse_Scsrmm2(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k, int64_t nnz, float alpha, float *csrvala, int *csrrowptra, int *csrcolinda, float *b, int64_t ldb, float beta, float *c, int64_t ldc)
{
  adjustLd(transb, m, n, k, &ldb, &ldc);
  hipsparseOperation_t opa = convertTransToCusparseOperation(transa);
  hipsparseOperation_t opb = convertTransToCusparseOperation(transb);

  THAssertMsg((m <= INT_MAX) && (n <= INT_MAX) && (k <= INT_MAX) && (nnz <= INT_MAX)  && (ldb <= INT_MAX) && (ldc <= INT_MAX),
    "hipsparseScsrmm2 only supports m, n, k, nnz, ldb, ldc with the bound [val] <= %d",
    INT_MAX);
  int i_m = (int)m;
  int i_n = (int)n;
  int i_k = (int)k;
  int i_nnz = (int)nnz;
  int i_ldb = (int)ldb;
  int i_ldc = (int)ldc;

  hipsparseHandle_t handle = THCState_getCurrentSparseHandle(state);
  hipsparseSetStream(handle, THCState_getCurrentStream(state));
  hipsparseMatDescr_t desc;
  hipsparseCreateMatDescr(&desc);
#if TH_INDEX_BASE == 1
  hipsparseSetMatIndexBase(&desc, HIPSPARSE_INDEX_BASE_ONE);
#endif
  THCusparseCheck(hipsparseScsrmm2(handle, opa, opb, i_m, i_n, i_k, i_nnz, &alpha, desc, csrvala, csrrowptra, csrcolinda, b, i_ldb, &beta, c, i_ldc));
}

void THCudaSparse_Dcsrmm2(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k, int64_t nnz, double alpha, double *csrvala, int *csrrowptra, int *csrcolinda, double *b, int64_t ldb, double beta, double *c, int64_t ldc)
{
  adjustLd(transb, m, n, k, &ldb, &ldc);
  hipsparseOperation_t opa = convertTransToCusparseOperation(transa);
  hipsparseOperation_t opb = convertTransToCusparseOperation(transb);

  THAssertMsg((m <= INT_MAX) && (n <= INT_MAX) && (k <= INT_MAX) && (nnz <= INT_MAX)  && (ldb <= INT_MAX) && (ldc <= INT_MAX),
    "hipsparseDcsrmm2 only supports m, n, k, nnz, ldb, ldc with the bound [val] <= %d",
    INT_MAX);
  int i_m = (int)m;
  int i_n = (int)n;
  int i_k = (int)k;
  int i_nnz = (int)nnz;
  int i_ldb = (int)ldb;
  int i_ldc = (int)ldc;

  hipsparseHandle_t handle = THCState_getCurrentSparseHandle(state);
  hipsparseSetStream(handle, THCState_getCurrentStream(state));
  hipsparseMatDescr_t desc;
  hipsparseCreateMatDescr(&desc);
#if TH_INDEX_BASE == 1
  hipsparseSetMatIndexBase(&desc, HIPSPARSE_INDEX_BASE_ONE);
#endif
  THCusparseCheck(hipsparseDcsrmm2(handle, opa, opb, i_m, i_n, i_k, i_nnz, &alpha, desc, csrvala, csrrowptra, csrcolinda, b, i_ldb, &beta, c, i_ldc));
}

/* format conversion */
void THCudaSparse_CreateIdentityPermutation(THCState *state, int64_t nnz, int *P) {
  THAssertMsg((nnz <= INT_MAX),
    "Xcsrsort_bufferSizeExt only supports m, n, nnz with the bound [val] <= %d",
    INT_MAX);
  int i_nnz = (int)nnz;

  hipsparseHandle_t handle = THCState_getCurrentSparseHandle(state);
  hipsparseSetStream(handle, THCState_getCurrentStream(state));
  hipsparseCreateIdentityPermutation(handle, i_nnz, P);
}

void THCudaSparse_Xcsrsort_bufferSizeExt(THCState *state, int64_t m, int64_t n, int64_t nnz, const int *csrRowPtr, const int *csrColInd, size_t *pBufferSizeInBytes)
{
  THAssertMsg((m <= INT_MAX) && (n <= INT_MAX) && (nnz <= INT_MAX),
    "Xcsrsort_bufferSizeExt only supports m, n, nnz with the bound [val] <= %d",
    INT_MAX);
  int i_m = (int)m;
  int i_n = (int)n;
  int i_nnz = (int)nnz;

  hipsparseHandle_t handle = THCState_getCurrentSparseHandle(state);
  hipsparseSetStream(handle, THCState_getCurrentStream(state));
  THCusparseCheck(hipsparseXcsrsort_bufferSizeExt(handle, i_m, i_n, i_nnz, csrRowPtr, csrColInd, pBufferSizeInBytes));
}

void THCudaSparse_Xcsrsort(THCState *state, int64_t m, int64_t n, int64_t nnz, const int *csrRowPtr, int *csrColInd, int *P, void *pBuffer)
{
  THAssertMsg((m <= INT_MAX) && (n <= INT_MAX) && (nnz <= INT_MAX),
    "Xcsrsort only supports m, n, nnz with the bound [val] <= %d",
    INT_MAX);
  int i_m = (int)m;
  int i_n = (int)n;
  int i_nnz = (int)nnz;

  hipsparseHandle_t handle = THCState_getCurrentSparseHandle(state);
  hipsparseSetStream(handle, THCState_getCurrentStream(state));
  hipsparseMatDescr_t desc;
  hipsparseCreateMatDescr(&desc);
#if TH_INDEX_BASE == 1
  hipsparseSetMatIndexBase(&desc, HIPSPARSE_INDEX_BASE_ONE);
#endif
  THCusparseCheck(hipsparseXcsrsort(handle, i_m, i_n, i_nnz, desc, csrRowPtr, csrColInd, P, pBuffer));
}

void THCudaSparse_Xcoosort_bufferSizeExt(THCState *state, int64_t m, int64_t n, int64_t nnz, const int *cooRows, const int *cooCols, size_t *pBufferSizeInBytes)
{
  THAssertMsg((m <= INT_MAX) && (n <= INT_MAX) && (nnz <= INT_MAX),
    "Xcoosort_bufferSizeExt only supports m, n, nnz with the bound [val] <= %d",
    INT_MAX);
  int i_m = (int)m;
  int i_n = (int)n;
  int i_nnz = (int)nnz;

  hipsparseHandle_t handle = THCState_getCurrentSparseHandle(state);
  hipsparseSetStream(handle, THCState_getCurrentStream(state));
  THCusparseCheck(hipsparseXcoosort_bufferSizeExt(handle, i_m, i_n, i_nnz, cooRows, cooCols, pBufferSizeInBytes));
}

void THCudaSparse_XcoosortByRow(THCState *state, int64_t m, int64_t n, int64_t nnz, int *cooRows, int *cooCols, int *P, void *pBuffer)
{
  THAssertMsg((m <= INT_MAX) && (n <= INT_MAX) && (nnz <= INT_MAX),
    "XcoosortByRow only supports m, n, nnz with the bound [val] <= %d",
    INT_MAX);
  int i_m = (int)m;
  int i_n = (int)n;
  int i_nnz = (int)nnz;

  hipsparseHandle_t handle = THCState_getCurrentSparseHandle(state);
  hipsparseSetStream(handle, THCState_getCurrentStream(state));
  THCusparseCheck(hipsparseXcoosortByRow(handle, i_m, i_n, i_nnz, cooRows, cooCols, P, pBuffer));
}
