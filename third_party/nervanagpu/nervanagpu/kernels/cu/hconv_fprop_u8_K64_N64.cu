
#include <hip/hip_runtime.h>
/*
 * Copyright 2014 Nervana Systems Inc. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *    http://www.apache.org/licenses/LICENSE-2.0
 * 
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

// nvcc -arch sm_50 -cubin hconv_fprop_u8_K64_N64.cu

extern "C"
__global__ void __launch_bounds__(64) hconv_fprop_u8_K64_N64
(
    int* param_Rand,
    unsigned short*       param_O,
    const unsigned char*  param_I,
    const unsigned short* param_F,
    float param_alpha,
    int param_flags,
    int param_N,
    int param_K,
    int param_D,
    int param_H,
    int param_W,
    int param_WN,
    int param_HWN,
    int param_DHWN,
    int param_C,
    int param_CRST,
    int param_RST,
    int param_magic_RST,
    int param_shift_RST,
    int param_RS,
    int param_magic_RS,
    int param_shift_RS,
    int param_S,
    int param_magic_S,
    int param_shift_S,
    int param_pad_d,
    int param_pad_h,
    int param_pad_w,
    int param_str_d,
    int param_str_h,
    int param_str_w,
    int param_P,
    int param_Q,
    int param_PQ,
    int param_QN,
    int param_PQN,
    int param_MPQN,
    int param_magic_Q,
    int param_shift_Q,
    int param_magic_PQ,
    int param_shift_PQ,
    int param_part_P,
    int param_part_Q,
    int param_part_PQ
)
{
    __shared__ float share[64*8*4 + 8];

    int tid = threadIdx.x;

    share[tid] = 1;

    *param_O = share[63-tid];
}
