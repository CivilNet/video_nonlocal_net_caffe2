
#include <hip/hip_runtime.h>
/*
 * Copyright 2014 Nervana Systems Inc. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *    http://www.apache.org/licenses/LICENSE-2.0
 * 
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

extern "C"
__global__ void __launch_bounds__(128) sgemm_nn_128x64
(
    unsigned*    param_Rand,
    const float* param_A,
    const float* param_B,
    float*       param_C,
    int          param_lda,  
    int          param_ldb8,  
    int          param_ldc,
    int          param_m,
    int          param_n,
    int          param_k,
    float        param_alpha,
    float        param_beta,
    int          param_flags,
    int          param_ldaz,
    int          param_ldbz,
    int          param_ldcz,
    int          param_batch_loops
)
{
    __shared__ float share[128*8*2 + 64*8*2 + 4];

    int tid = threadIdx.x;

    share[tid] = 1;

    param_C[tid] = share[127-tid];
}
